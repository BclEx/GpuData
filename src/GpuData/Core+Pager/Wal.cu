#include "hip/hip_runtime.h"
﻿// wal.c
#include "Core+Pager.cu.h"
#include <stddef.h>
using namespace Core;

namespace Core
{

#if defined(TEST) && defined(_DEBUG)
	bool _walTrace = false;
#define WALTRACE(X)  if (_walTrace) printf(X)
#else
#define WALTRACE(X)
#endif

#define WAL_MAX_VERSION      3007000
#define WALINDEX_MAX_VERSION 3007000
#define WAL_WRITE_LOCK         0
#define WAL_ALL_BUT_WRITE      1
#define WAL_CKPT_LOCK          1
#define WAL_RECOVER_LOCK       2
#define WAL_READ_LOCK(I)       (3 + (I))
#define WAL_NREADER            (VFile::SHM::SHM_MAX - 3)

	typedef struct WalIndexHeader WalIndexHeader;
	typedef struct WalIterator WalIterator;
	typedef struct WalCheckpointInfo WalCheckpointInfo;

	struct WalIndexHeader
	{
		uint32 Version;                 // Wal-index version
		uint32 Unused;					// Unused (padding) field
		uint32 Change;                  // Counter incremented each transaction
		bool IsInit;					// 1 when initialized
		bool BigEndianChecksum;			// True if checksums in WAL are big-endian
		uint16 SizePage;                // Database page size in bytes. 1==64K
		uint32 MaxFrame;                // Index of last valid frame in the WAL
		uint32 Pages;                   // Size of database in pages
		uint32 FrameChecksum[2];		// Checksum of last frame in log
		uint32 Salt[2];					// Two salt values copied from WAL header
		uint32 Checksum[2];				// Checksum over all prior fields
	};

	struct WalCheckpointInfo
	{
		uint32 Backfills;               // Number of WAL frames backfilled into DB
		uint32 ReadMarks[WAL_NREADER];  // Reader marks
	};

#define READMARK_NOT_USED 0xffffffff
#define WALINDEX_LOCK_OFFSET (sizeof(WalIndexHeader)*2 + sizeof(WalCheckpointInfo))
#define WALINDEX_LOCK_RESERVED 16
#define WALINDEX_HDR_SIZE (WALINDEX_LOCK_OFFSET+WALINDEX_LOCK_RESERVED)
#define WAL_FRAME_HDRSIZE 24
#define WAL_HDRSIZE 32
#define WAL_MAGIC 0x377f0682
#define walFrameOffset(Frame, SizePage) (WAL_HDRSIZE + ((Frame) - 1) * (int64)((SizePage) + WAL_FRAME_HDRSIZE))

	struct Wal
	{
		VFileSystem *Vfs;				// The VFS used to create pDbFd
		VFile *DBFile;					// File handle for the database file
		VFile *WalFile;					// File handle for WAL file
		uint32 Callback;				// Value to pass to log callback (or 0)
		int64 MaxWalSize;				// Truncate WAL to this size upon reset
		int SizeFirstBlock;				// Size of first block written to WAL file
		//int nWiData;					// Size of array apWiData
		volatile uint32 **WiData;		// Pointer to wal-index content in memory */
		uint32 SizePage;                // Database page size
		int16 ReadLock;					// Which read lock is being held.  -1 for none
		uint8 SyncFlags;				// Flags to use to sync header writes
		uint8 ExclusiveMode;			// Non-zero if connection is in exclusive mode
		uint8 WriteLock;				// True if in a write transaction
		uint8 CheckpointLock;           // True if holding a checkpoint lock
		uint8 ReadOnly;					// WAL_RDWR, WAL_RDONLY, or WAL_SHM_RDONLY
		uint8 TruncateOnCommit;			// True to truncate WAL file on commit
		uint8 SyncHeader;				// Fsync the WAL header if true
		uint8 PadToSectorBoundary;		// Pad transactions out to the next sector
		WalIndexHeader Header;			// Wal-index header for current transaction
		const char *WalName;			// Name of WAL file
		uint32 Checkpoints;				// Checkpoint sequence counter in the wal-header
#ifdef _DEBUG
		uint8 LockError;				// True if a locking error has occurred
#endif
	};

#define WAL_NORMAL_MODE     0
#define WAL_EXCLUSIVE_MODE  1     
#define WAL_HEAPMEMORY_MODE 2

#define WAL_RDWR        0    // Normal read/write connection
#define WAL_RDONLY      1    // The WAL file is readonly
#define WAL_SHM_RDONLY  2    // The SHM file is readonly

	// Each page of the wal-index mapping contains a hash-table made up of an array of HASHTABLE_NSLOT elements of the following type.
	typedef uint16 ht_slot;

	//
	// This structure is used to implement an iterator that loops through all frames in the WAL in database page order. Where two or more frames
	// correspond to the same database page, the iterator visits only the frame most recently written to the WAL (in other words, the frame with
	// the largest index).
	//
	// The internals of this structure are only accessed by:
	//
	//   walIteratorInit() - Create a new iterator,
	//   walIteratorNext() - Step an iterator,
	//   walIteratorFree() - Free an iterator.
	//
	// This functionality is used by the checkpoint code (see walCheckpoint()).
	struct WalIterator
	{
		int Prior;						// Last result returned from the iterator
		int SegmentsLength;             // Number of entries in aSegment[]
		struct WalSegment
		{
			int Next;					// Next slot in aIndex[] not yet returned
			ht_slot *Indexs;            // i0, i1, i2... such that aPgno[iN] ascend
			uint32 *IDs;				// Array of page numbers.
			int Entrys;                 // Nr. of entries in aPgno[] and aIndex[]
			int Zero;					// Frame number associated with aPgno[0]
		} Segments[1];					// One for every 32KB page in the wal-index
	};

	//
	// Define the parameters of the hash tables in the wal-index file. There is a hash-table following every HASHTABLE_NPAGE page numbers in the
	// wal-index.
	//
	// Changing any of these constants will alter the wal-index format and create incompatibilities.
#define HASHTABLE_NPAGE      4096                 // Must be power of 2
#define HASHTABLE_HASH_1     383                  // Should be prime
#define HASHTABLE_NSLOT      (HASHTABLE_NPAGE*2)  // Must be a power of 2

	// The block of page numbers associated with the first hash-table in a wal-index is smaller than usual. This is so that there is a complete
	// hash-table on each aligned 32KB page of the wal-index.
#define HASHTABLE_NPAGE_ONE (HASHTABLE_NPAGE - (WALINDEX_HDR_SIZE / sizeof(uint32)))

	// The wal-index is divided into pages of WALINDEX_PGSZ bytes each.
#define WALINDEX_PGSZ   (sizeof(ht_slot) * HASHTABLE_NSLOT + HASHTABLE_NPAGE * sizeof(uint32))

	static RC walIndexPage(Wal *wal, Pid id, volatile Pid **idOut)
	{
		// Enlarge the pWal->apWiData[] array if required
		if (__arrayLength(wal->WiData) <= id)
		{
			int bytes = sizeof(uint32 *) * (id + 1);
			volatile uint32 **newWiData = (volatile uint32 **)SysEx::Realloc((void *)wal->WiData, bytes);
			if (!newWiData)
			{
				*idOut = nullptr;
				return RC::NOMEM;
			}
			_memset((void *)&newWiData[__arrayLength(wal->WiData)], 0, sizeof(uint32 *) * (id + 1 - __arrayLength(wal->WiData)));
			__arraySet(wal->WiData, newWiData, id + 1);
		}

		// Request a pointer to the required page from the VFS
		RC rc = RC::OK;
		if (wal->WiData[id] == 0)
		{
			if (wal->ExclusiveMode == WAL_HEAPMEMORY_MODE)
			{
				wal->WiData[id] = (uint32 volatile *)SysEx::Alloc(WALINDEX_PGSZ, true);
				if (!wal->WiData[id]) rc = RC::NOMEM;
			}
			else
			{
				rc = wal->DBFile->ShmMap(id, WALINDEX_PGSZ, wal->WriteLock, (void volatile **)&wal->WiData[id]);
				if (rc == RC::READONLY)
				{
					wal->ReadOnly |= WAL_SHM_RDONLY;
					rc = RC::OK;
				}
			}
		}

		*idOut = wal->WiData[id];
		_assert(id == 0 || *idOut || rc != RC::OK);
		return rc;
	}

	static volatile WalCheckpointInfo *walCkptInfo(Wal *wal)
	{
		_assert(__arrayLength(wal->WiData) > 0 && wal->WiData[0]);
		return (volatile WalCheckpointInfo *) & (wal->WiData[0][sizeof(WalIndexHeader) / 2]);
	}

	static volatile WalIndexHeader *walIndexHeader(Wal *wal)
	{
		_assert(__arrayLength(wal->WiData) > 0 && wal->WiData[0]);
		return (volatile WalIndexHeader *)wal->WiData[0];
	}

#define BYTESWAP32(x) ((((x)&0x000000FF)<<24) + (((x)&0x0000FF00)<<8) + (((x)&0x00FF0000)>>8)  + (((x)&0xFF000000)>>24))

	static void walChecksumBytes(bool nativeChecksum,  uint8 *b, int length, const uint32 *checksum, uint32 *checksumOut)
	{
		uint32 s1, s2;
		if (checksum)
		{
			s1 = checksum[0];
			s2 = checksum[1];
		}
		else
			s1 = s2 = 0;

		_assert(length >= 8);
		_assert((length & 0x00000007) == 0);

		uint32 *data = (uint32 *)b;
		uint32 *end = (uint32 *)&b[length];
		if (nativeChecksum)
		{
			do
			{
				s1 += *data++ + s2;
				s2 += *data++ + s1;
			} while (data < end);
		}
		else
		{
			do
			{
				s1 += BYTESWAP32(data[0]) + s2;
				s2 += BYTESWAP32(data[1]) + s1;
				data += 2;
			} while (data < end);
		}

		checksumOut[0] = s1;
		checksumOut[1] = s2;
	}

	static void walShmBarrier(Wal *wal)
	{
		if (wal->ExclusiveMode != WAL_HEAPMEMORY_MODE)
			wal->DBFile->ShmBarrier();
	}

	static void walIndexWriteHdr(Wal *wal)
	{
		volatile WalIndexHeader *header = walIndexHeader(wal);
		const int checksumIdx = offsetof(WalIndexHeader, Checksum);

		_assert(wal->WriteLock);
		wal->Header.IsInit = true;
		wal->Header.Version = WALINDEX_MAX_VERSION;
		walChecksumBytes(1, (uint8 *)&wal->Header, checksumIdx, 0, wal->Header.Checksum);
		_memcpy((void *)&header[1], (void *)&wal->Header, sizeof(WalIndexHeader));
		walShmBarrier(wal);
		_memcpy((void *)&header[0], (void *)&wal->Header, sizeof(WalIndexHeader));
	}

	static void walEncodeFrame(Wal *wal, Pid id, uint32 truncate, uint8 *data, uint8 *frame)
	{
		uint32 *checksum = wal->Header.FrameChecksum;
		_assert(WAL_FRAME_HDRSIZE == 24);
		ConvertEx::Put4(&frame[0], id);
		ConvertEx::Put4(&frame[4], truncate);
		_memcpy(&frame[8], wal->Header.Salt, 8);

		bool nativeChecksum = (wal->Header.BigEndianChecksum == TYPE_BIGENDIAN); // True for native byte-order checksums
		walChecksumBytes(nativeChecksum, frame, 8, checksum, checksum);
		walChecksumBytes(nativeChecksum, data, wal->SizePage, checksum, checksum);

		ConvertEx::Put4(&frame[16], checksum[0]);
		ConvertEx::Put4(&frame[20], checksum[1]);
	}

	static int walDecodeFrame(Wal *wal, Pid *idOut, uint32 *truncateOut, uint8 *data, uint8 *frame)
	{
		uint32 *checksum = wal->Header.FrameChecksum;
		_assert(WAL_FRAME_HDRSIZE == 24);

		// A frame is only valid if the salt values in the frame-header match the salt values in the wal-header. 
		if (_memcmp(&wal->Header.Salt, &frame[8], 8) != 0)
			return false;

		// A frame is only valid if the page number is creater than zero.
		Pid id = ConvertEx::Get4(&frame[0]); // Page number of the frame
		if (id == 0)
			return false;

		// A frame is only valid if a checksum of the WAL header, all prior frams, the first 16 bytes of this frame-header, 
		// and the frame-data matches the checksum in the last 8 bytes of this frame-header.
		bool nativeChecksum = (wal->Header.BigEndianChecksum == TYPE_BIGENDIAN); // True for native byte-order checksums
		walChecksumBytes(nativeChecksum, frame, 8, checksum, checksum);
		walChecksumBytes(nativeChecksum, data, wal->SizePage, checksum, checksum);
		if (checksum[0] != ConvertEx::Get4(&frame[16]) || checksum[1]!=ConvertEx::Get4(&frame[20])) // Checksum failed.
			return false;

		// If we reach this point, the frame is valid.  Return the page number and the new database size.
		*idOut = id;
		*truncateOut = ConvertEx::Get4(&frame[4]);
		return true;
	}

#if defined(TEST) && defined(_DEBUG)
	static const char *walLockName(int lockIdx)
	{
		if (lockIdx == WAL_WRITE_LOCK)
			return "WRITE-LOCK";
		else if (lockIdx == WAL_CKPT_LOCK)
			return "CKPT-LOCK";
		else if (lockIdx == WAL_RECOVER_LOCK)
			return "RECOVER-LOCK";
		else
		{
			static char name[15];
			_snprintf(name, sizeof(name), "READ-LOCK[%d]", lockIdx - WAL_READ_LOCK(0));
			return name;
		}
	}
#endif

	static RC walLockShared(Wal *wal, int lockIdx)
	{
		if (wal->ExclusiveMode) return RC::OK;
		RC rc = wal->DBFile->ShmLock(lockIdx, 1, (VFile::SHM)(VFile::SHM::LOCK | VFile::SHM::SHARED));
		WALTRACE("WAL%p: acquire SHARED-%s %s\n", wal, walLockName(lockIdx), rc ? "failed" : "ok");
		return rc;
	}

	static void walUnlockShared(Wal *wal, int lockIdx)
	{
		if (wal->ExclusiveMode) return;
		wal->DBFile->ShmLock(lockIdx, 1, (VFile::SHM)(VFile::SHM::UNLOCK | VFile::SHM::SHARED));
		WALTRACE("WAL%p: release SHARED-%s\n", wal, walLockName(lockIdx));
	}

	static RC walLockExclusive(Wal *wal, int lockIdx, int n)
	{
		if (wal->ExclusiveMode) return RC::OK;
		RC rc = wal->DBFile->ShmLock(lockIdx, n, (VFile::SHM)(VFile::SHM::LOCK | VFile::SHM::EXCLUSIVE));
		WALTRACE("WAL%p: acquire EXCLUSIVE-%s cnt=%d %s\n", wal, walLockName(lockIdx), n, rc ? "failed" : "ok");
		return rc;
	}

	static void walUnlockExclusive(Wal *wal, int lockIdx, int n)
	{
		if (wal->ExclusiveMode) return;
		wal->DBFile->ShmLock(lockIdx, n, (VFile::SHM)(VFile::SHM::UNLOCK | VFile::SHM::EXCLUSIVE));
		WALTRACE("WAL%p: release EXCLUSIVE-%s cnt=%d\n", wal, walLockName(lockIdx), n);
	}

	static int walHash(uint id)
	{
		_assert(id > 0);
		_assert((HASHTABLE_NSLOT & (HASHTABLE_NSLOT-1)) == 0);
		return (id * HASHTABLE_HASH_1) & (HASHTABLE_NSLOT-1);
	}

	static int walNextHash(int priorHash)
	{
		return (priorHash + 1) & (HASHTABLE_NSLOT - 1);
	}

	static RC walHashGet(Wal *wal, int id, volatile ht_slot **hashOut, volatile Pid **idsOut, uint32 *zeroOut)
	{
		volatile Pid *ids;
		RC rc = walIndexPage(wal, id, &ids);
		_assert(rc == RC::OK || id > 0);

		if (rc == RC::OK)
		{
			Pid zero;
			volatile ht_slot *hash = (volatile ht_slot *)&ids[HASHTABLE_NPAGE];
			if (id == 0)
			{
				ids = &ids[WALINDEX_HDR_SIZE / sizeof(Pid)];
				zero = 0;
			}
			else
				zero = HASHTABLE_NPAGE_ONE + (id - 1) * HASHTABLE_NPAGE;

			*idsOut = &ids[-1];
			*hashOut = hash;
			*zeroOut = zero;
		}
		return rc;
	}

	static int walFramePage(uint32 frame)
	{
		int hash = (frame + HASHTABLE_NPAGE-HASHTABLE_NPAGE_ONE-1) / HASHTABLE_NPAGE;
		_assert((hash==0 || frame > HASHTABLE_NPAGE_ONE) && 
			(hash>=1 || frame <= HASHTABLE_NPAGE_ONE) && 
			(hash<=1 || frame > (HASHTABLE_NPAGE_ONE + HASHTABLE_NPAGE)) && 
			(hash>=2 || frame <= HASHTABLE_NPAGE_ONE + HASHTABLE_NPAGE) && 
			(hash<=2 || frame > (HASHTABLE_NPAGE_ONE + 2 * HASHTABLE_NPAGE)));
		return hash;
	}

	static uint32 walFramePgno(Wal *wal, uint32 frame)
	{
		int hash = walFramePage(frame);
		if (hash == 0)
			return wal->WiData[0][WALINDEX_HDR_SIZE / sizeof(uint32) + frame - 1];
		return wal->WiData[hash][(frame - 1 - HASHTABLE_NPAGE_ONE) % HASHTABLE_NPAGE];
	}

	static void walCleanupHash(Wal *wal)
	{
		_assert(wal->WriteLock);
		ASSERTCOVERAGE(wal->Header.MaxFrame == HASHTABLE_NPAGE_ONE - 1);
		ASSERTCOVERAGE(wal->Header.MaxFrame == HASHTABLE_NPAGE_ONE);
		ASSERTCOVERAGE(wal->Header.MaxFrame == HASHTABLE_NPAGE_ONE + 1);

		if (wal->Header.MaxFrame == 0) return;

		// Obtain pointers to the hash-table and page-number array containing the entry that corresponds to frame pWal->hdr.mxFrame. It is guaranteed
		// that the page said hash-table and array reside on is already mapped.
		_assert(__arrayLength(wal->WiData) > walFramePage(wal->Header.MaxFrame));
		_assert(wal->WiData[walFramePage(wal->Header.MaxFrame)] != 0);
		volatile ht_slot *hash = nullptr; // Pointer to hash table to clear
		volatile Pid *ids = nullptr; // Page number array for hash table
		int zero = 0; // frame == (aHash[x]+iZero)
		walHashGet(wal, walFramePage(wal->Header.MaxFrame), &hash, &ids, &zero);

		// Zero all hash-table entries that correspond to frame numbers greater than pWal->hdr.mxFrame.
		int limit = wal->Header.MaxFrame - zero; // Zero values greater than this
		_assert(limit > 0);
		for (int i = 0; i < HASHTABLE_NSLOT; i++)
			if (hash[i] > limit)
				hash[i] = 0;

		// Zero the entries in the aPgno array that correspond to frames with frame numbers greater than pWal->hdr.mxFrame. 
		int bytes = (int)((char *)hash - (char *)&ids[limit + 1]); // Number of bytes to zero in aPgno[]
		_memset((void *)&ids[limit + 1], 0, bytes);

#ifdef ENABLE_EXPENSIVE_ASSERT
		// Verify that the every entry in the mapping region is still reachable via the hash table even after the cleanup.
		int key; // Hash key
		if (limit)
			for (int i = 1; i <= limit; i++)
			{
				for (key = walHash(ids[i]); hash[key]; key = walNextHash(key))
					if (hash[key] == i) break;
				_assert(hash[key] == i);
			}
#endif
	}

	static RC walIndexAppend(Wal *wal, uint32 frame, Pid id)
	{
		volatile ht_slot *hash = nullptr; // Hash table
		volatile Pid *ids = nullptr; // Page number array
		uint zero = 0; // One less than frame number of aPgno[1]
		RC rc = walHashGet(wal, walFramePage(frame), &hash, &ids, &zero);

		// Assuming the wal-index file was successfully mapped, populate the page number array and hash table entry.
		if (rc == RC::OK)
		{
			int idx = frame - zero; // Value to write to hash-table slot
			_assert(idx <= HASHTABLE_NSLOT / 2 + 1 );

			// If this is the first entry to be added to this hash-table, zero the entire hash table and aPgno[] array before proceding. 
			if (idx == 1)
			{
				int bytes = (int)((uint8 *)&hash[HASHTABLE_NSLOT] - (uint8 *)&ids[1]);
				_memset((void*)&ids[1], 0, bytes);
			}

			// If the entry in aPgno[] is already set, then the previous writer must have exited unexpectedly in the middle of a transaction (after
			// writing one or more dirty pages to the WAL to free up memory). Remove the remnants of that writers uncommitted transaction from 
			// the hash-table before writing any new entries.
			if (ids[idx])
			{
				walCleanupHash(wal);
				_assert(!ids[idx]);
			}

			// Write the aPgno[] array entry and the hash-table slot.
			int collide = idx; // Number of hash collisions
			int key; // Hash table key
			for (key = walHash(id); hash[key]; key = walNextHash(key))
				if ((collide--) == 0) return SysEx_CORRUPT_BKPT;
			ids[idx] = id;
			hash[key] = (ht_slot)idx;

#ifdef ENABLE_EXPENSIVE_ASSERT
			// Verify that the number of entries in the hash table exactly equals the number of entries in the mapping region.
			{
				int entry = 0; // Number of entries in the hash table
				for (int i = 0; i < HASHTABLE_NSLOT; i++) { if (hash[i]) entry++; }
				_assert(entry == idx);
			}

			// Verify that the every entry in the mapping region is reachable via the hash table.  This turns out to be a really, really expensive
			// thing to check, so only do this occasionally - not on every iteration.
			if ((idx & 0x3ff) == 0)
				for (int i = 1; i <= idx; i++)
				{
					for(key = walHash(ids[i]); hash[key]; key = walNextHash(key))
						if (hash[key] == i) break;
					_assert(hash[key] == i);
				}
#endif
		}

		return rc;
	}

	static int walIndexRecover(Wal *wal)
	{
		uint32 frameChecksum[2] = {0, 0};

		// Obtain an exclusive lock on all byte in the locking range not already locked by the caller. The caller is guaranteed to have locked the
		// WAL_WRITE_LOCK byte, and may have also locked the WAL_CKPT_LOCK byte. If successful, the same bytes that are locked here are unlocked before
		// this function returns.
		_assert(wal->CheckpointLock == 1 || wal->CheckpointLock == 0);
		_assert(WAL_ALL_BUT_WRITE == WAL_WRITE_LOCK + 1);
		_assert(WAL_CKPT_LOCK == WAL_ALL_BUT_WRITE);
		_assert(wal->WriteLock);
		int lockIdx = WAL_ALL_BUT_WRITE + wal->CheckpointLock; // Lock offset to lock for checkpoint
		int locks = VFile::SHM::SHM_MAX - lockIdx; // Number of locks to hold
		RC rc = walLockExclusive(wal, lockIdx, locks);
		if (rc)
			return rc;
		WALTRACE("WAL%p: recovery begin...\n", wal);

		_memset(&wal->Header, 0, sizeof(WalIndexHeader));

		int64 size; // Size of log file
		rc = wal->WalFile->get_FileSize(&size);
		if (rc != RC::OK)
			goto recovery_error;

		if (size > WAL_HDRSIZE)
		{
			// Read in the WAL header.
			uint8 buf[WAL_HDRSIZE]; // Buffer to load WAL header into
			rc = wal->WalFile->Read(buf, WAL_HDRSIZE, 0);
			if (rc != RC::OK)
				goto recovery_error;

			// If the database page size is not a power of two, or is greater than SQLITE_MAX_PAGE_SIZE, conclude that the WAL file contains no valid 
			// data. Similarly, if the 'magic' value is invalid, ignore the whole WAL file.
			uint32 magic = ConvertEx::Get4(&buf[0]); // Magic value read from WAL header
			int sizePage = ConvertEx::Get4(&buf[8]); // Page size according to the log
			if ((magic & 0xFFFFFFFE) != WAL_MAGIC ||
				sizePage & (sizePage - 1) ||
				sizePage > MAX_PAGE_SIZE ||
				sizePage < 512)
				goto finished;
			wal->Header.BigEndianChecksum = (uint8)(magic & 0x00000001);
			wal->SizePage = sizePage;
			wal->Checkpoints = ConvertEx::Get4(&buf[12]);
			_memcpy(&wal->Header.Salt, &buf[16], 8);

			// Verify that the WAL header checksum is correct
			walChecksumBytes(wal->Header.BigEndianChecksum == TYPE_BIGENDIAN, buf, WAL_HDRSIZE - 2 * 4, 0, wal->Header.FrameChecksum);
			if (wal->Header.FrameChecksum[0] != ConvertEx::Get4(&buf[24]) || wal->Header.FrameChecksum[1] != ConvertEx::Get4(&buf[28]))
				goto finished;

			// Verify that the version number on the WAL format is one that are able to understand
			uint32 version = ConvertEx::Get4(&buf[4]); // Magic value read from WAL header
			if (version != WAL_MAX_VERSION)
			{
				rc = SysEx_CANTOPEN_BKPT;
				goto finished;
			}

			// Malloc a buffer to read frames into.
			int sizeFrame = sizePage + WAL_FRAME_HDRSIZE; // Number of bytes in buffer aFrame[]
			uint8 *frames = (uint8 *)SysEx::Alloc(sizeFrame); // Malloc'd buffer to load entire frame
			if (!frames)
			{
				rc = RC::NOMEM;
				goto recovery_error;
			}
			uint8 *data = &frames[WAL_FRAME_HDRSIZE]; // Pointer to data part of aFrame buffer

			// Read all frames from the log file.
			int frameIdx = 0; // Index of last frame read
			for (int64 offset = WAL_HDRSIZE; (offset + sizeFrame) <= size; offset += sizeFrame) // Next offset to read from log file
			{ 
				// Read and decode the next log frame.
				frameIdx++;
				rc = wal->WalFile->Read(frames, sizeFrame, offset);
				if (rc != RC::OK) break;
				Pid id; // Database page number for frame
				uint32 truncate; // dbsize field from frame header
				bool isValid = walDecodeFrame(wal, &id, &truncate, data, frames); // True if this frame is valid
				if (!isValid) break;
				rc = walIndexAppend(wal, frameIdx, id);
				if (rc != RC::OK) break;

				// If nTruncate is non-zero, this is a commit record.
				if (truncate)
				{
					wal->Header.MaxFrame = frameIdx;
					wal->Header.Pages = truncate;
					wal->Header.SizePage = (uint16)((sizePage & 0xff00) | (sizePage >> 16));
					ASSERTCOVERAGE(sizePage <= 32768);
					ASSERTCOVERAGE(sizePage >= 65536);
					frameChecksum[0] = wal->Header.FrameChecksum[0];
					frameChecksum[1] = wal->Header.FrameChecksum[1];
				}
			}

			SysEx::Free(frames);
		}

finished:
		if (rc == RC::OK)
		{
			volatile WalCheckpointInfo *info;
			wal->Header.FrameChecksum[0] = frameChecksum[0];
			wal->Header.FrameChecksum[1] = frameChecksum[1];
			walIndexWriteHdr(wal);

			// Reset the checkpoint-header. This is safe because this thread is currently holding locks that exclude all other readers, writers and checkpointers.
			info = walCkptInfo(wal);
			info->Backfills = 0;
			info->ReadMarks[0] = 0;
			for (int i = 1; i < WAL_NREADER; i++) info->ReadMarks[i] = READMARK_NOT_USED;
			if (wal->Header.MaxFrame) info->ReadMarks[1] = wal->Header.MaxFrame;

			// If more than one frame was recovered from the log file, report an event via sqlite3_log(). This is to help with identifying performance
			// problems caused by applications routinely shutting down without checkpointing the log file.
			if (wal->Header.Pages)
				SysEx_log(RC::OK, "Recovered %d frames from WAL file %s", wal->Header.Pages, wal->WalName);
		}

recovery_error:
		WALTRACE("WAL%p: recovery %s\n", wal, rc ? "failed" : "ok");
		walUnlockExclusive(wal, lockIdx, locks);
		return rc;
	}

	static void walIndexClose(Wal *wal, int isDelete)
	{
		if (wal->ExclusiveMode == WAL_HEAPMEMORY_MODE)
			for (int i = 0; i < __arrayLength(wal->WiData); i++)
			{
				SysEx::Free((void *)wal->WiData[i]);
				wal->WiData[i] = nullptr;
			}
		else
			wal->DBFile->ShmUnmap(isDelete);
	}

	RC Wal::Open(VFileSystem *vfs, VFile *dbFile, const char *walName, bool noShm, int64 maxWalSize, Wal **walOut)
	{
		_assert(walName && walName[0]);
		_assert(dbFile != nullptr);

		// In the amalgamation, the os_unix.c and os_win.c source files come before this source file.  Verify that the #defines of the locking byte offsets
		// in os_unix.c and os_win.c agree with the WALINDEX_LOCK_OFFSET value.
#ifdef WIN_SHM_BASE
		_assert(WIN_SHM_BASE == WALINDEX_LOCK_OFFSET);
#endif
#ifdef UNIX_SHM_BASE
		_assert(UNIX_SHM_BASE == WALINDEX_LOCK_OFFSET);
#endif

		// Allocate an instance of struct Wal to return.
		*walOut = nullptr;
		Wal *r = (Wal *)SysEx::Alloc(sizeof(Wal) + vfs->SizeOsFile, true); // Object to allocate and return
		if (!r)
			return RC::NOMEM;

		r->Vfs = vfs;
		r->WalFile = (VFile *)&r[1];
		r->DBFile = dbFile;
		r->ReadLock = -1;
		r->MaxWalSize = maxWalSize;
		r->WalName = walName;
		r->SyncHeader = 1;
		r->PadToSectorBoundary = 1;
		r->ExclusiveMode = (noShm ? WAL_HEAPMEMORY_MODE : WAL_NORMAL_MODE);

		// Open file handle on the write-ahead log file.
		VFileSystem::OPEN flags = (VFileSystem::OPEN)(VFileSystem::OPEN::OREADWRITE | VFileSystem::OPEN::CREATE | VFileSystem::OPEN::WAL);
		RC rc = vfs->Open(walName, r->WalFile, flags, &flags);
		if (rc == RC::OK && flags & VFileSystem::OPEN::READONLY)
			r->ReadOnly = WAL_RDONLY;

		if (rc != RC::OK)
		{
			walIndexClose(r, 0);
			r->WalFile->Close();
			SysEx::Free(r);
		}
		else
		{
			int dc = r->WalFile->get_DeviceCharacteristics();
			if (dc & VFile::IOCAP::SEQUENTIAL) { r->SyncHeader = 0; }
			if (dc & VFile::IOCAP::POWERSAFE_OVERWRITE)
				r->PadToSectorBoundary = 0;
			*walOut = r;
			WALTRACE("WAL%d: opened\n", r);
		}
		return rc;
	}

	void Wal::Limit(Wal *wal, int64 limit)
	{
		if (wal) wal->MaxWalSize = limit;
	}

	static int walIteratorNext(WalIterator *p, uint32 *page, uint32 *frame)
	{
		uint32 r = 0xFFFFFFFF; // 0xffffffff is never a valid page number
		uint32 min = p->Prior; // Result pgno must be greater than iMin
		_assert(min < 0xffffffff);
		for (int i = __arrayLength(p->Segments) - 1; i >= 0; i--)
		{
			struct WalSegment *segment = &p->Segments[i];
			while (segment->Next < segment->Entrys)
			{
				uint32 id = segment->IDs[segment->Indexs[segment->Next]];
				if (id > min)
				{
					if (id < r)
					{
						r = id;
						*frame = segment->Zero + segment->Indexs[segment->Next];
					}
					break;
				}
				segment->Next++;
			}
		}

		*page = p->Prior = r;
		return (r == 0xFFFFFFFF);
	}

	static void walMerge(const uint32 *content, ht_slot *lefts, int leftsLength, ht_slot **rightsOut, int *rightsLengthOut, ht_slot *tmp)
	{
		int left = 0; // Current index in aLeft
		int right = 0; // Current index in aRight
		int out = 0; // Current index in output buffer */
		int rightsLength = *rightsLengthOut;
		ht_slot *rights = *rightsOut;

		_assert(leftsLength > 0 && rightsLength > 0);
		while (right < rightsLength || left < leftsLength)
		{
			ht_slot logpage;
			if (left < leftsLength && (right >= rightsLength || content[lefts[left]] < content[rights[right]]))
				logpage = lefts[left++];
			else
				logpage = rights[right++];
			Pid dbpage = content[logpage];

			tmp[out++] = logpage;
			if (left < leftsLength && content[lefts[left]] == dbpage) left++;

			_assert(left >= leftsLength || content[lefts[left]] > dbpage);
			_assert(right >= rightsLength || content[rights[right]] > dbpage);
		}

		*rightsOut = lefts;
		*rightsLengthOut = out;
		_memcpy(lefts, tmp, sizeof(tmp[0]) * out);
	}

	static void walMergesort(const uint32 *content, ht_slot *buffer, ht_slot *list, int *listLengthRef)
	{
		struct Sublist
		{
			int ListLength; // Number of elements in aList
			ht_slot *List; // Pointer to sub-list content
		};

		const int listLength = *listLengthRef; // Size of input list
		int mergeLength = 0; // Number of elements in list aMerge
		ht_slot *merge = nullptr; // List to be merged
		int subIdx = 0; // Index into aSub array
		struct Sublist subs[13]; // Array of sub-lists

		_memset(subs, 0, sizeof(subs));
		_assert(listLength <= HASHTABLE_NPAGE && listLength > 0);
		_assert(HASHTABLE_NPAGE == (1 << (ArraySize(subs) - 1)));

		for (int listIdx = 0; listIdx < listLength; listIdx++) // Index into input list
		{
			mergeLength = 1;
			merge = &list[listIdx];
			for (subIdx = 0; listIdx & (1 << subIdx); subIdx++)
			{
				struct Sublist *p = &subs[subIdx];
				_assert(p->List && p->ListLength <= (1 << subIdx));
				_assert(p->List == &list[listIdx & ~((2 << subIdx) - 1)]);
				walMerge(content, p->List, p->ListLength, &merge, &mergeLength, buffer);
			}
			subs[subIdx].List = merge;
			subs[subIdx].ListLength = mergeLength;
		}

		for (subIdx++; subIdx < ArraySize(subs); subIdx++)
		{
			if (listLength & (1 << subIdx))
			{
				struct Sublist *p = &subs[subIdx];
				_assert(p->ListLength <= (1 << subIdx));
				_assert(p->List == &list[listLength & ~((2 << subIdx) - 1)]);
				walMerge(content, p->List, p->ListLength, &merge, &mergeLength, buffer);
			}
		}
		_assert(merge == list);
		*listLengthRef = mergeLength;

#ifdef _DEBUG
		for (int i = 1; i < *listLengthRef; i++)
			_assert(content[list[i]] > content[list[i - 1]]);
#endif
	}

	static void walIteratorFree(WalIterator *p)
	{
		sqlite3ScratchFree(p);
	}

	static RC walIteratorInit(Wal *wal, WalIterator **iteratorOut)
	{
		// This routine only runs while holding the checkpoint lock. And it only runs if there is actually content in the log (mxFrame>0).
		_assert(wal->CheckpointLock && wal->Header.MaxFrame > 0);
		uint32 lastFrame = wal->Header.MaxFrame; // Last frame in log

		// Allocate space for the WalIterator object.
		int segments = walFramePage(lastFrame) + 1; // Number of segments to merge
		int bytes = sizeof(WalIterator) + (segments - 1) * sizeof(struct WalSegment) + lastFrame * sizeof(ht_slot); // Number of bytes to allocate
		WalIterator *p = (WalIterator *)SysEx::ScratchAlloc(bytes); // Return value
		if (!p)
			return RC::NOMEM;
		_memset(p, 0, bytes);
		p->SegmentsLength = segments;

		// Allocate temporary space used by the merge-sort routine. This block of memory will be freed before this function returns.
		RC rc = RC::OK;
		ht_slot *tmp = (ht_slot *)SysEx::ScratchAlloc(sizeof(ht_slot) * (lastFrame > HASHTABLE_NPAGE ? HASHTABLE_NPAGE : lastFrame)); // Temp space used by merge-sort
		if (!tmp)
			rc = RC::NOMEM;

		for (int i = 0; rc == RC::OK && i < segments; i++)
		{
			volatile ht_slot *hash;
			volatile uint32 *ids;
			uint32 zero;
			rc = walHashGet(wal, i, &hash, &ids, &zero);
			if (rc == RC::OK)
			{
				ids++;
				int entrys; // Number of entries in this segment
				if ((i + 1) == segments)
					entrys = (int)(lastFrame - zero);
				else
					entrys = (int)((uint32 *)hash - (Pid *)ids);
				ht_slot *indexs = &((ht_slot *)&p->Segments[p->SegmentsLength])[zero]; // Sorted index for this segment
				zero++;

				for (int j = 0; j < entrys; j++)
					indexs[j] = (ht_slot)j;
				walMergesort((Pid *)ids, tmp, indexs, &entrys);
				p->Segments[i].Zero = zero;
				p->Segments[i].Entrys = entrys;
				p->Segments[i].Indexs = indexs;
				p->Segments[i].IDs = (Pid *)ids;
			}
		}
		SysEx::ScratchFree(tmp);

		if (rc != RC::OK)
			walIteratorFree(p);
		*iteratorOut = p;
		return rc;
	}

	static RC walBusyLock(Wal *wal, int (*busy)(void *), void *busyArg, int lockIdx, int n)
	{
		RC rc;
		do
		{
			rc = walLockExclusive(wal, lockIdx, n);
		} while (busy && rc == RC::BUSY && busy(busyArg));
		return rc;
	}

	static int walPagesize(Wal *wal)
	{
		return (wal->Header.SizePage & 0xfe00) + ((wal->Header.SizePage & 0x0001) << 16);
	}

	static int walCheckpoint(
		Wal *pWal,                      /* Wal connection */
		int eMode,                      /* One of PASSIVE, FULL or RESTART */
		int (*xBusyCall)(void*),        /* Function to call when busy */
		void *pBusyArg,                 /* Context argument for xBusyHandler */
		int sync_flags,                 /* Flags for OsSync() (or 0) */
		u8 *zBuf                        /* Temporary buffer to use */
		){
			int rc;                         /* Return code */
			int szPage;                     /* Database page-size */
			WalIterator *pIter = 0;         /* Wal iterator context */
			u32 iDbpage = 0;                /* Next database page to write */
			u32 iFrame = 0;                 /* Wal frame containing data for iDbpage */
			u32 mxSafeFrame;                /* Max frame that can be backfilled */
			u32 mxPage;                     /* Max database page to write */
			int i;                          /* Loop counter */
			volatile WalCkptInfo *pInfo;    /* The checkpoint status information */
			int (*xBusy)(void*) = 0;        /* Function to call when waiting for locks */

			szPage = walPagesize(pWal);
			testcase( szPage<=32768 );
			testcase( szPage>=65536 );
			pInfo = walCkptInfo(pWal);
			if( pInfo->nBackfill>=pWal->hdr.mxFrame ) return SQLITE_OK;

			/* Allocate the iterator */
			rc = walIteratorInit(pWal, &pIter);
			if( rc!=SQLITE_OK ){
				return rc;
			}
			assert( pIter );

			if( eMode!=SQLITE_CHECKPOINT_PASSIVE ) xBusy = xBusyCall;

			/* Compute in mxSafeFrame the index of the last frame of the WAL that is
			** safe to write into the database.  Frames beyond mxSafeFrame might
			** overwrite database pages that are in use by active readers and thus
			** cannot be backfilled from the WAL.
			*/
			mxSafeFrame = pWal->hdr.mxFrame;
			mxPage = pWal->hdr.nPage;
			for(i=1; i<WAL_NREADER; i++){
				u32 y = pInfo->aReadMark[i];
				if( mxSafeFrame>y ){
					assert( y<=pWal->hdr.mxFrame );
					rc = walBusyLock(pWal, xBusy, pBusyArg, WAL_READ_LOCK(i), 1);
					if( rc==SQLITE_OK ){
						pInfo->aReadMark[i] = (i==1 ? mxSafeFrame : READMARK_NOT_USED);
						walUnlockExclusive(pWal, WAL_READ_LOCK(i), 1);
					}else if( rc==SQLITE_BUSY ){
						mxSafeFrame = y;
						xBusy = 0;
					}else{
						goto walcheckpoint_out;
					}
				}
			}

			if( pInfo->nBackfill<mxSafeFrame
				&& (rc = walBusyLock(pWal, xBusy, pBusyArg, WAL_READ_LOCK(0), 1))==SQLITE_OK
				){
					i64 nSize;                    /* Current size of database file */
					u32 nBackfill = pInfo->nBackfill;

					/* Sync the WAL to disk */
					if( sync_flags ){
						rc = sqlite3OsSync(pWal->pWalFd, sync_flags);
					}

					/* If the database file may grow as a result of this checkpoint, hint
					** about the eventual size of the db file to the VFS layer. 
					*/
					if( rc==SQLITE_OK ){
						i64 nReq = ((i64)mxPage * szPage);
						rc = sqlite3OsFileSize(pWal->pDbFd, &nSize);
						if( rc==SQLITE_OK && nSize<nReq ){
							sqlite3OsFileControlHint(pWal->pDbFd, SQLITE_FCNTL_SIZE_HINT, &nReq);
						}
					}

					/* Iterate through the contents of the WAL, copying data to the db file. */
					while( rc==SQLITE_OK && 0==walIteratorNext(pIter, &iDbpage, &iFrame) ){
						i64 iOffset;
						assert( walFramePgno(pWal, iFrame)==iDbpage );
						if( iFrame<=nBackfill || iFrame>mxSafeFrame || iDbpage>mxPage ) continue;
						iOffset = walFrameOffset(iFrame, szPage) + WAL_FRAME_HDRSIZE;
						/* testcase( IS_BIG_INT(iOffset) ); // requires a 4GiB WAL file */
						rc = sqlite3OsRead(pWal->pWalFd, zBuf, szPage, iOffset);
						if( rc!=SQLITE_OK ) break;
						iOffset = (iDbpage-1)*(i64)szPage;
						testcase( IS_BIG_INT(iOffset) );
						rc = sqlite3OsWrite(pWal->pDbFd, zBuf, szPage, iOffset);
						if( rc!=SQLITE_OK ) break;
					}

					/* If work was actually accomplished... */
					if( rc==SQLITE_OK ){
						if( mxSafeFrame==walIndexHdr(pWal)->mxFrame ){
							i64 szDb = pWal->hdr.nPage*(i64)szPage;
							testcase( IS_BIG_INT(szDb) );
							rc = sqlite3OsTruncate(pWal->pDbFd, szDb);
							if( rc==SQLITE_OK && sync_flags ){
								rc = sqlite3OsSync(pWal->pDbFd, sync_flags);
							}
						}
						if( rc==SQLITE_OK ){
							pInfo->nBackfill = mxSafeFrame;
						}
					}

					/* Release the reader lock held while backfilling */
					walUnlockExclusive(pWal, WAL_READ_LOCK(0), 1);
			}

			if( rc==SQLITE_BUSY ){
				/* Reset the return code so as not to report a checkpoint failure
				** just because there are active readers.  */
				rc = SQLITE_OK;
			}

			/* If this is an SQLITE_CHECKPOINT_RESTART operation, and the entire wal
			** file has been copied into the database file, then block until all
			** readers have finished using the wal file. This ensures that the next
			** process to write to the database restarts the wal file.
			*/
			if( rc==SQLITE_OK && eMode!=SQLITE_CHECKPOINT_PASSIVE ){
				assert( pWal->writeLock );
				if( pInfo->nBackfill<pWal->hdr.mxFrame ){
					rc = SQLITE_BUSY;
				}else if( eMode==SQLITE_CHECKPOINT_RESTART ){
					assert( mxSafeFrame==pWal->hdr.mxFrame );
					rc = walBusyLock(pWal, xBusy, pBusyArg, WAL_READ_LOCK(1), WAL_NREADER-1);
					if( rc==SQLITE_OK ){
						walUnlockExclusive(pWal, WAL_READ_LOCK(1), WAL_NREADER-1);
					}
				}
			}

walcheckpoint_out:
			walIteratorFree(pIter);
			return rc;
	}

	static void walLimitSize(Wal *pWal, i64 nMax){
		i64 sz;
		int rx;
		sqlite3BeginBenignMalloc();
		rx = sqlite3OsFileSize(pWal->pWalFd, &sz);
		if( rx==SQLITE_OK && (sz > nMax ) ){
			rx = sqlite3OsTruncate(pWal->pWalFd, nMax);
		}
		sqlite3EndBenignMalloc();
		if( rx ){
			sqlite3_log(rx, "cannot limit WAL size: %s", pWal->zWalName);
		}
	}

	int sqlite3WalClose(
		Wal *pWal,                      /* Wal to close */
		int sync_flags,                 /* Flags to pass to OsSync() (or 0) */
		int nBuf,
		u8 *zBuf                        /* Buffer of at least nBuf bytes */
		){
			int rc = SQLITE_OK;
			if( pWal ){
				int isDelete = 0;             /* True to unlink wal and wal-index files */

				/* If an EXCLUSIVE lock can be obtained on the database file (using the
				** ordinary, rollback-mode locking methods, this guarantees that the
				** connection associated with this log file is the only connection to
				** the database. In this case checkpoint the database and unlink both
				** the wal and wal-index files.
				**
				** The EXCLUSIVE lock is not released before returning.
				*/
				rc = sqlite3OsLock(pWal->pDbFd, SQLITE_LOCK_EXCLUSIVE);
				if( rc==SQLITE_OK ){
					if( pWal->exclusiveMode==WAL_NORMAL_MODE ){
						pWal->exclusiveMode = WAL_EXCLUSIVE_MODE;
					}
					rc = sqlite3WalCheckpoint(
						pWal, SQLITE_CHECKPOINT_PASSIVE, 0, 0, sync_flags, nBuf, zBuf, 0, 0
						);
					if( rc==SQLITE_OK ){
						int bPersist = -1;
						sqlite3OsFileControlHint(
							pWal->pDbFd, SQLITE_FCNTL_PERSIST_WAL, &bPersist
							);
						if( bPersist!=1 ){
							/* Try to delete the WAL file if the checkpoint completed and
							** fsyned (rc==SQLITE_OK) and if we are not in persistent-wal
							** mode (!bPersist) */
							isDelete = 1;
						}else if( pWal->mxWalSize>=0 ){
							/* Try to truncate the WAL file to zero bytes if the checkpoint
							** completed and fsynced (rc==SQLITE_OK) and we are in persistent
							** WAL mode (bPersist) and if the PRAGMA journal_size_limit is a
							** non-negative value (pWal->mxWalSize>=0).  Note that we truncate
							** to zero bytes as truncating to the journal_size_limit might
							** leave a corrupt WAL file on disk. */
							walLimitSize(pWal, 0);
						}
					}
				}

				walIndexClose(pWal, isDelete);
				sqlite3OsClose(pWal->pWalFd);
				if( isDelete ){
					sqlite3BeginBenignMalloc();
					sqlite3OsDelete(pWal->pVfs, pWal->zWalName, 0);
					sqlite3EndBenignMalloc();
				}
				WALTRACE(("WAL%p: closed\n", pWal));
				sqlite3_free((void *)pWal->apWiData);
				sqlite3_free(pWal);
			}
			return rc;
	}

	static int walIndexTryHdr(Wal *pWal, int *pChanged){
		u32 aCksum[2];                  /* Checksum on the header content */
		WalIndexHdr h1, h2;             /* Two copies of the header content */
		WalIndexHdr volatile *aHdr;     /* Header in shared memory */

		/* The first page of the wal-index must be mapped at this point. */
		assert( pWal->nWiData>0 && pWal->apWiData[0] );

		/* Read the header. This might happen concurrently with a write to the
		** same area of shared memory on a different CPU in a SMP,
		** meaning it is possible that an inconsistent snapshot is read
		** from the file. If this happens, return non-zero.
		**
		** There are two copies of the header at the beginning of the wal-index.
		** When reading, read [0] first then [1].  Writes are in the reverse order.
		** Memory barriers are used to prevent the compiler or the hardware from
		** reordering the reads and writes.
		*/
		aHdr = walIndexHdr(pWal);
		memcpy(&h1, (void *)&aHdr[0], sizeof(h1));
		walShmBarrier(pWal);
		memcpy(&h2, (void *)&aHdr[1], sizeof(h2));

		if( memcmp(&h1, &h2, sizeof(h1))!=0 ){
			return 1;   /* Dirty read */
		}  
		if( h1.isInit==0 ){
			return 1;   /* Malformed header - probably all zeros */
		}
		walChecksumBytes(1, (u8*)&h1, sizeof(h1)-sizeof(h1.aCksum), 0, aCksum);
		if( aCksum[0]!=h1.aCksum[0] || aCksum[1]!=h1.aCksum[1] ){
			return 1;   /* Checksum does not match */
		}

		if( memcmp(&pWal->hdr, &h1, sizeof(WalIndexHdr)) ){
			*pChanged = 1;
			memcpy(&pWal->hdr, &h1, sizeof(WalIndexHdr));
			pWal->szPage = (pWal->hdr.szPage&0xfe00) + ((pWal->hdr.szPage&0x0001)<<16);
			testcase( pWal->szPage<=32768 );
			testcase( pWal->szPage>=65536 );
		}

		/* The header was successfully read. Return zero. */
		return 0;
	}

	static int walIndexReadHdr(Wal *pWal, int *pChanged){
		int rc;                         /* Return code */
		int badHdr;                     /* True if a header read failed */
		volatile u32 *page0;            /* Chunk of wal-index containing header */

		/* Ensure that page 0 of the wal-index (the page that contains the 
		** wal-index header) is mapped. Return early if an error occurs here.
		*/
		assert( pChanged );
		rc = walIndexPage(pWal, 0, &page0);
		if( rc!=SQLITE_OK ){
			return rc;
		};
		assert( page0 || pWal->writeLock==0 );

		/* If the first page of the wal-index has been mapped, try to read the
		** wal-index header immediately, without holding any lock. This usually
		** works, but may fail if the wal-index header is corrupt or currently 
		** being modified by another thread or process.
		*/
		badHdr = (page0 ? walIndexTryHdr(pWal, pChanged) : 1);

		/* If the first attempt failed, it might have been due to a race
		** with a writer.  So get a WRITE lock and try again.
		*/
		assert( badHdr==0 || pWal->writeLock==0 );
		if( badHdr ){
			if( pWal->readOnly & WAL_SHM_RDONLY ){
				if( SQLITE_OK==(rc = walLockShared(pWal, WAL_WRITE_LOCK)) ){
					walUnlockShared(pWal, WAL_WRITE_LOCK);
					rc = SQLITE_READONLY_RECOVERY;
				}
			}else if( SQLITE_OK==(rc = walLockExclusive(pWal, WAL_WRITE_LOCK, 1)) ){
				pWal->writeLock = 1;
				if( SQLITE_OK==(rc = walIndexPage(pWal, 0, &page0)) ){
					badHdr = walIndexTryHdr(pWal, pChanged);
					if( badHdr ){
						/* If the wal-index header is still malformed even while holding
						** a WRITE lock, it can only mean that the header is corrupted and
						** needs to be reconstructed.  So run recovery to do exactly that.
						*/
						rc = walIndexRecover(pWal);
						*pChanged = 1;
					}
				}
				pWal->writeLock = 0;
				walUnlockExclusive(pWal, WAL_WRITE_LOCK, 1);
			}
		}

		/* If the header is read successfully, check the version number to make
		** sure the wal-index was not constructed with some future format that
		** this version of SQLite cannot understand.
		*/
		if( badHdr==0 && pWal->hdr.iVersion!=WALINDEX_MAX_VERSION ){
			rc = SQLITE_CANTOPEN_BKPT;
		}

		return rc;
	}

#define WAL_RETRY  (-1)

	static int walTryBeginRead(Wal *pWal, int *pChanged, int useWal, int cnt){
		volatile WalCkptInfo *pInfo;    /* Checkpoint information in wal-index */
		u32 mxReadMark;                 /* Largest aReadMark[] value */
		int mxI;                        /* Index of largest aReadMark[] value */
		int i;                          /* Loop counter */
		int rc = SQLITE_OK;             /* Return code  */

		assert( pWal->readLock<0 );     /* Not currently locked */

		/* Take steps to avoid spinning forever if there is a protocol error.
		**
		** Circumstances that cause a RETRY should only last for the briefest
		** instances of time.  No I/O or other system calls are done while the
		** locks are held, so the locks should not be held for very long. But 
		** if we are unlucky, another process that is holding a lock might get
		** paged out or take a page-fault that is time-consuming to resolve, 
		** during the few nanoseconds that it is holding the lock.  In that case,
		** it might take longer than normal for the lock to free.
		**
		** After 5 RETRYs, we begin calling sqlite3OsSleep().  The first few
		** calls to sqlite3OsSleep() have a delay of 1 microsecond.  Really this
		** is more of a scheduler yield than an actual delay.  But on the 10th
		** an subsequent retries, the delays start becoming longer and longer, 
		** so that on the 100th (and last) RETRY we delay for 21 milliseconds.
		** The total delay time before giving up is less than 1 second.
		*/
		if( cnt>5 ){
			int nDelay = 1;                      /* Pause time in microseconds */
			if( cnt>100 ){
				VVA_ONLY( pWal->lockError = 1; )
					return SQLITE_PROTOCOL;
			}
			if( cnt>=10 ) nDelay = (cnt-9)*238;  /* Max delay 21ms. Total delay 996ms */
			sqlite3OsSleep(pWal->pVfs, nDelay);
		}

		if( !useWal ){
			rc = walIndexReadHdr(pWal, pChanged);
			if( rc==SQLITE_BUSY ){
				/* If there is not a recovery running in another thread or process
				** then convert BUSY errors to WAL_RETRY.  If recovery is known to
				** be running, convert BUSY to BUSY_RECOVERY.  There is a race here
				** which might cause WAL_RETRY to be returned even if BUSY_RECOVERY
				** would be technically correct.  But the race is benign since with
				** WAL_RETRY this routine will be called again and will probably be
				** right on the second iteration.
				*/
				if( pWal->apWiData[0]==0 ){
					/* This branch is taken when the xShmMap() method returns SQLITE_BUSY.
					** We assume this is a transient condition, so return WAL_RETRY. The
					** xShmMap() implementation used by the default unix and win32 VFS 
					** modules may return SQLITE_BUSY due to a race condition in the 
					** code that determines whether or not the shared-memory region 
					** must be zeroed before the requested page is returned.
					*/
					rc = WAL_RETRY;
				}else if( SQLITE_OK==(rc = walLockShared(pWal, WAL_RECOVER_LOCK)) ){
					walUnlockShared(pWal, WAL_RECOVER_LOCK);
					rc = WAL_RETRY;
				}else if( rc==SQLITE_BUSY ){
					rc = SQLITE_BUSY_RECOVERY;
				}
			}
			if( rc!=SQLITE_OK ){
				return rc;
			}
		}

		pInfo = walCkptInfo(pWal);
		if( !useWal && pInfo->nBackfill==pWal->hdr.mxFrame ){
			/* The WAL has been completely backfilled (or it is empty).
			** and can be safely ignored.
			*/
			rc = walLockShared(pWal, WAL_READ_LOCK(0));
			walShmBarrier(pWal);
			if( rc==SQLITE_OK ){
				if( memcmp((void *)walIndexHdr(pWal), &pWal->hdr, sizeof(WalIndexHdr)) ){
					/* It is not safe to allow the reader to continue here if frames
					** may have been appended to the log before READ_LOCK(0) was obtained.
					** When holding READ_LOCK(0), the reader ignores the entire log file,
					** which implies that the database file contains a trustworthy
					** snapshoT. Since holding READ_LOCK(0) prevents a checkpoint from
					** happening, this is usually correct.
					**
					** However, if frames have been appended to the log (or if the log 
					** is wrapped and written for that matter) before the READ_LOCK(0)
					** is obtained, that is not necessarily true. A checkpointer may
					** have started to backfill the appended frames but crashed before
					** it finished. Leaving a corrupt image in the database file.
					*/
					walUnlockShared(pWal, WAL_READ_LOCK(0));
					return WAL_RETRY;
				}
				pWal->readLock = 0;
				return SQLITE_OK;
			}else if( rc!=SQLITE_BUSY ){
				return rc;
			}
		}

		/* If we get this far, it means that the reader will want to use
		** the WAL to get at content from recent commits.  The job now is
		** to select one of the aReadMark[] entries that is closest to
		** but not exceeding pWal->hdr.mxFrame and lock that entry.
		*/
		mxReadMark = 0;
		mxI = 0;
		for(i=1; i<WAL_NREADER; i++){
			u32 thisMark = pInfo->aReadMark[i];
			if( mxReadMark<=thisMark && thisMark<=pWal->hdr.mxFrame ){
				assert( thisMark!=READMARK_NOT_USED );
				mxReadMark = thisMark;
				mxI = i;
			}
		}
		/* There was once an "if" here. The extra "{" is to preserve indentation. */
		{
			if( (pWal->readOnly & WAL_SHM_RDONLY)==0
				&& (mxReadMark<pWal->hdr.mxFrame || mxI==0)
				){
					for(i=1; i<WAL_NREADER; i++){
						rc = walLockExclusive(pWal, WAL_READ_LOCK(i), 1);
						if( rc==SQLITE_OK ){
							mxReadMark = pInfo->aReadMark[i] = pWal->hdr.mxFrame;
							mxI = i;
							walUnlockExclusive(pWal, WAL_READ_LOCK(i), 1);
							break;
						}else if( rc!=SQLITE_BUSY ){
							return rc;
						}
					}
			}
			if( mxI==0 ){
				assert( rc==SQLITE_BUSY || (pWal->readOnly & WAL_SHM_RDONLY)!=0 );
				return rc==SQLITE_BUSY ? WAL_RETRY : SQLITE_READONLY_CANTLOCK;
			}

			rc = walLockShared(pWal, WAL_READ_LOCK(mxI));
			if( rc ){
				return rc==SQLITE_BUSY ? WAL_RETRY : rc;
			}
			/* Now that the read-lock has been obtained, check that neither the
			** value in the aReadMark[] array or the contents of the wal-index
			** header have changed.
			**
			** It is necessary to check that the wal-index header did not change
			** between the time it was read and when the shared-lock was obtained
			** on WAL_READ_LOCK(mxI) was obtained to account for the possibility
			** that the log file may have been wrapped by a writer, or that frames
			** that occur later in the log than pWal->hdr.mxFrame may have been
			** copied into the database by a checkpointer. If either of these things
			** happened, then reading the database with the current value of
			** pWal->hdr.mxFrame risks reading a corrupted snapshot. So, retry
			** instead.
			**
			** This does not guarantee that the copy of the wal-index header is up to
			** date before proceeding. That would not be possible without somehow
			** blocking writers. It only guarantees that a dangerous checkpoint or 
			** log-wrap (either of which would require an exclusive lock on
			** WAL_READ_LOCK(mxI)) has not occurred since the snapshot was valid.
			*/
			walShmBarrier(pWal);
			if( pInfo->aReadMark[mxI]!=mxReadMark
				|| memcmp((void *)walIndexHdr(pWal), &pWal->hdr, sizeof(WalIndexHdr))
				){
					walUnlockShared(pWal, WAL_READ_LOCK(mxI));
					return WAL_RETRY;
			}else{
				assert( mxReadMark<=pWal->hdr.mxFrame );
				pWal->readLock = (i16)mxI;
			}
		}
		return rc;
	}

	int sqlite3WalBeginReadTransaction(Wal *pWal, int *pChanged){
		int rc;                         /* Return code */
		int cnt = 0;                    /* Number of TryBeginRead attempts */

		do{
			rc = walTryBeginRead(pWal, pChanged, 0, ++cnt);
		}while( rc==WAL_RETRY );
		testcase( (rc&0xff)==SQLITE_BUSY );
		testcase( (rc&0xff)==SQLITE_IOERR );
		testcase( rc==SQLITE_PROTOCOL );
		testcase( rc==SQLITE_OK );
		return rc;
	}

	void sqlite3WalEndReadTransaction(Wal *pWal){
		sqlite3WalEndWriteTransaction(pWal);
		if( pWal->readLock>=0 ){
			walUnlockShared(pWal, WAL_READ_LOCK(pWal->readLock));
			pWal->readLock = -1;
		}
	}

	int sqlite3WalRead(
		Wal *pWal,                      /* WAL handle */
		Pgno pgno,                      /* Database page number to read data for */
		int *pInWal,                    /* OUT: True if data is read from WAL */
		int nOut,                       /* Size of buffer pOut in bytes */
		u8 *pOut                        /* Buffer to write page data to */
		){
			u32 iRead = 0;                  /* If !=0, WAL frame to return data from */
			u32 iLast = pWal->hdr.mxFrame;  /* Last page in WAL for this reader */
			int iHash;                      /* Used to loop through N hash tables */

			/* This routine is only be called from within a read transaction. */
			assert( pWal->readLock>=0 || pWal->lockError );

			/* If the "last page" field of the wal-index header snapshot is 0, then
			** no data will be read from the wal under any circumstances. Return early
			** in this case as an optimization.  Likewise, if pWal->readLock==0, 
			** then the WAL is ignored by the reader so return early, as if the 
			** WAL were empty.
			*/
			if( iLast==0 || pWal->readLock==0 ){
				*pInWal = 0;
				return SQLITE_OK;
			}

			/* Search the hash table or tables for an entry matching page number
			** pgno. Each iteration of the following for() loop searches one
			** hash table (each hash table indexes up to HASHTABLE_NPAGE frames).
			**
			** This code might run concurrently to the code in walIndexAppend()
			** that adds entries to the wal-index (and possibly to this hash 
			** table). This means the value just read from the hash 
			** slot (aHash[iKey]) may have been added before or after the 
			** current read transaction was opened. Values added after the
			** read transaction was opened may have been written incorrectly -
			** i.e. these slots may contain garbage data. However, we assume
			** that any slots written before the current read transaction was
			** opened remain unmodified.
			**
			** For the reasons above, the if(...) condition featured in the inner
			** loop of the following block is more stringent that would be required 
			** if we had exclusive access to the hash-table:
			**
			**   (aPgno[iFrame]==pgno): 
			**     This condition filters out normal hash-table collisions.
			**
			**   (iFrame<=iLast): 
			**     This condition filters out entries that were added to the hash
			**     table after the current read-transaction had started.
			*/
			for(iHash=walFramePage(iLast); iHash>=0 && iRead==0; iHash--){
				volatile ht_slot *aHash;      /* Pointer to hash table */
				volatile u32 *aPgno;          /* Pointer to array of page numbers */
				u32 iZero;                    /* Frame number corresponding to aPgno[0] */
				int iKey;                     /* Hash slot index */
				int nCollide;                 /* Number of hash collisions remaining */
				int rc;                       /* Error code */

				rc = walHashGet(pWal, iHash, &aHash, &aPgno, &iZero);
				if( rc!=SQLITE_OK ){
					return rc;
				}
				nCollide = HASHTABLE_NSLOT;
				for(iKey=walHash(pgno); aHash[iKey]; iKey=walNextHash(iKey)){
					u32 iFrame = aHash[iKey] + iZero;
					if( iFrame<=iLast && aPgno[aHash[iKey]]==pgno ){
						/* assert( iFrame>iRead ); -- not true if there is corruption */
						iRead = iFrame;
					}
					if( (nCollide--)==0 ){
						return SQLITE_CORRUPT_BKPT;
					}
				}
			}

#ifdef SQLITE_ENABLE_EXPENSIVE_ASSERT
			/* If expensive assert() statements are available, do a linear search
			** of the wal-index file content. Make sure the results agree with the
			** result obtained using the hash indexes above.  */
			{
				u32 iRead2 = 0;
				u32 iTest;
				for(iTest=iLast; iTest>0; iTest--){
					if( walFramePgno(pWal, iTest)==pgno ){
						iRead2 = iTest;
						break;
					}
				}
				assert( iRead==iRead2 );
			}
#endif

			/* If iRead is non-zero, then it is the log frame number that contains the
			** required page. Read and return data from the log file.
			*/
			if( iRead ){
				int sz;
				i64 iOffset;
				sz = pWal->hdr.szPage;
				sz = (sz&0xfe00) + ((sz&0x0001)<<16);
				testcase( sz<=32768 );
				testcase( sz>=65536 );
				iOffset = walFrameOffset(iRead, sz) + WAL_FRAME_HDRSIZE;
				*pInWal = 1;
				/* testcase( IS_BIG_INT(iOffset) ); // requires a 4GiB WAL */
				return sqlite3OsRead(pWal->pWalFd, pOut, (nOut>sz ? sz : nOut), iOffset);
			}

			*pInWal = 0;
			return SQLITE_OK;
	}

	Pgno sqlite3WalDbsize(Wal *pWal){
		if( pWal && ALWAYS(pWal->readLock>=0) ){
			return pWal->hdr.nPage;
		}
		return 0;
	}

	int sqlite3WalBeginWriteTransaction(Wal *pWal){
		int rc;

		/* Cannot start a write transaction without first holding a read
		** transaction. */
		assert( pWal->readLock>=0 );

		if( pWal->readOnly ){
			return SQLITE_READONLY;
		}

		/* Only one writer allowed at a time.  Get the write lock.  Return
		** SQLITE_BUSY if unable.
		*/
		rc = walLockExclusive(pWal, WAL_WRITE_LOCK, 1);
		if( rc ){
			return rc;
		}
		pWal->writeLock = 1;

		/* If another connection has written to the database file since the
		** time the read transaction on this connection was started, then
		** the write is disallowed.
		*/
		if( memcmp(&pWal->hdr, (void *)walIndexHdr(pWal), sizeof(WalIndexHdr))!=0 ){
			walUnlockExclusive(pWal, WAL_WRITE_LOCK, 1);
			pWal->writeLock = 0;
			rc = SQLITE_BUSY;
		}

		return rc;
	}

	int sqlite3WalEndWriteTransaction(Wal *pWal){
		if( pWal->writeLock ){
			walUnlockExclusive(pWal, WAL_WRITE_LOCK, 1);
			pWal->writeLock = 0;
			pWal->truncateOnCommit = 0;
		}
		return SQLITE_OK;
	}

	int sqlite3WalUndo(Wal *pWal, int (*xUndo)(void *, Pgno), void *pUndoCtx){
		int rc = SQLITE_OK;
		if( ALWAYS(pWal->writeLock) ){
			Pgno iMax = pWal->hdr.mxFrame;
			Pgno iFrame;

			/* Restore the clients cache of the wal-index header to the state it
			** was in before the client began writing to the database. 
			*/
			memcpy(&pWal->hdr, (void *)walIndexHdr(pWal), sizeof(WalIndexHdr));

			for(iFrame=pWal->hdr.mxFrame+1; 
				ALWAYS(rc==SQLITE_OK) && iFrame<=iMax; 
				iFrame++
				){
					/* This call cannot fail. Unless the page for which the page number
					** is passed as the second argument is (a) in the cache and 
					** (b) has an outstanding reference, then xUndo is either a no-op
					** (if (a) is false) or simply expels the page from the cache (if (b)
					** is false).
					**
					** If the upper layer is doing a rollback, it is guaranteed that there
					** are no outstanding references to any page other than page 1. And
					** page 1 is never written to the log until the transaction is
					** committed. As a result, the call to xUndo may not fail.
					*/
					assert( walFramePgno(pWal, iFrame)!=1 );
					rc = xUndo(pUndoCtx, walFramePgno(pWal, iFrame));
			}
			if( iMax!=pWal->hdr.mxFrame ) walCleanupHash(pWal);
		}
		assert( rc==SQLITE_OK );
		return rc;
	}

	void sqlite3WalSavepoint(Wal *pWal, u32 *aWalData){
		assert( pWal->writeLock );
		aWalData[0] = pWal->hdr.mxFrame;
		aWalData[1] = pWal->hdr.aFrameCksum[0];
		aWalData[2] = pWal->hdr.aFrameCksum[1];
		aWalData[3] = pWal->nCkpt;
	}

	int sqlite3WalSavepointUndo(Wal *pWal, u32 *aWalData){
		int rc = SQLITE_OK;

		assert( pWal->writeLock );
		assert( aWalData[3]!=pWal->nCkpt || aWalData[0]<=pWal->hdr.mxFrame );

		if( aWalData[3]!=pWal->nCkpt ){
			/* This savepoint was opened immediately after the write-transaction
			** was started. Right after that, the writer decided to wrap around
			** to the start of the log. Update the savepoint values to match.
			*/
			aWalData[0] = 0;
			aWalData[3] = pWal->nCkpt;
		}

		if( aWalData[0]<pWal->hdr.mxFrame ){
			pWal->hdr.mxFrame = aWalData[0];
			pWal->hdr.aFrameCksum[0] = aWalData[1];
			pWal->hdr.aFrameCksum[1] = aWalData[2];
			walCleanupHash(pWal);
		}

		return rc;
	}

	static int walRestartLog(Wal *pWal){
		int rc = SQLITE_OK;
		int cnt;

		if( pWal->readLock==0 ){
			volatile WalCkptInfo *pInfo = walCkptInfo(pWal);
			assert( pInfo->nBackfill==pWal->hdr.mxFrame );
			if( pInfo->nBackfill>0 ){
				u32 salt1;
				sqlite3_randomness(4, &salt1);
				rc = walLockExclusive(pWal, WAL_READ_LOCK(1), WAL_NREADER-1);
				if( rc==SQLITE_OK ){
					/* If all readers are using WAL_READ_LOCK(0) (in other words if no
					** readers are currently using the WAL), then the transactions
					** frames will overwrite the start of the existing log. Update the
					** wal-index header to reflect this.
					**
					** In theory it would be Ok to update the cache of the header only
					** at this point. But updating the actual wal-index header is also
					** safe and means there is no special case for sqlite3WalUndo()
					** to handle if this transaction is rolled back.
					*/
					int i;                    /* Loop counter */
					u32 *aSalt = pWal->hdr.aSalt;       /* Big-endian salt values */

					pWal->nCkpt++;
					pWal->hdr.mxFrame = 0;
					sqlite3Put4byte((u8*)&aSalt[0], 1 + sqlite3Get4byte((u8*)&aSalt[0]));
					aSalt[1] = salt1;
					walIndexWriteHdr(pWal);
					pInfo->nBackfill = 0;
					pInfo->aReadMark[1] = 0;
					for(i=2; i<WAL_NREADER; i++) pInfo->aReadMark[i] = READMARK_NOT_USED;
					assert( pInfo->aReadMark[0]==0 );
					walUnlockExclusive(pWal, WAL_READ_LOCK(1), WAL_NREADER-1);
				}else if( rc!=SQLITE_BUSY ){
					return rc;
				}
			}
			walUnlockShared(pWal, WAL_READ_LOCK(0));
			pWal->readLock = -1;
			cnt = 0;
			do{
				int notUsed;
				rc = walTryBeginRead(pWal, &notUsed, 1, ++cnt);
			}while( rc==WAL_RETRY );
			assert( (rc&0xff)!=SQLITE_BUSY ); /* BUSY not possible when useWal==1 */
			testcase( (rc&0xff)==SQLITE_IOERR );
			testcase( rc==SQLITE_PROTOCOL );
			testcase( rc==SQLITE_OK );
		}
		return rc;
	}

	typedef struct WalWriter {
		Wal *pWal;                   /* The complete WAL information */
		sqlite3_file *pFd;           /* The WAL file to which we write */
		sqlite3_int64 iSyncPoint;    /* Fsync at this offset */
		int syncFlags;               /* Flags for the fsync */
		int szPage;                  /* Size of one page */
	} WalWriter;

	static int walWriteToLog(
		WalWriter *p,              /* WAL to write to */
		void *pContent,            /* Content to be written */
		int iAmt,                  /* Number of bytes to write */
		sqlite3_int64 iOffset      /* Start writing at this offset */
		){
			int rc;
			if( iOffset<p->iSyncPoint && iOffset+iAmt>=p->iSyncPoint ){
				int iFirstAmt = (int)(p->iSyncPoint - iOffset);
				rc = sqlite3OsWrite(p->pFd, pContent, iFirstAmt, iOffset);
				if( rc ) return rc;
				iOffset += iFirstAmt;
				iAmt -= iFirstAmt;
				pContent = (void*)(iFirstAmt + (char*)pContent);
				assert( p->syncFlags & (SQLITE_SYNC_NORMAL|SQLITE_SYNC_FULL) );
				rc = sqlite3OsSync(p->pFd, p->syncFlags);
				if( iAmt==0 || rc ) return rc;
			}
			rc = sqlite3OsWrite(p->pFd, pContent, iAmt, iOffset);
			return rc;
	}

	static int walWriteOneFrame(
		WalWriter *p,               /* Where to write the frame */
		PgHdr *pPage,               /* The page of the frame to be written */
		int nTruncate,              /* The commit flag.  Usually 0.  >0 for commit */
		sqlite3_int64 iOffset       /* Byte offset at which to write */
		){
			int rc;                         /* Result code from subfunctions */
			void *pData;                    /* Data actually written */
			u8 aFrame[WAL_FRAME_HDRSIZE];   /* Buffer to assemble frame-header in */
#if defined(SQLITE_HAS_CODEC)
			if( (pData = sqlite3PagerCodec(pPage))==0 ) return SQLITE_NOMEM;
#else
			pData = pPage->pData;
#endif
			walEncodeFrame(p->pWal, pPage->pgno, nTruncate, pData, aFrame);
			rc = walWriteToLog(p, aFrame, sizeof(aFrame), iOffset);
			if( rc ) return rc;
			/* Write the page data */
			rc = walWriteToLog(p, pData, p->szPage, iOffset+sizeof(aFrame));
			return rc;
	}


	int sqlite3WalFrames(
		Wal *pWal,                      /* Wal handle to write to */
		int szPage,                     /* Database page-size in bytes */
		PgHdr *pList,                   /* List of dirty pages to write */
		Pgno nTruncate,                 /* Database size after this commit */
		int isCommit,                   /* True if this is a commit */
		int sync_flags                  /* Flags to pass to OsSync() (or 0) */
		){
			int rc;                         /* Used to catch return codes */
			u32 iFrame;                     /* Next frame address */
			PgHdr *p;                       /* Iterator to run through pList with. */
			PgHdr *pLast = 0;               /* Last frame in list */
			int nExtra = 0;                 /* Number of extra copies of last page */
			int szFrame;                    /* The size of a single frame */
			i64 iOffset;                    /* Next byte to write in WAL file */
			WalWriter w;                    /* The writer */

			assert( pList );
			assert( pWal->writeLock );

			/* If this frame set completes a transaction, then nTruncate>0.  If
			** nTruncate==0 then this frame set does not complete the transaction. */
			assert( (isCommit!=0)==(nTruncate!=0) );

#if defined(SQLITE_TEST) && defined(SQLITE_DEBUG)
			{ int cnt; for(cnt=0, p=pList; p; p=p->pDirty, cnt++){}
			WALTRACE(("WAL%p: frame write begin. %d frames. mxFrame=%d. %s\n",
				pWal, cnt, pWal->hdr.mxFrame, isCommit ? "Commit" : "Spill"));
			}
#endif

			/* See if it is possible to write these frames into the start of the
			** log file, instead of appending to it at pWal->hdr.mxFrame.
			*/
			if( SQLITE_OK!=(rc = walRestartLog(pWal)) ){
				return rc;
			}

			/* If this is the first frame written into the log, write the WAL
			** header to the start of the WAL file. See comments at the top of
			** this source file for a description of the WAL header format.
			*/
			iFrame = pWal->hdr.mxFrame;
			if( iFrame==0 ){
				u8 aWalHdr[WAL_HDRSIZE];      /* Buffer to assemble wal-header in */
				u32 aCksum[2];                /* Checksum for wal-header */

				sqlite3Put4byte(&aWalHdr[0], (WAL_MAGIC | SQLITE_BIGENDIAN));
				sqlite3Put4byte(&aWalHdr[4], WAL_MAX_VERSION);
				sqlite3Put4byte(&aWalHdr[8], szPage);
				sqlite3Put4byte(&aWalHdr[12], pWal->nCkpt);
				if( pWal->nCkpt==0 ) sqlite3_randomness(8, pWal->hdr.aSalt);
				memcpy(&aWalHdr[16], pWal->hdr.aSalt, 8);
				walChecksumBytes(1, aWalHdr, WAL_HDRSIZE-2*4, 0, aCksum);
				sqlite3Put4byte(&aWalHdr[24], aCksum[0]);
				sqlite3Put4byte(&aWalHdr[28], aCksum[1]);

				pWal->szPage = szPage;
				pWal->hdr.bigEndCksum = SQLITE_BIGENDIAN;
				pWal->hdr.aFrameCksum[0] = aCksum[0];
				pWal->hdr.aFrameCksum[1] = aCksum[1];
				pWal->truncateOnCommit = 1;

				rc = sqlite3OsWrite(pWal->pWalFd, aWalHdr, sizeof(aWalHdr), 0);
				WALTRACE(("WAL%p: wal-header write %s\n", pWal, rc ? "failed" : "ok"));
				if( rc!=SQLITE_OK ){
					return rc;
				}

				/* Sync the header (unless SQLITE_IOCAP_SEQUENTIAL is true or unless
				** all syncing is turned off by PRAGMA synchronous=OFF).  Otherwise
				** an out-of-order write following a WAL restart could result in
				** database corruption.  See the ticket:
				**
				**     http://localhost:591/sqlite/info/ff5be73dee
				*/
				if( pWal->syncHeader && sync_flags ){
					rc = sqlite3OsSync(pWal->pWalFd, sync_flags & SQLITE_SYNC_MASK);
					if( rc ) return rc;
				}
			}
			assert( (int)pWal->szPage==szPage );

			/* Setup information needed to write frames into the WAL */
			w.pWal = pWal;
			w.pFd = pWal->pWalFd;
			w.iSyncPoint = 0;
			w.syncFlags = sync_flags;
			w.szPage = szPage;
			iOffset = walFrameOffset(iFrame+1, szPage);
			szFrame = szPage + WAL_FRAME_HDRSIZE;

			/* Write all frames into the log file exactly once */
			for(p=pList; p; p=p->pDirty){
				int nDbSize;   /* 0 normally.  Positive == commit flag */
				iFrame++;
				assert( iOffset==walFrameOffset(iFrame, szPage) );
				nDbSize = (isCommit && p->pDirty==0) ? nTruncate : 0;
				rc = walWriteOneFrame(&w, p, nDbSize, iOffset);
				if( rc ) return rc;
				pLast = p;
				iOffset += szFrame;
			}

			/* If this is the end of a transaction, then we might need to pad
			** the transaction and/or sync the WAL file.
			**
			** Padding and syncing only occur if this set of frames complete a
			** transaction and if PRAGMA synchronous=FULL.  If synchronous==NORMAL
			** or synchonous==OFF, then no padding or syncing are needed.
			**
			** If SQLITE_IOCAP_POWERSAFE_OVERWRITE is defined, then padding is not
			** needed and only the sync is done.  If padding is needed, then the
			** final frame is repeated (with its commit mark) until the next sector
			** boundary is crossed.  Only the part of the WAL prior to the last
			** sector boundary is synced; the part of the last frame that extends
			** past the sector boundary is written after the sync.
			*/
			if( isCommit && (sync_flags & WAL_SYNC_TRANSACTIONS)!=0 ){
				if( pWal->padToSectorBoundary ){
					int sectorSize = sqlite3SectorSize(pWal->pWalFd);
					w.iSyncPoint = ((iOffset+sectorSize-1)/sectorSize)*sectorSize;
					while( iOffset<w.iSyncPoint ){
						rc = walWriteOneFrame(&w, pLast, nTruncate, iOffset);
						if( rc ) return rc;
						iOffset += szFrame;
						nExtra++;
					}
				}else{
					rc = sqlite3OsSync(w.pFd, sync_flags & SQLITE_SYNC_MASK);
				}
			}

			/* If this frame set completes the first transaction in the WAL and
			** if PRAGMA journal_size_limit is set, then truncate the WAL to the
			** journal size limit, if possible.
			*/
			if( isCommit && pWal->truncateOnCommit && pWal->mxWalSize>=0 ){
				i64 sz = pWal->mxWalSize;
				if( walFrameOffset(iFrame+nExtra+1, szPage)>pWal->mxWalSize ){
					sz = walFrameOffset(iFrame+nExtra+1, szPage);
				}
				walLimitSize(pWal, sz);
				pWal->truncateOnCommit = 0;
			}

			/* Append data to the wal-index. It is not necessary to lock the 
			** wal-index to do this as the SQLITE_SHM_WRITE lock held on the wal-index
			** guarantees that there are no other writers, and no data that may
			** be in use by existing readers is being overwritten.
			*/
			iFrame = pWal->hdr.mxFrame;
			for(p=pList; p && rc==SQLITE_OK; p=p->pDirty){
				iFrame++;
				rc = walIndexAppend(pWal, iFrame, p->pgno);
			}
			while( rc==SQLITE_OK && nExtra>0 ){
				iFrame++;
				nExtra--;
				rc = walIndexAppend(pWal, iFrame, pLast->pgno);
			}

			if( rc==SQLITE_OK ){
				/* Update the private copy of the header. */
				pWal->hdr.szPage = (u16)((szPage&0xff00) | (szPage>>16));
				testcase( szPage<=32768 );
				testcase( szPage>=65536 );
				pWal->hdr.mxFrame = iFrame;
				if( isCommit ){
					pWal->hdr.iChange++;
					pWal->hdr.nPage = nTruncate;
				}
				/* If this is a commit, update the wal-index header too. */
				if( isCommit ){
					walIndexWriteHdr(pWal);
					pWal->iCallback = iFrame;
				}
			}

			WALTRACE(("WAL%p: frame write %s\n", pWal, rc ? "failed" : "ok"));
			return rc;
	}

	int sqlite3WalCheckpoint(Wal *wal, int mode, int (*busy)(void*), void *busyArg, int sync_flags, int bufferLength, u8 *bufuffer, int *logs, int *checkpoints)
	{
		int rc;                         /* Return code */
		int isChanged = 0;              /* True if a new wal-index header is loaded */
		int eMode2 = eMode;             /* Mode to pass to walCheckpoint() */

		_assert(wal->CheckpointLock == 0);
		_assert(wal->WriteLock == 0);

		if (wal->ReadOnly) return RC::READONLY;
		WALTRACE("WAL%p: checkpoint begins\n", wal);
		rc = walLockExclusive(wal, WAL_CKPT_LOCK, 1);
		if (rc) // Usually this is SQLITE_BUSY meaning that another thread or process is already running a checkpoint, or maybe a recovery.  But it might also be SQLITE_IOERR.
			return rc;
		wal->CheckpointLock = 1;

		// If this is a blocking-checkpoint, then obtain the write-lock as well to prevent any writers from running while the checkpoint is underway.
		// This has to be done before the call to walIndexReadHdr() below.
		//
		// If the writer lock cannot be obtained, then a passive checkpoint is run instead. Since the checkpointer is not holding the writer lock,
		// there is no point in blocking waiting for any readers. Assuming no other error occurs, this function will return SQLITE_BUSY to the caller.
		if (mode !=SQLITE_CHECKPOINT_PASSIVE)
		{
			rc = walBusyLock(wal, busy, busyArg, WAL_WRITE_LOCK, 1);
			if (rc == RC::OK)
				wal->WriteLock = 1;
			else if (rc == RC::BUSY)
			{
				mode2 = SQLITE_CHECKPOINT_PASSIVE;
				rc = RC::OK;
			}
		}

		// Read the wal-index header.
		if (rc == RC::OK)
			rc = walIndexReadHdr(wal, &isChanged);

		// Copy data from the log to the database file.
		if (rc == RC::OK)
		{
			if (wal->hdr.mxFrame && walPagesize(wal) != nBuf)
				rc = SQLITE_CORRUPT_BKPT;
			else
				rc = walCheckpoint(wal, mode2, busy, busyArg, sync_flags, bufffer);

			// If no error occurred, set the output variables.
			if (rc == RC::OK || rc == RC::BUSY)
			{
				if (logs) *logs = (int)wal->hdr.mxFrame;
				if (checkpoints) *checkpoints = (int)(walCkptInfo(wal)->nBackfill);
			}
		}

		if (isChanged)
		{
			// If a new wal-index header was loaded before the checkpoint was performed, then the pager-cache associated with pWal is now
			// out of date. So zero the cached wal-index header to ensure that next time the pager opens a snapshot on this database it knows that
			// the cache needs to be reset.
			memset(&wal->Hdr, 0, sizeof(WalIndexHdr));
		}

		// Release the locks.
		sqlite3WalEndWriteTransaction(wal);
		walUnlockExclusive(wal, WAL_CKPT_LOCK, 1);
		wal->CheckpointLock = 0;
		WALTRACE("WAL%p: checkpoint %s\n", wal, rc ? "failed" : "ok");
		return (rc == RC::OK && mode != mode2 ? RC::BUSY : rc);
	}

	int sqlite3WalCallback(Wal *wal)
	{
		uint32 ret = 0;
		if (wal)
		{
			ret = wal->Callback;
			wal->Callback = 0;
		}
		return (int)ret;
	}


	RC sqlite3WalExclusiveMode(Wal *wal, int op)
	{
		_assert(wal->WriteLock == 0);
		_assert(wal->ExclusiveMode != WAL_HEAPMEMORY::MODE || op == -1);

		// pWal->readLock is usually set, but might be -1 if there was a prior error while attempting to acquire are read-lock. This cannot 
		// happen if the connection is actually in exclusive mode (as no xShmLock locks are taken in this case). Nor should the pager attempt to
		// upgrade to exclusive-mode following such an error.
		_assert(wal->ReadLock >= 0 || wal->LockError);
		_assert(wal->ReadLock >= 0 || (op <= 0 && wal->ExclusiveMode == 0));

		RC rc;
		if (op == 0)
		{
			if (wal->ExclusiveMode)
			{
				wal->exclusiveMode = 0;
				if (walLockShared(wal, WAL_READ_LOCK(wal->ReadLock)) != RC::OK)
					wal->ExclusiveMode = 1;
				rc = wal->ExclusiveMode == 0;
			}
			else // Already in locking_mode=NORMAL
				rc = RC::OK;
		}
		else if (op > 0)
		{
			_assert(wal->ExclusiveMode == 0);
			_assert(wal->ReadLock >= 0);
			walUnlockShared(wal, WAL_READ_LOCK(wal->ReadLock));
			wal->ExclusiveMode = 1;
			rc = 1;
		}
		else
			rc = wal->ExclusiveMode == RC::OK;
		return rc;
	}

	int sqlite3WalHeapMemory(Wal *wal)
	{
		return (wal && wal->ExclusiveMode == WAL_HEAPMEMORY::MODE);
	}

#ifdef ENABLE_ZIPVFS
	int Wal::Framesize(Wal *wal)
	{
		_assert(wal == nullptr || wal->ReadLock >= 0);
		return (wal ? wal->SizePage : 0);
	}
#endif
