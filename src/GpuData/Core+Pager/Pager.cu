#include "hip/hip_runtime.h"
﻿// pager.c
#include "Core+Pager.cu.h"
using namespace Core;

namespace Core
{
	class Pager
	{
	public:
		enum PAGER : char
		{
			OPEN = 0,
			READER = 1,
			WRITER_LOCKED = 2,
			WRITER_CACHEMOD = 3,
			WRITER_DBMOD = 4,
			WRITER_FINISHED = 5,
			ERROR = 6,
		};

#define MAX_SECTOR_SIZE 0x10000

		typedef struct PagerSavepoint PagerSavepoint;
		struct PagerSavepoint
		{
			int64 Offset;             // Starting offset in main journal
			int64 HdrOffset;          // See above
			Bitvec *InSavepoint;      // Set of pages in this savepoint
			Pid Orig;                 // Original number of pages in file
			Pid SubRec;               // Index of first record in sub-journal
#ifndef OMIT_WAL
			uint32 WalData[WAL_SAVEPOINT_NDATA];  // WAL savepoint context
#endif
		};

		VFileSystem Vfs;			// OS functions to use for IO
		bool ExclusiveMode;			// Boolean. True if locking_mode==EXCLUSIVE
		uint8 JournalMode;			// One of the PAGER_JOURNALMODE_* values
		byte UseJournal;			// Use a rollback journal on this file
		byte NoSync;				// Do not sync the journal if true
		bool FullSync;				// Do extra syncs of the journal for robustness
		VFile::SYNC CkptSyncFlags;	// SYNC_NORMAL or SYNC_FULL for checkpoint
		VFile::SYNC WalSyncFlags;	// SYNC_NORMAL or SYNC_FULL for wal writes
		VFile::SYNC SyncFlags;		// SYNC_NORMAL or SYNC_FULL otherwise
		bool TempFile;				// zFilename is a temporary file
		bool ReadOnly;				// True for a read-only database
		bool MemoryDB;				// True to inhibit all file I/O
		// The following block contains those class members that change during routine opertion.  Class members not in this block are either fixed
		// when the pager is first created or else only change when there is a significant mode change (such as changing the page_size, locking_mode,
		// or the journal_mode).  From another view, these class members describe the "state" of the pager, while other class members describe the "configuration" of the pager.
		PAGER State;                // Pager state (OPEN, READER, WRITER_LOCKED..) */
		VFile::LOCK Lock;           // Current lock held on database file */
		bool changeCountDone;       // Set after incrementing the change-counter */
		bool SetMaster;             // True if a m-j name has been written to jrnl */
		byte DoNotSpill;            // Do not spill the cache when non-zero */
		byte DoNotSyncSpill;        // Do not do a spill that requires jrnl sync */
		byte SubjInMemory;          // True to use in-memory sub-journals */
		Pid DBSize;					// Number of pages in the database */
		Pid DBOrigSize;				// dbSize before the current transaction */
		Pid DBFileSize;				// Number of pages in the database file */
		Pid DBHintSize;				// Value passed to FCNTL_SIZE_HINT call */
		RC ErrorCode;               // One of several kinds of errors */
		int _nRec;                  // Pages journalled since last j-header written */
		uint32 CksumInit;           // Quasi-random value added to every checksum */
		uint32 SubRecords;          // Number of records written to sub-journal */
		Bitvec *InJournal;			// One bit for each page in the database file */
		VFile *File;				// File descriptor for database */
		VFile *JournalFile;			// File descriptor for main journal */
		VFile *SubJournalFile;		// File descriptor for sub-journal */
		int64 JournalOff;           // Current write offset in the journal file */
		int64 JournalHdr;           // Byte offset to previous journal header */
		IBackup *Backup;			// Pointer to list of ongoing backup processes */
		PagerSavepoint *Savepoint;	// Array of active savepoints */
		int nSavepoint;             // Number of elements in aSavepoint[] */
		char DBFileVersions[16];    // Changes whenever database file changes */
		// End of the routinely-changing class members
		u16 nExtra;                 // Add this many bytes to each in-memory page
		i16 nReserve;               // Number of unused bytes at end of each page
		u32 vfsFlags;               // Flags for sqlite3_vfs.xOpen()
		u32 sectorSize;             // Assumed sector size during rollback
		int pageSize;               // Number of bytes in a page
		Pgno mxPgno;                // Maximum allowed size of the database
		i64 journalSizeLimit;       // Size limit for persistent journal files
		char *zFilename;            // Name of the database file
		char *zJournal;             // Name of the journal file
		int (*xBusyHandler)(void*); // Function to call when busy
		void *pBusyHandlerArg;      // Context argument for xBusyHandler
		int aStat[3];               // Total cache hits, misses and writes
#ifdef TEST
		int nRead;                  // Database pages read
#endif
		void (*Reiniter)(IPage *);	// Call this routine when reloading pages
#ifdef HAS_CODEC
		void *(*Codec)(void*,void*,Pgno,int); // Routine for en/decoding data
		void (*CodecSizeChng)(void*,int,int); // Notify of page size changes
		void (*CodecFree)(void*);             // Destructor for the codec
		void *CodecArg;             // First argument to xCodec... methods
#endif
		char *TmpSpace;				// Pager.pageSize bytes of space for tmp use
		PCache *PCache;				// Pointer to page cache object
#ifndef OMIT_WAL
		Wal *Wal;					// Write-ahead log used by "journal_mode=wal"
		char *WalName;              // File name for write-ahead log
#endif
	};
}