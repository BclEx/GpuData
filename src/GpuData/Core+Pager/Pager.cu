#include "hip/hip_runtime.h"
﻿// pager.c
#include "Core+Pager.cu.h"
using namespace Core;

namespace Core
{
	static const unsigned char _journalMagic[] = { 0xd9, 0xd5, 0x05, 0xf9, 0x20, 0xa1, 0x63, 0xd7 };

	// sqliteLimit.h
#define MAX_PAGE_SIZE 65536
#define DEFAULT_PAGE_SIZE 1024
#define MAX_DEFAULT_PAGE_SIZE 8192
#define MAX_PAGE_COUNT 1073741823

	class Pager
	{
	public:
		enum PAGER : char
		{
			OPEN = 0,
			READER = 1,
			WRITER_LOCKED = 2,
			WRITER_CACHEMOD = 3,
			WRITER_DBMOD = 4,
			WRITER_FINISHED = 5,
			ERROR = 6,
		};

#define MAX_SECTOR_SIZE 0x10000

		struct PagerSavepoint
		{
			int64 Offset;             // Starting offset in main journal
			int64 HdrOffset;          // See above
			Bitvec *InSavepoint;      // Set of pages in this savepoint
			Pid Orig;                 // Original number of pages in file
			Pid SubRec;               // Index of first record in sub-journal
#ifndef OMIT_WAL
			uint32 WalData[WAL_SAVEPOINT_NDATA];  // WAL savepoint context
#endif
		};

		VFileSystem Vfs;			// OS functions to use for IO
		bool ExclusiveMode;			// Boolean. True if locking_mode==EXCLUSIVE
		uint8 JournalMode;			// One of the PAGER_JOURNALMODE_* values
		byte UseJournal;			// Use a rollback journal on this file
		byte NoSync;				// Do not sync the journal if true
		bool FullSync;				// Do extra syncs of the journal for robustness
		VFile::SYNC CheckpointSyncFlags;	// SYNC_NORMAL or SYNC_FULL for checkpoint
		VFile::SYNC WalSyncFlags;	// SYNC_NORMAL or SYNC_FULL for wal writes
		VFile::SYNC SyncFlags;		// SYNC_NORMAL or SYNC_FULL otherwise
		bool TempFile;				// zFilename is a temporary file
		bool ReadOnly;				// True for a read-only database
		bool MemoryDB;				// True to inhibit all file I/O
		// The following block contains those class members that change during routine opertion.  Class members not in this block are either fixed
		// when the pager is first created or else only change when there is a significant mode change (such as changing the page_size, locking_mode,
		// or the journal_mode).  From another view, these class members describe the "state" of the pager, while other class members describe the "configuration" of the pager.
		PAGER State;                // Pager state (OPEN, READER, WRITER_LOCKED..)
		VFile::LOCK Lock;           // Current lock held on database file
		bool ChangeCountDone;       // Set after incrementing the change-counter
		bool SetMaster;             // True if a m-j name has been written to jrnl
		byte DoNotSpill;            // Do not spill the cache when non-zero
		byte DoNotSyncSpill;        // Do not do a spill that requires jrnl sync
		byte SubjInMemory;          // True to use in-memory sub-journals
		Pid DBSize;					// Number of pages in the database
		Pid DBOrigSize;				// dbSize before the current transaction
		Pid DBFileSize;				// Number of pages in the database file
		Pid DBHintSize;				// Value passed to FCNTL_SIZE_HINT call
		RC ErrorCode;               // One of several kinds of errors
		int _nRec;                  // Pages journalled since last j-header written
		uint32 ChecksumInit;        // Quasi-random value added to every checksum
		uint32 SubRecords;          // Number of records written to sub-journal
		Bitvec *InJournal;			// One bit for each page in the database file
		VFile *File;				// File descriptor for database
		VFile *JournalFile;			// File descriptor for main journal
		VFile *SubJournalFile;		// File descriptor for sub-journal
		int64 JournalOffset;        // Current write offset in the journal file
		int64 JournalHeader;        // Byte offset to previous journal header
		IBackup *Backup;			// Pointer to list of ongoing backup processes
		//int _0;					// Number of elements in Savepoint[]
		PagerSavepoint *Savepoints;	// Array of active savepoints
		char DBFileVersions[16];    // Changes whenever database file changes
		// End of the routinely-changing class members
		uint16 ExtraBytes;          // Add this many bytes to each in-memory page
		int16 ReserveBytes;         // Number of unused bytes at end of each page
		uint32 VfsFlags;            // Flags for sqlite3_vfs.xOpen()
		uint32 SectorSize;          // Assumed sector size during rollback
		int PageSize;               // Number of bytes in a page
		Pid MaxPid;					// Maximum allowed size of the database
		int64 JournalSizeLimit;     // Size limit for persistent journal files
		char *Filename;            // Name of the database file
		char *Journal;             // Name of the journal file
		int (*BusyHandler)(void*);	// Function to call when busy
		void *BusyHandlerArg;		// Context argument for xBusyHandler
		int Stats[3];               // Total cache hits, misses and writes
#ifdef TEST
		int Reads;                  // Database pages read
#endif
		void (*Reiniter)(IPage *);	// Call this routine when reloading pages
#ifdef HAS_CODEC
		void *(*Codec)(void *,void *, Pid, int);	// Routine for en/decoding data
		void (*CodecSizeChange)(void *, int, int);	// Notify of page size changes
		void (*CodecFree)(void *);					// Destructor for the codec
		void *CodecArg;								// First argument to xCodec... methods
#endif
		unsigned char *TmpSpace;	// Pager.pageSize bytes of space for tmp use
		PCache *PCache;				// Pointer to page cache object
#ifndef OMIT_WAL
		Wal *Wal;					// Write-ahead log used by "journal_mode=wal"
		char *WalName;              // File name for write-ahead log
#endif

		enum STAT : char
		{
			HIT = 0,
			MISS = 1,
			WRITE = 2,
		};

#define JOURNAL_PG_SZ(pager) ((pager->PageSize) + 8)
#define JOURNAL_HDR_SZ(pager) (pager->SectorSize)

#define PAGER_MAX_PID 2147483647

#ifndef OMIT_WAL
		static int UseWal(Pager *pager) { return (pager->Wal != nullptr); }
#else
#define UseWal(x) 0
#define RollbackWal(x) 0
#define WalFrames(v,w,x,y) 0
#define OpenWalIfPresent(z) SQLITE_OK
#define BeginReadTransaction(z) SQLITE_OK
#endif

#pragma region Debug
#if _DEBUG 

		static int assert_pager_state(Pager *p)
		{
			// State must be valid.
			_assert(p->State == PAGER::OPEN ||
				p->State == PAGER::READER ||
				p->State == PAGER::WRITER_LOCKED ||
				p->State == PAGER::WRITER_CACHEMOD ||
				p->State == PAGER::WRITER_DBMOD ||
				p->State == PAGER::WRITER_FINISHED ||
				p->State == PAGER::ERROR);

			// Regardless of the current state, a temp-file connection always behaves as if it has an exclusive lock on the database file. It never updates
			// the change-counter field, so the changeCountDone flag is always set.
			_assert(p->TempFile == 0 || p->Lock == VFile::LOCK::EXCLUSIVE);
			_assert(p->TempFile == 0 || p->ChangeCountDone);

			// If the useJournal flag is clear, the journal-mode must be "OFF". And if the journal-mode is "OFF", the journal file must not be open.
			_assert(p->JournalMode == IPager::JOURNALMODE::OFF || p->UseJournal);
			_assert(p->JournalMode != IPager::JOURNALMODE::OFF || !p->JournalFile->Opened);

			// Check that MEMDB implies noSync. And an in-memory journal. Since this means an in-memory pager performs no IO at all, it cannot encounter 
			// either SQLITE_IOERR or SQLITE_FULL during rollback or while finalizing a journal file. (although the in-memory journal implementation may 
			// return SQLITE_IOERR_NOMEM while the journal file is being written). It is therefore not possible for an in-memory pager to enter the ERROR state.
			if (p->MemoryDB)
			{
				_assert(p->NoSync);
				_assert(p->JournalMode == IPager::JOURNALMODE::OFF || p->JournalMode == IPager::JOURNALMODE::JMEMORY);
				_assert(p->State != PAGER::ERROR && p->State != PAGER::OPEN);
				_assert(!UseWal(p));
			}

			// If changeCountDone is set, a RESERVED lock or greater must be held on the file.
			_assert(p->ChangeCountDone == 0 || p->Lock >= VFile::LOCK::RESERVED);
			_assert(p->Lock != VFile::LOCK::PENDING);

			switch (p->State)
			{
			case PAGER::OPEN:
				_assert(!p->MemoryDB);
				_assert(p->ErrorCode == RC::OK);
				_assert(PCache_RefCount(p->PCache) == 0 || p->TempFile);
				break;

			case PAGER::READER:
				_assert(p->ErrorCode == RC::OK);
				_assert(p->Lock != VFile::LOCK::UNKNOWN);
				_assert(p->Lock >= VFile::LOCK::SHARED);
				break;

			case PAGER::WRITER_LOCKED:
				_assert(p->Lock != VFile::LOCK::UNKNOWN);
				_assert(p->ErrorCode == RC::OK);
				if (!UseWal(p))
					_assert(p->Lock >= VFile::LOCK::RESERVED);
				_assert(p->DBSize == p->DBOrigSize);
				_assert(p->DBOrigSize == p->DBFileSize);
				_assert(p->DBOrigSize == p->DBHintSize);
				_assert(!p->SetMaster);
				break;

			case PAGER::WRITER_CACHEMOD:
				_assert(p->Lock != VFile::LOCK::UNKNOWN);
				_assert(p->ErrorCode == RC::OK);
				if (!UseWal(p))
				{
					// It is possible that if journal_mode=wal here that neither the journal file nor the WAL file are open. This happens during
					// a rollback transaction that switches from journal_mode=off to journal_mode=wal.
					_assert(p->Lock >= VFile::LOCK::RESERVED);
					_assert(p->JournalFile->Opened || p->JournalMode == IPager::JOURNALMODE::OFF || p->JournalMode == IPager::JOURNALMODE::WAL);
				}
				_assert(p->DBOrigSize == p->DBFileSize);
				_assert(p->DBOrigSize == p->DBHintSize);
				break;

			case PAGER::WRITER_DBMOD:
				_assert(p->Lock == VFile::LOCK::EXCLUSIVE);
				_assert(p->ErrorCode == RC::OK);
				_assert(!UseWal(p));
				_assert(p->Lock >= VFile::LOCK::EXCLUSIVE);
				_assert(p->JournalFile->Opened || p->JournalMode == IPager::JOURNALMODE::OFF || p->JournalMode == IPager::JOURNALMODE::WAL);
				_assert(p->DBOrigSize <= p->DBHintSize);
				break;

			case PAGER::WRITER_FINISHED:
				_assert(p->Lock == VFile::LOCK::EXCLUSIVE);
				_assert(p->ErrorCode == RC::OK);
				_assert(!UseWal(p));
				_assert(p->JournalFile->Opened || p->JournalMode == IPager::JOURNALMODE::OFF || p->JournalMode == IPager::JOURNALMODE::WAL);
				break;

			case PAGER::ERROR:
				// There must be at least one outstanding reference to the pager if in ERROR state. Otherwise the pager should have already dropped back to OPEN state.
				_assert(p->ErrorCode != RC::OK);
				_assert(PCache_RefCount(p->PCache) > 0);
				break;
			}

			return true;
		}

		static char *print_pager_state(Pager *p)
		{
			static char r[1024];
			_snprintf(r, 1024,
				"Filename:      %s\n"
				"State:         %s errCode=%d\n"
				"Lock:          %s\n"
				"Locking mode:  locking_mode=%s\n"
				"Journal mode:  journal_mode=%s\n"
				"Backing store: tempFile=%d memDb=%d useJournal=%d\n"
				"Journal:       journalOff=%lld journalHdr=%lld\n"
				"Size:          dbsize=%d dbOrigSize=%d dbFileSize=%d\n"
				, p->Filename
				, p->State == PAGER::OPEN ? "OPEN" :
				p->State == PAGER::READER ? "READER" :
				p->State == PAGER::WRITER_LOCKED ? "WRITER_LOCKED" :
				p->State == PAGER::WRITER_CACHEMOD ? "WRITER_CACHEMOD" :
				p->State == PAGER::WRITER_DBMOD ? "WRITER_DBMOD" :
				p->State == PAGER::WRITER_FINISHED ? "WRITER_FINISHED" :
				p->State == PAGER::ERROR ? "ERROR" : "?error?"
				, (int)p->ErrorCode
				, p->Lock == VFile::LOCK::NO ? "NO_LOCK" :
				p->Lock == VFile::LOCK::RESERVED ? "RESERVED" :
				p->Lock == VFile::LOCK::EXCLUSIVE ? "EXCLUSIVE" :
				p->Lock == VFile::LOCK::SHARED ? "SHARED" :
				p->Lock == VFile::LOCK::UNKNOWN ? "UNKNOWN" : "?error?"
				, p->ExclusiveMode ? "exclusive" : "normal"
				, p->JournalMode == IPager::JOURNALMODE::JMEMORY ? "memory" :
				p->JournalMode == IPager::JOURNALMODE::OFF ? "off" :
				p->JournalMode == IPager::JOURNALMODE::DELETE ? "delete" :
				p->JournalMode == IPager::JOURNALMODE::PERSIST ? "persist" :
				p->JournalMode == IPager::JOURNALMODE::TRUNCATE ? "truncate" :
				p->JournalMode == IPager::JOURNALMODE::WAL ? "wal" : "?error?"
				, (int)p->TempFile, (int)p->MemoryDB, (int)p->UseJournal
				, p->JournalOffset, p->JournalHeader
				, (int)p->DBSize, (int)p->DBOrigSize, (int)p->DBFileSize);
			return r;
		}

#endif
#pragma endregion

#pragma region Name1

		static bool subjRequiresPage(PgHdr *pg)
		{
			Pid id = pg->ID;
			Pager *pager = pg->Pager;
			for (int i = 0; i < __arrayLength(pager->Savepoints); i++)
			{
				PagerSavepoint *p = &pager->Savepoints[i];
				if (p->Orig >= id && !p->InSavepoint->Get(id))
					return true;
			}
			return false;
		}

		static bool pageInJournal(PgHdr *pg)
		{
			return pg->Pager->InJournal->Get(pg->ID);
		}

		static int pagerUnlockDb(Pager *pager, VFile::LOCK lock)
		{
			_assert(!pager->ExclusiveMode || pager->Lock == lock);
			_assert(lock == VFile::LOCK::NO || lock == VFile::LOCK::SHARED);
			_assert(lock != VFile::LOCK::NO || !UseWal(pager));
			int rc = RC::OK;
			if (pager->File->Opened)
			{
				_assert(pager->Lock >= lock);
				rc = pager->File->Unlock(lock);
				if (pager->Lock != VFile::LOCK::UNKNOWN)
					pager->Lock = lock;
				SysEx_IOTRACE("UNLOCK %p %d\n", pager, lock);
			}
			return rc;
		}

		static int pagerLockDb(Pager *pager, VFile::LOCK lock)
		{
			_assert(lock == VFile::LOCK::SHARED || lock == VFile::LOCK::RESERVED || lock == VFile::LOCK::EXCLUSIVE);
			int rc = RC::OK;
			if (pager->Lock < lock || pager->Lock == VFile::LOCK::UNKNOWN)
			{
				rc = pager->File->Lock(lock);
				if (rc == RC::OK && (pager->Lock != VFile::LOCK::UNKNOWN || lock == VFile::LOCK::EXCLUSIVE))
				{
					pager->Lock = lock;
					SysEx_IOTRACE("LOCK %p %d\n", pager, lock);
				}
			}
			return rc;
		}

#ifdef ENABLE_ATOMIC_WRITE
		static int jrnlBufferSize(Pager *pager)
		{
			_assert(!pager->MemoryDB);
			if (!pager->TempFile)
			{
				_assert(pager->File->Opened);
				int dc = pager->File->get_DeviceCharacteristics();
				int sectorSize = pager->SectorSize;
				int sizePage = pager->PageSize;
				_assert(IOCAP_ATOMIC512 == (512 >> 8));
				_assert(IOCAP_ATOMIC64K == (65536 >> 8));
				if ((dc & (IOCAP_ATOMIC | (sizePage >> 8)) || sectorSize > sizePage) == 0)
					return 0;
			}
			return JOURNAL_HDR_SZ(pager) + JOURNAL_PG_SZ(pager);
		}
#endif

#ifdef CHECK_PAGES
		static uint32 pager_datahash(int bytes, unsigned char *data)
		{
			uint32 hash = 0;
			for (int i = 0; i < bytes; i++)
				hash = (hash * 1039) + data[i];
			return hash;
		}
		static uint32 pager_pagehash(PgHdr *page) { return pager_datahash(page->Pager->PageSize, (unsigned char *)page->Data); }
		static void pager_set_pagehash(PgHdr *page) { page->PageHash = pager_pagehash(page); }
		//#define CHECK_PAGE(x) checkPage(x)
		static void checkPage(PgHdr *page)
		{
			Pager *pager = page->Pager;
			_assert(pager->State != PAGER::ERROR);
			_assert((page->Flags & PgHdr::PGHDR::DIRTY) || page->PageHash == pager_pagehash(page));
		}
#else
#define pager_datahash(X, Y) 0
#define pager_pagehash(X) 0
#define pager_set_pagehash(X)
#define CHECK_PAGE(x)
#endif

#pragma endregion

#pragma region Journal1

		static RC readMasterJournal(VFile *journalFile, char *master, uint32 masterLength)
		{
			uint32 nameLength;		// Length in bytes of master journal name
			int64 fileSize;			// Total size in bytes of journal file pJrnl
			uint32 checksum;		// MJ checksum value read from journal
			unsigned char magic[8]; // A buffer to hold the magic header
			master[0] = '\0';
			RC rc;
			if ((rc = journalFile->get_FileSize(fileSize)) != RC::OK ||
				fileSize < 16 ||
				(rc = journalFile->Read4(fileSize - 16, &nameLength)) != RC::OK ||
				nameLength >= masterLength ||
				(rc = journalFile->Read4(fileSize - 12, &checksum)) != RC::OK ||
				(rc = journalFile->Read(magic, 8, fileSize - 8)) != RC::OK ||
				_memcmp(magic, _journalMagic, 8) ||
				(rc = journalFile->Read(master, nameLength, fileSize - 16 - nameLength)) != RC::OK)
				return rc;
			// See if the checksum matches the master journal name
			for (uint32 u = 0; u < nameLength; u++)
				checksum -= master[u];
			if (checksum)
			{
				// If the checksum doesn't add up, then one or more of the disk sectors containing the master journal filename is corrupted. This means
				// definitely roll back, so just return SQLITE_OK and report a (nul) master-journal filename.
				nameLength = 0;
			}
			master[nameLength] = '\0';
			return RC::OK;
		}

		static int64 journalHdrOffset(Pager *pager)
		{
			int64 offset = 0;
			int64 c = pager->JournalOffset;
			if (c)
				offset = ((c-1) / JOURNAL_HDR_SZ(pager) + 1) * JOURNAL_HDR_SZ(pager);
			_assert(offset % JOURNAL_HDR_SZ(pager) == 0);
			_assert(offset >= c);
			_assert((offset - c) < JOURNAL_HDR_SZ(pager));
			return offset;
		}

		static RC zeroJournalHdr(Pager *pager, bool doTruncate)
		{
			_assert(pager->JournalFile->Opened);
			RC rc = RC::OK;
			if (pager->JournalOffset)
			{
				static const char zeroHeader[28] = { 0 };
				const int64 limit = pager->JournalSizeLimit; // Local cache of jsl
				SysEx_IOTRACE("JZEROHDR %p\n", pager);
				if (doTruncate || limit == 0)
					rc = pager->JournalFile->Truncate(0);
				else
					rc = pager->JournalFile->Write(zeroHeader, sizeof(zeroHeader), 0);
				if (rc == RC::OK && !pager->NoSync)
					rc = pager->JournalFile->Sync(VFile::SYNC::DATAONLY | pager->SyncFlags);
				// At this point the transaction is committed but the write lock is still held on the file. If there is a size limit configured for 
				// the persistent journal and the journal file currently consumes more space than that limit allows for, truncate it now. There is no need
				// to sync the file following this operation.
				if (rc == RC::OK && limit > 0)
				{
					int64 fileSize;
					rc = pager->JournalFile->get_FileSize(fileSize);
					if (rc == RC::OK && fileSize > limit)
						rc = pager->JournalFile->Truncate(limit);
				}
			}
			return rc;
		}

		static RC writeJournalHdr(Pager *pager)
		{
			_assert(pager->JournalFile->Opened); 
			unsigned char *header = pager->TmpSpace;		// Temporary space used to build header
			uint32 headerSize = (uint32)pager->PageSize;	// Size of buffer pointed to by zHeader
			if (headerSize > JOURNAL_HDR_SZ(pager))
				headerSize = JOURNAL_HDR_SZ(pager);

			// If there are active savepoints and any of them were created since the most recent journal header was written, update the PagerSavepoint.iHdrOffset fields now.
			for (int ii = 0; ii < __arrayLength(pager->Savepoints); ii++)
				if (pager->Savepoints[ii].HdrOffset == 0)
					pager->Savepoints[ii].HdrOffset = pager->JournalOffset;
			pager->JournalHeader = pager->JournalOffset = journalHdrOffset(pager);

			// Write the nRec Field - the number of page records that follow this journal header. Normally, zero is written to this value at this time.
			// After the records are added to the journal (and the journal synced, if in full-sync mode), the zero is overwritten with the true number
			// of records (see syncJournal()).
			//
			// A faster alternative is to write 0xFFFFFFFF to the nRec field. When reading the journal this value tells SQLite to assume that the
			// rest of the journal file contains valid page records. This assumption is dangerous, as if a failure occurred whilst writing to the journal
			// file it may contain some garbage data. There are two scenarios where this risk can be ignored:
			//   * When the pager is in no-sync mode. Corruption can follow a power failure in this case anyway.
			//   * When the SQLITE_IOCAP_SAFE_APPEND flag is set. This guarantees that garbage data is never appended to the journal file.
			_assert(pager->File->Opened || pager->NoSync);
			if (pager->NoSync || (pager->JournalMode == IPager::JOURNALMODE::JMEMORY) || (pager->File->get_DeviceCharacteristics() & VFile::IOCAP::SAFE_APPEND) != 0)
			{
				_memcpy(header, _journalMagic, sizeof(_journalMagic));
				ConvertEx::Put4(&header[sizeof(header)], 0xffffffff);
			}
			else
				_memset(header, 0, sizeof(_journalMagic) + 4);
			SysEx_MakeRandomness(sizeof(pager->ChecksumInit), &pager->ChecksumInit);
			ConvertEx::Put4(&header[sizeof(_journalMagic) + 4], pager->ChecksumInit);	// The random check-hash initializer
			ConvertEx::Put4(&header[sizeof(_journalMagic) + 8], pager->DBOrigSize);		// The initial database size
			ConvertEx::Put4(&header[sizeof(_journalMagic) + 12], pager->SectorSize);	// The assumed sector size for this process
			ConvertEx::Put4(&header[sizeof(_journalMagic) + 16], pager->PageSize);		// The page size
			// Initializing the tail of the buffer is not necessary.  Everything works find if the following memset() is omitted.  But initializing
			// the memory prevents valgrind from complaining, so we are willing to take the performance hit.
			_memset(&header[sizeof(_journalMagic) + 20], 0, headerSize - (sizeof(_journalMagic) + 20));

			// In theory, it is only necessary to write the 28 bytes that the journal header consumes to the journal file here. Then increment the 
			// Pager.journalOff variable by JOURNAL_HDR_SZ so that the next record is written to the following sector (leaving a gap in the file
			// that will be implicitly filled in by the OS).
			//
			// However it has been discovered that on some systems this pattern can be significantly slower than contiguously writing data to the file,
			// even if that means explicitly writing data to the block of (JOURNAL_HDR_SZ - 28) bytes that will not be used. So that is what is done. 
			//
			// The loop is required here in case the sector-size is larger than the database page size. Since the zHeader buffer is only Pager.pageSize
			// bytes in size, more than one call to sqlite3OsWrite() may be required to populate the entire journal header sector. 
			RC rc = RC::OK;
			for (uint32 headerWritten = 0; rc == RC::OK && headerWritten < JOURNAL_HDR_SZ(pager); headerWritten += headerSize)
			{
				SysEx_IOTRACE("JHDR %p %lld %d\n", pager, pager->JournalHeader, headerSize);
				rc = pager->JournalFile->Write(header, headerSize, pager->JournalOffset);
				_assert(pager->JournalHeader <= pager->JournalOffset);
				pager->JournalOffset += headerSize;
			}
			return rc;
		}

		static RC readJournalHdr(Pager *pager, int isHot, int64 journalSize, uint32 *recordsOut, uint32 *dbSizeOut)
		{
			_assert(pager->JournalFile->Opened);

			// Advance Pager.journalOff to the start of the next sector. If the journal file is too small for there to be a header stored at this
			// point, return SQLITE_DONE.
			pager->JournalOffset = journalHdrOffset(pager);
			if (pager->JournalOffset + JOURNAL_HDR_SZ(pager) > journalSize)
				return RC::DONE;
			int64 headerOffset = pager->JournalOffset;

			// Read in the first 8 bytes of the journal header. If they do not match the  magic string found at the start of each journal header, return
			// SQLITE_DONE. If an IO error occurs, return an error code. Otherwise, proceed.
			RC rc;
			unsigned char magic[8];
			if (isHot || headerOffset != pager->JournalHeader)
			{
				rc = pager->JournalFile->Read(magic, sizeof(magic), headerOffset);
				if (rc)
					return rc;
				if (_memcmp(magic, _journalMagic, sizeof(magic)) != 0)
					return RC::DONE;
			}

			// Read the first three 32-bit fields of the journal header: The nRec field, the checksum-initializer and the database size at the start
			// of the transaction. Return an error code if anything goes wrong.
			if ((rc = pager->JournalFile->Read4(headerOffset + 8, recordsOut)) != RC::OK ||
				(rc = pager->JournalFile->Read4(headerOffset + 12, &pager->ChecksumInit)) != RC::OK ||
				(rc = pager->JournalFile->Read4(headerOffset + 16, dbSizeOut)) != RC::OK)
				return rc;

			if (pager->JournalOffset == 0)
			{
				uint32 pageSize;	// Page-size field of journal header
				uint32 sectorSize;	// Sector-size field of journal header
				// Read the page-size and sector-size journal header fields.
				if ((rc = pager->JournalFile->Read4(headerOffset + 20, &sectorSize)) != RC::OK ||
					(rc = pager->JournalFile->Read4(headerOffset + 24, &pageSize)) != RC::OK)
					return rc;

				// Versions of SQLite prior to 3.5.8 set the page-size field of the journal header to zero. In this case, assume that the Pager.pageSize
				// variable is already set to the correct page size.
				if (pageSize == 0)
					pageSize = pager->PageSize;

				// Check that the values read from the page-size and sector-size fields are within range. To be 'in range', both values need to be a power
				// of two greater than or equal to 512 or 32, and not greater than their respective compile time maximum limits.
				if (pageSize < 512 || sectorSize < 32 ||
					pageSize > MAX_PAGE_SIZE || sectorSize > MAX_SECTOR_SIZE ||
					((pageSize - 1) & pageSize) != 0 || ((sectorSize - 1) & sectorSize) != 0)
					// If the either the page-size or sector-size in the journal-header is invalid, then the process that wrote the journal-header must have 
					// crashed before the header was synced. In this case stop reading the journal file here.
					return RC::DONE;

				// Update the page-size to match the value read from the journal. Use a testcase() macro to make sure that malloc failure within PagerSetPagesize() is tested.
				rc = SetPagesize(pager, &pageSize, -1);
				ASSERTCOVERAGE(rc != RC::OK);

				// Update the assumed sector-size to match the value used by the process that created this journal. If this journal was
				// created by a process other than this one, then this routine is being called from within pager_playback(). The local value
				// of Pager.sectorSize is restored at the end of that routine.
				pager->SectorSize = sectorSize;
			}

			pager->JournalOffset += JOURNAL_HDR_SZ(pager);
			return rc;
		}

		static int writeMasterJournal(Pager *pager, const char *master)
		{
			_assert(!pager->SetMaster);
			_assert(!UseWal(pager));

			if (!master ||
				pager->JournalMode == IPager::JOURNALMODE::JMEMORY ||
				pager->JournalMode == IPager::JOURNALMODE::OFF)
				return RC::OK;
			pager->SetMaster = true;
			_assert(pager->JournalFile->Opened);
			_assert(pager->JournalHeader <= pager->JournalOffset);

			// Calculate the length in bytes and the checksum of zMaster
			uint32 checksum = 0;	// Checksum of string zMaster
			int masterLength;		// Length of string zMaster
			for (masterLength = 0; master[masterLength]; masterLength++)
				checksum += master[masterLength];

			// If in full-sync mode, advance to the next disk sector before writing the master journal name. This is in case the previous page written to
			// the journal has already been synced.
			if (pager->FullSync)
				pager->JournalOffset = journalHdrOffset(pager);
			int64 headerOffset = pager->JournalOffset; // Offset of header in journal file

			// Write the master journal data to the end of the journal file. If an error occurs, return the error code to the caller.
			RC rc;
			if ((rc = pager->JournalFile->Write4(headerOffset, PAGER_MJ_PID(pager))) != RC::OK ||
				(rc = pager->JournalFile->Write(master, masterLength, headerOffset+4)) != RC::OK ||
				(rc = pager->JournalFile->Write4(headerOffset + 4 + masterLength, masterLength)) != RC::OK ||
				(rc = pager->JournalFile->Write4(headerOffset + 4 + masterLength + 4, checksum)) != RC::OK ||
				(rc = pager->JournalFile->Write(_journalMagic, 8, headerOffset + 4 + masterLength + 8)) != RC::OK)
				return rc;
			pager->JournalOffset += (masterLength + 20);

			// If the pager is in peristent-journal mode, then the physical journal-file may extend past the end of the master-journal name
			// and 8 bytes of magic data just written to the file. This is dangerous because the code to rollback a hot-journal file
			// will not be able to find the master-journal name to determine whether or not the journal is hot. 
			//
			// Easiest thing to do in this scenario is to truncate the journal file to the required size. 
			int64 journalSize;	// Size of journal file on disk
			if ((rc = pager->JournalFile->get_FileSize(journalSize)) == RC::OK && journalSize > pager->JournalOffset)
				rc = pager->JournalFile->Truncate(pager->JournalOffset);
			return rc;
		}

#pragma endregion

#pragma region Name2

		static PgHdr *pager_lookup(Pager *pager, Pid id)
		{
			PgHdr *p;
			// It is not possible for a call to PcacheFetch() with createFlag==0 to fail, since no attempt to allocate dynamic memory will be made.
			Pcache_Fetch(pager->PCache, id, 0, &p);
			return p;
		}

		// Discard the entire contents of the in-memory page-cache.
		static void pager_reset(Pager *pager)
		{
			pager->Backup->Restart();
			Pcache_Clear(pager->PCache);
		}

		// Free all structures in the Pager.aSavepoint[] array and set both Pager.aSavepoint and Pager.nSavepoint to zero. Close the sub-journal
		// if it is open and the pager is not in exclusive mode.
		static void releaseAllSavepoints(Pager *pager)
		{
			for (int ii = 0; ii < __arrayLength(pager->Savepoints); ii++)
				Bitvec::Destroy(pager->Savepoints[ii].InSavepoint);
			if (!pager->ExclusiveMode || pager->SubJournalFile->IsMemJournal)
				pager->SubJournalFile->Close();
			SysEx::Free(pager->Savepoints);
			pager->Savepoints = nullptr;
			pager->SubRecords = 0;
		}

		static RC addToSavepointBitvecs(Pager *pager, Pid id)
		{
			int rc = RC::OK;
			for (int ii = 0; ii < __arrayLength(pager->Savepoint); ii++)
			{
				PagerSavepoint *p = &pager->Savepoints[ii];
				if (id <= p->Orig)
				{
					rc |= p->InSavepoint->Set(id);
					ASSERTCOVERAGE(rc == RC::NOMEM);
					_assert(rc == RC::OK || rc == RC::NOMEM);
				}
			}
			return (RC)rc;
		}

		static void pager_unlock(Pager *pager)
		{
			_assert(pager->State==PAGER::READER ||
				pager->State==PAGER::OPEN ||
				pager->State==PAGER::ERROR);

			Bitvec::Destroy(pager->InJournal);
			pager->InJournal = nullptr;
			releaseAllSavepoints(pager);

			if (UseWal(pager))
			{
				_assert(!pager->JournalFile->Opened);
				WalEndReadTransaction(pager->Wal);
				pager->State = PAGER::OPEN;
			}
			else if (!pager->ExclusiveMode)
			{
				// If the operating system support deletion of open files, then close the journal file when dropping the database lock.  Otherwise
				// another connection with journal_mode=delete might delete the file out from under us.
				_assert((IPager::JOURNALMODE::JMEMORY & 5) != 1);
				_assert((IPager::JOURNALMODE::OFF & 5) != 1);
				_assert((IPager::JOURNALMODE::WAL & 5) != 1);
				_assert((IPager::JOURNALMODE::DELETE & 5) != 1);
				_assert((IPager::JOURNALMODE::TRUNCATE & 5) == 1);
				_assert((IPager::JOURNALMODE::PERSIST & 5) == 1);
				int dc = (pager->File->Opened ? pager->File->get_DeviceCharacteristics() : 0);
				if ((dc & IOCAP_UNDELETABLE_WHEN_OPEN) == 0 || (pager->JournalMode & 5) != 1)
					pager->JournalFile->Close();

				// If the pager is in the ERROR state and the call to unlock the database file fails, set the current lock to UNKNOWN_LOCK. See the comment
				// above the #define for UNKNOWN_LOCK for an explanation of why this is necessary.
				RC rc = pagerUnlockDb(pager, VFile::LOCK::NO);
				if (rc != RC::OK && pager->State == PAGER::ERROR)
					pager->Lock = VFile::LOCK::UNKNOWN;

				// The pager state may be changed from PAGER_ERROR to PAGER_OPEN here without clearing the error code. This is intentional - the error
				// code is cleared and the cache reset in the block below.
				_assert(pager->ErrorCode || pager->State != PAGER::ERROR);
				pager->ChangeCountDone = 0;
				pager->State = PAGER::OPEN;
			}

			// If Pager.errCode is set, the contents of the pager cache cannot be trusted. Now that there are no outstanding references to the pager,
			// it can safely move back to PAGER_OPEN state. This happens in both normal and exclusive-locking mode.
			if (pager->ErrorCode)
			{
				_assert(!pager->MemoryDB);
				pager_reset(pager);
				pager->ChangeCountDone = pager->TempFile;
				pager->State = PAGER::OPEN;
				pager->ErrorCode = RC::OK;
			}

			pager->JournalOffset = 0;
			pager->JournalHeader = 0;
			pager->SetMaster = false;
		}

		static RC pager_error(Pager *pager, RC rc)
		{
			RC rc2 = (rc & 0xff);
			_assert(rc == RC::OK || !pager->MemoryDB);
			_assert(pager->ErrorCode == RC::FULL ||
				pager->ErrorCode == RC::OK ||
				(pager->ErrorCode & 0xff) == RC::IOERR);
			if (rc2 == RC::FULL || rc2 == RC::IOERR)
			{
				pager->ErrorCode = rc;
				pager->State = PAGER::ERROR;
			}
			return rc;
		}

#pragma endregion

#pragma region Transaction1

		static int pager_truncate(Pager *pager, Pid pages);

		static int pager_end_transaction(Pager *pager, int hasMaster, bool commit)
		{
			// Do nothing if the pager does not have an open write transaction or at least a RESERVED lock. This function may be called when there
			// is no write-transaction active but a RESERVED or greater lock is held under two circumstances:
			//
			//   1. After a successful hot-journal rollback, it is called with eState==PAGER_NONE and eLock==EXCLUSIVE_LOCK.
			//
			//   2. If a connection with locking_mode=exclusive holding an EXCLUSIVE lock switches back to locking_mode=normal and then executes a
			//      read-transaction, this function is called with eState==PAGER_READER and eLock==EXCLUSIVE_LOCK when the read-transaction is closed.
			_assert(assert_pager_state(pager));
			_assert(pager->State != PAGER::ERROR);
			if (pager->State < PAGER::WRITER_LOCKED && pager->Lock < VFile::LOCK::RESERVED)
				return RC::OK;

			releaseAllSavepoints(pager);
			_assert(pager->JournalFile->Opened || pager->InJournal == nullptr);
			RC rc = RC::OK;
			if (pager->JournalFile->Opened)
			{
				_assert(!UseWal(pager));

				// Finalize the journal file.
				if (pager->JournalFile->IsMemJournal)
				{
					_assert(pager->JournalMode == IPager::JOURNALMODE::JMEMORY);
					pager->JournalFile->Close();
				}
				else if (pager->JournalMode == IPager::JOURNALMODE::TRUNCATE)
				{
					rc = (pager->JournalOffset == 0 ? RC::OK : pager->JournalFile->Truncate(0));
					pager->JournalOffset = 0;
				}
				else if (pager->JournalMode == IPager::JOURNALMODE::PERSIST || (pager->ExclusiveMode && pager->JournalMode != IPager::JOURNALMODE::WAL))
				{
					rc = zeroJournalHdr(pager, hasMaster);
					pager->JournalOffset = 0;
				}
				else
				{
					// This branch may be executed with Pager.journalMode==MEMORY if a hot-journal was just rolled back. In this case the journal
					// file should be closed and deleted. If this connection writes to the database file, it will do so using an in-memory journal. 
					bool delete_ = (!pager->TempFile && pager->JournalFile->JournalExists());
					_assert(pager->JournalMode == IPager::JOURNALMODE::DELETE ||
						pager->JournalMode == IPager::JOURNALMODE::JMEMORY ||
						pager->JournalMode == IPager::JOURNALMODE::WAL);
					pager->JournalFile->Close();
					if (delete_)
						pager->Vfs->Delete(pager->Journal, 0);
				}
			}

#ifdef CHECK_PAGES
			Pcache_IterateDirty(pager->PCache, pager_set_pagehash);
			if (pager->DBSize == 0 && Pcache_RefCount(pager->PCache) > 0)
			{
				PgHdr *p = pager_lookup(pager, 1);
				if (p)
				{
					p->PageHash = 0;
					Pager_Unref(p);
				}
			}
#endif

			Bitvec::Destroy(pager->InJournal); pager->InJournal = nullptr;
			pager->Records = 0;
			PCache_CleanAll(pager->PCache);
			PCache_Truncate(pager->PCache, pager->DBSize);

			RC rc2 = RC::OK;
			if (UseWal(pager))
			{
				// Drop the WAL write-lock, if any. Also, if the connection was in locking_mode=exclusive mode but is no longer, drop the EXCLUSIVE 
				// lock held on the database file.
				rc2 = WalEndWriteTransaction(pager->Wal);
				_assert(rc2 == RC::OK);
			}
			else if (rc == RC::OK && commit && pager->DBFileSize > pager->DBSize)
			{
				// This branch is taken when committing a transaction in rollback-journal mode if the database file on disk is larger than the database image.
				// At this point the journal has been finalized and the transaction successfully committed, but the EXCLUSIVE lock is still held on the
				// file. So it is safe to truncate the database file to its minimum required size.
				_assert(pager->Lock == VFile::LOCK::EXCLUSIVE);
				rc = pager_truncate(pager, pager->DBSize);
			}

			if (!pager->ExclusiveMode && (!UseWal(pager) || WalExclusiveMode(pager->Wal, 0)))
			{
				rc2 = pagerUnlockDb(pager, VFile::LOCK::SHARED);
				pager->ChangeCountDone = 0;
			}
			pager->State = PAGER::READER;
			pager->SetMaster = false;

			return (rc == RC::OK ? rc2 : rc);
		}

		static void pagerUnlockAndRollback(Pager *pager)
		{
			if (pager->State != PAGER::ERROR && pager->State != PAGER::OPEN)
			{
				_assert(assert_pager_state(pager));
				if (pager->State >= PAGER::WRITER_LOCKED)
				{
					SysEx::BeginBenignMalloc();
					sqlite3PagerRollback(pager);
					SysEx::EndBenignMalloc();
				}
				else if (!pager->ExclusiveMode)
				{
					_assert(pager->State == PAGER::READER);
					pager_end_transaction(pager, 0, 0);
				}
			}
			pager_unlock(pager);
		}

		static uint32 pager_cksum(Pager *pager, const uint8 *data)
		{
			uint32 checksum = pager->ChecksumInit;
			int i = pager->PageSize - 200;
			while (i > 0)
			{
				checksum += data[i];
				i -= 200;
			}
			return checksum;
		}

#ifdef HAS_CODEC
		static void pagerReportSize(Pager *pager)
		{
			if (pager->CodecSizeChange)
				pager->CodecSizeChange(pager->Codec, pager->PageSize, (int)pager->Reserve);
		}
#else
#define pagerReportSize(X)
#endif

		static int pager_playback_one_page(Pager *pager, int64 *offset, Bitvec *done, int isMainJournal, int isSavepoint)
		{
			_assert((isMainJournal & ~1) == 0);    // isMainJrnl is 0 or 1
			_assert((isSavepoint & ~1) == 0);     // isSavepnt is 0 or 1
			_assert(isMainJrnl || done);		// pDone always used on sub-journals
			_assert(isSavepnt || done == 0);	// pDone never used on non-savepoint

			char *data = pager->TmpSpace; // Temporary storage for the page
			_assert(data); // Temp storage must have already been allocated
			_assert(!UseWal(pager) || (!isMainJournal && isSavepoint));

			// Either the state is greater than PAGER_WRITER_CACHEMOD (a transaction or savepoint rollback done at the request of the caller) or this is
			// a hot-journal rollback. If it is a hot-journal rollback, the pager is in state OPEN and holds an EXCLUSIVE lock. Hot-journal rollback
			// only reads from the main journal, not the sub-journal.
			_assert(pager->State >= PAGER::WRITER_CACHEMOD || (pager->State == PAGER::OPEN && pager->Lock == VFile::LOCK::EXCLUSIVE));
			_assert(pager->State >= PAGER::WRITER_CACHEMOD || isMainJournal);

			// Read the page number and page data from the journal or sub-journal file. Return an error code to the caller if an IO error occurs.
			VFile *journalFile = (isMainJournal ? pager->JournalFile : pager->SubJournalFile); // The file descriptor for the journal file
			Pid id; // The page number of a page in journal
			RC rc = journalFile->Read4(*offset, &id);
			if (rc != RC::OK) return rc;
			rc = journalFile->Read((uint8 *)data, pager->PageSize, (*offset) + 4);
			if (rc != RC::OK) return rc;
			*offset += pager->PageSize + 4 + isMainJournal * 4;

			// Sanity checking on the page.  This is more important that I originally thought.  If a power failure occurs while the journal is being written,
			// it could cause invalid data to be written into the journal.  We need to detect this invalid data (with high probability) and ignore it.
			if (id == 0 || id == PAGER_MJ_PGNO(pager))
			{
				_assert(!isSavepoint);
				return RC::DONE;
			}
			if (id > (Pid)pager->DBSize || done->Get(id))
				return RC::OK;
			if (isMainJournal)
			{
				uint32 checksum; // Checksum used for sanity checking
				rc = journalFile->Read4((*offset) - 4, &checksum);
				if (rc) return rc;
				if (!isSavepnt && pager_cksum(pager, (uint8*)data) != checksum)
					return RC::DONE;
			}

			// If this page has already been played by before during the current rollback, then don't bother to play it back again.
			if (done && (rc = done->Set(id)) != RC::OK)
				return rc;

			// When playing back page 1, restore the nReserve setting
			if (id == 1 && pager->Reserves != ((uint8 *)data)[20])
			{
				pager->Reserves = ((uint8 *)data)[20];
				pagerReportSize(pager);
			}

			// If the pager is in CACHEMOD state, then there must be a copy of this page in the pager cache. In this case just update the pager cache,
			// not the database file. The page is left marked dirty in this case.
			//
			// An exception to the above rule: If the database is in no-sync mode and a page is moved during an incremental vacuum then the page may
			// not be in the pager cache. Later: if a malloc() or IO error occurs during a Movepage() call, then the page may not be in the cache
			// either. So the condition described in the above paragraph is not assert()able.
			//
			// If in WRITER_DBMOD, WRITER_FINISHED or OPEN state, then we update the pager cache if it exists and the main file. The page is then marked 
			// not dirty. Since this code is only executed in PAGER_OPEN state for a hot-journal rollback, it is guaranteed that the page-cache is empty
			// if the pager is in OPEN state.
			//
			// Ticket #1171:  The statement journal might contain page content that is different from the page content at the start of the transaction.
			// This occurs when a page is changed prior to the start of a statement then changed again within the statement.  When rolling back such a
			// statement we must not write to the original database unless we know for certain that original page contents are synced into the main rollback
			// journal.  Otherwise, a power loss might leave modified data in the database file without an entry in the rollback journal that can
			// restore the database to its original form.  Two conditions must be met before writing to the database files. (1) the database must be
			// locked.  (2) we know that the original page content is fully synced in the main journal either because the page is not in cache or else
			// the page is marked as needSync==0.
			//
			// 2008-04-14:  When attempting to vacuum a corrupt database file, it is possible to fail a statement on a database that does not yet exist.
			// Do not attempt to write if database file has never been opened.
			PgHdr *pg = (UseWal(pager) ? nullptr : pager_lookup(pager, id)); // An existing page in the cache
			_assert(pg || !pager->MemoryDB);
			_assert(pager->State != PAGER::OPEN || pg == 0);
			PAGERTRACE("PLAYBACK %d page %d hash(%08x) %s\n", PAGERID(pager), id, pager_datahash(pager->PageSize, (uint8 *)data), (isMainJournal ? "main-journal" : "sub-journal"));
			bool isSynced; // True if journal page is synced
			if (isMainJournal)
				isSynced = pager->NoSync || (*offset <= pager->JournalHeader);
			else
				isSynced = (pg == nullptr || (pg->Flags & PgHdr::PGHDR::NEED_SYNC) == 0);
			if (pager->File->Opened && (pager->State >= PAGER::WRITER_DBMOD || pager->State == PAGER::OPEN) && isSynced)
			{
				int64 offset = (id - 1) * (int64)pager->PageSize;
				ASSERTCOVERAGE(!isSavepoint && pg != nullptr && (pg->Flags & PgHdr::PGHDR::NEED_SYNC) != 0);
				_assert(!UseWal(pager));
				rc = pager->File->Write((uint8 *)data, pager->PageSize, offset);
				if (id > pager->DBFileSize)
					pager->DBFileSize = id;
				if (pager->Backup)
				{
					CODEC1(pager, data, id, 3, rc = RC::NOMEM);
					pager->Backup->Update(id, (uint8 *)data);
					CODEC2(pager, data, id, 7, rc = RC::NOMEM, data);
				}
			}
			else if (!isMainJournal && pg == nullptr)
			{
				// If this is a rollback of a savepoint and data was not written to the database and the page is not in-memory, there is a potential
				// problem. When the page is next fetched by the b-tree layer, it will be read from the database file, which may or may not be 
				// current. 
				//
				// There are a couple of different ways this can happen. All are quite obscure. When running in synchronous mode, this can only happen 
				// if the page is on the free-list at the start of the transaction, then populated, then moved using sqlite3PagerMovepage().
				//
				// The solution is to add an in-memory page to the cache containing the data just read from the sub-journal. Mark the page as dirty 
				// and if the pager requires a journal-sync, then mark the page as requiring a journal-sync before it is written.
				_assert(isSavepoint);
				_assert(pager->DoNotSpill == 0);
				pager->DoNotSpill++;
				rc = sqlite3PagerAcquire(pager, id, &pg, 1);
				_assert(pager->DoNotSpill == 1);
				pager->DoNotSpill--;
				if (rc != RC::OK) return rc;
				pg->Flags &= ~PgHdr::PGHDR::NEED_READ;
				Pcache_MakeDirty(pg);
			}
			if (pg)
			{
				// No page should ever be explicitly rolled back that is in use, except for page 1 which is held in use in order to keep the lock on the
				// database active. However such a page may be rolled back as a result of an internal error resulting in an automatic call to
				// sqlite3PagerRollback().
				void *pageData = pg->Data;
				_memcpy(pageData, (uint8 *)data, pager->PageSize);
				pager->Reiniter(pg);
				if (isMainJournal && (!isSavepoint || *offset <= pager->JournalHeader))
				{
					// If the contents of this page were just restored from the main journal file, then its content must be as they were when the 
					// transaction was first opened. In this case we can mark the page as clean, since there will be no need to write it out to the
					// database.
					//
					// There is one exception to this rule. If the page is being rolled back as part of a savepoint (or statement) rollback from an 
					// unsynced portion of the main journal file, then it is not safe to mark the page as clean. This is because marking the page as
					// clean will clear the PGHDR_NEED_SYNC flag. Since the page is already in the journal file (recorded in Pager.pInJournal) and
					// the PGHDR_NEED_SYNC flag is cleared, if the page is written to again within this transaction, it will be marked as dirty but
					// the PGHDR_NEED_SYNC flag will not be set. It could then potentially be written out into the database file before its journal file
					// segment is synced. If a crash occurs during or following this, database corruption may ensue.
					_assert(!UseWal(pager));
					Pcache_MakeClean(pg);
				}
				pager_set_pagehash(pg);

				// If this was page 1, then restore the value of Pager.dbFileVers. Do this before any decoding.
				if (id == 1)
					_memcpy(&pager->DBFileVers, &((uint8 *)pageData)[24], sizeof(pager->DBFileVers));

				// Decode the page just read from disk
				CODEC1(pager, pageData, pg->ID, 3, rc = RC::NOMEM);
				Pcache_Release(pg);
			}
			return rc;
		}

		static int pager_delmaster(Pager *pager, const char *master)
		{
			int rc;
			VFileSystem *vfs = pager->Vfs;

			// Allocate space for both the pJournal and pMaster file descriptors. If successful, open the master journal file for reading.         
			VFile *masterFile = (VFile *)SysEx::Alloc(vfs->SizeOsFile * 2, true); // Malloc'd master-journal file descriptor
			VFile *journalFile = (VFile *)(((uint8 *)masterFile) + vfs->SizeOsFile); // Malloc'd child-journal file descriptor
			if (!masterFile)
				rc = RC::NOMEM;
			else
			{
				const int flags = (SQLITE_OPEN_READONLY | SQLITE_OPEN_MASTER_JOURNAL);
				rc = vfs->Open(master, masterFile, flags, 0);
			}
			if (rc != RC::OK) goto delmaster_out;

			// Load the entire master journal file into space obtained from sqlite3_malloc() and pointed to by zMasterJournal.   Also obtain
			// sufficient space (in zMasterPtr) to hold the names of master journal files extracted from regular rollback-journals.
			int64 masterJournalSize; // Size of master journal file
			rc = masterFile->get_FileSize(masterJournalSize);
			if (rc != RC::OK) goto delmaster_out;
			int masterPtrSize = vfs->MaxPathName + 1; // Amount of space allocated to zMasterPtr[]
			char *masterJournal = (char *)SysEx::Alloc((int)masterJournalSize + masterPtrSize + 1); // Contents of master journal file
			if (!masterJournal)
			{
				rc = RC::NOMEM;
				goto delmaster_out;
			}
			char *masterPtr = &masterJournal[masterJournalSize + 1]; // Space to hold MJ filename from a journal file
			rc = masterFile->Read(masterJournal, (int)masterJournalSize, 0);
			if (rc != RC::OK) goto delmaster_out;
			masterJournal[masterJournalSize] = 0;

			char *journal = masterJournal; // Pointer to one journal within MJ file
			while ((journal - masterJournal) < masterJournalSize)
			{
				int exists;
				rc = vfs->Access(journal, SQLITE_ACCESS_EXISTS, &exists);
				if (rc != RC::OK)
					goto delmaster_out;
				if (exists)
				{
					// One of the journals pointed to by the master journal exists. Open it and check if it points at the master journal. If so, return without deleting the master journal file.
					int flags = (SQLITE_OPEN_READONLY | SQLITE_OPEN_MAIN_JOURNAL);
					rc = vfs->Open(journal, journalFile, flags, 0);
					if (rc != RC::OK)
						goto delmaster_out;

					rc = readMasterJournal(journalFile, masterPtr, masterPtrSize);
					journalFile->Close();
					if (rc != RC::OK)
						goto delmaster_out;

					int c = masterPtr[0] != 0 && _strcmp(masterPtr, master) == 0;
					if (c)
					{
						// We have a match. Do not delete the master journal file.
						goto delmaster_out;
					}
				}
				journal += (sqlite3Strlen30(journal) + 1);
			}

			masterFile->Close();
			rc = vfs->Delete(master, 0);

delmaster_out:
			SysEx::Free(masterJournal);
			if (masterFile != nullptr)
			{
				masterFile->Close();
				_assert(!journalFile->Opened);
				SysEx::Free(journalFile);
			}
			return rc;
		}


		static int pager_truncate(Pager *pPager, Pid pages)
		{
			RC rc = RC::OK;
			_assert(pager->State != PAGER::ERROR);
			_assert(pager->State != PAGER::READER);

			if (pager->File->Opened && pager->State >= PAGER::WRITER_DBMOD || pager->State == PAGER::OPEN)
			{
				newSize;
				int sizePage = pager->PageSize;
				_assert(pager->Lock == VFile::LOCK::EXCLUSIVE);
				// TODO: Is it safe to use Pager.dbFileSize here?
				int64 currentSize;
				rc = pager->File->get_FileSize(currentSize);
				int64 newSize = sizePage * (int64)pages;
				if (rc == RC::OK && currentSize != newSize)
				{
					if (currentSize > newSize)
						rc = pager->File->Truncate(newSize);
					else if ((currentSize + sizePage) <= newSize)
					{
						char *tmp = pager->TmpSpace;
						memset(tmp, 0, sizePage);
						ASSERTCOVERAGE((newSize - sizePage) == currentSize);
						ASSERTCOVERAGE((newSize - sizePage) > currentSize);
						rc = pager->File->Write(tmp, sizePage, newSize - sizePage);
					}
					if (rc == RC::OK)
						pager->DBFileSize = pages;
				}
			}
			return rc;
		}

#pragma endregion

#pragma region Transaction2

		int sqlite3SectorSize(VFile *file)
		{
			int ret = file->SectorSize;
			if (ret < 32)
				ret = 512;
			else if (ret > MAX_SECTOR_SIZE)
			{
				_assert(MAX_SECTOR_SIZE >= 512);
				ret = MAX_SECTOR_SIZE;
			}
			return ret;
		}

		static void setSectorSize(Pager *pager)
		{
			_assert(pager->File->Opened || pager->TempFile);
			if (pager->TempFile || (pager->File->get_DeviceCharacteristics() & SQLITE_IOCAP_POWERSAFE_OVERWRITE) != 0)
				pager->SectorSize = 512; // Sector size doesn't matter for temporary files. Also, the file may not have been opened yet, in which case the OsSectorSize() call will segfault.
			else
				pager->SectorSize = pager->File->SectorSize;
		}

		static int pager_playback(Pager *pPager, int isHot){
			sqlite3_vfs *pVfs = pPager->pVfs;
			i64 szJ;                 /* Size of the journal file in bytes */
			u32 nRec;                /* Number of Records in the journal */
			u32 u;                   /* Unsigned loop counter */
			Pgno mxPg = 0;           /* Size of the original file in pages */
			int rc;                  /* Result code of a subroutine */
			int res = 1;             /* Value returned by sqlite3OsAccess() */
			char *zMaster = 0;       /* Name of master journal file if any */
			int needPagerReset;      /* True to reset page prior to first page rollback */

			/* Figure out how many records are in the journal.  Abort early if
			** the journal is empty.
			*/
			assert( isOpen(pPager->jfd) );
			rc = sqlite3OsFileSize(pPager->jfd, &szJ);
			if( rc!=SQLITE_OK ){
				goto end_playback;
			}

			/* Read the master journal name from the journal, if it is present.
			** If a master journal file name is specified, but the file is not
			** present on disk, then the journal is not hot and does not need to be
			** played back.
			**
			** TODO: Technically the following is an error because it assumes that
			** buffer Pager.pTmpSpace is (mxPathname+1) bytes or larger. i.e. that
			** (pPager->pageSize >= pPager->pVfs->mxPathname+1). Using os_unix.c,
			**  mxPathname is 512, which is the same as the minimum allowable value
			** for pageSize.
			*/
			zMaster = pPager->pTmpSpace;
			rc = readMasterJournal(pPager->jfd, zMaster, pPager->pVfs->mxPathname+1);
			if( rc==SQLITE_OK && zMaster[0] ){
				rc = sqlite3OsAccess(pVfs, zMaster, SQLITE_ACCESS_EXISTS, &res);
			}
			zMaster = 0;
			if( rc!=SQLITE_OK || !res ){
				goto end_playback;
			}
			pPager->journalOff = 0;
			needPagerReset = isHot;

			/* This loop terminates either when a readJournalHdr() or 
			** pager_playback_one_page() call returns SQLITE_DONE or an IO error 
			** occurs. 
			*/
			while( 1 ){
				/* Read the next journal header from the journal file.  If there are
				** not enough bytes left in the journal file for a complete header, or
				** it is corrupted, then a process must have failed while writing it.
				** This indicates nothing more needs to be rolled back.
				*/
				rc = readJournalHdr(pPager, isHot, szJ, &nRec, &mxPg);
				if( rc!=SQLITE_OK ){ 
					if( rc==SQLITE_DONE ){
						rc = SQLITE_OK;
					}
					goto end_playback;
				}

				/* If nRec is 0xffffffff, then this journal was created by a process
				** working in no-sync mode. This means that the rest of the journal
				** file consists of pages, there are no more journal headers. Compute
				** the value of nRec based on this assumption.
				*/
				if( nRec==0xffffffff ){
					assert( pPager->journalOff==JOURNAL_HDR_SZ(pPager) );
					nRec = (int)((szJ - JOURNAL_HDR_SZ(pPager))/JOURNAL_PG_SZ(pPager));
				}

				/* If nRec is 0 and this rollback is of a transaction created by this
				** process and if this is the final header in the journal, then it means
				** that this part of the journal was being filled but has not yet been
				** synced to disk.  Compute the number of pages based on the remaining
				** size of the file.
				**
				** The third term of the test was added to fix ticket #2565.
				** When rolling back a hot journal, nRec==0 always means that the next
				** chunk of the journal contains zero pages to be rolled back.  But
				** when doing a ROLLBACK and the nRec==0 chunk is the last chunk in
				** the journal, it means that the journal might contain additional
				** pages that need to be rolled back and that the number of pages 
				** should be computed based on the journal file size.
				*/
				if( nRec==0 && !isHot &&
					pPager->journalHdr+JOURNAL_HDR_SZ(pPager)==pPager->journalOff ){
						nRec = (int)((szJ - pPager->journalOff) / JOURNAL_PG_SZ(pPager));
				}

				/* If this is the first header read from the journal, truncate the
				** database file back to its original size.
				*/
				if( pPager->journalOff==JOURNAL_HDR_SZ(pPager) ){
					rc = pager_truncate(pPager, mxPg);
					if( rc!=SQLITE_OK ){
						goto end_playback;
					}
					pPager->dbSize = mxPg;
				}

				/* Copy original pages out of the journal and back into the 
				** database file and/or page cache.
				*/
				for(u=0; u<nRec; u++){
					if( needPagerReset ){
						pager_reset(pPager);
						needPagerReset = 0;
					}
					rc = pager_playback_one_page(pPager,&pPager->journalOff,0,1,0);
					if( rc!=SQLITE_OK ){
						if( rc==SQLITE_DONE ){
							pPager->journalOff = szJ;
							break;
						}else if( rc==SQLITE_IOERR_SHORT_READ ){
							/* If the journal has been truncated, simply stop reading and
							** processing the journal. This might happen if the journal was
							** not completely written and synced prior to a crash.  In that
							** case, the database should have never been written in the
							** first place so it is OK to simply abandon the rollback. */
							rc = SQLITE_OK;
							goto end_playback;
						}else{
							/* If we are unable to rollback, quit and return the error
							** code.  This will cause the pager to enter the error state
							** so that no further harm will be done.  Perhaps the next
							** process to come along will be able to rollback the database.
							*/
							goto end_playback;
						}
					}
				}
			}
			/*NOTREACHED*/
			assert( 0 );

end_playback:
			/* Following a rollback, the database file should be back in its original
			** state prior to the start of the transaction, so invoke the
			** SQLITE_FCNTL_DB_UNCHANGED file-control method to disable the
			** assertion that the transaction counter was modified.
			*/
#ifdef SQLITE_DEBUG
			if( pPager->fd->pMethods ){
				sqlite3OsFileControlHint(pPager->fd,SQLITE_FCNTL_DB_UNCHANGED,0);
			}
#endif

			/* If this playback is happening automatically as a result of an IO or 
			** malloc error that occurred after the change-counter was updated but 
			** before the transaction was committed, then the change-counter 
			** modification may just have been reverted. If this happens in exclusive 
			** mode, then subsequent transactions performed by the connection will not
			** update the change-counter at all. This may lead to cache inconsistency
			** problems for other processes at some point in the future. So, just
			** in case this has happened, clear the changeCountDone flag now.
			*/
			pPager->changeCountDone = pPager->tempFile;

			if( rc==SQLITE_OK ){
				zMaster = pPager->pTmpSpace;
				rc = readMasterJournal(pPager->jfd, zMaster, pPager->pVfs->mxPathname+1);
				testcase( rc!=SQLITE_OK );
			}
			if( rc==SQLITE_OK
				&& (pPager->eState>=PAGER_WRITER_DBMOD || pPager->eState==PAGER_OPEN)
				){
					rc = sqlite3PagerSync(pPager);
			}
			if( rc==SQLITE_OK ){
				rc = pager_end_transaction(pPager, zMaster[0]!='\0', 0);
				testcase( rc!=SQLITE_OK );
			}
			if( rc==SQLITE_OK && zMaster[0] && res ){
				/* If there was a master journal and this routine will return success,
				** see if it is possible to delete the master journal.
				*/
				rc = pager_delmaster(pPager, zMaster);
				testcase( rc!=SQLITE_OK );
			}

			/* The Pager.sectorSize variable may have been updated while rolling
			** back a journal created by a process with a different sector size
			** value. Reset it to the correct value for this process.
			*/
			setSectorSize(pPager);
			return rc;
		}

		static int readDbPage(PgHdr *pPg){
			Pager *pPager = pPg->pPager; /* Pager object associated with page pPg */
			Pgno pgno = pPg->pgno;       /* Page number to read */
			int rc = SQLITE_OK;          /* Return code */
			int isInWal = 0;             /* True if page is in log file */
			int pgsz = pPager->pageSize; /* Number of bytes to read */

			assert( pPager->eState>=PAGER_READER && !MEMDB );
			assert( isOpen(pPager->fd) );

			if( NEVER(!isOpen(pPager->fd)) ){
				assert( pPager->tempFile );
				memset(pPg->pData, 0, pPager->pageSize);
				return SQLITE_OK;
			}

			if( pagerUseWal(pPager) ){
				/* Try to pull the page from the write-ahead log. */
				rc = sqlite3WalRead(pPager->pWal, pgno, &isInWal, pgsz, pPg->pData);
			}
			if( rc==SQLITE_OK && !isInWal ){
				i64 iOffset = (pgno-1)*(i64)pPager->pageSize;
				rc = sqlite3OsRead(pPager->fd, pPg->pData, pgsz, iOffset);
				if( rc==SQLITE_IOERR_SHORT_READ ){
					rc = SQLITE_OK;
				}
			}

			if( pgno==1 ){
				if( rc ){
					/* If the read is unsuccessful, set the dbFileVers[] to something
					** that will never be a valid file version.  dbFileVers[] is a copy
					** of bytes 24..39 of the database.  Bytes 28..31 should always be
					** zero or the size of the database in page. Bytes 32..35 and 35..39
					** should be page numbers which are never 0xffffffff.  So filling
					** pPager->dbFileVers[] with all 0xff bytes should suffice.
					**
					** For an encrypted database, the situation is more complex:  bytes
					** 24..39 of the database are white noise.  But the probability of
					** white noising equaling 16 bytes of 0xff is vanishingly small so
					** we should still be ok.
					*/
					memset(pPager->dbFileVers, 0xff, sizeof(pPager->dbFileVers));
				}else{
					u8 *dbFileVers = &((u8*)pPg->pData)[24];
					memcpy(&pPager->dbFileVers, dbFileVers, sizeof(pPager->dbFileVers));
				}
			}
			CODEC1(pPager, pPg->pData, pgno, 3, rc = SQLITE_NOMEM);

			PAGER_INCR(sqlite3_pager_readdb_count);
			PAGER_INCR(pPager->nRead);
			IOTRACE(("PGIN %p %d\n", pPager, pgno));
			PAGERTRACE(("FETCH %d page %d hash(%08x)\n",
				PAGERID(pPager), pgno, pager_pagehash(pPg)));

			return rc;
		}

		static void pager_write_changecounter(PgHdr *pPg){
			u32 change_counter;

			/* Increment the value just read and write it back to byte 24. */
			change_counter = sqlite3Get4byte((u8*)pPg->pPager->dbFileVers)+1;
			put32bits(((char*)pPg->pData)+24, change_counter);

			/* Also store the SQLite version number in bytes 96..99 and in
			** bytes 92..95 store the change counter for which the version number
			** is valid. */
			put32bits(((char*)pPg->pData)+92, change_counter);
			put32bits(((char*)pPg->pData)+96, SQLITE_VERSION_NUMBER);
		}

#ifndef OMIT_WAL
		static int pagerUndoCallback(void *pCtx, Pgno iPg){
			int rc = SQLITE_OK;
			Pager *pPager = (Pager *)pCtx;
			PgHdr *pPg;

			pPg = sqlite3PagerLookup(pPager, iPg);
			if( pPg ){
				if( sqlite3PcachePageRefcount(pPg)==1 ){
					sqlite3PcacheDrop(pPg);
				}else{
					rc = readDbPage(pPg);
					if( rc==SQLITE_OK ){
						pPager->xReiniter(pPg);
					}
					sqlite3PagerUnref(pPg);
				}
			}

			/* Normally, if a transaction is rolled back, any backup processes are
			** updated as data is copied out of the rollback journal and into the
			** database. This is not generally possible with a WAL database, as
			** rollback involves simply truncating the log file. Therefore, if one
			** or more frames have already been written to the log (and therefore 
			** also copied into the backup databases) as part of this transaction,
			** the backups must be restarted.
			*/
			sqlite3BackupRestart(pPager->pBackup);

			return rc;
		}

		static int pagerRollbackWal(Pager *pPager){
			int rc;                         /* Return Code */
			PgHdr *pList;                   /* List of dirty pages to revert */

			/* For all pages in the cache that are currently dirty or have already
			** been written (but not committed) to the log file, do one of the 
			** following:
			**
			**   + Discard the cached page (if refcount==0), or
			**   + Reload page content from the database (if refcount>0).
			*/
			pPager->dbSize = pPager->dbOrigSize;
			rc = sqlite3WalUndo(pPager->pWal, pagerUndoCallback, (void *)pPager);
			pList = sqlite3PcacheDirtyList(pPager->pPCache);
			while( pList && rc==SQLITE_OK ){
				PgHdr *pNext = pList->pDirty;
				rc = pagerUndoCallback((void *)pPager, pList->pgno);
				pList = pNext;
			}

			return rc;
		}

		static int pagerWalFrames(
			Pager *pPager,                  /* Pager object */
			PgHdr *pList,                   /* List of frames to log */
			Pgno nTruncate,                 /* Database size after this commit */
			int isCommit                    /* True if this is a commit */
			){
				int rc;                         /* Return code */
				int nList;                      /* Number of pages in pList */
#if defined(SQLITE_DEBUG) || defined(SQLITE_CHECK_PAGES)
				PgHdr *p;                       /* For looping over pages */
#endif

				assert( pPager->pWal );
				assert( pList );
#ifdef SQLITE_DEBUG
				/* Verify that the page list is in accending order */
				for(p=pList; p && p->pDirty; p=p->pDirty){
					assert( p->pgno < p->pDirty->pgno );
				}
#endif

				assert( pList->pDirty==0 || isCommit );
				if( isCommit ){
					/* If a WAL transaction is being committed, there is no point in writing
					** any pages with page numbers greater than nTruncate into the WAL file.
					** They will never be read by any client. So remove them from the pDirty
					** list here. */
					PgHdr *p;
					PgHdr **ppNext = &pList;
					nList = 0;
					for(p=pList; (*ppNext = p)!=0; p=p->pDirty){
						if( p->pgno<=nTruncate ){
							ppNext = &p->pDirty;
							nList++;
						}
					}
					assert( pList );
				}else{
					nList = 1;
				}
				pPager->aStat[PAGER_STAT_WRITE] += nList;

				if( pList->pgno==1 ) pager_write_changecounter(pList);
				rc = sqlite3WalFrames(pPager->pWal, 
					pPager->pageSize, pList, nTruncate, isCommit, pPager->walSyncFlags
					);
				if( rc==SQLITE_OK && pPager->pBackup ){
					PgHdr *p;
					for(p=pList; p; p=p->pDirty){
						sqlite3BackupUpdate(pPager->pBackup, p->pgno, (u8 *)p->pData);
					}
				}

#ifdef SQLITE_CHECK_PAGES
				pList = sqlite3PcacheDirtyList(pPager->pPCache);
				for(p=pList; p; p=p->pDirty){
					pager_set_pagehash(p);
				}
#endif

				return rc;
		}

		static int pagerBeginReadTransaction(Pager *pPager){
			int rc;                         /* Return code */
			int changed = 0;                /* True if cache must be reset */

			assert( pagerUseWal(pPager) );
			assert( pPager->eState==PAGER_OPEN || pPager->eState==PAGER_READER );

			/* sqlite3WalEndReadTransaction() was not called for the previous
			** transaction in locking_mode=EXCLUSIVE.  So call it now.  If we
			** are in locking_mode=NORMAL and EndRead() was previously called,
			** the duplicate call is harmless.
			*/
			sqlite3WalEndReadTransaction(pPager->pWal);

			rc = sqlite3WalBeginReadTransaction(pPager->pWal, &changed);
			if( rc!=SQLITE_OK || changed ){
				pager_reset(pPager);
			}

			return rc;
		}
#endif

		static int pagerPagecount(Pager *pPager, Pgno *pnPage){
			Pgno nPage;                     /* Value to return via *pnPage */

			/* Query the WAL sub-system for the database size. The WalDbsize()
			** function returns zero if the WAL is not open (i.e. Pager.pWal==0), or
			** if the database size is not available. The database size is not
			** available from the WAL sub-system if the log file is empty or
			** contains no valid committed transactions.
			*/
			assert( pPager->eState==PAGER_OPEN );
			assert( pPager->eLock>=SHARED_LOCK );
			nPage = sqlite3WalDbsize(pPager->pWal);

			/* If the database size was not available from the WAL sub-system,
			** determine it based on the size of the database file. If the size
			** of the database file is not an integer multiple of the page-size,
			** round down to the nearest page. Except, any file larger than 0
			** bytes in size is considered to contain at least one page.
			*/
			if( nPage==0 ){
				i64 n = 0;                    /* Size of db file in bytes */
				assert( isOpen(pPager->fd) || pPager->tempFile );
				if( isOpen(pPager->fd) ){
					int rc = sqlite3OsFileSize(pPager->fd, &n);
					if( rc!=SQLITE_OK ){
						return rc;
					}
				}
				nPage = (Pgno)((n+pPager->pageSize-1) / pPager->pageSize);
			}

			/* If the current number of pages in the file is greater than the
			** configured maximum pager number, increase the allowed limit so
			** that the file can be read.
			*/
			if( nPage>pPager->mxPgno ){
				pPager->mxPgno = (Pgno)nPage;
			}

			*pnPage = nPage;
			return SQLITE_OK;
		}

#ifndef OMIT_WAL
		static int pagerOpenWalIfPresent(Pager *pPager){
			int rc = SQLITE_OK;
			assert( pPager->eState==PAGER_OPEN );
			assert( pPager->eLock>=SHARED_LOCK );

			if( !pPager->tempFile ){
				int isWal;                    /* True if WAL file exists */
				Pgno nPage;                   /* Size of the database file */

				rc = pagerPagecount(pPager, &nPage);
				if( rc ) return rc;
				if( nPage==0 ){
					rc = sqlite3OsDelete(pPager->pVfs, pPager->zWal, 0);
					if( rc==SQLITE_IOERR_DELETE_NOENT ) rc = SQLITE_OK;
					isWal = 0;
				}else{
					rc = sqlite3OsAccess(
						pPager->pVfs, pPager->zWal, SQLITE_ACCESS_EXISTS, &isWal
						);
				}
				if( rc==SQLITE_OK ){
					if( isWal ){
						testcase( sqlite3PcachePagecount(pPager->pPCache)==0 );
						rc = sqlite3PagerOpenWal(pPager, 0);
					}else if( pPager->journalMode==PAGER_JOURNALMODE_WAL ){
						pPager->journalMode = PAGER_JOURNALMODE_DELETE;
					}
				}
			}
			return rc;
		}
#endif

		static int pagerPlaybackSavepoint(Pager *pPager, PagerSavepoint *pSavepoint){
			i64 szJ;                 /* Effective size of the main journal */
			i64 iHdrOff;             /* End of first segment of main-journal records */
			int rc = SQLITE_OK;      /* Return code */
			Bitvec *pDone = 0;       /* Bitvec to ensure pages played back only once */

			assert( pPager->eState!=PAGER_ERROR );
			assert( pPager->eState>=PAGER_WRITER_LOCKED );

			/* Allocate a bitvec to use to store the set of pages rolled back */
			if( pSavepoint ){
				pDone = sqlite3BitvecCreate(pSavepoint->nOrig);
				if( !pDone ){
					return SQLITE_NOMEM;
				}
			}

			/* Set the database size back to the value it was before the savepoint 
			** being reverted was opened.
			*/
			pPager->dbSize = pSavepoint ? pSavepoint->nOrig : pPager->dbOrigSize;
			pPager->changeCountDone = pPager->tempFile;

			if( !pSavepoint && pagerUseWal(pPager) ){
				return pagerRollbackWal(pPager);
			}

			/* Use pPager->journalOff as the effective size of the main rollback
			** journal.  The actual file might be larger than this in
			** PAGER_JOURNALMODE_TRUNCATE or PAGER_JOURNALMODE_PERSIST.  But anything
			** past pPager->journalOff is off-limits to us.
			*/
			szJ = pPager->journalOff;
			assert( pagerUseWal(pPager)==0 || szJ==0 );

			/* Begin by rolling back records from the main journal starting at
			** PagerSavepoint.iOffset and continuing to the next journal header.
			** There might be records in the main journal that have a page number
			** greater than the current database size (pPager->dbSize) but those
			** will be skipped automatically.  Pages are added to pDone as they
			** are played back.
			*/
			if( pSavepoint && !pagerUseWal(pPager) ){
				iHdrOff = pSavepoint->iHdrOffset ? pSavepoint->iHdrOffset : szJ;
				pPager->journalOff = pSavepoint->iOffset;
				while( rc==SQLITE_OK && pPager->journalOff<iHdrOff ){
					rc = pager_playback_one_page(pPager, &pPager->journalOff, pDone, 1, 1);
				}
				assert( rc!=SQLITE_DONE );
			}else{
				pPager->journalOff = 0;
			}

			/* Continue rolling back records out of the main journal starting at
			** the first journal header seen and continuing until the effective end
			** of the main journal file.  Continue to skip out-of-range pages and
			** continue adding pages rolled back to pDone.
			*/
			while( rc==SQLITE_OK && pPager->journalOff<szJ ){
				u32 ii;            /* Loop counter */
				u32 nJRec = 0;     /* Number of Journal Records */
				u32 dummy;
				rc = readJournalHdr(pPager, 0, szJ, &nJRec, &dummy);
				assert( rc!=SQLITE_DONE );

				/*
				** The "pPager->journalHdr+JOURNAL_HDR_SZ(pPager)==pPager->journalOff"
				** test is related to ticket #2565.  See the discussion in the
				** pager_playback() function for additional information.
				*/
				if( nJRec==0 
					&& pPager->journalHdr+JOURNAL_HDR_SZ(pPager)==pPager->journalOff
					){
						nJRec = (u32)((szJ - pPager->journalOff)/JOURNAL_PG_SZ(pPager));
				}
				for(ii=0; rc==SQLITE_OK && ii<nJRec && pPager->journalOff<szJ; ii++){
					rc = pager_playback_one_page(pPager, &pPager->journalOff, pDone, 1, 1);
				}
				assert( rc!=SQLITE_DONE );
			}
			assert( rc!=SQLITE_OK || pPager->journalOff>=szJ );

			/* Finally,  rollback pages from the sub-journal.  Page that were
			** previously rolled back out of the main journal (and are hence in pDone)
			** will be skipped.  Out-of-range pages are also skipped.
			*/
			if( pSavepoint ){
				u32 ii;            /* Loop counter */
				i64 offset = (i64)pSavepoint->iSubRec*(4+pPager->pageSize);

				if( pagerUseWal(pPager) ){
					rc = sqlite3WalSavepointUndo(pPager->pWal, pSavepoint->aWalData);
				}
				for(ii=pSavepoint->iSubRec; rc==SQLITE_OK && ii<pPager->nSubRec; ii++){
					assert( offset==(i64)ii*(4+pPager->pageSize) );
					rc = pager_playback_one_page(pPager, &offset, pDone, 0, 1);
				}
				assert( rc!=SQLITE_DONE );
			}

			sqlite3BitvecDestroy(pDone);
			if( rc==SQLITE_OK ){
				pPager->journalOff = szJ;
			}

			return rc;
		}

#pragma endregion

#if 0
#pragma region Name3

		void sqlite3PagerSetCachesize(Pager *pPager, int mxPage)
		{
			sqlite3PcacheSetCachesize(pPager->pPCache, mxPage);
		}

		void sqlite3PagerShrink(Pager *pPager)
		{
			sqlite3PcacheShrink(pPager->pPCache);
		}

#ifndef OMIT_PAGER_PRAGMAS
		void sqlite3PagerSetSafetyLevel(
			Pager *pPager,        /* The pager to set safety level for */
			int level,            /* PRAGMA synchronous.  1=OFF, 2=NORMAL, 3=FULL */  
			int bFullFsync,       /* PRAGMA fullfsync */
			int bCkptFullFsync    /* PRAGMA checkpoint_fullfsync */
			){
				assert( level>=1 && level<=3 );
				pPager->noSync =  (level==1 || pPager->tempFile) ?1:0;
				pPager->fullSync = (level==3 && !pPager->tempFile) ?1:0;
				if( pPager->noSync ){
					pPager->syncFlags = 0;
					pPager->ckptSyncFlags = 0;
				}else if( bFullFsync ){
					pPager->syncFlags = SQLITE_SYNC_FULL;
					pPager->ckptSyncFlags = SQLITE_SYNC_FULL;
				}else if( bCkptFullFsync ){
					pPager->syncFlags = SQLITE_SYNC_NORMAL;
					pPager->ckptSyncFlags = SQLITE_SYNC_FULL;
				}else{
					pPager->syncFlags = SQLITE_SYNC_NORMAL;
					pPager->ckptSyncFlags = SQLITE_SYNC_NORMAL;
				}
				pPager->walSyncFlags = pPager->syncFlags;
				if( pPager->fullSync ){
					pPager->walSyncFlags |= WAL_SYNC_TRANSACTIONS;
				}
		}
#endif

		// The following global variable is incremented whenever the library attempts to open a temporary file.  This information is used for testing and analysis only.  
#ifdef TEST
		int sqlite3_opentemp_count = 0;
#endif

		static int pagerOpentemp(
			Pager *pPager,        /* The pager object */
			sqlite3_file *pFile,  /* Write the file descriptor here */
			int vfsFlags          /* Flags passed through to the VFS */
			){
				int rc;               /* Return code */

#ifdef TEST
				sqlite3_opentemp_count++;  // Used for testing and analysis only
#endif
				vfsFlags |=  SQLITE_OPEN_READWRITE | SQLITE_OPEN_CREATE |
					SQLITE_OPEN_EXCLUSIVE | SQLITE_OPEN_DELETEONCLOSE;
				rc = sqlite3OsOpen(pPager->pVfs, 0, pFile, vfsFlags, 0);
				assert( rc!=SQLITE_OK || isOpen(pFile) );
				return rc;
		}

		void sqlite3PagerSetBusyhandler(
			Pager *pPager,                       /* Pager object */
			int (*xBusyHandler)(void *),         /* Pointer to busy-handler function */
			void *pBusyHandlerArg                /* Argument to pass to xBusyHandler */
			){
				pPager->xBusyHandler = xBusyHandler;
				pPager->pBusyHandlerArg = pBusyHandlerArg;

				if( isOpen(pPager->fd) ){
					void **ap = (void **)&pPager->xBusyHandler;
					assert( ((int(*)(void *))(ap[0]))==xBusyHandler );
					assert( ap[1]==pBusyHandlerArg );
					sqlite3OsFileControlHint(pPager->fd, SQLITE_FCNTL_BUSYHANDLER, (void *)ap);
				}
		}

		int sqlite3PagerSetPagesize(Pager *pager, uint32 *pageSize, int reserves)
		{
			// It is not possible to do a full assert_pager_state() here, as this function may be called from within PagerOpen(), before the state
			// of the Pager object is internally consistent.
			//
			// At one point this function returned an error if the pager was in PAGER_ERROR state. But since PAGER_ERROR state guarantees that
			// there is at least one outstanding page reference, this function is a no-op for that case anyhow.
			uint32 pageSize_ = *pageSize;
			_assert(pageSize_ == 0 || (pageSize_ >= 512 && pageSize_ <= MAX_PAGE_SIZE));
			int rc = RC::OK;
			if ((!pager->MemoryDB || pager->DBSize == 0) && PCache_RefCount(pager->PCache) == 0 && pageSize_ && pageSize_ != (uint32)pager->PageSize)
			{
				char *tempSpace = nullptr; // New temp space
				i64 bytes = 0;
				if (pager->State > PAGER::OPEN && pager->File->Opened)
					rc = pager->File->get_FileSize(&bytes);
				if (rc == RC::OK)
				{
					tempSpace = (char *)sqlite3PageMalloc(pageSize);
					if (!tempSpace) rc = RC::NOMEM;
				}
				if (rc == RC::OK)
				{
					pager_reset(pager);
					pager->DBSize = (Pid)((bytes + pageSize_ - 1) / pageSize_);
					pager->PageSize = pageSize_;
					sqlite3PageFree(pager->TmpSpace);
					pager->TmpSpace = tempSpace;
					PCache_SetPageSize(pager->PCache, pageSize_);
				}
			}
			*pageSize = pager->PageSize;
			if (rc == RC::OK)
			{
				if (reserves < 0) reserves = pager->Reserves;
				_assert(reserves >= 0 && reserves < 1000);
				pager->Reserves = (int16)reserves;
				pagerReportSize(pager);
			}
			return rc;
		}

		void *sqlite3PagerTempSpace(Pager *pager)
		{
			return pager->TmpSpace;
		}

		int sqlite3PagerMaxPageCount(Pager *pager, int maxPage)
		{
			if (maxPage > 0)
				pager->MaxPid = maxPage;
			_assert(pager->State != PAGER::OPEN);		// Called only by OP_MaxPgcnt
			_assert(pager->MaxPid >= pager->DBSize);	// OP_MaxPgcnt enforces this
			return pager->MaxPid;
		}


#ifdef TEST
		extern int sqlite3_io_error_pending;
		extern int sqlite3_io_error_hit;
		static int saved_cnt;
		void disable_simulated_io_errors(void){
			saved_cnt = sqlite3_io_error_pending;
			sqlite3_io_error_pending = -1;
		}
		void enable_simulated_io_errors(void){
			sqlite3_io_error_pending = saved_cnt;
		}
#else
# define disable_simulated_io_errors()
# define enable_simulated_io_errors()
#endif

		int sqlite3PagerReadFileheader(Pager *pPager, int N, unsigned char *pDest)
		{
			int rc = SQLITE_OK;
			memset(pDest, 0, N);
			assert( isOpen(pPager->fd) || pPager->tempFile );

			// This routine is only called by btree immediately after creating the Pager object.  There has not been an opportunity to transition to WAL mode yet.
			assert( !pagerUseWal(pPager) );

			if( isOpen(pPager->fd) ){
				IOTRACE(("DBHDR %p 0 %d\n", pPager, N))
					rc = sqlite3OsRead(pPager->fd, pDest, N, 0);
				if( rc==SQLITE_IOERR_SHORT_READ ){
					rc = SQLITE_OK;
				}
			}
			return rc;
		}

		void sqlite3PagerPagecount(Pager *pPager, int *pnPage)
		{
			assert( pPager->eState>=PAGER_READER );
			assert( pPager->eState!=PAGER_WRITER_FINISHED );
			*pnPage = (int)pPager->dbSize;
		}


		static int pager_wait_on_lock(Pager *pPager, int locktype)
		{
			int rc;

			/* Check that this is either a no-op (because the requested lock is already held, or one of the transistions that the busy-handler
			** may be invoked during, according to the comment above sqlite3PagerSetBusyhandler().
			*/
			assert( (pPager->eLock>=locktype)
				|| (pPager->eLock==NO_LOCK && locktype==SHARED_LOCK)
				|| (pPager->eLock==RESERVED_LOCK && locktype==EXCLUSIVE_LOCK)
				);

			do {
				rc = pagerLockDb(pPager, locktype);
			}while( rc==SQLITE_BUSY && pPager->xBusyHandler(pPager->pBusyHandlerArg) );
			return rc;
		}
#if defined(_DEBUG)
		static void assertTruncateConstraintCb(PgHdr *pPg){
			assert( pPg->flags&PGHDR_DIRTY );
			assert( !subjRequiresPage(pPg) || pPg->pgno<=pPg->pPager->dbSize );
		}
		static void assertTruncateConstraint(Pager *pPager){
			sqlite3PcacheIterateDirty(pPager->pPCache, assertTruncateConstraintCb);
		}
#else
# define assertTruncateConstraint(pPager)
#endif

		void sqlite3PagerTruncateImage(Pager *pPager, Pgno nPage){
			assert( pPager->dbSize>=nPage );
			assert( pPager->eState>=PAGER_WRITER_CACHEMOD );
			pPager->dbSize = nPage;

			/* At one point the code here called assertTruncateConstraint() to
			** ensure that all pages being truncated away by this operation are,
			** if one or more savepoints are open, present in the savepoint 
			** journal so that they can be restored if the savepoint is rolled
			** back. This is no longer necessary as this function is now only
			** called right before committing a transaction. So although the 
			** Pager object may still have open savepoints (Pager.nSavepoint!=0), 
			** they cannot be rolled back. So the assertTruncateConstraint() call
			** is no longer correct. */
		}

		static int pagerSyncHotJournal(Pager *pPager){
			int rc = SQLITE_OK;
			if( !pPager->noSync ){
				rc = sqlite3OsSync(pPager->jfd, SQLITE_SYNC_NORMAL);
			}
			if( rc==SQLITE_OK ){
				rc = sqlite3OsFileSize(pPager->jfd, &pPager->journalHdr);
			}
			return rc;
		}

		int sqlite3PagerClose(Pager *pPager){
			u8 *pTmp = (u8 *)pPager->pTmpSpace;

			assert( assert_pager_state(pPager) );
			disable_simulated_io_errors();
			sqlite3BeginBenignMalloc();
			/* pPager->errCode = 0; */
			pPager->exclusiveMode = 0;
#ifndef SQLITE_OMIT_WAL
			sqlite3WalClose(pPager->pWal, pPager->ckptSyncFlags, pPager->pageSize, pTmp);
			pPager->pWal = 0;
#endif
			pager_reset(pPager);
			if( MEMDB ){
				pager_unlock(pPager);
			}else{
				/* If it is open, sync the journal file before calling UnlockAndRollback.
				** If this is not done, then an unsynced portion of the open journal 
				** file may be played back into the database. If a power failure occurs 
				** while this is happening, the database could become corrupt.
				**
				** If an error occurs while trying to sync the journal, shift the pager
				** into the ERROR state. This causes UnlockAndRollback to unlock the
				** database and close the journal file without attempting to roll it
				** back or finalize it. The next database user will have to do hot-journal
				** rollback before accessing the database file.
				*/
				if( isOpen(pPager->jfd) ){
					pager_error(pPager, pagerSyncHotJournal(pPager));
				}
				pagerUnlockAndRollback(pPager);
			}
			sqlite3EndBenignMalloc();
			enable_simulated_io_errors();
			PAGERTRACE(("CLOSE %d\n", PAGERID(pPager)));
			IOTRACE(("CLOSE %p\n", pPager))
				sqlite3OsClose(pPager->jfd);
			sqlite3OsClose(pPager->fd);
			sqlite3PageFree(pTmp);
			sqlite3PcacheClose(pPager->pPCache);

#ifdef SQLITE_HAS_CODEC
			if( pPager->xCodecFree ) pPager->xCodecFree(pPager->pCodec);
#endif

			assert( !pPager->aSavepoint && !pPager->pInJournal );
			assert( !isOpen(pPager->jfd) && !isOpen(pPager->sjfd) );

			sqlite3_free(pPager);
			return SQLITE_OK;
		}

#if !defined(_DEBUG) || defined(TEST)
		Pid sqlite3PagerPagenumber(DbPage *pg)
		{
			return pg->ID;
		}
#endif

		void sqlite3PagerRef(DbPage *pg)
		{
			sqlite3PcacheRef(pg);
		}

		static int syncJournal(Pager *pPager, int newHdr){
			int rc;                         /* Return code */

			assert( pPager->eState==PAGER_WRITER_CACHEMOD
				|| pPager->eState==PAGER_WRITER_DBMOD
				);
			assert( assert_pager_state(pPager) );
			assert( !pagerUseWal(pPager) );

			rc = sqlite3PagerExclusiveLock(pPager);
			if( rc!=SQLITE_OK ) return rc;

			if( !pPager->noSync ){
				assert( !pPager->tempFile );
				if( isOpen(pPager->jfd) && pPager->journalMode!=PAGER_JOURNALMODE_MEMORY ){
					const int iDc = sqlite3OsDeviceCharacteristics(pPager->fd);
					assert( isOpen(pPager->jfd) );

					if( 0==(iDc&SQLITE_IOCAP_SAFE_APPEND) ){
						/* This block deals with an obscure problem. If the last connection
						** that wrote to this database was operating in persistent-journal
						** mode, then the journal file may at this point actually be larger
						** than Pager.journalOff bytes. If the next thing in the journal
						** file happens to be a journal-header (written as part of the
						** previous connection's transaction), and a crash or power-failure 
						** occurs after nRec is updated but before this connection writes 
						** anything else to the journal file (or commits/rolls back its 
						** transaction), then SQLite may become confused when doing the 
						** hot-journal rollback following recovery. It may roll back all
						** of this connections data, then proceed to rolling back the old,
						** out-of-date data that follows it. Database corruption.
						**
						** To work around this, if the journal file does appear to contain
						** a valid header following Pager.journalOff, then write a 0x00
						** byte to the start of it to prevent it from being recognized.
						**
						** Variable iNextHdrOffset is set to the offset at which this
						** problematic header will occur, if it exists. aMagic is used 
						** as a temporary buffer to inspect the first couple of bytes of
						** the potential journal header.
						*/
						i64 iNextHdrOffset;
						u8 aMagic[8];
						u8 zHeader[sizeof(aJournalMagic)+4];

						memcpy(zHeader, aJournalMagic, sizeof(aJournalMagic));
						put32bits(&zHeader[sizeof(aJournalMagic)], pPager->nRec);

						iNextHdrOffset = journalHdrOffset(pPager);
						rc = sqlite3OsRead(pPager->jfd, aMagic, 8, iNextHdrOffset);
						if( rc==SQLITE_OK && 0==memcmp(aMagic, aJournalMagic, 8) ){
							static const u8 zerobyte = 0;
							rc = sqlite3OsWrite(pPager->jfd, &zerobyte, 1, iNextHdrOffset);
						}
						if( rc!=SQLITE_OK && rc!=SQLITE_IOERR_SHORT_READ ){
							return rc;
						}

						/* Write the nRec value into the journal file header. If in
						** full-synchronous mode, sync the journal first. This ensures that
						** all data has really hit the disk before nRec is updated to mark
						** it as a candidate for rollback.
						**
						** This is not required if the persistent media supports the
						** SAFE_APPEND property. Because in this case it is not possible 
						** for garbage data to be appended to the file, the nRec field
						** is populated with 0xFFFFFFFF when the journal header is written
						** and never needs to be updated.
						*/
						if( pPager->fullSync && 0==(iDc&SQLITE_IOCAP_SEQUENTIAL) ){
							PAGERTRACE(("SYNC journal of %d\n", PAGERID(pPager)));
							IOTRACE(("JSYNC %p\n", pPager))
								rc = sqlite3OsSync(pPager->jfd, pPager->syncFlags);
							if( rc!=SQLITE_OK ) return rc;
						}
						IOTRACE(("JHDR %p %lld\n", pPager, pPager->journalHdr));
						rc = sqlite3OsWrite(
							pPager->jfd, zHeader, sizeof(zHeader), pPager->journalHdr
							);
						if( rc!=SQLITE_OK ) return rc;
					}
					if( 0==(iDc&SQLITE_IOCAP_SEQUENTIAL) ){
						PAGERTRACE(("SYNC journal of %d\n", PAGERID(pPager)));
						IOTRACE(("JSYNC %p\n", pPager))
							rc = sqlite3OsSync(pPager->jfd, pPager->syncFlags| 
							(pPager->syncFlags==SQLITE_SYNC_FULL?SQLITE_SYNC_DATAONLY:0)
							);
						if( rc!=SQLITE_OK ) return rc;
					}

					pPager->journalHdr = pPager->journalOff;
					if( newHdr && 0==(iDc&SQLITE_IOCAP_SAFE_APPEND) ){
						pPager->nRec = 0;
						rc = writeJournalHdr(pPager);
						if( rc!=SQLITE_OK ) return rc;
					}
				}else{
					pPager->journalHdr = pPager->journalOff;
				}
			}

			/* Unless the pager is in noSync mode, the journal file was just 
			** successfully synced. Either way, clear the PGHDR_NEED_SYNC flag on 
			** all pages.
			*/
			sqlite3PcacheClearSyncFlags(pPager->pPCache);
			pPager->eState = PAGER_WRITER_DBMOD;
			assert( assert_pager_state(pPager) );
			return SQLITE_OK;
		}

		static int pager_write_pagelist(Pager *pPager, PgHdr *pList){
			int rc = SQLITE_OK;                  /* Return code */

			/* This function is only called for rollback pagers in WRITER_DBMOD state. */
			assert( !pagerUseWal(pPager) );
			assert( pPager->eState==PAGER_WRITER_DBMOD );
			assert( pPager->eLock==EXCLUSIVE_LOCK );

			/* If the file is a temp-file has not yet been opened, open it now. It
			** is not possible for rc to be other than SQLITE_OK if this branch
			** is taken, as pager_wait_on_lock() is a no-op for temp-files.
			*/
			if( !isOpen(pPager->fd) ){
				assert( pPager->tempFile && rc==SQLITE_OK );
				rc = pagerOpentemp(pPager, pPager->fd, pPager->vfsFlags);
			}

			/* Before the first write, give the VFS a hint of what the final
			** file size will be.
			*/
			assert( rc!=SQLITE_OK || isOpen(pPager->fd) );
			if( rc==SQLITE_OK && pPager->dbSize>pPager->dbHintSize ){
				sqlite3_int64 szFile = pPager->pageSize * (sqlite3_int64)pPager->dbSize;
				sqlite3OsFileControlHint(pPager->fd, SQLITE_FCNTL_SIZE_HINT, &szFile);
				pPager->dbHintSize = pPager->dbSize;
			}

			while( rc==SQLITE_OK && pList ){
				Pgno pgno = pList->pgno;

				/* If there are dirty pages in the page cache with page numbers greater
				** than Pager.dbSize, this means sqlite3PagerTruncateImage() was called to
				** make the file smaller (presumably by auto-vacuum code). Do not write
				** any such pages to the file.
				**
				** Also, do not write out any page that has the PGHDR_DONT_WRITE flag
				** set (set by sqlite3PagerDontWrite()).
				*/
				if( pgno<=pPager->dbSize && 0==(pList->flags&PGHDR_DONT_WRITE) ){
					i64 offset = (pgno-1)*(i64)pPager->pageSize;   /* Offset to write */
					char *pData;                                   /* Data to write */    

					assert( (pList->flags&PGHDR_NEED_SYNC)==0 );
					if( pList->pgno==1 ) pager_write_changecounter(pList);

					/* Encode the database */
					CODEC2(pPager, pList->pData, pgno, 6, return SQLITE_NOMEM, pData);

					/* Write out the page data. */
					rc = sqlite3OsWrite(pPager->fd, pData, pPager->pageSize, offset);

					/* If page 1 was just written, update Pager.dbFileVers to match
					** the value now stored in the database file. If writing this 
					** page caused the database file to grow, update dbFileSize. 
					*/
					if( pgno==1 ){
						memcpy(&pPager->dbFileVers, &pData[24], sizeof(pPager->dbFileVers));
					}
					if( pgno>pPager->dbFileSize ){
						pPager->dbFileSize = pgno;
					}
					pPager->aStat[PAGER_STAT_WRITE]++;

					/* Update any backup objects copying the contents of this pager. */
					sqlite3BackupUpdate(pPager->pBackup, pgno, (u8*)pList->pData);

					PAGERTRACE(("STORE %d page %d hash(%08x)\n",
						PAGERID(pPager), pgno, pager_pagehash(pList)));
					IOTRACE(("PGOUT %p %d\n", pPager, pgno));
					PAGER_INCR(sqlite3_pager_writedb_count);
				}else{
					PAGERTRACE(("NOSTORE %d page %d\n", PAGERID(pPager), pgno));
				}
				pager_set_pagehash(pList);
				pList = pList->pDirty;
			}

			return rc;
		}

		static int openSubJournal(Pager *pPager){
			int rc = SQLITE_OK;
			if( !isOpen(pPager->sjfd) ){
				if( pPager->journalMode==PAGER_JOURNALMODE_MEMORY || pPager->subjInMemory ){
					sqlite3MemJournalOpen(pPager->sjfd);
				}else{
					rc = pagerOpentemp(pPager, pPager->sjfd, SQLITE_OPEN_SUBJOURNAL);
				}
			}
			return rc;
		}

		static int subjournalPage(PgHdr *pPg){
			int rc = SQLITE_OK;
			Pager *pPager = pPg->pPager;
			if( pPager->journalMode!=PAGER_JOURNALMODE_OFF ){

				/* Open the sub-journal, if it has not already been opened */
				assert( pPager->useJournal );
				assert( isOpen(pPager->jfd) || pagerUseWal(pPager) );
				assert( isOpen(pPager->sjfd) || pPager->nSubRec==0 );
				assert( pagerUseWal(pPager) 
					|| pageInJournal(pPg) 
					|| pPg->pgno>pPager->dbOrigSize 
					);
				rc = openSubJournal(pPager);

				/* If the sub-journal was opened successfully (or was already open),
				** write the journal record into the file.  */
				if( rc==SQLITE_OK ){
					void *pData = pPg->pData;
					i64 offset = (i64)pPager->nSubRec*(4+pPager->pageSize);
					char *pData2;

					CODEC2(pPager, pData, pPg->pgno, 7, return SQLITE_NOMEM, pData2);
					PAGERTRACE(("STMT-JOURNAL %d page %d\n", PAGERID(pPager), pPg->pgno));
					rc = write32bits(pPager->sjfd, offset, pPg->pgno);
					if( rc==SQLITE_OK ){
						rc = sqlite3OsWrite(pPager->sjfd, pData2, pPager->pageSize, offset+4);
					}
				}
			}
			if( rc==SQLITE_OK ){
				pPager->nSubRec++;
				assert( pPager->nSavepoint>0 );
				rc = addToSavepointBitvecs(pPager, pPg->pgno);
			}
			return rc;
		}

		static int pagerStress(void *p, PgHdr *pPg){
			Pager *pPager = (Pager *)p;
			int rc = SQLITE_OK;

			assert( pPg->pPager==pPager );
			assert( pPg->flags&PGHDR_DIRTY );

			/* The doNotSyncSpill flag is set during times when doing a sync of
			** journal (and adding a new header) is not allowed.  This occurs
			** during calls to sqlite3PagerWrite() while trying to journal multiple
			** pages belonging to the same sector.
			**
			** The doNotSpill flag inhibits all cache spilling regardless of whether
			** or not a sync is required.  This is set during a rollback.
			**
			** Spilling is also prohibited when in an error state since that could
			** lead to database corruption.   In the current implementaton it 
			** is impossible for sqlite3PcacheFetch() to be called with createFlag==1
			** while in the error state, hence it is impossible for this routine to
			** be called in the error state.  Nevertheless, we include a NEVER()
			** test for the error state as a safeguard against future changes.
			*/
			if( NEVER(pPager->errCode) ) return SQLITE_OK;
			if( pPager->doNotSpill ) return SQLITE_OK;
			if( pPager->doNotSyncSpill && (pPg->flags & PGHDR_NEED_SYNC)!=0 ){
				return SQLITE_OK;
			}

			pPg->pDirty = 0;
			if( pagerUseWal(pPager) ){
				/* Write a single frame for this page to the log. */
				if( subjRequiresPage(pPg) ){ 
					rc = subjournalPage(pPg); 
				}
				if( rc==SQLITE_OK ){
					rc = pagerWalFrames(pPager, pPg, 0, 0);
				}
			}else{

				/* Sync the journal file if required. */
				if( pPg->flags&PGHDR_NEED_SYNC 
					|| pPager->eState==PAGER_WRITER_CACHEMOD
					){
						rc = syncJournal(pPager, 1);
				}

				/* If the page number of this page is larger than the current size of
				** the database image, it may need to be written to the sub-journal.
				** This is because the call to pager_write_pagelist() below will not
				** actually write data to the file in this case.
				**
				** Consider the following sequence of events:
				**
				**   BEGIN;
				**     <journal page X>
				**     <modify page X>
				**     SAVEPOINT sp;
				**       <shrink database file to Y pages>
				**       pagerStress(page X)
				**     ROLLBACK TO sp;
				**
				** If (X>Y), then when pagerStress is called page X will not be written
				** out to the database file, but will be dropped from the cache. Then,
				** following the "ROLLBACK TO sp" statement, reading page X will read
				** data from the database file. This will be the copy of page X as it
				** was when the transaction started, not as it was when "SAVEPOINT sp"
				** was executed.
				**
				** The solution is to write the current data for page X into the 
				** sub-journal file now (if it is not already there), so that it will
				** be restored to its current value when the "ROLLBACK TO sp" is 
				** executed.
				*/
				if( NEVER(
					rc==SQLITE_OK && pPg->pgno>pPager->dbSize && subjRequiresPage(pPg)
					) ){
						rc = subjournalPage(pPg);
				}

				/* Write the contents of the page out to the database file. */
				if( rc==SQLITE_OK ){
					assert( (pPg->flags&PGHDR_NEED_SYNC)==0 );
					rc = pager_write_pagelist(pPager, pPg);
				}
			}

			/* Mark the page as clean. */
			if( rc==SQLITE_OK ){
				PAGERTRACE(("STRESS %d page %d\n", PAGERID(pPager), pPg->pgno));
				sqlite3PcacheMakeClean(pPg);
			}

			return pager_error(pPager, rc); 
		}

		int sqlite3PagerOpen(
			sqlite3_vfs *pVfs,       /* The virtual file system to use */
			Pager **ppPager,         /* OUT: Return the Pager structure here */
			const char *zFilename,   /* Name of the database file to open */
			int nExtra,              /* Extra bytes append to each in-memory page */
			int flags,               /* flags controlling this file */
			int vfsFlags,            /* flags passed through to sqlite3_vfs.xOpen() */
			void (*xReinit)(DbPage*) /* Function to reinitialize pages */
			){
				u8 *pPtr;
				Pager *pPager = 0;       /* Pager object to allocate and return */
				int rc = SQLITE_OK;      /* Return code */
				int tempFile = 0;        /* True for temp files (incl. in-memory files) */
				int memDb = 0;           /* True if this is an in-memory file */
				int readOnly = 0;        /* True if this is a read-only file */
				int journalFileSize;     /* Bytes to allocate for each journal fd */
				char *zPathname = 0;     /* Full path to database file */
				int nPathname = 0;       /* Number of bytes in zPathname */
				int useJournal = (flags & PAGER_OMIT_JOURNAL)==0; /* False to omit journal */
				int pcacheSize = sqlite3PcacheSize();       /* Bytes to allocate for PCache */
				u32 szPageDflt = SQLITE_DEFAULT_PAGE_SIZE;  /* Default page size */
				const char *zUri = 0;    /* URI args to copy */
				int nUri = 0;            /* Number of bytes of URI args at *zUri */

				/* Figure out how much space is required for each journal file-handle
				** (there are two of them, the main journal and the sub-journal). This
				** is the maximum space required for an in-memory journal file handle 
				** and a regular journal file-handle. Note that a "regular journal-handle"
				** may be a wrapper capable of caching the first portion of the journal
				** file in memory to implement the atomic-write optimization (see 
				** source file journal.c).
				*/
				if( sqlite3JournalSize(pVfs)>sqlite3MemJournalSize() ){
					journalFileSize = ROUND8(sqlite3JournalSize(pVfs));
				}else{
					journalFileSize = ROUND8(sqlite3MemJournalSize());
				}

				/* Set the output variable to NULL in case an error occurs. */
				*ppPager = 0;

#ifndef SQLITE_OMIT_MEMORYDB
				if( flags & PAGER_MEMORY ){
					memDb = 1;
					if( zFilename && zFilename[0] ){
						zPathname = sqlite3DbStrDup(0, zFilename);
						if( zPathname==0  ) return SQLITE_NOMEM;
						nPathname = sqlite3Strlen30(zPathname);
						zFilename = 0;
					}
				}
#endif

				/* Compute and store the full pathname in an allocated buffer pointed
				** to by zPathname, length nPathname. Or, if this is a temporary file,
				** leave both nPathname and zPathname set to 0.
				*/
				if( zFilename && zFilename[0] ){
					const char *z;
					nPathname = pVfs->mxPathname+1;
					zPathname = sqlite3DbMallocRaw(0, nPathname*2);
					if( zPathname==0 ){
						return SQLITE_NOMEM;
					}
					zPathname[0] = 0; /* Make sure initialized even if FullPathname() fails */
					rc = sqlite3OsFullPathname(pVfs, zFilename, nPathname, zPathname);
					nPathname = sqlite3Strlen30(zPathname);
					z = zUri = &zFilename[sqlite3Strlen30(zFilename)+1];
					while( *z ){
						z += sqlite3Strlen30(z)+1;
						z += sqlite3Strlen30(z)+1;
					}
					nUri = (int)(&z[1] - zUri);
					assert( nUri>=0 );
					if( rc==SQLITE_OK && nPathname+8>pVfs->mxPathname ){
						/* This branch is taken when the journal path required by
						** the database being opened will be more than pVfs->mxPathname
						** bytes in length. This means the database cannot be opened,
						** as it will not be possible to open the journal file or even
						** check for a hot-journal before reading.
						*/
						rc = SQLITE_CANTOPEN_BKPT;
					}
					if( rc!=SQLITE_OK ){
						sqlite3DbFree(0, zPathname);
						return rc;
					}
				}

				/* Allocate memory for the Pager structure, PCache object, the
				** three file descriptors, the database file name and the journal 
				** file name. The layout in memory is as follows:
				**
				**     Pager object                    (sizeof(Pager) bytes)
				**     PCache object                   (sqlite3PcacheSize() bytes)
				**     Database file handle            (pVfs->szOsFile bytes)
				**     Sub-journal file handle         (journalFileSize bytes)
				**     Main journal file handle        (journalFileSize bytes)
				**     Database file name              (nPathname+1 bytes)
				**     Journal file name               (nPathname+8+1 bytes)
				*/
				pPtr = (u8 *)sqlite3MallocZero(
					ROUND8(sizeof(*pPager)) +      /* Pager structure */
					ROUND8(pcacheSize) +           /* PCache object */
					ROUND8(pVfs->szOsFile) +       /* The main db file */
					journalFileSize * 2 +          /* The two journal files */ 
					nPathname + 1 + nUri +         /* zFilename */
					nPathname + 8 + 2              /* zJournal */
#ifndef SQLITE_OMIT_WAL
					+ nPathname + 4 + 2            /* zWal */
#endif
					);
				assert( EIGHT_BYTE_ALIGNMENT(SQLITE_INT_TO_PTR(journalFileSize)) );
				if( !pPtr ){
					sqlite3DbFree(0, zPathname);
					return SQLITE_NOMEM;
				}
				pPager =              (Pager*)(pPtr);
				pPager->pPCache =    (PCache*)(pPtr += ROUND8(sizeof(*pPager)));
				pPager->fd =   (sqlite3_file*)(pPtr += ROUND8(pcacheSize));
				pPager->sjfd = (sqlite3_file*)(pPtr += ROUND8(pVfs->szOsFile));
				pPager->jfd =  (sqlite3_file*)(pPtr += journalFileSize);
				pPager->zFilename =    (char*)(pPtr += journalFileSize);
				assert( EIGHT_BYTE_ALIGNMENT(pPager->jfd) );

				/* Fill in the Pager.zFilename and Pager.zJournal buffers, if required. */
				if( zPathname ){
					assert( nPathname>0 );
					pPager->zJournal =   (char*)(pPtr += nPathname + 1 + nUri);
					memcpy(pPager->zFilename, zPathname, nPathname);
					if( nUri ) memcpy(&pPager->zFilename[nPathname+1], zUri, nUri);
					memcpy(pPager->zJournal, zPathname, nPathname);
					memcpy(&pPager->zJournal[nPathname], "-journal\000", 8+2);
					sqlite3FileSuffix3(pPager->zFilename, pPager->zJournal);
#ifndef SQLITE_OMIT_WAL
					pPager->zWal = &pPager->zJournal[nPathname+8+1];
					memcpy(pPager->zWal, zPathname, nPathname);
					memcpy(&pPager->zWal[nPathname], "-wal\000", 4+1);
					sqlite3FileSuffix3(pPager->zFilename, pPager->zWal);
#endif
					sqlite3DbFree(0, zPathname);
				}
				pPager->pVfs = pVfs;
				pPager->vfsFlags = vfsFlags;

				/* Open the pager file.
				*/
				if( zFilename && zFilename[0] ){
					int fout = 0;                    /* VFS flags returned by xOpen() */
					rc = sqlite3OsOpen(pVfs, pPager->zFilename, pPager->fd, vfsFlags, &fout);
					assert( !memDb );
					readOnly = (fout&SQLITE_OPEN_READONLY);

					/* If the file was successfully opened for read/write access,
					** choose a default page size in case we have to create the
					** database file. The default page size is the maximum of:
					**
					**    + SQLITE_DEFAULT_PAGE_SIZE,
					**    + The value returned by sqlite3OsSectorSize()
					**    + The largest page size that can be written atomically.
					*/
					if( rc==SQLITE_OK && !readOnly ){
						setSectorSize(pPager);
						assert(SQLITE_DEFAULT_PAGE_SIZE<=SQLITE_MAX_DEFAULT_PAGE_SIZE);
						if( szPageDflt<pPager->sectorSize ){
							if( pPager->sectorSize>SQLITE_MAX_DEFAULT_PAGE_SIZE ){
								szPageDflt = SQLITE_MAX_DEFAULT_PAGE_SIZE;
							}else{
								szPageDflt = (u32)pPager->sectorSize;
							}
						}
#ifdef SQLITE_ENABLE_ATOMIC_WRITE
						{
							int iDc = sqlite3OsDeviceCharacteristics(pPager->fd);
							int ii;
							assert(SQLITE_IOCAP_ATOMIC512==(512>>8));
							assert(SQLITE_IOCAP_ATOMIC64K==(65536>>8));
							assert(SQLITE_MAX_DEFAULT_PAGE_SIZE<=65536);
							for(ii=szPageDflt; ii<=SQLITE_MAX_DEFAULT_PAGE_SIZE; ii=ii*2){
								if( iDc&(SQLITE_IOCAP_ATOMIC|(ii>>8)) ){
									szPageDflt = ii;
								}
							}
						}
#endif
					}
				}else{
					/* If a temporary file is requested, it is not opened immediately.
					** In this case we accept the default page size and delay actually
					** opening the file until the first call to OsWrite().
					**
					** This branch is also run for an in-memory database. An in-memory
					** database is the same as a temp-file that is never written out to
					** disk and uses an in-memory rollback journal.
					*/ 
					tempFile = 1;
					pPager->eState = PAGER_READER;
					pPager->eLock = EXCLUSIVE_LOCK;
					readOnly = (vfsFlags&SQLITE_OPEN_READONLY);
				}

				/* The following call to PagerSetPagesize() serves to set the value of 
				** Pager.pageSize and to allocate the Pager.pTmpSpace buffer.
				*/
				if( rc==SQLITE_OK ){
					assert( pPager->memDb==0 );
					rc = sqlite3PagerSetPagesize(pPager, &szPageDflt, -1);
					testcase( rc!=SQLITE_OK );
				}

				/* If an error occurred in either of the blocks above, free the 
				** Pager structure and close the file.
				*/
				if( rc!=SQLITE_OK ){
					assert( !pPager->pTmpSpace );
					sqlite3OsClose(pPager->fd);
					sqlite3_free(pPager);
					return rc;
				}

				/* Initialize the PCache object. */
				assert( nExtra<1000 );
				nExtra = ROUND8(nExtra);
				sqlite3PcacheOpen(szPageDflt, nExtra, !memDb,
					!memDb?pagerStress:0, (void *)pPager, pPager->pPCache);

				PAGERTRACE(("OPEN %d %s\n", FILEHANDLEID(pPager->fd), pPager->zFilename));
				IOTRACE(("OPEN %p %s\n", pPager, pPager->zFilename))

					pPager->useJournal = (u8)useJournal;
				/* pPager->stmtOpen = 0; */
				/* pPager->stmtInUse = 0; */
				/* pPager->nRef = 0; */
				/* pPager->stmtSize = 0; */
				/* pPager->stmtJSize = 0; */
				/* pPager->nPage = 0; */
				pPager->mxPgno = SQLITE_MAX_PAGE_COUNT;
				/* pPager->state = PAGER_UNLOCK; */
#if 0
				assert( pPager->state == (tempFile ? PAGER_EXCLUSIVE : PAGER_UNLOCK) );
#endif
				/* pPager->errMask = 0; */
				pPager->tempFile = (u8)tempFile;
				assert( tempFile==PAGER_LOCKINGMODE_NORMAL 
					|| tempFile==PAGER_LOCKINGMODE_EXCLUSIVE );
				assert( PAGER_LOCKINGMODE_EXCLUSIVE==1 );
				pPager->exclusiveMode = (u8)tempFile; 
				pPager->changeCountDone = pPager->tempFile;
				pPager->memDb = (u8)memDb;
				pPager->readOnly = (u8)readOnly;
				assert( useJournal || pPager->tempFile );
				pPager->noSync = pPager->tempFile;
				if( pPager->noSync ){
					assert( pPager->fullSync==0 );
					assert( pPager->syncFlags==0 );
					assert( pPager->walSyncFlags==0 );
					assert( pPager->ckptSyncFlags==0 );
				}else{
					pPager->fullSync = 1;
					pPager->syncFlags = SQLITE_SYNC_NORMAL;
					pPager->walSyncFlags = SQLITE_SYNC_NORMAL | WAL_SYNC_TRANSACTIONS;
					pPager->ckptSyncFlags = SQLITE_SYNC_NORMAL;
				}
				/* pPager->pFirst = 0; */
				/* pPager->pFirstSynced = 0; */
				/* pPager->pLast = 0; */
				pPager->nExtra = (u16)nExtra;
				pPager->journalSizeLimit = SQLITE_DEFAULT_JOURNAL_SIZE_LIMIT;
				assert( isOpen(pPager->fd) || tempFile );
				setSectorSize(pPager);
				if( !useJournal ){
					pPager->journalMode = PAGER_JOURNALMODE_OFF;
				}else if( memDb ){
					pPager->journalMode = PAGER_JOURNALMODE_MEMORY;
				}
				/* pPager->xBusyHandler = 0; */
				/* pPager->pBusyHandlerArg = 0; */
				pPager->xReiniter = xReinit;
				/* memset(pPager->aHash, 0, sizeof(pPager->aHash)); */

				*ppPager = pPager;
				return SQLITE_OK;
		}

		static int hasHotJournal(Pager *pPager, int *pExists){
			sqlite3_vfs * const pVfs = pPager->pVfs;
			int rc = SQLITE_OK;           /* Return code */
			int exists = 1;               /* True if a journal file is present */
			int jrnlOpen = !!isOpen(pPager->jfd);

			assert( pPager->useJournal );
			assert( isOpen(pPager->fd) );
			assert( pPager->eState==PAGER_OPEN );

			assert( jrnlOpen==0 || ( sqlite3OsDeviceCharacteristics(pPager->jfd) &
				SQLITE_IOCAP_UNDELETABLE_WHEN_OPEN
				));

			*pExists = 0;
			if( !jrnlOpen ){
				rc = sqlite3OsAccess(pVfs, pPager->zJournal, SQLITE_ACCESS_EXISTS, &exists);
			}
			if( rc==SQLITE_OK && exists ){
				int locked = 0;             /* True if some process holds a RESERVED lock */

				/* Race condition here:  Another process might have been holding the
				** the RESERVED lock and have a journal open at the sqlite3OsAccess() 
				** call above, but then delete the journal and drop the lock before
				** we get to the following sqlite3OsCheckReservedLock() call.  If that
				** is the case, this routine might think there is a hot journal when
				** in fact there is none.  This results in a false-positive which will
				** be dealt with by the playback routine.  Ticket #3883.
				*/
				rc = sqlite3OsCheckReservedLock(pPager->fd, &locked);
				if( rc==SQLITE_OK && !locked ){
					Pgno nPage;                 /* Number of pages in database file */

					/* Check the size of the database file. If it consists of 0 pages,
					** then delete the journal file. See the header comment above for 
					** the reasoning here.  Delete the obsolete journal file under
					** a RESERVED lock to avoid race conditions and to avoid violating
					** [H33020].
					*/
					rc = pagerPagecount(pPager, &nPage);
					if( rc==SQLITE_OK ){
						if( nPage==0 ){
							sqlite3BeginBenignMalloc();
							if( pagerLockDb(pPager, RESERVED_LOCK)==SQLITE_OK ){
								sqlite3OsDelete(pVfs, pPager->zJournal, 0);
								if( !pPager->exclusiveMode ) pagerUnlockDb(pPager, SHARED_LOCK);
							}
							sqlite3EndBenignMalloc();
						}else{
							/* The journal file exists and no other connection has a reserved
							** or greater lock on the database file. Now check that there is
							** at least one non-zero bytes at the start of the journal file.
							** If there is, then we consider this journal to be hot. If not, 
							** it can be ignored.
							*/
							if( !jrnlOpen ){
								int f = SQLITE_OPEN_READONLY|SQLITE_OPEN_MAIN_JOURNAL;
								rc = sqlite3OsOpen(pVfs, pPager->zJournal, pPager->jfd, f, &f);
							}
							if( rc==SQLITE_OK ){
								u8 first = 0;
								rc = sqlite3OsRead(pPager->jfd, (void *)&first, 1, 0);
								if( rc==SQLITE_IOERR_SHORT_READ ){
									rc = SQLITE_OK;
								}
								if( !jrnlOpen ){
									sqlite3OsClose(pPager->jfd);
								}
								*pExists = (first!=0);
							}else if( rc==SQLITE_CANTOPEN ){
								/* If we cannot open the rollback journal file in order to see if
								** its has a zero header, that might be due to an I/O error, or
								** it might be due to the race condition described above and in
								** ticket #3883.  Either way, assume that the journal is hot.
								** This might be a false positive.  But if it is, then the
								** automatic journal playback and recovery mechanism will deal
								** with it under an EXCLUSIVE lock where we do not need to
								** worry so much with race conditions.
								*/
								*pExists = 1;
								rc = SQLITE_OK;
							}
						}
					}
				}
			}

			return rc;
		}

		int sqlite3PagerSharedLock(Pager *pPager){
			int rc = SQLITE_OK;                /* Return code */

			/* This routine is only called from b-tree and only when there are no
			** outstanding pages. This implies that the pager state should either
			** be OPEN or READER. READER is only possible if the pager is or was in 
			** exclusive access mode.
			*/
			assert( sqlite3PcacheRefCount(pPager->pPCache)==0 );
			assert( assert_pager_state(pPager) );
			assert( pPager->eState==PAGER_OPEN || pPager->eState==PAGER_READER );
			if( NEVER(MEMDB && pPager->errCode) ){ return pPager->errCode; }

			if( !pagerUseWal(pPager) && pPager->eState==PAGER_OPEN ){
				int bHotJournal = 1;          /* True if there exists a hot journal-file */

				assert( !MEMDB );

				rc = pager_wait_on_lock(pPager, SHARED_LOCK);
				if( rc!=SQLITE_OK ){
					assert( pPager->eLock==NO_LOCK || pPager->eLock==UNKNOWN_LOCK );
					goto failed;
				}

				/* If a journal file exists, and there is no RESERVED lock on the
				** database file, then it either needs to be played back or deleted.
				*/
				if( pPager->eLock<=SHARED_LOCK ){
					rc = hasHotJournal(pPager, &bHotJournal);
				}
				if( rc!=SQLITE_OK ){
					goto failed;
				}
				if( bHotJournal ){
					if( pPager->readOnly ){
						rc = SQLITE_READONLY_ROLLBACK;
						goto failed;
					}

					/* Get an EXCLUSIVE lock on the database file. At this point it is
					** important that a RESERVED lock is not obtained on the way to the
					** EXCLUSIVE lock. If it were, another process might open the
					** database file, detect the RESERVED lock, and conclude that the
					** database is safe to read while this process is still rolling the 
					** hot-journal back.
					** 
					** Because the intermediate RESERVED lock is not requested, any
					** other process attempting to access the database file will get to 
					** this point in the code and fail to obtain its own EXCLUSIVE lock 
					** on the database file.
					**
					** Unless the pager is in locking_mode=exclusive mode, the lock is
					** downgraded to SHARED_LOCK before this function returns.
					*/
					rc = pagerLockDb(pPager, EXCLUSIVE_LOCK);
					if( rc!=SQLITE_OK ){
						goto failed;
					}

					/* If it is not already open and the file exists on disk, open the 
					** journal for read/write access. Write access is required because 
					** in exclusive-access mode the file descriptor will be kept open 
					** and possibly used for a transaction later on. Also, write-access 
					** is usually required to finalize the journal in journal_mode=persist 
					** mode (and also for journal_mode=truncate on some systems).
					**
					** If the journal does not exist, it usually means that some 
					** other connection managed to get in and roll it back before 
					** this connection obtained the exclusive lock above. Or, it 
					** may mean that the pager was in the error-state when this
					** function was called and the journal file does not exist.
					*/
					if( !isOpen(pPager->jfd) ){
						sqlite3_vfs * const pVfs = pPager->pVfs;
						int bExists;              /* True if journal file exists */
						rc = sqlite3OsAccess(
							pVfs, pPager->zJournal, SQLITE_ACCESS_EXISTS, &bExists);
						if( rc==SQLITE_OK && bExists ){
							int fout = 0;
							int f = SQLITE_OPEN_READWRITE|SQLITE_OPEN_MAIN_JOURNAL;
							assert( !pPager->tempFile );
							rc = sqlite3OsOpen(pVfs, pPager->zJournal, pPager->jfd, f, &fout);
							assert( rc!=SQLITE_OK || isOpen(pPager->jfd) );
							if( rc==SQLITE_OK && fout&SQLITE_OPEN_READONLY ){
								rc = SQLITE_CANTOPEN_BKPT;
								sqlite3OsClose(pPager->jfd);
							}
						}
					}

					/* Playback and delete the journal.  Drop the database write
					** lock and reacquire the read lock. Purge the cache before
					** playing back the hot-journal so that we don't end up with
					** an inconsistent cache.  Sync the hot journal before playing
					** it back since the process that crashed and left the hot journal
					** probably did not sync it and we are required to always sync
					** the journal before playing it back.
					*/
					if( isOpen(pPager->jfd) ){
						assert( rc==SQLITE_OK );
						rc = pagerSyncHotJournal(pPager);
						if( rc==SQLITE_OK ){
							rc = pager_playback(pPager, 1);
							pPager->eState = PAGER_OPEN;
						}
					}else if( !pPager->exclusiveMode ){
						pagerUnlockDb(pPager, SHARED_LOCK);
					}

					if( rc!=SQLITE_OK ){
						/* This branch is taken if an error occurs while trying to open
						** or roll back a hot-journal while holding an EXCLUSIVE lock. The
						** pager_unlock() routine will be called before returning to unlock
						** the file. If the unlock attempt fails, then Pager.eLock must be
						** set to UNKNOWN_LOCK (see the comment above the #define for 
						** UNKNOWN_LOCK above for an explanation). 
						**
						** In order to get pager_unlock() to do this, set Pager.eState to
						** PAGER_ERROR now. This is not actually counted as a transition
						** to ERROR state in the state diagram at the top of this file,
						** since we know that the same call to pager_unlock() will very
						** shortly transition the pager object to the OPEN state. Calling
						** assert_pager_state() would fail now, as it should not be possible
						** to be in ERROR state when there are zero outstanding page 
						** references.
						*/
						pager_error(pPager, rc);
						goto failed;
					}

					assert( pPager->eState==PAGER_OPEN );
					assert( (pPager->eLock==SHARED_LOCK)
						|| (pPager->exclusiveMode && pPager->eLock>SHARED_LOCK)
						);
				}

				if( !pPager->tempFile 
					&& (pPager->pBackup || sqlite3PcachePagecount(pPager->pPCache)>0) 
					){
						/* The shared-lock has just been acquired on the database file
						** and there are already pages in the cache (from a previous
						** read or write transaction).  Check to see if the database
						** has been modified.  If the database has changed, flush the
						** cache.
						**
						** Database changes is detected by looking at 15 bytes beginning
						** at offset 24 into the file.  The first 4 of these 16 bytes are
						** a 32-bit counter that is incremented with each change.  The
						** other bytes change randomly with each file change when
						** a codec is in use.
						** 
						** There is a vanishingly small chance that a change will not be 
						** detected.  The chance of an undetected change is so small that
						** it can be neglected.
						*/
						Pgno nPage = 0;
						char dbFileVers[sizeof(pPager->dbFileVers)];

						rc = pagerPagecount(pPager, &nPage);
						if( rc ) goto failed;

						if( nPage>0 ){
							IOTRACE(("CKVERS %p %d\n", pPager, sizeof(dbFileVers)));
							rc = sqlite3OsRead(pPager->fd, &dbFileVers, sizeof(dbFileVers), 24);
							if( rc!=SQLITE_OK ){
								goto failed;
							}
						}else{
							memset(dbFileVers, 0, sizeof(dbFileVers));
						}

						if( memcmp(pPager->dbFileVers, dbFileVers, sizeof(dbFileVers))!=0 ){
							pager_reset(pPager);
						}
				}

				/* If there is a WAL file in the file-system, open this database in WAL
				** mode. Otherwise, the following function call is a no-op.
				*/
				rc = pagerOpenWalIfPresent(pPager);
#ifndef SQLITE_OMIT_WAL
				assert( pPager->pWal==0 || rc==SQLITE_OK );
#endif
			}

			if( pagerUseWal(pPager) ){
				assert( rc==SQLITE_OK );
				rc = pagerBeginReadTransaction(pPager);
			}

			if( pPager->eState==PAGER_OPEN && rc==SQLITE_OK ){
				rc = pagerPagecount(pPager, &pPager->dbSize);
			}

failed:
			if( rc!=SQLITE_OK ){
				assert( !MEMDB );
				pager_unlock(pPager);
				assert( pPager->eState==PAGER_OPEN );
			}else{
				pPager->eState = PAGER_READER;
			}
			return rc;
		}

		static void pagerUnlockIfUnused(Pager *pPager){
			if( (sqlite3PcacheRefCount(pPager->pPCache)==0) ){
				pagerUnlockAndRollback(pPager);
			}
		}

		int sqlite3PagerAcquire(
			Pager *pPager,      /* The pager open on the database file */
			Pgno pgno,          /* Page number to fetch */
			DbPage **ppPage,    /* Write a pointer to the page here */
			int noContent       /* Do not bother reading content from disk if true */
			){
				int rc;
				PgHdr *pPg;

				assert( pPager->eState>=PAGER_READER );
				assert( assert_pager_state(pPager) );

				if( pgno==0 ){
					return SQLITE_CORRUPT_BKPT;
				}

				/* If the pager is in the error state, return an error immediately. 
				** Otherwise, request the page from the PCache layer. */
				if( pPager->errCode!=SQLITE_OK ){
					rc = pPager->errCode;
				}else{
					rc = sqlite3PcacheFetch(pPager->pPCache, pgno, 1, ppPage);
				}

				if( rc!=SQLITE_OK ){
					/* Either the call to sqlite3PcacheFetch() returned an error or the
					** pager was already in the error-state when this function was called.
					** Set pPg to 0 and jump to the exception handler.  */
					pPg = 0;
					goto pager_acquire_err;
				}
				assert( (*ppPage)->pgno==pgno );
				assert( (*ppPage)->pPager==pPager || (*ppPage)->pPager==0 );

				if( (*ppPage)->pPager && !noContent ){
					/* In this case the pcache already contains an initialized copy of
					** the page. Return without further ado.  */
					assert( pgno<=PAGER_MAX_PGNO && pgno!=PAGER_MJ_PGNO(pPager) );
					pPager->aStat[PAGER_STAT_HIT]++;
					return SQLITE_OK;

				}else{
					/* The pager cache has created a new page. Its content needs to 
					** be initialized.  */

					pPg = *ppPage;
					pPg->pPager = pPager;

					/* The maximum page number is 2^31. Return SQLITE_CORRUPT if a page
					** number greater than this, or the unused locking-page, is requested. */
					if( pgno>PAGER_MAX_PGNO || pgno==PAGER_MJ_PGNO(pPager) ){
						rc = SQLITE_CORRUPT_BKPT;
						goto pager_acquire_err;
					}

					if( MEMDB || pPager->dbSize<pgno || noContent || !isOpen(pPager->fd) ){
						if( pgno>pPager->mxPgno ){
							rc = SQLITE_FULL;
							goto pager_acquire_err;
						}
						if( noContent ){
							/* Failure to set the bits in the InJournal bit-vectors is benign.
							** It merely means that we might do some extra work to journal a 
							** page that does not need to be journaled.  Nevertheless, be sure 
							** to test the case where a malloc error occurs while trying to set 
							** a bit in a bit vector.
							*/
							sqlite3BeginBenignMalloc();
							if( pgno<=pPager->dbOrigSize ){
								TESTONLY( rc = ) sqlite3BitvecSet(pPager->pInJournal, pgno);
								testcase( rc==SQLITE_NOMEM );
							}
							TESTONLY( rc = ) addToSavepointBitvecs(pPager, pgno);
							testcase( rc==SQLITE_NOMEM );
							sqlite3EndBenignMalloc();
						}
						memset(pPg->pData, 0, pPager->pageSize);
						IOTRACE(("ZERO %p %d\n", pPager, pgno));
					}else{
						assert( pPg->pPager==pPager );
						pPager->aStat[PAGER_STAT_MISS]++;
						rc = readDbPage(pPg);
						if( rc!=SQLITE_OK ){
							goto pager_acquire_err;
						}
					}
					pager_set_pagehash(pPg);
				}

				return SQLITE_OK;

pager_acquire_err:
				assert( rc!=SQLITE_OK );
				if( pPg ){
					sqlite3PcacheDrop(pPg);
				}
				pagerUnlockIfUnused(pPager);

				*ppPage = 0;
				return rc;
		}

		DbPage *sqlite3PagerLookup(Pager *pPager, Pgno pgno){
			PgHdr *pPg = 0;
			assert( pPager!=0 );
			assert( pgno!=0 );
			assert( pPager->pPCache!=0 );
			assert( pPager->eState>=PAGER_READER && pPager->eState!=PAGER_ERROR );
			sqlite3PcacheFetch(pPager->pPCache, pgno, 0, &pPg);
			return pPg;
		}

		void sqlite3PagerUnref(DbPage *pPg){
			if( pPg ){
				Pager *pPager = pPg->pPager;
				sqlite3PcacheRelease(pPg);
				pagerUnlockIfUnused(pPager);
			}
		}

		static int pager_open_journal(Pager *pPager){
			int rc = SQLITE_OK;                        /* Return code */
			sqlite3_vfs * const pVfs = pPager->pVfs;   /* Local cache of vfs pointer */

			assert( pPager->eState==PAGER_WRITER_LOCKED );
			assert( assert_pager_state(pPager) );
			assert( pPager->pInJournal==0 );

			/* If already in the error state, this function is a no-op.  But on
			** the other hand, this routine is never called if we are already in
			** an error state. */
			if( NEVER(pPager->errCode) ) return pPager->errCode;

			if( !pagerUseWal(pPager) && pPager->journalMode!=PAGER_JOURNALMODE_OFF ){
				pPager->pInJournal = sqlite3BitvecCreate(pPager->dbSize);
				if( pPager->pInJournal==0 ){
					return SQLITE_NOMEM;
				}

				/* Open the journal file if it is not already open. */
				if( !isOpen(pPager->jfd) ){
					if( pPager->journalMode==PAGER_JOURNALMODE_MEMORY ){
						sqlite3MemJournalOpen(pPager->jfd);
					}else{
						const int flags =                   /* VFS flags to open journal file */
							SQLITE_OPEN_READWRITE|SQLITE_OPEN_CREATE|
							(pPager->tempFile ? 
							(SQLITE_OPEN_DELETEONCLOSE|SQLITE_OPEN_TEMP_JOURNAL):
						(SQLITE_OPEN_MAIN_JOURNAL)
							);
#ifdef SQLITE_ENABLE_ATOMIC_WRITE
						rc = sqlite3JournalOpen(
							pVfs, pPager->zJournal, pPager->jfd, flags, jrnlBufferSize(pPager)
							);
#else
						rc = sqlite3OsOpen(pVfs, pPager->zJournal, pPager->jfd, flags, 0);
#endif
					}
					assert( rc!=SQLITE_OK || isOpen(pPager->jfd) );
				}


				/* Write the first journal header to the journal file and open 
				** the sub-journal if necessary.
				*/
				if( rc==SQLITE_OK ){
					/* TODO: Check if all of these are really required. */
					pPager->nRec = 0;
					pPager->journalOff = 0;
					pPager->setMaster = 0;
					pPager->journalHdr = 0;
					rc = writeJournalHdr(pPager);
				}
			}

			if( rc!=SQLITE_OK ){
				sqlite3BitvecDestroy(pPager->pInJournal);
				pPager->pInJournal = 0;
			}else{
				assert( pPager->eState==PAGER_WRITER_LOCKED );
				pPager->eState = PAGER_WRITER_CACHEMOD;
			}

			return rc;
		}

		int sqlite3PagerBegin(Pager *pPager, int exFlag, int subjInMemory){
			int rc = SQLITE_OK;

			if( pPager->errCode ) return pPager->errCode;
			assert( pPager->eState>=PAGER_READER && pPager->eState<PAGER_ERROR );
			pPager->subjInMemory = (u8)subjInMemory;

			if( ALWAYS(pPager->eState==PAGER_READER) ){
				assert( pPager->pInJournal==0 );

				if( pagerUseWal(pPager) ){
					/* If the pager is configured to use locking_mode=exclusive, and an
					** exclusive lock on the database is not already held, obtain it now.
					*/
					if( pPager->exclusiveMode && sqlite3WalExclusiveMode(pPager->pWal, -1) ){
						rc = pagerLockDb(pPager, EXCLUSIVE_LOCK);
						if( rc!=SQLITE_OK ){
							return rc;
						}
						sqlite3WalExclusiveMode(pPager->pWal, 1);
					}

					/* Grab the write lock on the log file. If successful, upgrade to
					** PAGER_RESERVED state. Otherwise, return an error code to the caller.
					** The busy-handler is not invoked if another connection already
					** holds the write-lock. If possible, the upper layer will call it.
					*/
					rc = sqlite3WalBeginWriteTransaction(pPager->pWal);
				}else{
					/* Obtain a RESERVED lock on the database file. If the exFlag parameter
					** is true, then immediately upgrade this to an EXCLUSIVE lock. The
					** busy-handler callback can be used when upgrading to the EXCLUSIVE
					** lock, but not when obtaining the RESERVED lock.
					*/
					rc = pagerLockDb(pPager, RESERVED_LOCK);
					if( rc==SQLITE_OK && exFlag ){
						rc = pager_wait_on_lock(pPager, EXCLUSIVE_LOCK);
					}
				}

				if( rc==SQLITE_OK ){
					/* Change to WRITER_LOCKED state.
					**
					** WAL mode sets Pager.eState to PAGER_WRITER_LOCKED or CACHEMOD
					** when it has an open transaction, but never to DBMOD or FINISHED.
					** This is because in those states the code to roll back savepoint 
					** transactions may copy data from the sub-journal into the database 
					** file as well as into the page cache. Which would be incorrect in 
					** WAL mode.
					*/
					pPager->eState = PAGER_WRITER_LOCKED;
					pPager->dbHintSize = pPager->dbSize;
					pPager->dbFileSize = pPager->dbSize;
					pPager->dbOrigSize = pPager->dbSize;
					pPager->journalOff = 0;
				}

				assert( rc==SQLITE_OK || pPager->eState==PAGER_READER );
				assert( rc!=SQLITE_OK || pPager->eState==PAGER_WRITER_LOCKED );
				assert( assert_pager_state(pPager) );
			}

			PAGERTRACE(("TRANSACTION %d\n", PAGERID(pPager)));
			return rc;
		}

		static int pager_write(PgHdr *pg)
		{
			void *data = pg->Data;
			Pager *pager = pg->Pager;
			RC rc = RC::OK;

			// This routine is not called unless a write-transaction has already been started. The journal file may or may not be open at this point. It is never called in the ERROR state.
			_assert(pager->State == PAGER::WRITER_LOCKED
				|| pager->State == PAGER::WRITER_CACHEMOD
				|| pager->State == PAGER::WRITER_DBMOD);
			_assert(assert_pager_state(pager));

			// If an error has been previously detected, report the same error again. This should not happen, but the check provides robustness.
			if (SysEx_NEVER(pager->ErrorCode)) return pager->ErrorCode;

			// Higher-level routines never call this function if database is not writable.  But check anyway, just for robustness.
			if (SysEx_NEVER(pager->ReadOnly)) return RC::PERM;

			CHECK_PAGE(pg);

			// The journal file needs to be opened. Higher level routines have already obtained the necessary locks to begin the write-transaction, but the
			// rollback journal might not yet be open. Open it now if this is the case.
			//
			// This is done before calling sqlite3PcacheMakeDirty() on the page. Otherwise, if it were done after calling sqlite3PcacheMakeDirty(), then
			// an error might occur and the pager would end up in WRITER_LOCKED state with pages marked as dirty in the cache.
			if (pager->State == PAGER::WRITER_LOCKED)
			{
				rc = pager_open_journal(pager);
				if (rc != RC::OK) return rc;
			}
			_assert(pager->State >= PAGER::WRITER_CACHEMOD);
			_assert(assert_pager_state(pager));

			// Mark the page as dirty.  If the page has already been written to the journal then we can return right away.
			sqlite3PcacheMakeDirty(pg);
			if (pageInJournal(pg) && !subjRequiresPage(pg))
				_assert(!UseWal(pager));
			else
			{
				// The transaction journal now exists and we have a RESERVED or an EXCLUSIVE lock on the main database file.  Write the current page to the transaction journal if it is not there already.
				if (!pageInJournal(pg) && !UseWal(pager))
				{
					_assert(UseWal(pager) == 0);
					if (pg->ID <= pager->DBOrigSize && pager->JournalFile->Opened)
					{
						// We should never write to the journal file the page that contains the database locks.  The following assert verifies that we do not.
						_assert(pg->ID != PAGER_MJ_PGNO(pager));

						_assert(pager->JournalHeader <= pager->JournalOffset);
						CODEC2(pager, data, pg->ID, 7, return RC::NOMEM, data2);
						char *data2;
						uint32 cksum = pager_cksum(pager, (uint8 *)data2);

						// Even if an IO or diskfull error occurs while journalling the page in the block above, set the need-sync flag for the page.
						// Otherwise, when the transaction is rolled back, the logic in playback_one_page() will think that the page needs to be restored
						// in the database file. And if an IO error occurs while doing so, then corruption may follow.
						pg->Flags |= PgHdr::PGHDR::NEED_SYNC;

						int64 offset = pager->JournalOffset;
						rc = pPager->JournalFile->Write4(offset, pg->ID);
						if (rc != RC::OK) return rc;
						rc = pager->JournalFile->Write(data2, pager->PageSize, offset + 4);
						if (rc != RC::OK) return rc;
						rc = pager->JournalFile->Write4(offset + pager->PageSize + 4, checksum);
						if (rc != RC::OK) return rc;

						IOTRACE("JOUT %p %d %lld %d\n", pager, pg->ID, pager->JournalOffset, pager->PageSize);
						PAGER_INCR(sqlite3_pager_writej_count);
						PAGERTRACE("JOURNAL %d page %d needSync=%d hash(%08x)\n", PAGERID(pager), pg->ID, ((pg->Flags & PgHdr::PGHDR::NEED_SYNC) ? 1 : 0), pager_pagehash(pg)));

						pager->JournalOff += 8 + pager->PageSize;
						pager->Records++;
						_assert(pager->InJournal != 0);
						rc = pager->InJournal->Set(pg->ID);
						ASSERTCOVERAGE(rc == RC::SQLITE);
						_assert(rc == RC::OK || rc == RC::NOMEM);
						rc |= addToSavepointBitvecs(pager, pg->ID);
						if (rc != RC::OK)
						{
							_assert(rc == RC::NOMEM);
							return rc;
						}
					}
					else
					{
						if (pager->State != PAGER::WRITER_DBMOD)
							pg->Flags |= PgHdr::PGHDR::NEED_SYNC;
						PAGERTRACE("APPEND %d page %d needSync=%d\n", PAGERID(pager), pg->ID, ((pg->Flags & PgHdr::PGHDR_NEED_SYNC) ? 1 : 0)));
					}
				}

				// If the statement journal is open and the page is not in it, then write the current page to the statement journal.  Note that
				// the statement journal format differs from the standard journal format in that it omits the checksums and the header.
				if (subjRequiresPage(pg))
					rc = subjournalPage(pg);
			}

			// Update the database size and return.
			if (pager->DBSize < pg->ID)
				pager->DBSize = pg->ID;
			return rc;
		}

		int sqlite3PagerWrite(DbPage *dbPage)
		{
			RC rc = RC::OK;

			PgHdr *pg = dbPage;
			Pager *pager = pg->Pager;
			Pid pagePerSector = (pager->SectorSize / pager->PageSize);

			_assert(pager->State >= PAGER::WRITER_LOCKED);
			_assert(pager->State != PAGER::ERROR);
			_assert(assert_pager_state(pager));

			if (pagePerSector > 1)
			{
				// Set the doNotSyncSpill flag to 1. This is because we cannot allow a journal header to be written between the pages journaled by this function.
				_assert(!pager->MemoryDB);
				_assert(pager->DoNotSyncSpill == 0);
				pager->DoNotSyncSpill++;

				// This trick assumes that both the page-size and sector-size are an integer power of 2. It sets variable pg1 to the identifier of the first page of the sector pPg is located on.
				Pid pg1 = ((pg->ID - 1) & ~(nagePerSector - 1)) + 1; // First page of the sector pPg is located on.

				int pages = 0; // Number of pages starting at pg1 to journal
				Pid pageCount = pager->DBSize; // Total number of pages in database file
				if (pg->ID > pageCount)
					pages = (pg->ID - pg1) + 1;
				else if ((pg1 + pagePerSector - 1) > nageCount)
					pages = pageCount + 1 - pg1;
				else
					pages = pagePerSector;
				_assert(pages > 0);
				_assert(pg1 <= pg->ID);
				_assert((pg1 + pages) > pg->ID);

				bool needSync = false; // True if any page has PGHDR_NEED_SYNC
				for (int ii = 0; ii < pages && rc == RC::OK; ii++)
				{
					PID id = pg1 + ii;
					PgHdr *page;
					if (id == pg->ID || !sqlite3BitvecTest(pager->InJournal->Get(id))
					{
						if (id != PAGER_MJ_PGNO(pager))
						{
							rc = sqlite3PagerGet(pager, id, &page);
							if (rc == RC::OK)
							{
								rc = pager_write(page);
								if ( age->Flags & PgHdr::PGHDR::NEED_SYNC)
									needSync = true;
								sqlite3PagerUnref(page);
							}
						}
					}
					else if ((page = pager_lookup(pager, id)) != 0)
					{
						if (page->Flags & PgHdr::PGHDR::NEED_SYNC)
							needSync = true;
						sqlite3PagerUnref(page);
					}
				}

				// If the PGHDR_NEED_SYNC flag is set for any of the nPage pages starting at pg1, then it needs to be set for all of them. Because
				// writing to any of these nPage pages may damage the others, the journal file must contain sync()ed copies of all of them
				// before any of them can be written out to the database file.
				if (rc == RC::OK && needSync)
				{
					_assert(!pager->MemoryDB);
					for (int ii = 0; ii < pages; ii++)
					{
						PgHdr *page = pager_lookup(pager, pg1 + ii);
						if (page)
						{
							page->Flags |= PgHdr::PGHDR::NEED_SYNC;
							sqlite3PagerUnref(page);
						}
					}
				}

				_assert(pager->DoNotSyncSpill == 1);
				pager->DoNotSyncSpill--;
			}
			else
				rc = pager_write(dbPage);
			return rc;
		}

#ifndef DEBUG
		int sqlite3PagerIswriteable(DbPage *pg)
		{
			return pg->Flags & PGHDR_DIRTY;
		}
#endif

		void sqlite3PagerDontWrite(PgHdr *pg)
		{
			Pager *pager = pg->Pager;
			if ((pg->Flags & PgHdr::PGHDR::DIRTY) && __arrayLength(pager->Savepoints) == 0)
			{
				PAGERTRACE("DONT_WRITE page %d of %d\n", pg->ID, PAGERID(pager));
				IOTRACE("CLEAN %p %d\n", pager, pg->ID);
				pg->Flags |= PgHdr::PGHDR::DONT_WRITE;
				pager_set_pagehash(pg);
			}
		}

		static RC pager_incr_changecounter(Pager *pager, int isDirectMode)
		{
			_assert(pager->State == PAGER::WRITER_CACHEMOD
				|| pager->State == PAGER::WRITER_DBMOD);
			_assert(assert_pager_state(pager));

			// Declare and initialize constant integer 'isDirect'. If the atomic-write optimization is enabled in this build, then isDirect
			// is initialized to the value passed as the isDirectMode parameter to this function. Otherwise, it is always set to zero.
			//
			// The idea is that if the atomic-write optimization is not enabled at compile time, the compiler can omit the tests of
			// 'isDirect' below, as well as the block enclosed in the "if( isDirect )" condition.
#ifndef ENABLE_ATOMIC_WRITE
#define DIRECT_MODE 0
			assert(isDirectMode==0);
			UNUSED_PARAMETER(isDirectMode);
#else
#define DIRECT_MODE isDirectMode
#endif
			RC rc = RC::OK;
			if (!pager->ChangeCountDone && SysEx::ALWAYS(pager->DBSize > 0))
			{
				_assert(!pager->TempFile && pager->File->Opened);

				// Open page 1 of the file for writing.
				PgHdr *pgHdr; // Reference to page 1
				rc = sqlite3PagerGet(pager, 1, &pgHdr);
				_assert(pgHdr == nullptr || rc == RC::OK);

				// If page one was fetched successfully, and this function is not operating in direct-mode, make page 1 writable.  When not in 
				// direct mode, page 1 is always held in cache and hence the PagerGet() above is always successful - hence the ALWAYS on rc==SQLITE_OK.
				if (!DIRECT_MODE && SysEx_ALWAYS(rc == RC::OK))
					rc = sqlite3PagerWrite(pgHdr);

				if (rc == RC::OK)
				{
					// Actually do the update of the change counter
					pager_write_changecounter(pgHdr);

					// If running in direct mode, write the contents of page 1 to the file.
					if (DIRECT_MODE)
					{
						const void *buf;
						assert(pager->DBFileSize > 0);
						CODEC2(pager, pgHdr->Data, 1, 6, rc = RC::NOMEM, buf);
						if (rc == RC::OK)
						{
							rc = pager->File->Write(buf, pager->PageSize, 0);
							pager->Stats[IPager::STAT::WRITE]++;
						}
						if (rc == RC::OK)
							pager->ChangeCountDone = 1;
					}
					else
					{
						pager->ChangeCountDone = 1;
					}
				}

				// Release the page reference.
				sqlite3PagerUnref(pgHdr);
			}
			return rc;
		}

		RC sqlite3PagerSync(Pager *pager)
		{
			RC rc = RC::OK;
			if (!pager->NSync)
			{
				_assert(!pager->MemoryDB);
				rc = pager->File->Sync(pager->SyncFlags);
			}
			else if (pager->File->Opened)
			{
				_assert(!pager->MemoryDB);
				rc = pager->File->FileControl(FCNT::SYNC_OMITTED, 0);
				if (rc == RC::NOTFOUND)
					rc = RC::OK;
			}
			return rc;
		}

		RC sqlite3PagerExclusiveLock(Pager *pager)
		{
			_assert(pager->State == PAGER::WRITER_CACHEMOD || 
				pager->State == PAGER::WRITER_DBMOD ||
				pager->State == PAGER::WRITER_LOCKED);
			_assert(assert_pager_state(pager));
			RC rc = RC::OK;
			if (!UseWal(pager))
				rc = pager_wait_on_lock(pager, EXCLUSIVE_LOCK);
			return rc;
		}

		RC sqlite3PagerCommitPhaseOne(Pager *pager, const char *master, int noSync)
		{
			_assert(pager->State == PAGER::WRITER_LOCKED ||
				pager->State == PAGER::WRITER_CACHEMOD ||
				pager->State == PAGER::WRITER_DBMOD ||
				pager->State == PAGER::ERROR);
			_assert(assert_pager_state(pager));

			// If a prior error occurred, report that error again.
			if (SysEx_NEVER(pager->ErrorCode)) return pager->ErrorCode;

			PAGERTRACE("DATABASE SYNC: File=%s zMaster=%s nSize=%d\n", pager->Filename, zaster, pager->DBSize);

			// If no database changes have been made, return early.
			if (pager->State < PAGER::WRITER_CACHEMOD) return RC::OK;

			RC rc = RC::OK;
			if (pager->MemoryDB)
			{
				// If this is an in-memory db, or no pages have been written to, or this function has already been called, it is mostly a no-op.  However, any
				// backup in progress needs to be restarted.
				sqlite3BackupRestart(pager->Backup);
			}
			else
			{
				if (UseWal(pager))
				{
					PgHdr *list = PCache_DirtyList(pager->PCache);
					PgHdr *pageOne = nullptr;
					if (list == 0)
					{
						// Must have at least one page for the WAL commit flag. Ticket [2d1a5c67dfc2363e44f29d9bbd57f] 2011-05-18
						rc = sqlite3PagerGet(pager, 1, &pageOne);
						pist = pageOne;
						pist->Dirty = nullptr;
					}
					_assert(rc == RC::OK);
					if (SysEx_ALWAYS(list))
						rc = pagerWalFrames(pager, list, pager->DBSize, 1);
					sqlite3PagerUnref(pageOne);
					if (rc == RC::OK)
					{
						PCache_CleanAll(pager->PCache);
					}
				}
				else
				{
					// The following block updates the change-counter. Exactly how it does this depends on whether or not the atomic-update optimization
					// was enabled at compile time, and if this transaction meets the runtime criteria to use the operation: 
					//
					//    * The file-system supports the atomic-write property for blocks of size page-size, and 
					//    * This commit is not part of a multi-file transaction, and
					//    * Exactly one page has been modified and store in the journal file.
					//
					// If the optimization was not enabled at compile time, then the pager_incr_changecounter() function is called to update the change
					// counter in 'indirect-mode'. If the optimization is compiled in but is not applicable to this transaction, call sqlite3JournalCreate()
					// to make sure the journal file has actually been created, then call pager_incr_changecounter() to update the change-counter in indirect mode. 
					//
					// Otherwise, if the optimization is both enabled and applicable, then call pager_incr_changecounter() to update the change-counter
					// in 'direct' mode. In this case the journal file will never be created for this transaction.
#ifdef ENABLE_ATOMIC_WRITE
					PgHdr *pg;
					_assert(pager->JournalFile->Opened ||
						pager->JournalMode == PAGER::JOURNALMODE_OFF ||
						pager->JournalMode == PAGER::JOURNALMODE_WAL);
					if (!master && pager->JournalFile->Opened &&
						pager->JournalOffset == jrnlBufferSize(pager) &&
						pager->DBSize >= pager->DBOrigSize &&
						((pg = Pcache_DirtyList(pager->PCache)) == nullptr || pg->Dirty == nullptr))
					{
						// Update the db file change counter via the direct-write method. The following call will modify the in-memory representation of page 1 
						// to include the updated change counter and then write page 1 directly to the database file. Because of the atomic-write 
						// property of the host file-system, this is safe.
						rc = pager_incr_changecounter(pager, 1);
					}
					else
					{
						rc = sqlite3JournalCreate(pPager->jfd);
						if (rc RC::OK)
							rc = pager_incr_changecounter(pager, 0);
					}
#else
					rc = pager_incr_changecounter(pager, 0);
#endif
					if (rc != RC::OK) goto commit_phase_one_exit;

					// Write the master journal name into the journal file. If a master journal file name has already been written to the journal file, 
					// or if zMaster is NULL (no master journal), then this call is a no-op.
					rc = writeMasterJournal(pager, master);
					if (rc != RC::OK) goto commit_phase_one_exit;

					// Sync the journal file and write all dirty pages to the database. If the atomic-update optimization is being used, this sync will not 
					// create the journal file or perform any real IO.
					//
					// Because the change-counter page was just modified, unless the atomic-update optimization is used it is almost certain that the
					// journal requires a sync here. However, in locking_mode=exclusive on a system under memory pressure it is just possible that this is 
					// not the case. In this case it is likely enough that the redundant xSync() call will be changed to a no-op by the OS anyhow. 
					rc = syncJournal(pager, 0);
					if (rc != RC::OK) goto commit_phase_one_exit;

					rc = pager_write_pagelist(pager, PCache_DirtyList(pager->PCache));
					if (rc != RC::OK)
					{
						_assert(rc != RC::IOERR_BLOCKED);
						goto commit_phase_one_exit;
					}
					sqlite3PcacheCleanAll(pager->PCache);

					// If the file on disk is smaller than the database image, use pager_truncate to grow the file here. This can happen if the database
					// image was extended as part of the current transaction and then the last page in the db image moved to the free-list. In this case the
					// last page is never written out to disk, leaving the database file undersized. Fix this now if it is the case.
					if (pager->DBSize > pager->DBFileSize)
					{
						Pid newID = pager->DBSize - (pager->DBSize == PAGER_MJ_PGNO(pager));
						_assert(pager->State == PAGER::WRITER_DBMOD);
						rc = pager_truncate(pager, newID);
						if (rc != RC::OK) goto commit_phase_one_exit;
					}

					// Finally, sync the database file.
					if (!noSync)
						rc = sqlite3PagerSync(pager);
					IOTRACE("DBSYNC %p\n", pager);
				}
			}

commit_phase_one_exit:
			if (rc == RC::OK && !UseWal(pager))
				pager->State = PAGER::WRITER_FINISHED;
			return rc;
		}

		RC sqlite3PagerCommitPhaseTwo(Pager *pager)
		{
			// This routine should not be called if a prior error has occurred. But if (due to a coding error elsewhere in the system) it does get
			// called, just return the same error code without doing anything.
			if (SysEx_NEVER(pager->ErrorCode)) return pager->ErrorCode;
			_assert(pager->State == PAGER::WRITER_LOCKED ||
				pager->State == PAGER::WRITER_FINISHED ||
				(UseWal(pager) && pager->State == PAGER::WRITER_CACHEMOD));
			_assert(assert_pager_state(pager));

			// An optimization. If the database was not actually modified during this transaction, the pager is running in exclusive-mode and is
			// using persistent journals, then this function is a no-op.
			//
			// The start of the journal file currently contains a single journal header with the nRec field set to 0. If such a journal is used as
			// a hot-journal during hot-journal rollback, 0 changes will be made to the database file. So there is no need to zero the journal 
			// header. Since the pager is in exclusive mode, there is no need to drop any locks either.
			if (pager->State == PAGER::WRITER_LOCKED &&
				pager->ExclusiveMode &&
				pager->JournalMode == PAGER::JOURNALMODE_PERSIST)
			{
				_assert(pager->JournalOffset == JOURNAL_HDR_SZ(pager) || !pager->JournalOffset);
				pager->State = PAGER::READER;
				return RC::OK;
			}

			PAGERTRACE(("COMMIT %d\n", PAGERID(pager)));
			RC rc = pager_end_transaction(pager, pager->SetMaster, 1);
			return pager_error(pager, rc);
		}

		RC sqlite3PagerRollback(Pager *pager)
		{
			PAGERTRACE("ROLLBACK %d\n", PAGERID(pager));
			// PagerRollback() is a no-op if called in READER or OPEN state. If the pager is already in the ERROR state, the rollback is not attempted here. Instead, the error code is returned to the caller.
			_assert(assert_pager_state(pager));
			if (pager->State == PAGER::ERROR) return pager->ErrorCode;
			if (pager->State <= PAGER::READER) return RC::OK;

			RC rc = RC::OK;
			if (UseWal(pager))
			{
				rc = sqlite3PagerSavepoint(pager, SAVEPOINT::ROLLBACK, -1);
				RC rc2 = pager_end_transaction(pager, pager->SetMaster, 0);
				if (rc == RC::OK) rc = rc2;
			}
			else if (!pager->JournalFile->Opened || pager->State == PAGER::WRITER_LOCKED)
			{
				PAGER state = pager->State;
				rc = pager_end_transaction(pager, 0, 0);
				if (!pager->MemoryDB && state > PAGER::WRITER_LOCKED)
				{
					// This can happen using journal_mode=off. Move the pager to the error state to indicate that the contents of the cache may not be trusted. Any active readers will get SQLITE_ABORT.
					pager->ErrorCode = RC::ABORT;
					pager->State = PAGER::ERROR;
					return rc;
				}
			}
			else
				rc = pager_playback(pPager, 0);

			_assert(pager->State == PAGER::READER || rc != RC::OK);
			_assert(rc == RC::OK || rc == RC::FULL || rc == RC::NOMEM || (rc & 0xFF) == RC::IOERR);

			// If an error occurs during a ROLLBACK, we can no longer trust the pager cache. So call pager_error() on the way out to make any error persistent.
			return pager_error(pPager, rc);
		}

		uint8 sqlite3PagerIsreadonly(Pager *pager)
		{
			return pager->ReadOnly;
		}

		int sqlite3PagerRefcount(Pager *pager)
		{
			return PCache_RefCount(pager->PCache);
		}

		int sqlite3PagerMemUsed(Pager *pager)
		{
			int perPageSize = pager->PageSize + pager->Extras + sizeof(PgHdr) + 5 * sizeof(void *);
			return perPageSize * PCache_Pagecount(pager->PCache) + SysEx::AllocSize(pager) + pager->PageSize;
		}

		int sqlite3PagerPageRefcount(DbPage *page)
		{
			return sqlite3PcachePageRefcount(page);
		}

#ifdef TEST
		int *sqlite3PagerStats(Pager *pager)
		{
			static int a[11];
			a[0] = PCache_RefCount(pager->PCache);
			a[1] = PCache_Pagecount(pager->PCache);
			a[2] = PCache_GetCachesize(pager->PCache);
			a[3] = pager->eState == (PAGER::OPEN ? -1 : (int)pager->DBSize);
			a[4] = pager->eState;
			a[5] = pager->errCode;
			a[6] = pager->aStat[PAGER_STAT_HIT];
			a[7] = pager->aStat[PAGER_STAT_MISS];
			a[8] = 0;  // Used to be pPager->nOvfl
			a[9] = pager->Reads;
			a[10] = pager->Stats[PAGER_STAT_WRITE];
			return a;
		}
#endif

		void sqlite3PagerCacheStat(Pager *pager, DBSTATUS dbStatus, bool reset, int *valueOut)
		{
			_assert(dbStatus == DBSTATUS::CACHE_HIT ||
				dbStatus == DBSTATUS::CACHE_MISS ||
				dbStatus == DBSTATUS::CACHE_WRITE);
			_assert(DBSTATUS::CACHE_HIT + 1 == DBSTATUS::CACHE_MISS);
			_assert(DBSTATUS::CACHE_HIT + 2 == DBSTATUS::CACHE_WRITE);
			_assert(PAGER_STAT_HIT == 0 &&
				PAGER_STAT_MISS == 1 &&
				PAGER_STAT_WRITE == 2);

			*valueOut += pager->Stats[dbStatus - DBSTATUS::CACHE_HIT];
			if (reset)
				pager->Stats[dbStatus - DBSTATUS::CACHE_HIT] = 0;
		}

		int sqlite3PagerIsMemdb(Pager *pager)
		{
			return pager->MemoryDB;
		}

		int sqlite3PagerOpenSavepoint(Pager *pager, int savepoints)
		{
			_assert(pager->State >= PAGER::WRITER_LOCKED);
			_assert(assert_pager_state(pager));

			RC rc = RC::OK;
			int currentSavepoints = __arrayLength(pager->Savepoints); // Current number of savepoints
			if (savepoints > currentSavepoints && pager->UseJournal)
			{
				// Grow the Pager.aSavepoint array using realloc(). Return SQLITE_NOMEM if the allocation fails. Otherwise, zero the new portion in case a 
				// malloc failure occurs while populating it in the for(...) loop below.
				PagerSavepoint *newSavepoints = (PagerSavepoint *)SysEx::Realloc(pager->Savepoints, sizeof(PagerSavepoint) * savepoints); // New Pager.Savepoints array
				if (!newSavepoints)
					return RC::NOMEM;
				_memset(&newSavepoints[currentSavepoints], 0, (savepoints - currentSavepoints) * sizeof(PagerSavepoint));
				pager->Savepoints = newSavepoints;

				// Populate the PagerSavepoint structures just allocated.
				for (int ii = currentSavepoints; ii < savepoints; ii++)
				{
					newSavepoints[ii].Orig = pager->DBSize;
					newSavepoints[ii].Offset = (pager->JournalFile->Opened && pager->JournalOffset > 0 ? pager->JournalOffset : JOURNAL_HDR_SZ(pager));
					newSavepoints[ii].SubRecords = pager->SubRecords;
					newSavepoints[ii].InSavepoint = Bitvec::Create(pager->DBSize);
					if (!newSavepoints[ii].InSavepoint)
						return RC::NOMEM;
					if (UseWal(pager))
						sqlite3WalSavepoint(pager->Wal, newSavepoints[ii].WalData);
					__arraySetLength(pager->Savepoints, ii + 1);
				}
				_assert(__arrayLength(pager->Savepoints) == savepoints);
				assertTruncateConstraint(pager);
			}
			return rc;
		}

		int sqlite3PagerSavepoint(Pager *pager, int op, int savepoints)
		{
			_assert(op == SAVEPOINT::RELEASE || op == SAVEPOINT::ROLLBACK);
			_assert(savepoints >= 0 || op == SAVEPOINT::ROLLBACK);
			RC rc = pager->ErrorCode;
			if (rc == RC::OK && savepoints < __arrayLength(pager->Savepoints))
			{
				// Figure out how many savepoints will still be active after this operation. Store this value in nNew. Then free resources associated 
				// with any savepoints that are destroyed by this operation.
				int newLength = savepoints + (op == SAVEPOINT::RELEASE ? 0 : 1); // Number of remaining savepoints after this op.
				for (int ii = newLength; ii < __arrayLength(pager->Savepoints); ii++)
					Bitvec::Destroy(pager->Savepoints[ii].InSavepoint);
				__arraySetLength(pager->Savepoints, newLength);

				// If this is a release of the outermost savepoint, truncate the sub-journal to zero bytes in size.
				if (op == SAVEPOINT::RELEASE)
				{
					if (newLength == 0 && pager->SubJournalFile->Opened)
					{
						// Only truncate if it is an in-memory sub-journal.
						if (pager->SubJournalFile->IsMemJournal)
						{
							rc = pager->SubJournalFile->Truncate(0);
							_assert(rc == RC::OK);
						}
						pager->SubRecords = 0;
					}
				}
				// Else this is a rollback operation, playback the specified savepoint. If this is a temp-file, it is possible that the journal file has
				// not yet been opened. In this case there have been no changes to the database file, so the playback operation can be skipped.
				else if (UseWal(pager) || pager->JournalFile->Opened)
				{
					PagerSavepoint *savepoint = (newLength == 0 ? nullptr : &pager->Savepoints[newLength - 1]);
					rc = pagerPlaybackSavepoint(pager, savepoint);
					_assert(rc != RC::DONE);
				}
			}
			return rc;
		}

		const char *sqlite3PagerFilename(Pager *pager, int nullIfMemDb)
		{
			return (nullIfMemDb && pager->MemoryDB ? "" : pager->Filename);
		}

		const VFileSystem *sqlite3PagerVfs(Pager *pager)
		{
			return pager->Vfs;
		}

		VFile *sqlite3PagerFile(Pager *pager)
		{
			return pager->File;
		}

		const char *sqlite3PagerJournalname(Pager *pager)
		{
			return pager->Journal;
		}

		int sqlite3PagerNosync(Pager *pager)
		{
			return pager->NoSync;
		}

#ifndef HAS_CODEC
		void sqlite3PagerSetCodec(Pager *pager, void *(*codec)(void *,void *, Pid, int), void (*codecSizeChange)(void *, int, int), void (*codecFree)(void *), void *codecArg)
		{
			if (pager->CodecFree) pager->CodecFree(pager->Codec);
			pager->Codec = (pager->MemoryDB ? nullptr : codec);
			pager->CodecSizeChng = codecSizeChange;
			pager->CodecFree = codecFree;
			pager->CodecArg = codecArg;
			pagerReportSize(pager);
		}
		void *sqlite3PagerGetCodec(Pager *pager)
		{
			return pager->Codec;
		}
#endif

#ifndef OMIT_AUTOVACUUM
		int sqlite3PagerMovepage(Pager *pager, DbPage *pg, Pid id, int isCommit)
		{
			_assert(pg->Refs > 0);
			_assert(pager->State == PAGER::WRITER_CACHEMOD ||
				pager->State==PAGER::WRITER_DBMOD);
			_assert(assert_pager_state(pager));

			// In order to be able to rollback, an in-memory database must journal the page we are moving from.
			RC rc;
			if (pager->MemoryDB)
			{
				rc = sqlite3PagerWrite(pPg);
				if (rc) return rc;
			}

			// If the page being moved is dirty and has not been saved by the latest savepoint, then save the current contents of the page into the 
			// sub-journal now. This is required to handle the following scenario:
			//
			//   BEGIN;
			//     <journal page X, then modify it in memory>
			//     SAVEPOINT one;
			//       <Move page X to location Y>
			//     ROLLBACK TO one;
			//
			// If page X were not written to the sub-journal here, it would not be possible to restore its contents when the "ROLLBACK TO one"
			// statement were is processed.
			//
			// subjournalPage() may need to allocate space to store pPg->pgno into one or more savepoint bitvecs. This is the reason this function
			// may return SQLITE_NOMEM.
			if (pg->Flags & PgHdr::PGHDR_DIRTY
				&& subjRequiresPage(pg)
				&& (rc = subjournalPage(pg) != RC::OK))
				return rc;

			SysEx_PAGERTRACE(("MOVE %d page %d (needSync=%d) moves to %d\n", PAGERID(pager), pg->ID, (pg->Flags & PgHdr::PGHDR::NEED_SYNC) ? 1 : 0, id));
			SysEx_IOTRACE(("MOVE %p %d %d\n", pager, pg->ID, id));

			// If the journal needs to be sync()ed before page pPg->pgno can be written to, store pPg->pgno in local variable needSyncPgno.
			//
			// If the isCommit flag is set, there is no need to remember that the journal needs to be sync()ed before database page pPg->pgno 
			// can be written to. The caller has already promised not to write to it.
			Pid needSyncID = 0; // Old value of pPg->pgno, if sync is required
			if ((pg->Flags & PgHdr::PGHDR::NEED_SYNC) && !isCommit)
			{
				needSyncID = pg->ID;
				_assert(pager->JournalMode == PAGER::JOURNALMODE_OFF || pageInJournal(pg) || pg->ID > pager->DBOrigSize);
				_assert(pg->Flags & PgHdr::PGHDR::DIRTY);
			}

			// If the cache contains a page with page-number pgno, remove it from its hash chain. Also, if the PGHDR_NEED_SYNC flag was set for 
			// page pgno before the 'move' operation, it needs to be retained for the page moved there.
			pg->Flags &= ~PgHdr::PGHDR::NEED_SYNC;
			PgHdr *pgOld = pager_lookup(pager, id); // The page being overwritten.
			_assert(!pgOld || pgOld->Refs == 1);
			if (pgOld)
			{
				pg->Flags |= (pgOld->Flags & PgHdr::PGHDR::NEED_SYNC);
				if (pager->MemoryDB)
				{
					// Do not discard pages from an in-memory database since we might need to rollback later.  Just move the page out of the way.
					sqlite3PcacheMove(pgOld, pager->DBSize + 1);
				}
				else
					sqlite3PcacheDrop(pPgOld);
			}

			Pid origID = pg->ID; // The original page number
			PCache_Move(pg, id);
			PCache_MakeDirty(pg);

			// For an in-memory database, make sure the original page continues to exist, in case the transaction needs to roll back.  Use pPgOld
			// as the original page since it has already been allocated.
			if (pager->MemoryDB)
			{
				assert(pgOld);
				PCache_Move(pgOld, origID);
				sqlite3PagerUnref(pgOld);
			}

			if (needSyncPgno)
			{
				// If needSyncPgno is non-zero, then the journal file needs to be sync()ed before any data is written to database file page needSyncPgno.
				// Currently, no such page exists in the page-cache and the "is journaled" bitvec flag has been set. This needs to be remedied by
				// loading the page into the pager-cache and setting the PGHDR_NEED_SYNC flag.
				//
				// If the attempt to load the page into the page-cache fails, (due to a malloc() or IO failure), clear the bit in the pInJournal[]
				// array. Otherwise, if the page is loaded and written again in this transaction, it may be written to the database file before
				// it is synced into the journal file. This way, it may end up in the journal file twice, but that is not a problem.
				PgHdr *pgHdr;
				rc = sqlite3PagerGet(pager, needSyncID, &pgHdr);
				if (rc != RC::OK)
				{
					if (needSyncID <= pager->DBOrigSize)
					{
						_assert(pager->TmpSpace != nullptr);
						pager->InJournal->Clear(needSyncID, pager->TmpSpace);
					}
					return rc;
				}
				pgHdr->Flags |= PgHdr::PGHDR::NEED_SYNC;
				PCache_MakeDirty(pgHdr);
				sqlite3PagerUnref(pgHdr);
			}
			return RC::OK;
		}
#endif

		void *sqlite3PagerGetData(DbPage *pg)
		{
			_assert(pg->Ref > 0 || pg->Pager->MemoryDB);
			return pg->Data;
		}

		void *sqlite3PagerGetExtra(DbPage *pg)
		{
			return pg->Extra;
		}

		int sqlite3PagerLockingMode(Pager *pager, int mode)
		{
			_assert(mode == PAGER::LOCKINGMODE_QUERY ||
				mode == PAGER::LOCKINGMODE_NORMAL ||
				mode == PAGER::LOCKINGMODE_EXCLUSIVE);
			_assert(PAGER::LOCKINGMODE_QUERY < 0);
			_assert(PAGER::LOCKINGMODE_NORMAL >= 0 && PAGER::LOCKINGMODE_EXCLUSIVE >= 0);
			_assert(pager->ExclusiveMode || sqlite3WalHeapMemory(pager->Wal) == 0);
			if (mode >= 0 && !pager->TempFile && !sqlite3WalHeapMemory(pager->Wal))
				pager->ExclusiveMode = (uint8)mode;
			return (int)pager->ExclusiveMode;
		}

		int sqlite3PagerSetJournalMode(Pager *pPager, int mode)
		{
			uint8 old = pager->JournalMode; // Prior journalmode

#ifdef _DEBUG
			// The print_pager_state() routine is intended to be used by the debugger only.  We invoke it once here to suppress a compiler warning.
			print_pager_state(pager);
#endif

			// The mode parameter is always valid
			_assert(mode == IPager::JOURNALMODE::DELETE ||
				mode == IPager::JOURNALMODE::TRUNCATE ||
				mode == IPager::JOURNALMODE::PERSIST ||
				mode == IPager::JOURNALMODE::OFF ||
				mode == IPager::JOURNALMODE::WAL ||
				mode == IPager::JOURNALMODE::JMEMORY);

			// This routine is only called from the OP_JournalMode opcode, and the logic there will never allow a temporary file to be changed to WAL mode.
			_assert(pager->TempFile == nullptr || mode != IPager::JOURNALMODE::WAL);

			// Do allow the journalmode of an in-memory database to be set to anything other than MEMORY or OFF
			if (pager->MemoryDB)
			{
				_assert(old == IPager::JOURNALMODE::JMEMORY || old == IPager::JOURNALMODE::OFF);
				if (mode != IPager::JOURNALMODE::JMEMORY && mode != IPager::JOURNALMODE::OFF)
					mode = old;
			}

			if (mode != old)
			{
				// Change the journal mode
				_assert(pager->State != PAGER::ERROR);
				pager->JournalMode = (uint8)mode;

				// When transistioning from TRUNCATE or PERSIST to any other journal mode except WAL, unless the pager is in locking_mode=exclusive mode,
				// delete the journal file.
				_assert((IPager::JOURNALMODE::TRUNCATE & 5) == 1);
				_assert((IPager::JOURNALMODE::PERSIST & 5) == 1);
				_assert((IPager::JOURNALMODE::DELETE & 5) == 0);
				_assert((IPager::JOURNALMODE::MEMORY & 5) == 4);
				_assert((IPager::JOURNALMODE::OFF & 5) == 0);
				_assert((IPager::JOURNALMODE::WAL & 5) == 5);

				_assert(pager->File->Opened || pager->ExclusiveMode);
				if (!pager->ExclusiveMode && (eOld & 5) == 1 && (eMode & 1) == 0)
				{
					// In this case we would like to delete the journal file. If it is not possible, then that is not a problem. Deleting the journal file
					// here is an optimization only.
					//
					// Before deleting the journal file, obtain a RESERVED lock on the database file. This ensures that the journal file is not deleted
					// while it is in use by some other client.
					pPager->JournalFile->Close();
					if (pager->Lock >= VFile::LOCK::RESERVED)
						pager->Vfs->Delete(pager->Journal, 0);
					else
					{
						RCt rc = RC::OK;
						PAGER state = pager->State;
						_assert(state == PAGER::OPEN || state == PAGER::READER);
						if (state == PAGER::OPEN)
							rc = sqlite3PagerSharedLock(pager);
						if (pager->State == PAGER::READER)
						{
							_assert(rc == RC::OK);
							rc = pagerLockDb(pager, VFile::LOCK::RESERVED);
						}
						if (rc == RC::OK)
							pager->Vfs->Delete(pager->Journal, 0);
						if (rc == RC::OK && state == PAGER::READER)
							pagerUnlockDb(pager, VFile::LOCK::SHARED);
						else if (state == PAGER_OPEN)
							pager_unlock(pager);
						assert(state == pager->State);
					}
				}
			}

			// Return the new journal mode
			return (int)pager->JournalMode;
		}

		int sqlite3PagerGetJournalMode(Pager *pager)
		{
			return (int)pager->JournalMode;
		}

		int sqlite3PagerOkToChangeJournalMode(Pager *pager)
		{
			_assert(assert_pager_state(pager));
			if (pager->State >= PAGER::WRITER_CACHEMOD) return 0;
			return (SysEx::NEVER(pager->JournalFile->Opened && pager->JournalOffset > 0) ? 0 : 1);
		}

		int64 sqlite3PagerJournalSizeLimit(Pager *pager, int64 limit)
		{
			if (limit >= -1)
			{
				pager->JournalSizeLimit = limit;
				sqlite3WalLimit(pager->Wal, limit);
			}
			return pager->JournalSizeLimit;
		}

		IBackup **sqlite3PagerBackupPtr(Pager *pager)
		{
			return &pager->Backup;
		}

#ifndef OMIT_VACUUM
		void sqlite3PagerClearCache(Pager *pager)
		{
			if (!pager->MemoryDB && pager->TempFile == nullptr)
				pager_reset(pPager);
		}
#endif

#pragma endregion
#endif

	};
}