#include "hip/hip_runtime.h"
﻿// pager.c
#include "Core+Pager.cu.h"
using namespace Core;

namespace Core
{
	class Pager
	{
	public:
		enum PAGER : char
		{
			OPEN = 0,
			READER = 1,
			WRITER_LOCKED = 2,
			WRITER_CACHEMOD = 3,
			WRITER_DBMOD = 4,
			WRITER_FINISHED = 5,
			ERROR = 6,
		};

#define MAX_SECTOR_SIZE 0x10000

		struct PagerSavepoint
		{
			int64 Offset;             // Starting offset in main journal
			int64 HdrOffset;          // See above
			Bitvec *InSavepoint;      // Set of pages in this savepoint
			Pid Orig;                 // Original number of pages in file
			Pid SubRec;               // Index of first record in sub-journal
#ifndef OMIT_WAL
			uint32 WalData[WAL_SAVEPOINT_NDATA];  // WAL savepoint context
#endif
		};

		VFileSystem Vfs;			// OS functions to use for IO
		bool ExclusiveMode;			// Boolean. True if locking_mode==EXCLUSIVE
		uint8 JournalMode;			// One of the PAGER_JOURNALMODE_* values
		byte UseJournal;			// Use a rollback journal on this file
		byte NoSync;				// Do not sync the journal if true
		bool FullSync;				// Do extra syncs of the journal for robustness
		VFile::SYNC CheckpointSyncFlags;	// SYNC_NORMAL or SYNC_FULL for checkpoint
		VFile::SYNC WalSyncFlags;	// SYNC_NORMAL or SYNC_FULL for wal writes
		VFile::SYNC SyncFlags;		// SYNC_NORMAL or SYNC_FULL otherwise
		bool TempFile;				// zFilename is a temporary file
		bool ReadOnly;				// True for a read-only database
		bool MemoryDB;				// True to inhibit all file I/O
		// The following block contains those class members that change during routine opertion.  Class members not in this block are either fixed
		// when the pager is first created or else only change when there is a significant mode change (such as changing the page_size, locking_mode,
		// or the journal_mode).  From another view, these class members describe the "state" of the pager, while other class members describe the "configuration" of the pager.
		PAGER State;                // Pager state (OPEN, READER, WRITER_LOCKED..)
		VFile::LOCK Lock;           // Current lock held on database file
		bool ChangeCountDone;       // Set after incrementing the change-counter
		bool SetMaster;             // True if a m-j name has been written to jrnl
		byte DoNotSpill;            // Do not spill the cache when non-zero
		byte DoNotSyncSpill;        // Do not do a spill that requires jrnl sync
		byte SubjInMemory;          // True to use in-memory sub-journals
		Pid DBSize;					// Number of pages in the database
		Pid DBOrigSize;				// dbSize before the current transaction
		Pid DBFileSize;				// Number of pages in the database file
		Pid DBHintSize;				// Value passed to FCNTL_SIZE_HINT call
		RC ErrorCode;               // One of several kinds of errors
		int _nRec;                  // Pages journalled since last j-header written
		uint32 ChecksumInit;        // Quasi-random value added to every checksum
		uint32 SubRecords;          // Number of records written to sub-journal
		Bitvec *InJournal;			// One bit for each page in the database file
		VFile *File;				// File descriptor for database
		VFile *JournalFile;			// File descriptor for main journal
		VFile *SubJournalFile;		// File descriptor for sub-journal
		int64 JournalOffset;        // Current write offset in the journal file
		int64 JournalHeader;        // Byte offset to previous journal header
		IBackup *Backup;			// Pointer to list of ongoing backup processes
		//int _0;					// Number of elements in Savepoint[]
		PagerSavepoint *Savepoint;	// Array of active savepoints
		char DBFileVersions[16];    // Changes whenever database file changes
		// End of the routinely-changing class members
		uint16 ExtraBytes;          // Add this many bytes to each in-memory page
		int16 ReserveBytes;         // Number of unused bytes at end of each page
		uint32 VfsFlags;            // Flags for sqlite3_vfs.xOpen()
		uint32 SectorSize;          // Assumed sector size during rollback
		int PageSize;               // Number of bytes in a page
		Pid MaxPid;					// Maximum allowed size of the database
		int64 JournalSizeLimit;     // Size limit for persistent journal files
		char *Filename;            // Name of the database file
		char *Journal;             // Name of the journal file
		int (*BusyHandler)(void*);	// Function to call when busy
		void *BusyHandlerArg;		// Context argument for xBusyHandler
		int Stats[3];               // Total cache hits, misses and writes
#ifdef TEST
		int Reads;                  // Database pages read
#endif
		void (*Reiniter)(IPage *);	// Call this routine when reloading pages
#ifdef HAS_CODEC
		void *(*Codec)(void *,void *, Pid, int);	// Routine for en/decoding data
		void (*CodecSizeChange)(void *, int, int);	// Notify of page size changes
		void (*CodecFree)(void *);					// Destructor for the codec
		void *CodecArg;								// First argument to xCodec... methods
#endif
		char *TmpSpace;				// Pager.pageSize bytes of space for tmp use
		PCache *PCache;				// Pointer to page cache object
#ifndef OMIT_WAL
		Wal *Wal;					// Write-ahead log used by "journal_mode=wal"
		char *WalName;              // File name for write-ahead log
#endif

		enum STAT : char
		{
			HIT = 0,
			MISS = 1,
			WRITE = 2,
		};

#define PAGER_MAX_PID 2147483647

#ifndef OMIT_WAL
		static int UseWal(Pager *pager) { return (pager->Wal != nullptr); }
#else
#define UseWal(x) 0
#define RollbackWal(x) 0
#define WalFrames(v,w,x,y) 0
#define OpenWalIfPresent(z) SQLITE_OK
#define BeginReadTransaction(z) SQLITE_OK
#endif


#pragma region Debug

#ifndef NDEBUG 

		static int assert_pager_state(Pager *p){
			Pager *pPager = p;

			// State must be valid.
			_assert(p->State == PAGER::OPEN ||
				p->State == PAGER::READER ||
				p->State == PAGER::WRITER_LOCKED ||
				p->State == PAGER::WRITER_CACHEMOD ||
				p->State == PAGER::WRITER_DBMOD ||
				p->State == PAGER::WRITER_FINISHED ||
				p->State == PAGER::ERROR);

			// Regardless of the current state, a temp-file connection always behaves as if it has an exclusive lock on the database file. It never updates
			// the change-counter field, so the changeCountDone flag is always set.
			_assert( p->TempFile == 0 || p->Lock == VFile::LOCK::EXCLUSIVE);
			_assert( p->TempFile == 0 || pager->ChangeCountDone);

			// If the useJournal flag is clear, the journal-mode must be "OFF". And if the journal-mode is "OFF", the journal file must not be open.
			_assert( p->journalMode==PAGER_JOURNALMODE_OFF || p->useJournal );
			_assert( p->journalMode!=PAGER_JOURNALMODE_OFF || !isOpen(p->jfd) );

			/* Check that MEMDB implies noSync. And an in-memory journal. Since 
			** this means an in-memory pager performs no IO at all, it cannot encounter 
			** either SQLITE_IOERR or SQLITE_FULL during rollback or while finalizing 
			** a journal file. (although the in-memory journal implementation may 
			** return SQLITE_IOERR_NOMEM while the journal file is being written). It 
			** is therefore not possible for an in-memory pager to enter the ERROR 
			** state.
			*/
			if( MEMDB ){
				assert( p->noSync );
				assert( p->journalMode==PAGER_JOURNALMODE_OFF 
					|| p->journalMode==PAGER_JOURNALMODE_MEMORY 
					);
				assert( p->eState!=PAGER_ERROR && p->eState!=PAGER_OPEN );
				assert( pagerUseWal(p)==0 );
			}

			/* If changeCountDone is set, a RESERVED lock or greater must be held
			** on the file.
			*/
			assert( pPager->changeCountDone==0 || pPager->eLock>=RESERVED_LOCK );
			assert( p->eLock!=PENDING_LOCK );

			switch( p->eState ){
			case PAGER_OPEN:
				assert( !MEMDB );
				assert( pPager->errCode==SQLITE_OK );
				assert( sqlite3PcacheRefCount(pPager->pPCache)==0 || pPager->tempFile );
				break;

			case PAGER_READER:
				assert( pPager->errCode==SQLITE_OK );
				assert( p->eLock!=UNKNOWN_LOCK );
				assert( p->eLock>=SHARED_LOCK );
				break;

			case PAGER_WRITER_LOCKED:
				assert( p->eLock!=UNKNOWN_LOCK );
				assert( pPager->errCode==SQLITE_OK );
				if( !pagerUseWal(pPager) ){
					assert( p->eLock>=RESERVED_LOCK );
				}
				assert( pPager->dbSize==pPager->dbOrigSize );
				assert( pPager->dbOrigSize==pPager->dbFileSize );
				assert( pPager->dbOrigSize==pPager->dbHintSize );
				assert( pPager->setMaster==0 );
				break;

			case PAGER_WRITER_CACHEMOD:
				assert( p->eLock!=UNKNOWN_LOCK );
				assert( pPager->errCode==SQLITE_OK );
				if( !pagerUseWal(pPager) ){
					/* It is possible that if journal_mode=wal here that neither the
					** journal file nor the WAL file are open. This happens during
					** a rollback transaction that switches from journal_mode=off
					** to journal_mode=wal.
					*/
					assert( p->eLock>=RESERVED_LOCK );
					assert( isOpen(p->jfd) 
						|| p->journalMode==PAGER_JOURNALMODE_OFF 
						|| p->journalMode==PAGER_JOURNALMODE_WAL 
						);
				}
				assert( pPager->dbOrigSize==pPager->dbFileSize );
				assert( pPager->dbOrigSize==pPager->dbHintSize );
				break;

			case PAGER_WRITER_DBMOD:
				assert( p->eLock==EXCLUSIVE_LOCK );
				assert( pPager->errCode==SQLITE_OK );
				assert( !pagerUseWal(pPager) );
				assert( p->eLock>=EXCLUSIVE_LOCK );
				assert( isOpen(p->jfd) 
					|| p->journalMode==PAGER_JOURNALMODE_OFF 
					|| p->journalMode==PAGER_JOURNALMODE_WAL 
					);
				assert( pPager->dbOrigSize<=pPager->dbHintSize );
				break;

			case PAGER_WRITER_FINISHED:
				assert( p->eLock==EXCLUSIVE_LOCK );
				assert( pPager->errCode==SQLITE_OK );
				assert( !pagerUseWal(pPager) );
				assert( isOpen(p->jfd) 
					|| p->journalMode==PAGER_JOURNALMODE_OFF 
					|| p->journalMode==PAGER_JOURNALMODE_WAL 
					);
				break;

			case PAGER_ERROR:
				/* There must be at least one outstanding reference to the pager if
				** in ERROR state. Otherwise the pager should have already dropped
				** back to OPEN state.
				*/
				assert( pPager->errCode!=SQLITE_OK );
				assert( sqlite3PcacheRefCount(pPager->pPCache)>0 );
				break;
			}

			return 1;
		}
#endif /* ifndef NDEBUG */

#pragma endregion


#pragma region X



#pragma endregion



	};
}