﻿// pcache.c
#include "Core+Pager.cu.h"
using namespace Core;

namespace Core
{
	struct PCache
	{
		PgHdr *Dirty, *DirtyTail;   // List of dirty pages in LRU order
		PgHdr *Synced;              // Last synced page in dirty page list
		int Refs;                   // Number of referenced pages
		int SizeCache;              // Configured cache size
		int SizePage;               // Size of every page in this cache
		int SizeExtra;              // Size of extra space for each page
		bool Purgeable;             // True if pages are on backing store
		RC (*Stress)(void *, PgHdr *);// Call to try make a page clean
		void *StressArg;            // Argument to xStress
		IPCache *Cache;				// Pluggable cache module
		PgHdr *Page1;				// Reference to page 1
	public:
		static int Initialize();
		static void Shutdown();
		//	static int SizeOf();
		void Open(int sizePage, int sizeExtra, bool purgeable, RC (*stress)(void *, PgHdr *), void *stressArg, PCache *p);
		void SetPageSize(int sizePage);
		int Fetch(Pid id, bool createFlag, PgHdr **pageOut);
		//	void Release(PgHdr *);
		//	void Drop(PgHdr *);         // Remove page from cache
		//	void MakeDirty(PgHdr *);    // Make sure page is marked dirty
		//	void MakeClean(PgHdr *);	// Mark a single page as clean
		//	void CleanAll(PCache *);	// Mark all dirty list pages as clean
		//	void Move(PgHdr*, Pid);
		//	void Truncate(PCache*, Pid);
		//	PgHdr *DirtyList(PCache *);
		//	void Close(PCache *);
		//	void ClearSyncFlags(PCache *);
		//	void Clear(PCache *);
		//	int RefCount(PCache *);
		//	void Ref(PgHdr *);
		//	int PageRefcount(PgHdr *);
		//	int Pagecount(PCache *);
		//#if defined(CHECK_PAGES) || defined(DEBUG)
		//	void IterateDirty(PCache *cache, void (*iter)(PgHdr *));
		//#endif
		//	void SetCachesize(PCache *, int);
		//#ifdef TEST
		//	int GetCachesize(PCache *);
		//#endif
		//	void Shrink(PCache *);
		//#ifdef ENABLE_MEMORY_MANAGEMENT
		//	int ReleaseMemory(int);
		//#endif
		//	void BufferSetup(void *, int sz, int n);

	};

#pragma region Linked List

#if EXPENSIVE_ASSERT
	static bool CheckSynced(PCache *cache)
	{
		PgHdr *p;
		for (p = cache->DirtyTail; p != cache->Synced; p = p->DirtyPrev)
			_assert(p->Refs || (p->Flags & PgHdr::PGHDR::NEED_SYNC));
		return (p == nullptr || p->Refs || (p->Flags & PgHdr::PGHDR::NEED_SYNC) == 0);
	}
#endif

	static void RemoveFromDirtyList(PgHdr *page)
	{
		PCache *p = page->Cache;
		_assert(page->DirtyNext || page == p->DirtyTail);
		_assert(page->DirtyPrev || page == p->Dirty);
		// Update the PCache1.Synced variable if necessary.
		if (p->Synced == page)
		{
			PgHdr *synced = page->DirtyPrev;
			while (synced && (synced->Flags & PgHdr::PGHDR::NEED_SYNC))
				synced = synced->DirtyPrev;
			p->Synced = synced;
		}
		if (page->DirtyNext)
			page->DirtyNext->DirtyPrev = page->DirtyPrev;
		else
		{
			_assert(page == p->DirtyTail);
			p->DirtyTail = page->DirtyPrev;
		}
		if (page->DirtyPrev)
			page->DirtyPrev->DirtyNext = page->DirtyNext;
		else
		{
			_assert(page == p->Dirty);
			p->Dirty = page->DirtyNext;
		}
		page->DirtyNext = nullptr;
		page->DirtyPrev = nullptr;
#if EXPENSIVE_ASSERT
		_assert(CheckSynced(p));
#endif
	}

	static void AddToDirtyList(PgHdr *page)
	{
		PCache *p = page->Cache;
		_assert(page->DirtyNext == nullptr && page->DirtyPrev == nullptr && p->Dirty != page);
		page->DirtyNext = p->Dirty;
		if (page->DirtyNext)
		{
			_assert(page->DirtyNext->DirtyPrev == nullptr);
			page->DirtyNext->DirtyPrev = page;
		}
		p->Dirty = page;
		if (!p->DirtyTail)
			p->DirtyTail = page;
		if (!p->Synced && (page->Flags & PgHdr::PGHDR::NEED_SYNC) == 0)
			p->Synced = page;
#if EXPENSIVE_ASSERT
		_assert(CheckSynced(p));
#endif
	}

	static void Unpin(PgHdr *p)
	{
		PCache *cache = p->Cache;
		if (cache->Purgeable)
		{
			if (p->ID == 1)
				cache->Page1 = nullptr;
			cache->Cache->Unpin(p->Page, false);
		}
	}

#pragma endregion

#pragma region Interface

	static IPCache *_pcache;

	int PCache::Initialize() 
	{ 
		//if (_pcache == nullptr)
		//	_pcache = new PCache1();
		return _pcache->Init(); 
	}
	void PCache::Shutdown()
	{
		_pcache->Shutdown(); 
	}
	//int PCache::SizeOf() { return sizeof(PCache); }

	void PCache::Open(int sizePage, int sizeExtra, bool purgeable, RC (*stress)(void *, PgHdr *), void *stressArg, PCache *p)
	{
		_memset(p, 0, sizeof(PCache));
		p->SizePage = sizePage;
		p->SizeExtra = sizeExtra;
		p->Purgeable = purgeable;
		p->Stress = stress;
		p->StressArg = stressArg;
		p->SizeCache = 100;
	}

	void PCache::SetPageSize(int sizePage)
	{
		_assert(Refs == 0 && Dirty == nullptr);
		if (Cache)
		{
			_pcache->Destroy(Cache);
			Cache = nullptr;
			Page1 = nullptr;
		}
		SizePage = sizePage;
	}

	static uint NumberOfCachePages(PCache *p)
	{
		if (p->SizeCache >= 0)
			return (uint)p->SizeCache;
		return (uint)((-1024 * (int64)p->SizeCache) / (p->SizePage + p->SizeExtra));
	}

	int PCache::Fetch(Pid id, bool createFlag, PgHdr **pageOut)
	{
		_assert(id > 0);
		// If the pluggable cache (sqlite3_pcache*) has not been allocated, allocate it now.
		if (!Cache && createFlag)
		{
			IPCache *p = _pcache->Create(SizePage, SizeExtra + sizeof(PgHdr), Purgeable);
			if (!p)
				return RC::NOMEM;
			p->Cachesize(NumberOfCachePages(this));
			Cache = p;
		}
		IPage *page = nullptr;
		int create = createFlag * (1 + (!Purgeable || !Dirty));
		if (Cache)
			page = Cache->Fetch(id, create);
		if (!page && create)
		{
			// Find a dirty page to write-out and recycle. First try to find a page that does not require a journal-sync (one with PGHDR_NEED_SYNC
			// cleared), but if that is not possible settle for any other unreferenced dirty page.
#if EXPENSIVE_ASSERT
			CheckSynced(this);
#endif
			PgHdr *pg;
			for (pg = Synced; pg && (pg->Refs || (pg->Flags & PgHdr::PGHDR::NEED_SYNC)); pg = pg->DirtyPrev) ;
			Synced = pg;
			if (!pg)
				for (pg = DirtyTail; pg && pg->Refs; pg = pg->DirtyPrev) ;
			if (pg)
			{
#ifdef LOG_CACHE_SPILL
				SysEx::Log(RC::FULL, "spill page %d making room for %d - cache used: %d/%d", pg->ID, id, _pcache->Pagecount(), NumberOfCachePages(this));
#endif
				RC rc = Stress(StressArg, pg);
				if (rc != RC::OK && rc != RC::BUSY)
					return rc;
			}
			page = Cache->Fetch(id, 2);
		}
		PgHdr *pgHdr = nullptr;
		if (page)
		{
			pgHdr = (PgHdr *)page->Extra;
			if (!pgHdr->Page)
			{
				_memset(pgHdr, 0, sizeof(PgHdr));
				pgHdr->Page = page;
				pgHdr->Data = page->Buffer;
				pgHdr->Extra = (void *)&pgHdr[1];
				_memset(pgHdr->Extra, 0, SizeExtra);
				pgHdr->Cache = this;
				pgHdr->ID = id;
			}
			_assert(pgHdr->Cache == this);
			_assert(pgHdr->ID == id);
			_assert(pgHdr->Data == page->Buffer);
			_assert(pgHdr->Extra == (void *)&pgHdr[1]);
			if (pgHdr->Refs == 0)
				Refs++;
			pgHdr->Refs++;
			if (id == 1)
				Page1 = pgHdr;
		}
		*pageOut = pgHdr;
		return (pgHdr == nullptr && create ? RC::NOMEM : RC::OK);
	}

	void Release(PgHdr *p)
	{
		_assert(p->Refs > 0);
		p->Refs--;
		if (p->Refs == 0)
		{
			PCache *cache = p->Cache;
			cache->Refs--;
			if ((p->Flags & PgHdr::PGHDR::DIRTY) == 0)
				Unpin(p);
			else
			{
				// Move the page to the head of the dirty list.
				RemoveFromDirtyList(p);
				AddToDirtyList(p);
			}
		}
	}

	void Ref(PgHdr *p)
	{
		_assert(p->Refs > 0);
		p->Refs++;
	}

	void Drop(PgHdr *p)
	{
		_assert(p->Refs == 1);
		if (p->Flags & PgHdr::PGHDR::DIRTY)
			RemoveFromDirtyList(p);
		PCache *cache = p->Cache;
		cache->Refs--;
		if (p->ID == 1)
			cache->Page1 = nullptr;
		cache->Cache->Unpin(p->Page, true);
	}

	void MakeDirty(PgHdr *p)
	{
		p->Flags &= ~PgHdr::PGHDR::DONT_WRITE;
		_assert(p->Refs > 0);
		if ((p->Flags & PgHdr::PGHDR::DIRTY) == 0)
		{
			p->Flags |= PgHdr::PGHDR::DIRTY;
			AddToDirtyList(p);
		}
	}

	void MakeClean(PgHdr *p)
	{
		if ((p->Flags & PgHdr::PGHDR::DIRTY))
		{
			RemoveFromDirtyList(p);
			p->Flags &= ~(PgHdr::PGHDR::DIRTY | PgHdr::PGHDR::NEED_SYNC);
			if (p->Refs == 0)
				Unpin(p);
		}
	}

	void CleanAll(PCache *cache)
	{
		PgHdr *p;
		while ((p = cache->Dirty) != nullptr)
			MakeClean(p);
	}

	void ClearSyncFlags(PCache *cache)
	{
		for (PgHdr *p = cache->Dirty; p; p = p->DirtyNext)
			p->Flags &= ~PgHdr::PGHDR::NEED_SYNC;
		cache->Synced = cache->DirtyTail;
	}

	void Move(PgHdr *p, Pid newID)
	{
		PCache *cache = p->Cache;
		_assert(p->Refs > 0);
		_assert(newID > 0);
		cache->Cache->Rekey(p->Page, p->ID, newID);
		p->ID = newID;
		if ((p->Flags & PgHdr::PGHDR::DIRTY) && (p->Flags & PgHdr::PGHDR::NEED_SYNC))
		{
			RemoveFromDirtyList(p);
			AddToDirtyList(p);
		}
	}

	void Truncate(PCache *cache, Pid id)
	{
		if (cache->Cache)
		{
			PgHdr *p;
			PgHdr *next;
			for (p = cache->Dirty; p; p = next)
			{
				next = p->DirtyNext;
				// This routine never gets call with a positive pgno except right after sqlite3PcacheCleanAll().  So if there are dirty pages, it must be that pgno==0.
				_assert(p->ID > 0);
				if (SysEx_ALWAYS(p->ID > id))
				{
					_assert(p->Flags & PgHdr::PGHDR::DIRTY);
					MakeClean(p);
				}
			}
			if (id == 0 && cache->Page1)
			{
				_memset(cache->Page1->Data, 0, cache->SizePage);
				id = 1;
			}
			cache->Cache->Truncate(id + 1);
		}
	}

	void Close(PCache *cache)
	{
		if (cache->Cache)
			_pcache->Destroy(cache->Cache);
	}

	void Clear(PCache *cache)
	{
		Truncate(cache, 0); 
	}

	static PgHdr *MergeDirtyList(PgHdr *a, PgHdr *b)
	{
		PgHdr result;
		PgHdr *tail = &result;
		while (a && b)
		{
			if (a->ID < b->ID)
			{
				tail->Dirty = a;
				tail = a;
				a = a->Dirty;
			}
			else
			{
				tail->Dirty = b;
				tail = b;
				b = b->Dirty;
			}
		}
		if (a)
			tail->Dirty = a;
		else if (b)
			tail->Dirty = b;
		else
			tail->Dirty = nullptr;
		return result.Dirty;
	}

#define N_SORT_BUCKET 32

	static PgHdr *SortDirtyList(PgHdr *in)
	{
		PgHdr *a[N_SORT_BUCKET], *p;
		_memset(a, 0, sizeof(a));
		int i;
		while (in)
		{
			p = in;
			in = p->Dirty;
			p->Dirty = nullptr;
			for (i = 0; SysEx_ALWAYS(i < N_SORT_BUCKET - 1); i++)
			{
				if (a[i] == nullptr)
				{
					a[i] = p;
					break;
				}
				else
				{
					p = MergeDirtyList(a[i], p);
					a[i] = nullptr;
				}
			}
			if (SysEx_NEVER(i == N_SORT_BUCKET - 1))
				// To get here, there need to be 2^(N_SORT_BUCKET) elements in the input list.  But that is impossible.
					a[i] = MergeDirtyList(a[i], p);
		}
		p = a[0];
		for (i = 1; i < N_SORT_BUCKET; i++)
			p = MergeDirtyList(p, a[i]);
		return p;
	}

	PgHdr *DirtyList(PCache *cache)
	{
		for (PgHdr *p = cache->Dirty; p; p = p->DirtyNext)
			p->Dirty = p->DirtyNext;
		return SortDirtyList(cache->Dirty);
	}

	int RefCount(PCache *cache)
	{
		return cache->Refs;
	}

	int PageRefcount(PgHdr *p)
	{
		return p->Refs;
	}

	int Pagecount(PCache *cache)
	{
		return (cache->Cache ? cache->Cache->Pagecount() : 0);
	}

	void SetCachesize(PCache *cache, int maxPage)
	{
		cache->SizeCache = maxPage;
		if (cache->Cache)
			cache->Cache->Cachesize(NumberOfCachePages(cache));
	}

	void Shrink(PCache *cache)
	{
		if (cache->Cache)
			cache->Cache->Shrink();
	}

#if defined(CHECK_PAGES) || defined(DEBUG)
	void IterateDirty(PCache *cache, void (*iter)(PgHdr *))
	{
		for (PgHdr *dirty = cache->Dirty; dirty; dirty = dirty->DirtyNext)
			iter(dirty);
	}
#endif

#pragma endregion

#pragma region Test 
#ifdef TEST

	int GetCachesize(PCache *cache)
	{
		return NumberOfCachePages(cache);
	}

#endif
#pragma endregion
}