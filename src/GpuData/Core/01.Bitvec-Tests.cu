﻿//bitvec.c
#include "Core.cu.h"
using namespace Core;

namespace Core
{

#define SETBIT(V,I) V[I>>3] |= (1<<(I&7))
#define CLEARBIT(V,I) V[I>>3] &= ~(1<<(I&7))
#define TESTBIT(V,I) (V[I>>3]&(1<<(I&7)))!=0

	__device__ int Bitvec_BuiltinTest(int size, int *aOp)
	{
		int rc = -1;
		// Allocate the Bitvec to be tested and a linear array of bits to act as the reference
		Bitvec *bitvec = new Bitvec(size);
		unsigned char *v = (unsigned char *)SysEx::Alloc((size + 7) / 8 + 1, true);
		void *tmpSpace = SysEx::Alloc(BITVEC_SZ);
		if (!bitvec || !v || !tmpSpace)
			goto bitvec_end;

		// Run the program
		int pc = 0;
		int i, nx, op;
		while ((op = aOp[pc]))
		{
			switch (op)
			{
			case 1:
			case 2:
			case 5:
				{
					nx = 4;
					i = aOp[pc + 2] - 1;
					aOp[pc + 2] += aOp[pc + 3];
					break;
				}
			case 3:
			case 4: 
			default:
				{
					nx = 2;
					SysEx::SetRandom(sizeof(i), &i);
					break;
				}
			}
			if ((--aOp[pc + 1]) > 0) nx = 0;
			pc += nx;
			i = (i & 0x7fffffff) % size;
			if ((op & 1) !=0)
			{
				SETBIT(v, (i + 1));
				if (op != 5)
					if (bitvec->Set(i + 1)) goto bitvec_end;
			}
			else
			{
				CLEARBIT(v, (i + 1));
				bitvec->Clear(i + 1, tmpSpace);
			}
		}

		// Test to make sure the linear array exactly matches the Bitvec object.  Start with the assumption that they do
		// match (rc==0).  Change rc to non-zero if a discrepancy is found.
		rc = bitvec->Get(size + 1)
			+ bitvec->Get(0)
			+ (bitvec->get_Length() - size);
		for (i = 1; i <= size; i++)
		{
			if ((TESTBIT(v,i)) != bitvec->Get(i))
			{
				rc = i;
				break;
			}
		}

		// Free allocated structure
bitvec_end:
		SysEx::Free(tmpSpace);
		SysEx::Free(v);
		Bitvec::Destroy(bitvec);
		return rc;
	}
}
