﻿#include "Core.cu.h"
#include <stdarg.h>
using namespace Core;

namespace Core
{
	//void SysEx::Log(RC rc, const char *format, ...)
	//{
	//	va_list a;
	//	va_start(a, format);
	//	vprintf(format, a);
	//	va_end(a);
	//}

	static uint8 randomByte()
	{
		return 1;
		//	unsigned char t;
		//	if(!wsdPrng.isInit)
		//	{
		//		int i;
		//		char k[256];
		//		wsdPrng.j = 0;
		//		wsdPrng.i = 0;
		//		sqlite3OsRandomness(sqlite3_vfs_find(0), 256, k);
		//		for(i=0; i<256; i++)
		//			wsdPrng.s[i] = (u8)i;
		//		for(i=0; i<256; i++)
		//		{
		//			wsdPrng.j += wsdPrng.s[i] + k[i];
		//			t = wsdPrng.s[wsdPrng.j];
		//			wsdPrng.s[wsdPrng.j] = wsdPrng.s[i];
		//			wsdPrng.s[i] = t;
		//		}
		//		wsdPrng.isInit = 1;
		//	}
		//	wsdPrng.i++;
		//	t = wsdPrng.s[wsdPrng.i];
		//	wsdPrng.j += t;
		//	wsdPrng.s[wsdPrng.i] = wsdPrng.s[wsdPrng.j];
		//	wsdPrng.s[wsdPrng.j] = t;
		//	t += wsdPrng.s[wsdPrng.i];
		//	return wsdPrng.s[t];
	}

	void SysEx::PutRandom(int length, void *buffer)
	{
		unsigned char *b = (unsigned char *)buffer;
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX::STATIC_PRNG);
		MutexEx::Enter(mutex);
		while (length--)
			*(b++) = randomByte();
		MutexEx::Leave(mutex);
	}
}
