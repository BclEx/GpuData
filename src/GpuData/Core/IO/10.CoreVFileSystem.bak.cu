﻿// ow_win.c
#include "../Core.cu.h"
#include <Windows.h>

namespace Core { namespace IO
{

	// winFile
	class CoreVFile : public VFile
	{
	public:
		VSystem *Vfs;		// The VFS used to open this file
		HANDLE h;               // Handle for accessing the file
		uint8 locktype;         // Type of lock currently held on this file
		short sharedLockByte;   // Randomly chosen byte used as a shared lock
		uint8 ctrlFlags;        // Flags.  See WINFILE_* below
		DWORD lastErrno;        // The Windows errno from the last I/O error
#ifndef SQLITE_OMIT_WAL
		winShm *pShm;           // Instance of shared memory on this file */
#endif
		const char *zPath;      // Full pathname of this file
		int szChunk;            // Chunk size configured by FCNTL_CHUNK_SIZE
#if OS_WINCE
		LPWSTR zDeleteOnClose;  // Name of file to delete when closing
		HANDLE hMutex;          // Mutex used to control access to shared lock
		HANDLE hShared;         // Shared memory segment used for locking
		winceLock local;        // Locks obtained by this instance of winFile
		winceLock *shared;      // Global shared lock memory for the file
#endif
	};

	class CoreVFileSystem : public VSystem
	{
	private:
	public:
		__device__ virtual RC Open(const char *name, VFile *id, OPEN flags, OPEN *outFlags);
		__device__ virtual RC Write(const void *buffer, int amount, int64 offset);
	};

	RC CoreVFileSystem::Open(const char *name, VFile *id, OPEN flags, OPEN *outFlags)
	{
//		HANDLE h;
//		DWORD lastErrno;
//		DWORD dwDesiredAccess;
//		DWORD dwShareMode;
//		DWORD dwCreationDisposition;
//		DWORD dwFlagsAndAttributes = 0;
//#if OS_WINCE
//		int isTemp = 0;
//#endif
//			void *converted; // Filename in OS encoding
//		const char *utf8Name = name; // Filename in UTF-8 encoding
//		int cnt = 0;
		_assert(id != nullptr);

		// If argument zPath is a NULL pointer, this function is required to open a temporary file. Use this buffer to store the file name in.
		char tmpname[MAX_PATH + 2]; // Buffer used to create temp filename

		RC rc = RC::OK;
		OPEN type = (OPEN)(flags & 0xFFFFFF00); // Type of file to open
		bool exclusive = (flags & OPEN::EXCLUSIVE);
		bool delete_ = (flags & OPEN::DELETEONCLOSE);
		bool create = (flags & OPEN::CREATE);
		bool readonly = (flags & OPEN::READONLY);
		bool readWrite = (flags & OPEN::OREADWRITE);
		bool openJournal = (create && (type == OPEN::MASTER_JOURNAL || type == OPEN::MAIN_JOURNAL || type == OPEN::WAL));

		// Check the following statements are true: 
		//
		//   (a) Exactly one of the READWRITE and READONLY flags must be set, and 
		//   (b) if CREATE is set, then READWRITE must also be set, and
		//   (c) if EXCLUSIVE is set, then CREATE must also be set.
		//   (d) if DELETEONCLOSE is set, then CREATE must also be set.
		_assert((!readonly|| !readWrite) && (readWrite || readonly));
		_assert(!create || readWrite);
		_assert(!exclusive || create);
		_assert(!delete_ || create);

		// The main DB, main journal, WAL file and master journal are never automatically deleted. Nor are they ever temporary files.
		_assert((!delete_ && name) || type != OPEN::MAIN_DB);
		_assert((!delete_ && name) || type != OPEN::MAIN_JOURNAL);
		_assert((!delete_ && name) || type != OPEN::MASTER_JOURNAL);
		_assert((!delete_ && name) || type != OPEN::WAL);

		// Assert that the upper layer has set one of the "file-type" flags.
		_assert(type == OPEN::MAIN_DB || type == OPEN::TEMP_DB ||
			type == OPEN::MAIN_JOURNAL || type == OPEN::TEMP_JOURNAL ||
			type == OPEN::SUBJOURNAL || type == OPEN::MASTER_JOURNAL ||
			type == OPEN::TRANSIENT_DB || type == OPEN::WAL);

		CoreVFile *file = (CoreVFile *)id;
		_memset(file, 0, sizeof(CoreVFile));
		file->h = INVALID_HANDLE_VALUE;

#if !OS_WINRT
		if (!sqlite3_temp_directory)
			sqlite3_log(SQLITE_ERROR, "sqlite3_temp_directory variable should be set for WinRT");
#endif

		/// If the second argument to this function is NULL, generate a temporary file name to use 
		if (!utf8Name)
		{
			_assert(delete_ && !openJournal);
			_memset(tmpname, 0, MAX_PATH + 2);
			rc = getTempname(MAX_PATH + 2, tmpname);
			if (rc != RC::OK)
				return rc;
			utf8Name = tmpname;
		}

		// Database filenames are double-zero terminated if they are not URIs with parameters.  Hence, they can always be passed into
		// sqlite3_uri_parameter().
		_assert((type != OPEN::MAIN_DB) || (flags & OPEN::URI) || utf8Name[strlen(utf8Name) + 1] == 0);

		/* Convert the filename to the system encoding. */
		zConverted = convertUtf8Filename(zUtf8Name);
		if( zConverted==0 ){
			return SQLITE_IOERR_NOMEM;
		}

		if( winIsDir(zConverted) ){
			sqlite3_free(zConverted);
			return SQLITE_CANTOPEN_ISDIR;
		}

		if( isReadWrite ){
			dwDesiredAccess = GENERIC_READ | GENERIC_WRITE;
		}else{
			dwDesiredAccess = GENERIC_READ;
		}

		/* SQLITE_OPEN_EXCLUSIVE is used to make sure that a new file is 
		** created. SQLite doesn't use it to indicate "exclusive access" 
		** as it is usually understood.
		*/
		if( isExclusive ){
			/* Creates a new file, only if it does not already exist. */
			/* If the file exists, it fails. */
			dwCreationDisposition = CREATE_NEW;
		}else if( isCreate ){
			/* Open existing file, or create if it doesn't exist */
			dwCreationDisposition = OPEN_ALWAYS;
		}else{
			/* Opens a file, only if it exists. */
			dwCreationDisposition = OPEN_EXISTING;
		}

		dwShareMode = FILE_SHARE_READ | FILE_SHARE_WRITE;

		if( isDelete ){
#if SQLITE_OS_WINCE
			dwFlagsAndAttributes = FILE_ATTRIBUTE_HIDDEN;
			isTemp = 1;
#else
			dwFlagsAndAttributes = FILE_ATTRIBUTE_TEMPORARY
				| FILE_ATTRIBUTE_HIDDEN
				| FILE_FLAG_DELETE_ON_CLOSE;
#endif
		}else{
			dwFlagsAndAttributes = FILE_ATTRIBUTE_NORMAL;
		}
		/* Reports from the internet are that performance is always
		** better if FILE_FLAG_RANDOM_ACCESS is used.  Ticket #2699. */
#if SQLITE_OS_WINCE
		dwFlagsAndAttributes |= FILE_FLAG_RANDOM_ACCESS;
#endif

		if( isNT() ){
#if SQLITE_OS_WINRT
			CREATEFILE2_EXTENDED_PARAMETERS extendedParameters;
			extendedParameters.dwSize = sizeof(CREATEFILE2_EXTENDED_PARAMETERS);
			extendedParameters.dwFileAttributes =
				dwFlagsAndAttributes & FILE_ATTRIBUTE_MASK;
			extendedParameters.dwFileFlags = dwFlagsAndAttributes & FILE_FLAG_MASK;
			extendedParameters.dwSecurityQosFlags = SECURITY_ANONYMOUS;
			extendedParameters.lpSecurityAttributes = NULL;
			extendedParameters.hTemplateFile = NULL;
			while( (h = osCreateFile2((LPCWSTR)zConverted,
				dwDesiredAccess,
				dwShareMode,
				dwCreationDisposition,
				&extendedParameters))==INVALID_HANDLE_VALUE &&
				retryIoerr(&cnt, &lastErrno) ){
					/* Noop */
			}
#else
			while( (h = osCreateFileW((LPCWSTR)zConverted,
				dwDesiredAccess,
				dwShareMode, NULL,
				dwCreationDisposition,
				dwFlagsAndAttributes,
				NULL))==INVALID_HANDLE_VALUE &&
				retryIoerr(&cnt, &lastErrno) ){
					/* Noop */
			}
#endif
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			while( (h = osCreateFileA((LPCSTR)zConverted,
				dwDesiredAccess,
				dwShareMode, NULL,
				dwCreationDisposition,
				dwFlagsAndAttributes,
				NULL))==INVALID_HANDLE_VALUE &&
				retryIoerr(&cnt, &lastErrno) ){
					/* Noop */
			}
		}
#endif
		logIoerr(cnt);

		OSTRACE(("OPEN %d %s 0x%lx %s\n", 
			h, zName, dwDesiredAccess, 
			h==INVALID_HANDLE_VALUE ? "failed" : "ok"));

		if( h==INVALID_HANDLE_VALUE ){
			pFile->lastErrno = lastErrno;
			winLogError(SQLITE_CANTOPEN, pFile->lastErrno, "winOpen", zUtf8Name);
			sqlite3_free(zConverted);
			if( isReadWrite && !isExclusive ){
				return winOpen(pVfs, zName, id, 
					((flags|SQLITE_OPEN_READONLY) &
					~(SQLITE_OPEN_CREATE|SQLITE_OPEN_READWRITE)),
					pOutFlags);
			}else{
				return SQLITE_CANTOPEN_BKPT;
			}
		}

		if( pOutFlags ){
			if( isReadWrite ){
				*pOutFlags = SQLITE_OPEN_READWRITE;
			}else{
				*pOutFlags = SQLITE_OPEN_READONLY;
			}
		}

#if SQLITE_OS_WINCE
		if( isReadWrite && eType==SQLITE_OPEN_MAIN_DB
			&& (rc = winceCreateLock(zName, pFile))!=SQLITE_OK
			){
				osCloseHandle(h);
				sqlite3_free(zConverted);
				return rc;
		}
		if( isTemp ){
			pFile->zDeleteOnClose = zConverted;
		}else
#endif
		{
			sqlite3_free(zConverted);
		}

		pFile->pMethod = &winIoMethod;
		pFile->pVfs = pVfs;
		pFile->h = h;
		if( sqlite3_uri_boolean(zName, "psow", SQLITE_POWERSAFE_OVERWRITE) ){
			pFile->ctrlFlags |= WINFILE_PSOW;
		}
		pFile->lastErrno = NO_ERROR;
		pFile->zPath = zName;

		OpenCounter(+1);
		return rc;
	}


	static int winDelete(sqlite3_vfs *pVfs, const char *zFilename, int syncDir)
	{
		int cnt = 0;
		int rc;
		DWORD attr;
		DWORD lastErrno;
		void *zConverted;
		UNUSED_PARAMETER(pVfs);
		UNUSED_PARAMETER(syncDir);

		SimulateIOError(return SQLITE_IOERR_DELETE);
		zConverted = convertUtf8Filename(zFilename);
		if( zConverted==0 ){
			return SQLITE_IOERR_NOMEM;
		}
		if( isNT() ){
			do {
#if SQLITE_OS_WINRT
				WIN32_FILE_ATTRIBUTE_DATA sAttrData;
				memset(&sAttrData, 0, sizeof(sAttrData));
				if ( osGetFileAttributesExW(zConverted, GetFileExInfoStandard,
					&sAttrData) ){
						attr = sAttrData.dwFileAttributes;
				}else{
					lastErrno = osGetLastError();
					if( lastErrno==ERROR_FILE_NOT_FOUND
						|| lastErrno==ERROR_PATH_NOT_FOUND ){
							rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
					}else{
						rc = SQLITE_ERROR;
					}
					break;
				}
#else
				attr = osGetFileAttributesW(zConverted);
#endif
				if ( attr==INVALID_FILE_ATTRIBUTES ){
					lastErrno = osGetLastError();
					if( lastErrno==ERROR_FILE_NOT_FOUND
						|| lastErrno==ERROR_PATH_NOT_FOUND ){
							rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
					}else{
						rc = SQLITE_ERROR;
					}
					break;
				}
				if ( attr&FILE_ATTRIBUTE_DIRECTORY ){
					rc = SQLITE_ERROR; /* Files only. */
					break;
				}
				if ( osDeleteFileW(zConverted) ){
					rc = SQLITE_OK; /* Deleted OK. */
					break;
				}
				if ( !retryIoerr(&cnt, &lastErrno) ){
					rc = SQLITE_ERROR; /* No more retries. */
					break;
				}
			} while(1);
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			do {
				attr = osGetFileAttributesA(zConverted);
				if ( attr==INVALID_FILE_ATTRIBUTES ){
					lastErrno = osGetLastError();
					if( lastErrno==ERROR_FILE_NOT_FOUND
						|| lastErrno==ERROR_PATH_NOT_FOUND ){
							rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
					}else{
						rc = SQLITE_ERROR;
					}
					break;
				}
				if ( attr&FILE_ATTRIBUTE_DIRECTORY ){
					rc = SQLITE_ERROR; /* Files only. */
					break;
				}
				if ( osDeleteFileA(zConverted) ){
					rc = SQLITE_OK; /* Deleted OK. */
					break;
				}
				if ( !retryIoerr(&cnt, &lastErrno) ){
					rc = SQLITE_ERROR; /* No more retries. */
					break;
				}
			} while(1);
		}
#endif
		if( rc && rc!=SQLITE_IOERR_DELETE_NOENT ){
			rc = winLogError(SQLITE_IOERR_DELETE, lastErrno,
				"winDelete", zFilename);
		}else{
			logIoerr(cnt);
		}
		sqlite3_free(zConverted);
		OSTRACE(("DELETE \"%s\" %s\n", zFilename, (rc ? "failed" : "ok" )));
		return rc;
	}

	static int winAccess(sqlite3_vfs *pVfs, const char *zFilename, int flags, int *pResOut)
	{
		DWORD attr;
		int rc = 0;
		DWORD lastErrno;
		void *zConverted;
		UNUSED_PARAMETER(pVfs);

		SimulateIOError( return SQLITE_IOERR_ACCESS; );
		zConverted = convertUtf8Filename(zFilename);
		if( zConverted==0 ){
			return SQLITE_IOERR_NOMEM;
		}
		if( isNT() ){
			int cnt = 0;
			WIN32_FILE_ATTRIBUTE_DATA sAttrData;
			memset(&sAttrData, 0, sizeof(sAttrData));
			while( !(rc = osGetFileAttributesExW((LPCWSTR)zConverted,
				GetFileExInfoStandard, 
				&sAttrData)) && retryIoerr(&cnt, &lastErrno) ){}
			if( rc ){
				/* For an SQLITE_ACCESS_EXISTS query, treat a zero-length file
				** as if it does not exist.
				*/
				if(    flags==SQLITE_ACCESS_EXISTS
					&& sAttrData.nFileSizeHigh==0 
					&& sAttrData.nFileSizeLow==0 ){
						attr = INVALID_FILE_ATTRIBUTES;
				}else{
					attr = sAttrData.dwFileAttributes;
				}
			}else{
				logIoerr(cnt);
				if( lastErrno!=ERROR_FILE_NOT_FOUND && lastErrno!=ERROR_PATH_NOT_FOUND ){
					winLogError(SQLITE_IOERR_ACCESS, lastErrno, "winAccess", zFilename);
					sqlite3_free(zConverted);
					return SQLITE_IOERR_ACCESS;
				}else{
					attr = INVALID_FILE_ATTRIBUTES;
				}
			}
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			attr = osGetFileAttributesA((char*)zConverted);
		}
#endif
		sqlite3_free(zConverted);
		switch( flags ){
		case SQLITE_ACCESS_READ:
		case SQLITE_ACCESS_EXISTS:
			rc = attr!=INVALID_FILE_ATTRIBUTES;
			break;
		case SQLITE_ACCESS_READWRITE:
			rc = attr!=INVALID_FILE_ATTRIBUTES &&
				(attr & FILE_ATTRIBUTE_READONLY)==0;
			break;
		default:
			assert(!"Invalid flags argument");
		}
		*pResOut = rc;
		return SQLITE_OK;
	}


	static BOOL winIsVerbatimPathname(
		const char *zPathname
		){
			/*
			** If the path name starts with a forward slash or a backslash, it is either
			** a legal UNC name, a volume relative path, or an absolute path name in the
			** "Unix" format on Windows.  There is no easy way to differentiate between
			** the final two cases; therefore, we return the safer return value of TRUE
			** so that callers of this function will simply use it verbatim.
			*/
			if ( zPathname[0]=='/' || zPathname[0]=='\\' ){
				return TRUE;
			}

			/*
			** If the path name starts with a letter and a colon it is either a volume
			** relative path or an absolute path.  Callers of this function must not
			** attempt to treat it as a relative path name (i.e. they should simply use
			** it verbatim).
			*/
			if ( sqlite3Isalpha(zPathname[0]) && zPathname[1]==':' ){
				return TRUE;
			}

			/*
			** If we get to this point, the path name should almost certainly be a purely
			** relative one (i.e. not a UNC name, not absolute, and not volume relative).
			*/
			return FALSE;
	}

	static int winFullPathname(sqlite3_vfs *pVfs, const char *zRelative, int nFull, char *zFull)
	{

#if defined(__CYGWIN__)
		SimulateIOError( return SQLITE_ERROR );
		UNUSED_PARAMETER(nFull);
		assert( pVfs->mxPathname>=MAX_PATH );
		assert( nFull>=pVfs->mxPathname );
		if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
			/*
			** NOTE: We are dealing with a relative path name and the data
			**       directory has been set.  Therefore, use it as the basis
			**       for converting the relative path name to an absolute
			**       one by prepending the data directory and a slash.
			*/
			char zOut[MAX_PATH+1];
			memset(zOut, 0, MAX_PATH+1);
			cygwin_conv_path(CCP_POSIX_TO_WIN_A|CCP_RELATIVE, zRelative, zOut,
				MAX_PATH+1);
			sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
				sqlite3_data_directory, zOut);
		}else{
			cygwin_conv_path(CCP_POSIX_TO_WIN_A, zRelative, zFull, nFull);
		}
		return SQLITE_OK;
#endif

#if (SQLITE_OS_WINCE || SQLITE_OS_WINRT) && !defined(__CYGWIN__)
		SimulateIOError( return SQLITE_ERROR );
		/* WinCE has no concept of a relative pathname, or so I am told. */
		/* WinRT has no way to convert a relative path to an absolute one. */
		if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
			/*
			** NOTE: We are dealing with a relative path name and the data
			**       directory has been set.  Therefore, use it as the basis
			**       for converting the relative path name to an absolute
			**       one by prepending the data directory and a backslash.
			*/
			sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
				sqlite3_data_directory, zRelative);
		}else{
			sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s", zRelative);
		}
		return SQLITE_OK;
#endif

#if !SQLITE_OS_WINCE && !SQLITE_OS_WINRT && !defined(__CYGWIN__)
		DWORD nByte;
		void *zConverted;
		char *zOut;

		/* If this path name begins with "/X:", where "X" is any alphabetic
		** character, discard the initial "/" from the pathname.
		*/
		if( zRelative[0]=='/' && sqlite3Isalpha(zRelative[1]) && zRelative[2]==':' ){
			zRelative++;
		}

		/* It's odd to simulate an io-error here, but really this is just
		** using the io-error infrastructure to test that SQLite handles this
		** function failing. This function could fail if, for example, the
		** current working directory has been unlinked.
		*/
		SimulateIOError( return SQLITE_ERROR );
		if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
			/*
			** NOTE: We are dealing with a relative path name and the data
			**       directory has been set.  Therefore, use it as the basis
			**       for converting the relative path name to an absolute
			**       one by prepending the data directory and a backslash.
			*/
			sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
				sqlite3_data_directory, zRelative);
			return SQLITE_OK;
		}
		zConverted = convertUtf8Filename(zRelative);
		if( zConverted==0 ){
			return SQLITE_IOERR_NOMEM;
		}
		if( isNT() ){
			LPWSTR zTemp;
			nByte = osGetFullPathNameW((LPCWSTR)zConverted, 0, 0, 0);
			if( nByte==0 ){
				winLogError(SQLITE_ERROR, osGetLastError(),
					"GetFullPathNameW1", zConverted);
				sqlite3_free(zConverted);
				return SQLITE_CANTOPEN_FULLPATH;
			}
			nByte += 3;
			zTemp = sqlite3MallocZero( nByte*sizeof(zTemp[0]) );
			if( zTemp==0 ){
				sqlite3_free(zConverted);
				return SQLITE_IOERR_NOMEM;
			}
			nByte = osGetFullPathNameW((LPCWSTR)zConverted, nByte, zTemp, 0);
			if( nByte==0 ){
				winLogError(SQLITE_ERROR, osGetLastError(),
					"GetFullPathNameW2", zConverted);
				sqlite3_free(zConverted);
				sqlite3_free(zTemp);
				return SQLITE_CANTOPEN_FULLPATH;
			}
			sqlite3_free(zConverted);
			zOut = unicodeToUtf8(zTemp);
			sqlite3_free(zTemp);
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			char *zTemp;
			nByte = osGetFullPathNameA((char*)zConverted, 0, 0, 0);
			if( nByte==0 ){
				winLogError(SQLITE_ERROR, osGetLastError(),
					"GetFullPathNameA1", zConverted);
				sqlite3_free(zConverted);
				return SQLITE_CANTOPEN_FULLPATH;
			}
			nByte += 3;
			zTemp = sqlite3MallocZero( nByte*sizeof(zTemp[0]) );
			if( zTemp==0 ){
				sqlite3_free(zConverted);
				return SQLITE_IOERR_NOMEM;
			}
			nByte = osGetFullPathNameA((char*)zConverted, nByte, zTemp, 0);
			if( nByte==0 ){
				winLogError(SQLITE_ERROR, osGetLastError(),
					"GetFullPathNameA2", zConverted);
				sqlite3_free(zConverted);
				sqlite3_free(zTemp);
				return SQLITE_CANTOPEN_FULLPATH;
			}
			sqlite3_free(zConverted);
			zOut = sqlite3_win32_mbcs_to_utf8(zTemp);
			sqlite3_free(zTemp);
		}
#endif
		if( zOut ){
			sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s", zOut);
			sqlite3_free(zOut);
			return SQLITE_OK;
		}else{
			return SQLITE_IOERR_NOMEM;
		}
#endif
	}

#ifndef OMIT_LOAD_EXTENSION

	static void *CoreVFileSystem::DllOpen(const char *filename)
	{
		void *converted = convertUtf8Filename(filename);
		if (!converted)
			return nullptr;
		HANDLE h;
		if (isNT())
#if OS_WINRT
			h = osLoadPackagedLibrary((LPCWSTR)converted, 0);
#else
			h = osLoadLibraryW((LPCWSTR)converted);
#endif
#ifdef WIN32_HAS_ANSI
		else
			h = osLoadLibraryA((char *)converted);
#endif
		SysEx::Free(converted);
		return (void*)h;
	}

	static void CoreVFileSystem::DllError(int bufLength, char *bufOut)
	{
		getLastErrorMsg(osGetLastError(), bufLength, bufOut);
	}

	static void (*DllSym(void *handle, const char *sym))(void)
	{
		return (void(*)(void))osGetProcAddressA((HANDLE)h, sym);
	}

	static void DllClose(void *handle)
	{
		osFreeLibrary((HANDLE)pHandle);
	}

#else
#define winDlOpen  0
#define winDlError 0
#define winDlSym   0
#define winDlClose 0
#endif




}}

