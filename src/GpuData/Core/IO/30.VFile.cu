﻿#include "../Core.cu.h"

namespace Core { namespace IO
{

	RC VFile::Lock(LOCK lock) { return RC::OK; }
	RC VFile::Unlock(LOCK lock) { return RC::OK; }
	RC VFile::CheckReservedLock(int &lock) { return RC::OK; }
	RC VFile::FileControl(FCNTL op, void *arg) { return RC::NOTFOUND; }

	uint VFile::get_SectorSize() { return 0; }
	VFile::IOCAP VFile::get_DeviceCharacteristics() { return (VFile::IOCAP)0; }

	RC VFile::ShmLock(int offset, int n, SHM flags) { return RC::OK; }
	void VFile::ShmBarrier() { }
	RC VFile::ShmUnmap(bool deleteFlag) { return RC::OK; }
	RC VFile::ShmMap(int region, int sizeRegion, bool isWrite, void volatile **pp) { return RC::OK; }
}}