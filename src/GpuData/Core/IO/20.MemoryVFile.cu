﻿// memjournal.c
#include "../Core.cu.h"
using namespace Core;

namespace Core
{
#define JOURNAL_CHUNKSIZE ((int)(1024 - sizeof(FileChunk *)))

	typedef struct FilePoint FilePoint;
	typedef struct FileChunk FileChunk;

	struct FileChunk
	{
		FileChunk *Next;				// Next chunk in the journal
		uint8 Chunk[JOURNAL_CHUNKSIZE];	// Content of this chunk
	};

	struct FilePoint
	{
		int64 Offset;		// Offset from the beginning of the file
		FileChunk *Chunk;	// Specific chunk into which cursor points
	};

	class MemoryVFile : VFile
	{
	private:
		FileChunk *First;       // Head of in-memory chunk-list
		FilePoint _endpoint;    // Pointer to the end of the file
		FilePoint _readpoint;   // Pointer to the end of the last xRead()
		void Open();
	public:
		virtual int Read(void *buffer, int amount, int64 offset);
		virtual int Write(const void *buffer, int amount, int64 offset);
		virtual int Truncate(int64 size);
		virtual int Close();
		virtual int Sync(int flags);
		virtual int get_FileSize(int64 &size);
	};

	int MemoryVFile::Read(void *buffer, int amount, int64 offset)
	{
		// SQLite never tries to read past the end of a rollback journal file
		_assert(offset + amount <= _endpoint.Offset);
		FileChunk *chunk;
		if (_readpoint.Offset != offset || offset == 0)
		{
			int64 offset2 = 0;
			for (chunk = First; SysEx_ALWAYS(chunk) && (offset2 + JOURNAL_CHUNKSIZE) <= offset; chunk = chunk->Next)
				offset2 += JOURNAL_CHUNKSIZE;
		}
		else
			chunk = _readpoint.Chunk;
		int chunkOffset = (int)(offset % JOURNAL_CHUNKSIZE);
		uint8 *out = (uint8 *)buffer;
		int read = amount;
		do
		{
			int space = JOURNAL_CHUNKSIZE - chunkOffset;
			int copy = MIN(read, (JOURNAL_CHUNKSIZE - chunkOffset));
			_memcpy(out, &chunk->Chunk[chunkOffset], copy);
			out += copy;
			read -= space;
			chunkOffset = 0;
		} while (read >= 0 && (chunk = chunk->Next) && read > 0);
		_readpoint.Offset = offset + amount;
		_readpoint.Chunk = chunk;
		return RC::OK;
	}

	int MemoryVFile::Write(const void *buffer, int amount, int64 offset)
	{
		// An in-memory journal file should only ever be appended to. Random access writes are not required by sqlite.
		_assert(offset == _endpoint.Offset);
		uint8 *b = (uint8 *)buffer;
		while (amount > 0)
		{
			FileChunk *chunk = _endpoint.Chunk;
			int chunkOffset = (int)(_endpoint.Offset % JOURNAL_CHUNKSIZE);
			int space = MIN(amount, JOURNAL_CHUNKSIZE - chunkOffset);
			if (chunkOffset == 0)
			{
				// New chunk is required to extend the file
				FileChunk *newChunk = new FileChunk();
				if (!newChunk)
					return RC::IOERR_NOMEM;
				newChunk->Next = nullptr;
				if (chunk) { _assert(First); chunk->Next = newChunk; }
				else { _assert(!First); First = newChunk; }
				_endpoint.Chunk = newChunk;
			}
			_memcpy(&_endpoint.Chunk->Chunk[chunkOffset], b, space);
			b += space;
			amount -= space;
			_endpoint.Offset += space;
		}
		return RC::OK;
	}

	int MemoryVFile::Truncate(int64 size)
	{
		_assert(size == 0);
		FileChunk *chunk = First;
		while (chunk)
		{
			FileChunk *tmp = chunk;
			chunk = chunk->Next;
			SysEx::Free(tmp);
		}
		Open();
		return RC::OK;
	}

	int MemoryVFile::Close()
	{
		Truncate(0);
		return RC::OK;
	}

	int MemoryVFile::Sync(int flags)
	{
		return RC::OK;
	}

	int MemoryVFile::get_FileSize(int64 &size)
	{
		size = (int64)_endpoint.Offset;
		return RC::OK;
	}

	void MemoryVFile::Open()
	{
		_assert(SysEx_HASALIGNMENT8(this));
		_memset(this, 0, sizeof(MemoryVFile));
	}
}
