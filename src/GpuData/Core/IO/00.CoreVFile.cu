
#include <hip/hip_runtime.h>
﻿// journal.c
// 
namespace Core
{

#ifdef OMIT_WSD
# define PENDING_BYTE     (0x40000000)
#else
# define PENDING_BYTE      sqlite3PendingByte
#endif
#define RESERVED_BYTE     (PENDING_BYTE+1)
#define SHARED_FIRST      (PENDING_BYTE+2)
#define SHARED_SIZE       510

}
