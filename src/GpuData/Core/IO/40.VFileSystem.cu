﻿// os.c
#include "../Core.cu.h"

namespace Core { namespace IO
{
	__device__ static VFileSystem *_vfsList = nullptr;

	VFileSystem *VFileSystem::Find(const char *name)
	{
		VFileSystem *vfs = nullptr;
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX::STATIC_MASTER);
		MutexEx::Enter(mutex);
		for (vfs = _vfsList; vfs && _strcmp(name, vfs->Name); vfs = vfs->Next) { }
		MutexEx::Leave(mutex);
		return vfs;
	}

	__device__ static void UnlinkVfs(VFileSystem *vfs)
	{
		_assert(MutexEx::Held(MutexEx::Alloc(MutexEx::MUTEX::STATIC_MASTER)));
		if (!vfs) { }
		else if (_vfsList == vfs)
			_vfsList = vfs->Next;
		else if (_vfsList)
		{
			VFileSystem *p = _vfsList;
			while (p->Next && p->Next != vfs)
				p = p->Next;
			if (p->Next == vfs)
				p->Next = vfs->Next;
		}
	}

	int VFileSystem::RegisterVfs(VFileSystem *vfs, bool _default)
	{
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX::STATIC_MASTER);
		MutexEx::Enter(mutex);
		UnlinkVfs(vfs);
		if (_default || !_vfsList)
		{
			vfs->Next = _vfsList;
			_vfsList = vfs;
		}
		else
		{
			vfs->Next = _vfsList->Next;
			_vfsList->Next = vfs;
		}
		_assert(_vfsList != nullptr);
		MutexEx::Leave(mutex);
		return RC::OK;
	}

	int VFileSystem::UnregisterVfs(VFileSystem *vfs)
	{
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX::STATIC_MASTER);
		MutexEx::Enter(mutex);
		UnlinkVfs(vfs);
		MutexEx::Leave(mutex);
		return RC::OK;
	}
}}