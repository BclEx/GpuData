﻿// status.c
#include "Core.cu.h"
using namespace Core;

namespace Core
{
	typedef struct StatType StatType;
	static struct StatType
	{
		int nowValue[10];         // Current value
		int mxValue[10];          // Maximum value
	} Stat = { {0,}, {0,} };

	int StatusEx::StatusValue(StatusEx::STATUS op)
	{
		wsdStatInit;
		_assert(op >= 0 && op < ArraySize(wsdStat.nowValue));
		return wsdStat.nowValue[op];
	}

	void StatusEx::StatusAdd(int op, int N)
	{
		wsdStatInit;
		_assert(op >= 0 && op < ArraySize(wsdStat.nowValue));
		wsdStat.nowValue[op] += N;
		if (wsdStat.nowValue[op]>wsdStat.mxValue[op])
			wsdStat.mxValue[op] = wsdStat.nowValue[op];
	}

	void StatusEx::StatusSet(int op, int X){
		wsdStatInit;
		_assert(op >= 0 && op < ArraySize(wsdStat.nowValue));
		wsdStat.nowValue[op] = X;
		if (wsdStat.nowValue[op]>wsdStat.mxValue[op])
			wsdStat.mxValue[op] = wsdStat.nowValue[op];
	}

	int StatusEx::Status(int op, int *pCurrent, int *pHighwater, int resetFlag)
	{
		wsdStatInit;
		if (op < 0 || op >= ArraySize(wsdStat.nowValue))
			return RC::MISUSE_BKPT;
		*pCurrent = wsdStat.nowValue[op];
		*pHighwater = wsdStat.mxValue[op];
		if (resetFlag)
			wsdStat.mxValue[op] = wsdStat.nowValue[op];
		return RC::OK;
	}


}
