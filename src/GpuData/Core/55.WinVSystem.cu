#include "hip/hip_runtime.h"
﻿// ow_win.c
#include "../Core.cu.h"
#if 1 //OS_WIN // This file is used for Windows only
#include <Windows.h>

namespace Core { namespace IO
{
#pragma region Polyfill

#if !OS_WINNT && !defined(OMIT_WAL) // Compiling and using WAL mode requires several APIs that are only available in Windows platforms based on the NT kernel.
#error "WAL mode requires support from the Windows NT kernel, compile with OMIT_WAL."
#endif

	// Are most of the Win32 ANSI APIs available (i.e. with certain exceptions based on the sub-platform)?
#ifdef __CYGWIN__
#include <sys/cygwin.h>
#endif
#if !OS_WINCE && !OS_WINRT 
#define WIN32_HAS_ANSI
#endif
#if OS_WINCE || OS_WINNT || OS_WINRT
#define WIN32_HAS_WIDE
#endif
#if WIN32_FILEMAPPING_API && !defined(OMIT_WAL)
#if OS_WINRT
	WINBASEAPI HANDLE WINAPI CreateFileMappingFromApp(HANDLE, LPSECURITY_ATTRIBUTES, ULONG, ULONG64, LPCWSTR);
	WINBASEAPI LPVOID WINAPI MapViewOfFileFromApp(HANDLE, ULONG, ULONG64, SIZE_T);
#else
#if defined(WIN32_HAS_ANSI)
	WINBASEAPI HANDLE WINAPI CreateFileMappingA(HANDLE, LPSECURITY_ATTRIBUTES, DWORD, DWORD, DWORD, LPCSTR);
#endif
#if defined(WIN32_HAS_WIDE)
	WINBASEAPI HANDLE WINAPI CreateFileMappingW(HANDLE, LPSECURITY_ATTRIBUTES, DWORD, DWORD, DWORD, LPCWSTR);
#endif
	WINBASEAPI LPVOID WINAPI MapViewOfFile(HANDLE, DWORD, DWORD, DWORD, SIZE_T);
#endif
	WINBASEAPI BOOL WINAPI UnmapViewOfFile(LPCVOID);
#endif
#if OS_WINCE // WinCE lacks native support for file locking so we have to fake it with some code of our own.
	typedef struct winceLock
	{
		int Readers;       // Number of reader locks obtained
		bool Pending;      // Indicates a pending lock has been obtained
		bool Reserved;     // Indicates a reserved lock has been obtained
		bool Exclusive;    // Indicates an exclusive lock has been obtained
	} winceLock;
#endif

	// Some Microsoft compilers lack this definition.
#ifndef INVALID_FILE_ATTRIBUTES
#define INVALID_FILE_ATTRIBUTES ((DWORD)-1) 
#endif
#ifndef FILE_FLAG_MASK
#define FILE_FLAG_MASK (0xFF3C0000)
#endif
#ifndef FILE_ATTRIBUTE_MASK
#define FILE_ATTRIBUTE_MASK (0x0003FFF7)
#endif
#ifndef INVALID_SET_FILE_POINTER
#define INVALID_SET_FILE_POINTER ((DWORD)-1)
#endif

	// The following variable is (normally) set once and never changes thereafter.  It records whether the operating system is Win9x or WinNT.
	// 0:   Operating system unknown.
	// 1:   Operating system is Win9x.
	// 2:   Operating system is WinNT.
	// In order to facilitate testing on a WinNT system, the test fixture can manually set this value to 1 to emulate Win98 behavior.
#ifdef TEST
	int os_type = 0;
#else
	static int os_type = 0;
#endif
#if OS_WINCE || OS_WINRT
#define isNT() (true)
#elif !defined(WIN32_HAS_WIDE)
#define isNT() (false)
#else
	static bool isNT()
	{
		if (os_type == 0)
		{
			OSVERSIONINFOA sInfo;
			sInfo.dwOSVersionInfoSize = sizeof(sInfo);
			osGetVersionExA(&sInfo);
			os_type = (sInfo.dwPlatformId == VER_PLATFORM_WIN32_NT ? 2 : 1);
		}
		return (os_type == 2);
	}
#endif

#pragma endregion


	// LOCKFILE_FAIL_IMMEDIATELY is undefined on some Windows systems.
#ifndef LOCKFILE_FAIL_IMMEDIATELY
#define LOCKFILE_FAIL_IMMEDIATELY 1
#endif
#ifndef LOCKFILE_EXCLUSIVE_LOCK
#define LOCKFILE_EXCLUSIVE_LOCK 2
#endif

	/*
	** Historically, SQLite has used both the LockFile and LockFileEx functions.
	** When the LockFile function was used, it was always expected to fail
	** immediately if the lock could not be obtained.  Also, it always expected to
	** obtain an exclusive lock.  These flags are used with the LockFileEx function
	** and reflect those expectations; therefore, they should not be changed.
	*/
#ifndef SQLITE_LOCKFILE_FLAGS
#define SQLITE_LOCKFILE_FLAGS (LOCKFILE_FAIL_IMMEDIATELY | LOCKFILE_EXCLUSIVE_LOCK)
#endif
#ifndef SQLITE_LOCKFILEEX_FLAGS
#define SQLITE_LOCKFILEEX_FLAGS (LOCKFILE_FAIL_IMMEDIATELY)
#endif


#pragma region WinVFile

#ifndef OMIT_WAL // Forward references
	typedef struct winShm winShm;           // A connection to shared-memory
	typedef struct winShmNode winShmNode;   // A region of shared-memory
#endif

	// winFile
	class WinVFile : public VFile
	{
	public:
		enum class WINFILE : uint8
		{
			PERSIST_WA = 0x04,  // Persistent WAL mode
			PSOW = 0x10,		// SQLITE_IOCAP_POWERSAFE_OVERWRITE
		};

		VSystem *Vfs;			// The VFS used to open this file
		HANDLE H;               // Handle for accessing the file
		LOCK Lock;				// Type of lock currently held on this file
		short SharedLockByte;   // Randomly chosen byte used as a shared lock
		WINFILE CtrlFlags;      // Flags.  See WINFILE_* below
		DWORD LastErrno;        // The Windows errno from the last I/O error
#ifndef OMIT_WAL
		winShm *Shm;			// Instance of shared memory on this file
#endif
		const char *Path;		// Full pathname of this file
		int SizeChunk;          // Chunk size configured by FCNTL_CHUNK_SIZE
#if OS_WINCE
		LPWSTR DeleteOnClose;  // Name of file to delete when closing
		HANDLE Mutex;			// Mutex used to control access to shared lock
		HANDLE SharedHandle;	// Shared memory segment used for locking
		winceLock Local;        // Locks obtained by this instance of winFile
		winceLock *Shared;      // Global shared lock memory for the file
#endif
	};

#pragma endregion

#pragma region Win32

#ifndef WIN32_DBG_BUF_SIZE // The size of the buffer used by sqlite3_win32_write_debug().
#define WIN32_DBG_BUF_SIZE ((int)(4096 - sizeof(DWORD)))
#endif
#ifndef WIN32_DATA_DIRECTORY_TYPE // The value used with sqlite3_win32_set_directory() to specify that the data directory should be changed.
#define WIN32_DATA_DIRECTORY_TYPE (1)
#endif
#ifndef WIN32_TEMP_DIRECTORY_TYPE // The value used with sqlite3_win32_set_directory() to specify that the temporary directory should be changed.
#define WIN32_TEMP_DIRECTORY_TYPE (2) 
#endif

#pragma endregion

#pragma region Syscall

	typedef void (*syscall_ptr)(void);
#ifndef SYSCALL
#define SYSCALL syscall_ptr
#endif

	static struct win_syscall {
		const char *Name;            // Name of the system call
		syscall_ptr Current; // Current value of the system call
		syscall_ptr Default; // Default value
	} Syscalls[] = 
	{
#if !OS_WINCE && !OS_WINRT
		{"AreFileApisANSI", (SYSCALL)AreFileApisANSI, nullptr},
#else
		{"AreFileApisANSI", (SYSCALL)nullptr, nullptr},
#endif
#if OS_WINCE || OS_WINRT // This function is not available on Windows CE or WinRT.
#define osAreFileApisANSI() 1
#else
#define osAreFileApisANSI ((BOOL(WINAPI*)(VOID))Syscalls[0].Current)
#endif
#if OS_WINCE && defined(WIN32_HAS_WIDE)
		{"CharLowerW", (SYSCALL)CharLowerW, nullptr},
#else
		{"CharLowerW", (SYSCALL)nullptr, nullptr},
#endif
#define osCharLowerW ((LPWSTR(WINAPI*)(LPWSTR))aSyscall[1].pCurrent)
#if OS_WINCE && defined(WIN32_HAS_WIDE)
		{"CharUpperW", (SYSCALL)CharUpperW, nullptr},
#else
		{"CharUpperW", (SYSCALL)nullptr, nullptr},
#endif
#define osCharUpperW ((LPWSTR(WINAPI *)(LPWSTR))Syscalls[2].Current)
		{"CloseHandle", (SYSCALL)CloseHandle, nullptr},
#define osCloseHandle ((BOOL(WINAPI *)(HANDLE))Syscalls[3].Current)
#if defined(WIN32_HAS_ANSI)
		{"CreateFileA", (SYSCALL)CreateFileA, nullptr},
#else
		{"CreateFileA", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFileA ((HANDLE(WINAPI *)(LPCSTR,DWORD,DWORD,LPSECURITY_ATTRIBUTES,DWORD,DWORD,HANDLE))Syscalls[4].Current)
#if !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"CreateFileW", (SYSCALL)CreateFileW, nullptr},
#else
		{"CreateFileW", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFileW ((HANDLE(WINAPI*)(LPCWSTR,DWORD,DWORD,LPSECURITY_ATTRIBUTES,DWORD,DWORD,HANDLE))Syscalls[5].Current)
#if (!OS_WINRT && defined(WIN32_HAS_ANSI) && !defined(OMIT_WAL))
		{"CreateFileMappingA", (SYSCALL)CreateFileMappingA, nullptr},
#else
		{"CreateFileMappingA", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFileMappingA ((HANDLE(WINAPI *)(HANDLE,LPSECURITY_ATTRIBUTES,DWORD,DWORD,DWORD,LPCSTR))Syscalls[6].Current)

#if OS_WINCE || (!OS_WINRT && defined(WIN32_HAS_WIDE) && !defined(OMIT_WAL))
		{"CreateFileMappingW", (SYSCALL)CreateFileMappingW, nullptr},
#else
		{"CreateFileMappingW", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFileMappingW ((HANDLE(WINAPI *)(HANDLE,LPSECURITY_ATTRIBUTES,DWORD,DWORD,DWORD,LPCWSTR))Syscalls[7].Current)
#if !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"CreateMutexW", (SYSCALL)CreateMutexW, nullptr},
#else
		{"CreateMutexW", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateMutexW ((HANDLE(WINAPI *)(LPSECURITY_ATTRIBUTES,BOOL,LPCWSTR))Syscalls[8].Current)
#if defined(WIN32_HAS_ANSI)
		{"DeleteFileA", (SYSCALL)DeleteFileA, nullptr},
#else
		{"DeleteFileA", (SYSCALL)nullptr, nullptr},
#endif
#define osDeleteFileA ((BOOL(WINAPI *)(LPCSTR))Syscalls[9].Current)
#if defined(SQLITE_WIN32_HAS_WIDE)
		{"DeleteFileW", (SYSCALL)DeleteFileW, nullptr},
#else
		{"DeleteFileW", (SYSCALL)nullptr, nullptr},
#endif
#define osDeleteFileW ((BOOL(WINAPI *)(LPCWSTR))Syscalls[10].Current)
#if OS_WINCE
		{"FileTimeToLocalFileTime", (SYSCALL)FileTimeToLocalFileTime, nullptr},
#else
		{"FileTimeToLocalFileTime", (SYSCALL)nullptr, nullptr},
#endif
#define osFileTimeToLocalFileTime ((BOOL(WINAPI *)(CONST FILETIME*,LPFILETIME))Syscalls[11].Current)
#if OS_WINCE
		{"FileTimeToSystemTime", (SYSCALL)FileTimeToSystemTime, nullptr},
#else
		{"FileTimeToSystemTime", (SYSCALL)nullptr, nullptr},
#endif
#define osFileTimeToSystemTime ((BOOL(WINAPI *)(CONST FILETIME*,LPSYSTEMTIME))Syscalls[12].Current)
		{"FlushFileBuffers", (SYSCALL)FlushFileBuffers, nullptr},
#define osFlushFileBuffers ((BOOL(WINAPI *)(HANDLE))Syscalls[13].Current)
#if defined(WIN32_HAS_ANSI)
		{"FormatMessageA", (SYSCALL)FormatMessageA, nullptr},
#else
		{"FormatMessageA", (SYSCALL)nullptr, nullptr},
#endif
#define osFormatMessageA ((DWORD(WINAPI *)(DWORD,LPCVOID,DWORD,DWORD,LPSTR,DWORD,va_list*))Syscalls[14].Current)
#if defined(WIN32_HAS_WIDE)
		{"FormatMessageW", (SYSCALL)FormatMessageW, nullptr},
#else
		{"FormatMessageW", (SYSCALL)nullptr, nullptr},
#endif
#define osFormatMessageW ((DWORD(WINAPI *)(DWORD,LPCVOID,DWORD,DWORD,LPWSTR,DWORD,va_list*))Syscalls[15].Current)
#if !defined(OMIT_LOAD_EXTENSION)
		{"FreeLibrary", (SYSCALL)FreeLibrary, nullptr},
#else
		{"FreeLibrary", (SYSCALL)nullptr, nullptr},
#endif
#define osFreeLibrary ((BOOL(WINAPI *)(HMODULE))Syscalls[16].Current)
		{"GetCurrentProcessId", (SYSCALL)GetCurrentProcessId, nullptr},
#define osGetCurrentProcessId ((DWORD(WINAPI *)(VOID))Syscalls[17].Current)
#if !OS_WINCE && defined(WIN32_HAS_ANSI)
		{"GetDiskFreeSpaceA", (SYSCALL)GetDiskFreeSpaceA, nullptr},
#else
		{"GetDiskFreeSpaceA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetDiskFreeSpaceA ((BOOL(WINAPI *)(LPCSTR,LPDWORD,LPDWORD,LPDWORD,LPDWORD))Syscalls[18].Current)
#if !OS_WINCE && !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"GetDiskFreeSpaceW", (SYSCALL)GetDiskFreeSpaceW, nullptr},
#else
		{"GetDiskFreeSpaceW", (SYSCALL)nullptr, nullptr},
#endif
#define osGetDiskFreeSpaceW ((BOOL(WINAPI*)(LPCWSTR,LPDWORD,LPDWORD,LPDWORD,LPDWORD))Syscalls[19].Current)
#if defined(WIN32_HAS_ANSI)
		{"GetFileAttributesA", (SYSCALL)GetFileAttributesA, nullptr},
#else
		{"GetFileAttributesA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFileAttributesA ((DWORD(WINAPI *)(LPCSTR))Syscalls[20].Current)
#if !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"GetFileAttributesW", (SYSCALL)GetFileAttributesW, nullptr},
#else
		{"GetFileAttributesW", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFileAttributesW ((DWORD(WINAPI *)(LPCWSTR))Syscalls[21].Current)

#if defined(WIN32_HAS_WIDE)
		{"GetFileAttributesExW", (SYSCALL)GetFileAttributesExW, nullptr},
#else
		{"GetFileAttributesExW", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFileAttributesExW ((BOOL(WINAPI*)(LPCWSTR,GET_FILEEX_INFO_LEVELS,LPVOID))Syscalls[22].Current)
#if !OS_WINRT
		{"GetFileSize", (SYSCALL)GetFileSize, nullptr},
#else
		{"GetFileSize", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFileSize ((DWORD(WINAPI *)(HANDLE,LPDWORD))Syscalls[23].Current)
#if !OS_WINCE && defined(WIN32_HAS_ANSI)
		{"GetFullPathNameA", (SYSCALL)GetFullPathNameA, nullptr},
#else
		{"GetFullPathNameA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFullPathNameA ((DWORD(WINAPI*)(LPCSTR,DWORD,LPSTR,LPSTR*))Syscalls[24].Current)
#if !OS_WINCE && !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"GetFullPathNameW", (SYSCALL)GetFullPathNameW, nullptr},
#else
		{"GetFullPathNameW", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFullPathNameW ((DWORD(WINAPI *)(LPCWSTR,DWORD,LPWSTR,LPWSTR*))Syscalls[25].Current)
		{"GetLastError", (SYSCALL)GetLastError, nullptr},
#define osGetLastError ((DWORD(WINAPI *)(VOID))Syscalls[26].Current)
#if !defined(OMIT_LOAD_EXTENSION)
#if OS_WINCE
		// The GetProcAddressA() routine is only available on Windows CE.
		{"GetProcAddressA", (SYSCALL)GetProcAddressA, nullptr},
#else
		// All other Windows platforms expect GetProcAddress() to take an ANSI string regardless of the _UNICODE setting
		{"GetProcAddressA", (SYSCALL)GetProcAddress, nullptr},
#endif
#else
		{"GetProcAddressA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetProcAddressA ((FARPROC(WINAPI *)(HMODULE,LPCSTR))Syscalls[27].Current)
#if !OS_WINRT
		{"GetSystemInfo", (SYSCALL)GetSystemInfo, nullptr},
#else
		{"GetSystemInfo", (SYSCALL)nullptr, nullptr},
#endif
#define osGetSystemInfo ((VOID(WINAPI *)(LPSYSTEM_INFO))Syscalls[28].Current)
		{"GetSystemTime", (SYSCALL)GetSystemTime, nullptr},
#define osGetSystemTime ((VOID(WINAPIs*)(LPSYSTEMTIME))Syscalls[29].Current)
#if !OS_WINCE
		{"GetSystemTimeAsFileTime", (SYSCALL)GetSystemTimeAsFileTime, nullptr},
#else
		{"GetSystemTimeAsFileTime", (SYSCALL)nullptr, nullptr},
#endif
#define osGetSystemTimeAsFileTime ((VOID(WINAPI *)(LPFILETIME))Syscalls[30].Current)
#if defined(WIN32_HAS_ANSI)
		{"GetTempPathA", (SYSCALL)GetTempPathA, nullptr},
#else
		{"GetTempPathA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetTempPathA ((DWORD(WINAPI*)(DWORD,LPSTR))Syscalls[31].Current)
#if !OS_WINRT && defined(WIN32_HAS_WIDE)
		{"GetTempPathW", (SYSCALL)GetTempPathW, nullptr},
#else
		{"GetTempPathW", (SYSCALL)nullptr, nullptr},
#endif
#define osGetTempPathW ((DWORD(WINAPI *)(DWORD,LPWSTR))Syscalls[32].Current)
#if !OS_WINRT
		{"GetTickCount", (SYSCALL)GetTickCount, nullptr},
#else
		{"GetTickCount", (SYSCALL)nullptr, nullptr},
#endif
#define osGetTickCount ((DWORD(WINAPI *)(VOID))Syscalls[33].Current)
#if defined(SQLITE_WIN32_HAS_ANSI)
		{"GetVersionExA", (SYSCALL)GetVersionExA, nullptr},
#else
		{"GetVersionExA", (SYSCALL)nullptr, nullptr},
#endif
#define osGetVersionExA ((BOOL(WINAPI *)(LPOSVERSIONINFOA))Syscalls[34].Current)
		{"HeapAlloc", (SYSCALL)HeapAlloc, nullptr},
#define osHeapAlloc ((LPVOID(WINAPI *)(HANDLE,DWORD,SIZE_T))Syscalls[35].Current)
#if !OS_WINRT
		{"HeapCreate", (SYSCALL)HeapCreate, nullptr},
#else
		{"HeapCreate", (SYSCALL)nullptr, nullptr},
#endif
#define osHeapCreate ((HANDLE(WINAPI *)(DWORD,SIZE_T,SIZE_T))Syscalls[36].Current)
#if !OS_WINRT
		{"HeapDestroy", (SYSCALL)HeapDestroy, nullptr},
#else
		{"HeapDestroy", (SYSCALL)nullptr, nullptr},
#endif
#define osHeapDestroy ((BOOL(WINAPI *)(HANDLE))Syscalls[37].Current)
		{"HeapFree", (SYSCALL)HeapFree, nullptr},
#define osHeapFree ((BOOL(WINAPI *)(HANDLE,DWORD,LPVOID))Syscalls[38].Current)
		{"HeapReAlloc", (SYSCALL)HeapReAlloc, nullptr},
#define osHeapReAlloc ((LPVOID(WINAPI *)(HANDLE,DWORD,LPVOID,SIZE_T))Syscalls[39].Current)
		{"HeapSize", (SYSCALL)HeapSize, nullptr},

#define osHeapSize ((SIZE_T(WINAPI *)(HANDLE,DWORD,LPCVOID))Syscalls[40].Current)
#if !OS_WINRT
		{"HeapValidate", (SYSCALL)HeapValidate, nullptr},
#else
		{"HeapValidate", (SYSCALL)nullptr, nullptr},
#endif
#define osHeapValidate ((BOOL(WINAPI *)(HANDLE,DWORD,LPCVOID))Syscalls[41].Current)
#if defined(WIN32_HAS_ANSI) && !defined(OMIT_LOAD_EXTENSION)
		{"LoadLibraryA", (SYSCALL)LoadLibraryA, nullptr},
#else
		{"LoadLibraryA", (SYSCALL)nullptr, nullptr},
#endif
#define osLoadLibraryA ((HMODULE(WINAPI *)(LPCSTR))Syscalls[42].Current)
#if !OS_WINRT && defined(WIN32_HAS_WIDE) && !defined(OMIT_LOAD_EXTENSION)
		{"LoadLibraryW", (SYSCALL)LoadLibraryW, nullptr},
#else
		{"LoadLibraryW", (SYSCALL)nullptr, nullptr},
#endif
#define osLoadLibraryW ((HMODULE(WINAPI *)(LPCWSTR))Syscalls[43].Current)
#if !OS_WINRT
		{"LocalFree", (SYSCALL)LocalFree, nullptr},
#else
		{"LocalFree", (SYSCALL)nullptr, nullptr},
#endif
#define osLocalFree ((HLOCAL(WINAPI *)(HLOCAL))Syscalls[44].Current)
#if !OS_WINCE && !OS_WINRT
		{"LockFile", (SYSCALL)LockFile, nullptr},
#else
		{"LockFile", (SYSCALL)nullptr, nullptr},
#endif
#ifndef osLockFile
#define osLockFile ((BOOL(WINAPI *)(HANDLE,DWORD,DWORD,DWORD,DWORD))Syscalls[45].Current)
#endif
#if !OS_WINCE
		{"LockFileEx", (SYSCALL)LockFileEx, nullptr},
#else
		{"LockFileEx", (SYSCALL)nullptr, nullptr},
#endif
#ifndef osLockFileEx
#define osLockFileEx ((BOOL(WINAPI *)(HANDLE,DWORD,DWORD,DWORD,DWORD,LPOVERLAPPED))Syscalls[46].Current)
#endif
#if OS_WINCE || (!OS_WINRT && !defined(OMIT_WAL))
		{"MapViewOfFile", (SYSCALL)MapViewOfFile,nullptr},
#else
		{"MapViewOfFile", (SYSCALL)nullptr,nullptr},
#endif
#define osMapViewOfFile ((LPVOID(WINAPI *)(HANDLE,DWORD,DWORD,DWORD,SIZE_T))Syscalls[47].Current)
		{"MultiByteToWideChar", (SYSCALL)MultiByteToWideChar, nullptr},
#define osMultiByteToWideChar ((int(WINAPI *)(UINT,DWORD,LPCSTR,int,LPWSTR,int))Syscalls[48].Current)
		{"QueryPerformanceCounter", (SYSCALL)QueryPerformanceCounter, nullptr},
#define osQueryPerformanceCounter ((BOOL(WINAPI *)(LARGE_INTEGER*))Syscalls[49].Current)
		{"ReadFile", (SYSCALL)ReadFile, nullptr},
#define osReadFile ((BOOL(WINAPI *)(HANDLE,LPVOID,DWORD,LPDWORD,LPOVERLAPPED))Syscalls[50].Current)
		{"SetEndOfFile", (SYSCALL)SetEndOfFile, nullptr},
#define osSetEndOfFile ((BOOL(WINAPI *)(HANDLE))Syscalls[51].Current)
#if !OS_WINRT
		{"SetFilePointer", (SYSCALL)SetFilePointer, nullptr},
#else
		{"SetFilePointer", (SYSCALL)nullptr, nullptr},
#endif
#define osSetFilePointer ((DWORD(WINAPI *)(HANDLE,LONG,PLONG,DWORD))Syscalls[52].Current)
#if !OS_WINRT
		{"Sleep", (SYSCALL)Sleep, nullptr},
#else
		{"Sleep", (SYSCALL)nullptr, nullptr},
#endif
#define osSleep ((VOID(WINAPI *)(DWORD))Syscalls[53].Current)
		{"SystemTimeToFileTime", (SYSCALL)SystemTimeToFileTime, nullptr},
#define osSystemTimeToFileTime ((BOOL(WINAPI *)(CONST SYSTEMTIME*,LPFILETIME))Syscalls[54].Current)
#if !OS_WINCE && !OS_WINRT
		{"UnlockFile", (SYSCALL)UnlockFile, nullptr},
#else
		{"UnlockFile", (SYSCALL)nullptr, nullptr},
#endif
#ifndef osUnlockFile
#define osUnlockFile ((BOOL(WINAPI *)(HANDLE,DWORD,DWORD,DWORD,DWORD))Syscalls[55].Current)
#endif
#if !OS_WINCE
		{"UnlockFileEx", (SYSCALL)UnlockFileEx, nullptr},
#else
		{"UnlockFileEx", (SYSCALL)nullptr, nullptr},
#endif
#define osUnlockFileEx ((BOOL(WINAPI *)(HANDLE,DWORD,DWORD,DWORD,LPOVERLAPPED))Syscalls[56].Current)
#if OS_WINCE || !defined(OMIT_WAL)
		{"UnmapViewOfFile", (SYSCALL)UnmapViewOfFile, nullptr},
#else
		{"UnmapViewOfFile", (SYSCALL)nullptr, nullptr},
#endif
#define osUnmapViewOfFile ((BOOL(WINAPI *)(LPCVOID))Syscalls[57].Current)
		{"WideCharToMultiByte", (SYSCALL)WideCharToMultiByte, nullptr},
#define osWideCharToMultiByte ((int(WINAPI *)(UINT,DWORD,LPCWSTR,int,LPSTR,int,LPCSTR,LPBOOL))Syscalls[58].Current)
		{"WriteFile", (SYSCALL)WriteFile, nullptr},
#define osWriteFile ((BOOL(WINAPI *)(HANDLE,LPCVOID,DWORD,LPDWORD,LPOVERLAPPED))Syscalls[59].Current)
#if OS_WINRT
		{"CreateEventExW", (SYSCALL)CreateEventExW, nullptr},
#else
		{"CreateEventExW", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateEventExW ((HANDLE(WINAPI *)(LPSECURITY_ATTRIBUTES,LPCWSTR,DWORD,DWORD))Syscalls[60].Current)
#if !OS_WINRT
		{"WaitForSingleObject", (SYSCALL)WaitForSingleObject, nullptr},
#else
		{"WaitForSingleObject", (SYSCALL)nullptr, nullptr},
#endif
#define osWaitForSingleObject ((DWORD(WINAPI *)(HANDLE,DWORD))Syscalls[61].Current)
#if OS_WINRT
		{"WaitForSingleObjectEx", (SYSCALL)WaitForSingleObjectEx, nullptr},
#else
		{"WaitForSingleObjectEx", (SYSCALL)nullptr, nullptr},
#endif
#define osWaitForSingleObjectEx ((DWORD(WINAPI *)(HANDLE,DWORD,BOOL))Syscalls[62].Current)
#if OS_WINRT
		{"SetFilePointerEx", (SYSCALL)SetFilePointerEx, nullptr},
#else
		{"SetFilePointerEx", (SYSCALL)nullptr, nullptr},
#endif
#define osSetFilePointerEx ((BOOL(WINAPI *)(HANDLE,LARGE_INTEGER,PLARGE_INTEGER,DWORD))Syscalls[63].Current)
#if OS_WINRT
		{"GetFileInformationByHandleEx", (SYSCALL)GetFileInformationByHandleEx, nullptr},
#else
		{"GetFileInformationByHandleEx", (SYSCALL)nullptr, nullptr},
#endif
#define osGetFileInformationByHandleEx ((BOOL(WINAPI *)(HANDLE,FILE_INFO_BY_HANDLE_CLASS,LPVOID,DWORD))Syscalls[64].Current)
#if OS_WINRT && !defined(OMIT_WAL)
		{"MapViewOfFileFromApp", (SYSCALL)MapViewOfFileFromApp, nullptr},
#else
		{"MapViewOfFileFromApp", (SYSCALL)nullptr, nullptr},
#endif
#define osMapViewOfFileFromApp ((LPVOID(WINAPI *)(HANDLE,ULONG,ULONG64,SIZE_T))Syscalls[65].Current)
#if OS_WINRT
		{"CreateFile2", (SYSCALL)CreateFile2, nullptr},
#else
		{"CreateFile2", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFile2 ((HANDLE(WINAPI *)(LPCWSTR,DWORD,DWORD,DWORD,LPCREATEFILE2_EXTENDED_PARAMETERS))Syscalls[66].Current)
#if OS_WINRT && !defined(OMIT_LOAD_EXTENSION)
		{"LoadPackagedLibrary", (SYSCALL)LoadPackagedLibrary, nullptr},
#else
		{"LoadPackagedLibrary", (SYSCALL)nullptr, nullptr},
#endif
#define osLoadPackagedLibrary ((HMODULE(WINAPI *)(LPCWSTR,DWORD))Syscalls[67].Current)
#if OS_WINRT
		{"GetTickCount64", (SYSCALL)GetTickCount64, nullptr},
#else
		{"GetTickCount64", (SYSCALL)nullptr, nullptr},
#endif
#define osGetTickCount64 ((ULONGLONG(WINAPI *)(VOID))Syscalls[68].Current)
#if OS_WINRT
		{"GetNativeSystemInfo", (SYSCALL)GetNativeSystemInfo, nullptr},
#else
		{"GetNativeSystemInfo", (SYSCALL)nullptr, nullptr},
#endif
#define osGetNativeSystemInfo ((VOID(WINAPI *)(LPSYSTEM_INFO))Syscalls[69].Current)
#if defined(WIN32_HAS_ANSI)
		{"OutputDebugStringA", (SYSCALL)OutputDebugStringA, nullptr},
#else
		{"OutputDebugStringA", (SYSCALL)nullptr, nullptr},
#endif
#define osOutputDebugStringA ((VOID(WINAPI *)(LPCSTR))Syscalls[70].Current)
#if defined(WIN32_HAS_WIDE)
		{"OutputDebugStringW", (SYSCALL)OutputDebugStringW, nullptr},
#else
		{"OutputDebugStringW", (SYSCALL)nullptr, nullptr},
#endif
#define osOutputDebugStringW ((VOID(WINAPI *)(LPCWSTR))Syscalls[71].Current)
		{"GetProcessHeap", (SYSCALL)GetProcessHeap, nullptr},
#define osGetProcessHeap ((HANDLE(WINAPI *)(VOID))Syscalls[72].Current)
#if OS_WINRT && !defined(OMIT_WAL)
		{"CreateFileMappingFromApp", (SYSCALL)CreateFileMappingFromApp, nullptr},
#else
		{"CreateFileMappingFromApp", (SYSCALL)nullptr, nullptr},
#endif
#define osCreateFileMappingFromApp ((HANDLE(WINAPI *)(HANDLE,LPSECURITY_ATTRIBUTES,ULONG,ULONG64,LPCWSTR))Syscalls[73].Current)
	}; // End of the overrideable system calls

	static int winSetSystemCall(VSystem *notUsed, const char *name, syscall_ptr newFunc)
	{
		RC rc = RC::NOTFOUND;
		if (name == nullptr)
		{
			/// If no zName is given, restore all system calls to their default settings and return NULL
			rc = RC::OK;
			for (int i = 0; i < __arrayStaticLength(Syscalls); i++)
				if (Syscalls[i].Default)
					Syscalls[i].Current = Syscalls[i].Default;
			return rc;
		}
		// If zName is specified, operate on only the one system call specified.
		for (int i = 0; i < __arrayStaticLength(Syscalls); i++)
		{
			if (_!strcmp(name, Syscalls[i].Name))
			{
				if (!Syscalls[i].Default)
					Syscalls[i].Default = Syscalls[i].Current;
				rc = RC::OK;
				if (!newFunc) newFunc = Syscalls[i].Default;
				Syscalls[i].Current = newFunc;
				break;
			}
		}
		return rc;
	}

	static syscall_ptr winGetSystemCall(VSystem *notUsed, const char *name)
	{
		for (int i = 0; i < __arrayStaticLength(Syscalls); i++)
			if (!_strcmp(name, Syscalls[i].Name)) return Syscalls[i].Current;
		return nullptr;
	}

	static const char *winNextSystemCall(VSystem *notUsed, const char *name)
	{
		int i = -1;
		if (name)
			for (i = 0; i < __arrayStaticLength(Syscalls)-1; i++)
				if (!_strcmp(name, Syscalls[i].Name)) break;
		for (i++; i < __arrayStaticLength(Syscalls); i++)
			if (Syscalls[i].Current) return Syscalls[i].Name;
		return 0;
	}

#pragma endregion

#pragma region Win32

	void win32_WriteDebug(const char *buf, int bufLength)
	{
		char dbgBuf[WIN32_DBG_BUF_SIZE];
		int min = MIN(bufLength, (WIN32_DBG_BUF_SIZE - 1)); // may be negative.
		if (min < -1) min = -1; // all negative values become -1.
		_assert(min == -1 || min == 0 || min < WIN32_DBG_BUF_SIZE);
#if defined(WIN32_HAS_ANSI)
		if (min > 0)
		{
			memset(dbgBuf, 0, WIN32_DBG_BUF_SIZE);
			memcpy(dbgBuf, buf, min);
			osOutputDebugStringA(dbgBuf);
		}
		else
			osOutputDebugStringA(buf);
#elif defined(WIN32_HAS_WIDE)
		memset(dbgBuf, 0, WIN32_DBG_BUF_SIZE);
		if (osMultiByteToWideChar(osAreFileApisANSI() ? CP_ACP : CP_OEMCP, 0, buf, min, (LPWSTR)dbgBuf, WIN32_DBG_BUF_SIZE/sizeof(WCHAR)) <= 0)
			return;
		osOutputDebugStringW((LPCWSTR)dbgBuf);
#else
		if (min > 0)
		{
			memset(dbgBuf, 0, WIN32_DBG_BUF_SIZE);
			memcpy(dbgBuf, buf, min);
			fprintf(stderr, "%s", dbgBuf);
		}
		else
			fprintf(stderr, "%s", buf);
#endif
	}

#if OS_WINRT
	static HANDLE sleepObj = NULL;
#endif
	void win32_Sleep(DWORD milliseconds)
	{
#if OS_WINRT
		if (sleepObj == NULL)
			sleepObj = osCreateEventExW(NULL, NULL, CREATE_EVENT_MANUAL_RESET, SYNCHRONIZE);
		_assert(sleepObj != NULL);
		osWaitForSingleObjectEx(sleepObj, milliseconds, FALSE);
#else
		osSleep(milliseconds);
#endif
	}

#pragma endregion

#pragma region WIN32_MALLOC
#ifdef WIN32_MALLOC

	// If compiled with WIN32_MALLOC on Windows, we will use the various Win32 API heap functions instead of our own.

	// If this is non-zero, an isolated heap will be created by the native Win32 allocator subsystem; otherwise, the default process heap will be used.  This
	// setting has no effect when compiling for WinRT.  By default, this is enabled and an isolated heap will be created to store all allocated data.
	//
	//*****************************************************************************
	// WARNING: It is important to note that when this setting is non-zero and the winMemShutdown function is called (e.g. by the sqlite3_shutdown
	//          function), all data that was allocated using the isolated heap will be freed immediately and any attempt to access any of that freed
	//          data will almost certainly result in an immediate access violation.
	//*****************************************************************************
#ifndef WIN32_HEAP_CREATE
#define WIN32_HEAP_CREATE (TRUE)
#endif
#ifndef WIN32_HEAP_INIT_SIZE // The initial size of the Win32-specific heap.  This value may be zero.
#define WIN32_HEAP_INIT_SIZE ((DEFAULT_CACHE_SIZE) * (DEFAULT_PAGE_SIZE) + 4194304)
#endif
#ifndef WIN32_HEAP_MAX_SIZE // The maximum size of the Win32-specific heap.  This value may be zero.
#define WIN32_HEAP_MAX_SIZE (0)
#endif
#ifndef WIN32_HEAP_FLAGS // The extra flags to use in calls to the Win32 heap APIs. This value may be zero for the default behavior.
#define WIN32_HEAP_FLAGS (0)
#endif

	// The winMemData structure stores information required by the Win32-specific sqlite3_mem_methods implementation.
	typedef struct WinMemData
	{
#ifdef _DEBUG
		uint32 Magic;    // Magic number to detect structure corruption.
#endif
		HANDLE Heap; // The handle to our heap.
		BOOL Owned;  // Do we own the heap (i.e. destroy it on shutdown)?
	} WinMemData;

#ifdef _DEBUG
#define WINMEM_MAGIC 0x42b2830b
#endif

	static struct WinMemData winMemData_ = {
#ifdef _DEBUG
		WINMEM_MAGIC,
#endif
		NULL, FALSE
	};

#ifdef _DEBUG
#define winMemAssertMagic() _assert(winMemData_.Magic == WINMEM_MAGIC)
#else
#define winMemAssertMagic()
#endif
#define winMemGetHeap() winMemData_.Heap

	void *WinMem::Malloc(int bytes)
	{
		winMemAssertMagic();
		HANDLE heap = winMemGetHeap();
		_assert(heap != 0);
		_assert(heap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(WIN32_MALLOC_VALIDATE)
		_assert(osHeapValidate(heap, WIN32_HEAP_FLAGS, NULL));
#endif
		_assert(bytes >=0);
		void *p = osHeapAlloc(heap, WIN32_HEAP_FLAGS, (SIZE_T)bytes);
		if (!p)
			SysEx_LOG(RC::NOMEM, "failed to HeapAlloc %u bytes (%d), heap=%p", bytes, osGetLastError(), (void*)heap);
		return p;
	}

	void WinMem::Free(void *prior)
	{
		winMemAssertMagic();
		HANDLE heap = winMemGetHeap();
		_assert(heap != 0);
		_assert(heap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(WIN32_MALLOC_VALIDATE)
		_assert(osHeapValidate(heap, WIN32_HEAP_FLAGS, prior));
#endif
		if (!prior) return; // Passing NULL to HeapFree is undefined.
		if (!osHeapFree(heap, WIN32_HEAP_FLAGS, prior))
			SysEx_LOG(RC::NOMEM, "failed to HeapFree block %p (%d), heap=%p", prior, osGetLastError(), (void*)heap);
	}

	void *WinMem::Realloc(void *prior, int bytes)
	{
		winMemAssertMagic();
		HANDLE heap = winMemGetHeap();
		_assert(hHeap != 0 );
		_assert(hHeap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(SQLITE_WIN32_MALLOC_VALIDATE)
		_assert(osHeapValidate(heap, WIN32_HEAP_FLAGS, prior));
#endif
		_assert(bytes >= 0);
		void *p;
		if (!prior)
			p = osHeapAlloc(heap, WIN32_HEAP_FLAGS, (SIZE_T)bytes);
		else
			p = osHeapReAlloc(heap, WIN32_HEAP_FLAGS, prior, (SIZE_T)bytes);
		if (!p)
			SysEx_LOG(RC::NOMEM, "failed to %s %u bytes (%d), heap=%p", (prior ? "HeapReAlloc" : "HeapAlloc"), bytes, osGetLastError(), (void*)heap);
		return p;
	}

	int WinMem::Size(void *p)
	{
		winMemAssertMagic();
		HANDLE heap = winMemGetHeap();
		vassert(heap != 0);
		_assert(heap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(SQLITE_WIN32_MALLOC_VALIDATE)
		_assert(osHeapValidate(heap, WIN32_HEAP_FLAGS, NULL));
#endif
		if (!p) return 0;
		SIZE_T n = osHeapSize(heap, WIN32_HEAP_FLAGS, p);
		if (n == (SIZE_T)-1)
		{
			SysEx_LOG(RC::NOMEM, "failed to HeapSize block %p (%d), heap=%p", p, osGetLastError(), (void*)heap);
			return 0;
		}
		return (int)n;
	}

	int WinMem::Roundup(int bytes)
	{
		return bytes;
	}

	RC WinMem::Init(void *appData)
	{
		WinMemData *winMemData = (winMemData *)appData;
		if (!winMemData) return RC::ERROR;
		_assert(winMemData->Magic == WINMEM_MAGIC);
#if !OS_WINRT && WIN32_HEAP_CREATE
		if (!winMemData->Heap)
		{
			winMemData->Heap = osHeapCreate(WIN32_HEAP_FLAGS, WIN32_HEAP_INIT_SIZE, WIN32_HEAP_MAX_SIZE);
			if (!winMemData->Heap)
			{
				SysEx_LOG(RC::NOMEM, "failed to HeapCreate (%d), flags=%u, initSize=%u, maxSize=%u", osGetLastError(), WIN32_HEAP_FLAGS, WIN32_HEAP_INIT_SIZE, WIN32_HEAP_MAX_SIZE);
				return RC::NOMEM;
			}
			winMemData->Owned = TRUE;
			_assert(winMemData->Owned);
		}
#else
		winMemData->Heap = osGetProcessHeap();
		if (!winMemData->Heap)
		{
			SysEx_LOG(RC::NOMEM, "failed to GetProcessHeap (%d)", osGetLastError());
			return RC::NOMEM;
		}
		winMemData->Owned = FALSE;
		_assert(!winMemData->Owned);
#endif
		_assert(winMemData->Heap != 0);
		_assert(winMemData->Heap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(WIN32_MALLOC_VALIDATE)
		_assert(osHeapValidate(winMemData->Heap, WIN32_HEAP_FLAGS, NULL));
#endif
		return RC::OK;
	}

	void WinMem::Shutdown(void *appData)
	{
		WinMemData *winMemData = (winMemData *)appData;
		if (!winMemData) return;
		if (winMemData->Heap)
		{
			_assert(winMemData->Heap != INVALID_HANDLE_VALUE);
#if !OS_WINRT && defined(WIN32_MALLOC_VALIDATE)
			_assert(osHeapValidate(winMemData->Heap, WIN32_HEAP_FLAGS, NULL));
#endif
			if (winMemData->Owned)
			{
				if (!osHeapDestroy(winMemData->Heap))
					SysEx_LOG(RC::NOMEM, "failed to HeapDestroy (%d), heap=%p", osGetLastError(), (void*)winMemData->Heap);
				winMemData->Owned = FALSE;
			}
			winMemData->Heap = NULL;
		}
	}

#endif
#pragma endregion

#pragma region String Converters

	static LPWSTR Utf8ToUnicode(const char *name)
	{
		int c = osMultiByteToWideChar(CP_UTF8, 0, name, -1, NULL, 0);
		if (!c)
			return nullptr;
		LPWSTR wideName = SysEx::Alloc(c*sizeof(wideName[0]), true);
		if (!wideName)
			return nullptr;
		c = osMultiByteToWideChar(CP_UTF8, 0, name, -1, wideName, c);
		if (!c)
		{
			SysEx::Free(wideName);
			wideName = nullptr;
		}
		return wideName;
	}

	static char *UnicodeToUtf8(LPCWSTR wideName)
	{
		int c = osWideCharToMultiByte(CP_UTF8, 0, wideName, -1, 0, 0, 0, 0);
		if (!c)
			return nullptr;
		char *name = SysEx::Alloc(c, true);
		if (!name)
			return nullptr;
		c = osWideCharToMultiByte(CP_UTF8, 0, wideName, -1, filename, c, 0, 0);
		if (!c)
		{
			SysEx::Free(name);
			name = nullptr;
		}
		return name;
	}

	static LPWSTR MbcsToUnicode(const char *name)
	{
		int codepage = (osAreFileApisANSI() ? CP_ACP : CP_OEMCP);
		int c = osMultiByteToWideChar(codepage, 0, name, -1, NULL, 0)*sizeof(WCHAR);
		if (!c)
			return nullptr;
		LPWSTR mbcsName = SysEx::Alloc(c*sizeof(mbcsName[0]), true);
		if (!mbcsName)
			return nullptr;
		c = osMultiByteToWideChar(codepage, 0, name, -1, mbcsName, c);
		if (!c)
		{
			SysEx::Free(mbcsName);
			mbcsName = nullptr;
		}
		return mbcsName;
	}

	static char *UnicodeToMbcs(LPCWSTR wideName)
	{
		int codepage = (osAreFileApisANSI() ? CP_ACP : CP_OEMCP);
		int c = osWideCharToMultiByte(codepage, 0, wideName, -1, 0, 0, 0, 0);
		if (!c)
			return nullptr;
		char *name = SysEx::Alloc(c, true);
		if (!ame)
			return nullptr;
		c = osWideCharToMultiByte(codepage, 0, wideName, -1, filename, c, 0, 0);
		if (!c)
		{
			SysEx::Free(name);
			name = nullptr;
		}
		return name;
	}

	char *win32_MbcsToUtf8(const char *name)
	{
		LPWSTR tmpWide = MbcsToUnicode(name);
		if (!tmpWide)
			return nullptr;
		char *nameUtf8 = UnicodeToUtf8(tmpWide);
		SysEx::Free(tmpWide);
		return nameUtf8;
	}

	char *win32_Utf8ToMbcs(const char *name)
	{
		LPWSTR tmpWide = Utf8ToUnicode(name);
		if (!tmpWide)
			return nullptr;
		char *nameMbcs = UnicodeToMbcs(tmpWide);
		SysEx::Free(tmpWide);
		return nameMbcs;
	}

#pragma endregion

#pragma region Win32

	RC win32_SetDirectory(DWORD type, LPCWSTR value)
	{
#ifndef OMIT_AUTOINIT
		RC rc = sqlite3_initialize();
		if (rc) return rc;
#endif
		char **directory = nullptr;
		if (type == WIN32_DATA_DIRECTORY_TYPE)
			directory = &data_directory;
		else if (type == WIN32_TEMP_DIRECTORY_TYPE)
			directory = &temp_directory;
		_assert(!directory || type == WIN32_DATA_DIRECTORY_TYPE || type == WIN32_TEMP_DIRECTORY_TYPE);
		_assert(!directory || SysEx::MemdebugHasType(*directory, SysEx::MEMTYPE::HEAP));
		if (directory)
		{
			char *valueUtf8 = nullptr;
			if (value && value[0])
			{
				valueUtf8 = UnicodeToUtf8(value);
				if (!valueUtf8)
					return RC::NOMEM;
			}
			SysEx::Free(*directory);
			*directory = valueUtf8;
			return RC::OK;
		}
		return RC::ERROR;
	}

#pragma endregion

#pragma region OS Errors

	static RC getLastErrorMsg(DWORD lastErrno, int bufLength, char *buf)
	{
		// FormatMessage returns 0 on failure.  Otherwise it returns the number of TCHARs written to the output
		// buffer, excluding the terminating null char.
		DWORD dwLen = 0;
		char *out = nullptr;
		if (isNT())
		{
#if OS_WINRT
			WCHAR tempWide[MAX_PATH + 1]; // NOTE: Somewhat arbitrary.
			dwLen = osFormatMessageW(FORMAT_MESSAGE_FROM_SYSTEM | FORMAT_MESSAGE_IGNORE_INSERTS, NULL, lastErrno, 0, tempWide, MAX_PATH, 0);
#else
			LPWSTR tempWide = NULL;
			dwLen = osFormatMessageW(FORMAT_MESSAGE_ALLOCATE_BUFFER | FORMAT_MESSAGE_FROM_SYSTEM | FORMAT_MESSAGE_IGNORE_INSERTS, NULL, lastErrno, 0, (LPWSTR)&tempWide, 0, 0);
#endif
			if (dwLen > 0)
			{
				// allocate a buffer and convert to UTF8
				SysEx::BeginBenignAlloc();
				out = UnicodeToUtf8(tempWide);
				SysEx::EndBenignAlloc();
#if !OS_WINRT
				// free the system buffer allocated by FormatMessage
				osLocalFree(tempWide);
#endif
			}
		}
#ifdef WIN32_HAS_ANSI
		else
		{
			char *temp = NULL;
			dwLen = osFormatMessageA(FORMAT_MESSAGE_ALLOCATE_BUFFER | FORMAT_MESSAGE_FROM_SYSTEM | FORMAT_MESSAGE_IGNORE_INSERTS, NULL, lastErrno, 0, (LPSTR)&temp, 0, 0);
			if (dwLen > 0)
			{
				// allocate a buffer and convert to UTF8
				SysEx::BeginBenignAlloc();
				out = win32MbcsToUtf8(temp);
				SysEx::EndBenignAlloc();
				// free the system buffer allocated by FormatMessage
				osLocalFree(temp);
			}
		}
#endif
		if (!dwLen)
			_snprintf(bufLength, buf, "OsError 0x%x (%u)", lastErrno, lastErrno);
		else
		{
			// copy a maximum of nBuf chars to output buffer
			_snprintf(bufLength, buf, "%s", out);
			// free the UTF8 buffer
			SysEx::Free(out);
		}
		return RC::OK;
	}

#define winLogError(a,b,c,d) winLogErrorAtLine(a,b,c,d,__LINE__)
	static RC winLogErrorAtLine(RC errcode, DWORD lastErrno, const char *func, const char *path, int line)
	{
		char msg[500]; // Human readable error text
		msg[0] = 0;
		getLastErrorMsg(lastErrno, sizeof(msg), msg);
		_assert(errcode != RC::OK);
		if (!path) path = "";
		int i;
		for (i = 0; msg[i] && mMsg[i] != '\r' && msg[i] != '\n'; i++) { }
		msg[i] = 0;
		SysEx_LOG(errcode, "os_win.c:%d: (%d) %s(%s) - %s", line, lastErrno, func, path, msg);
		return errcode;
	}

#ifndef WIN32_IOERR_RETRY
#define WIN32_IOERR_RETRY 10
#endif
#ifndef WIN32_IOERR_RETRY_DELAY
#define WIN32_IOERR_RETRY_DELAY 25
#endif
	static int retryIoerr(int *retry, DWORD *error)
	{
		static int win32IoerrRetry = WIN32_IOERR_RETRY;
		static int win32IoerrRetryDelay = WIN32_IOERR_RETRY_DELAY;

		DWORD e = osGetLastError();
		if (*retry >= win32IoerrRetry)
		{
			if (error)
				*error = e;
			return 0;
		}
		if (e == ERROR_ACCESS_DENIED || e == ERROR_LOCK_VIOLATION || e == ERROR_SHARING_VIOLATION)
		{
			win32_Sleep(win32IoerrRetryDelay*(1+*retry));
			++*retry;
			return 1;
		}
		if (error)
			*error = e;
		return 0;
	}

	static void logIoerr(int retry)
	{
		if (retry)
			SysEx_LOG(RC::IOERR, "delayed %dms for lock/sharing conflict", win32IoerrRetryDelay*nRetry*(retry+1)/2);
	}

#pragma endregion

#pragma region WinCE Only
#if OS_WINCE

#define HANDLE_TO_WINFILE(a) (WinVFile*)&((char*)a)[-(int)offsetof(WinVFile,h)]

#if !defined(MSVC_LOCALTIME_API) || !MSVC_LOCALTIME_API
	// The MSVC CRT on Windows CE may not have a localtime() function.  So create a substitute.
#include <time.h>
	struct tm *__cdecl localtime(const time_t *t)
	{
		static struct tm y;
		FILETIME uTm, lTm;
		SYSTEMTIME pTm;
		sqlite3_int64 t64;
		t64 = *t;
		t64 = (t64 + 11644473600)*10000000;
		uTm.dwLowDateTime = (DWORD)(t64 & 0xFFFFFFFF);
		uTm.dwHighDateTime= (DWORD)(t64 >> 32);
		osFileTimeToLocalFileTime(&uTm,&lTm);
		osFileTimeToSystemTime(&lTm,&pTm);
		y.tm_year = pTm.wYear - 1900;
		y.tm_mon = pTm.wMonth - 1;
		y.tm_wday = pTm.wDayOfWeek;
		y.tm_mday = pTm.wDay;
		y.tm_hour = pTm.wHour;
		y.tm_min = pTm.wMinute;
		y.tm_sec = pTm.wSecond;
		return &y;
	}
#endif

	static void winceMutexAcquire(HANDLE h)
	{
		DWORD err;
		do
		{
			err = osWaitForSingleObject(h, INFINITE);
		} while (err != WAIT_OBJECT_0 && err != WAIT_ABANDONED);
	}

#define winceMutexRelease(h) ReleaseMutex(h)

	static RC winceCreateLock(const char *filename, WinVFile *file)
	{
		LPWSTR name = Utf8ToUnicode(filename);
		if (!name)
			return RC::IOERR_NOMEM;
		// Initialize the local lockdata
		memset(&file->Local, 0, sizeof(file->Local));
		// Replace the backslashes from the filename and lowercase it to derive a mutex name.
		LPWSTR tok = osCharLowerW(name);
		for (; *tok; tok++)
			if (*tok == '\\') *tok = '_';
		// Create/open the named mutex
		file->Mutex = osCreateMutexW(NULL, FALSE, name);
		if (!file->Mutex)
		{
			file->LastErrno = osGetLastError();
			winLogError(RC::IOERR, file->LastErrno, "winceCreateLock1", filename);
			SysEx::Free(name);
			return RC::IOERR;
		}
		// Acquire the mutex before continuing
		winceMutexAcquire(file->Mutex);
		// Since the names of named mutexes, semaphores, file mappings etc are case-sensitive, take advantage of that by uppercasing the mutex name
		// and using that as the shared filemapping name.
		osCharUpperW(name);
		file->SharedHandle = osCreateFileMappingW(INVALID_HANDLE_VALUE, NULL, PAGE_READWRITE, 0, sizeof(winceLock), name);  
		// Set a flag that indicates we're the first to create the memory so it must be zero-initialized
		bool init = true;
		DWORD lastErrno = osGetLastError();
		if (lastErrno == ERROR_ALREADY_EXISTS)
			init = false;
		SysEx::Free(name);

		// If we succeeded in making the shared memory handle, map it.
		bool logged = false;
		if (file->SharedHandle)
		{
			file->Shared = (winceLock *)osMapViewOfFile(file->SharedHandle, FILE_MAP_READ | FILE_MAP_WRITE, 0, 0, sizeof(winceLock));
			// If mapping failed, close the shared memory handle and erase it
			if (!file->Shared)
			{
				file->LastErrno = osGetLastError();
				winLogError(RC::IOERR, file->LastErrno, "winceCreateLock2", filename);
				logged = true;
				osCloseHandle(file->SharedHandle);
				file->SharedHandle = NULL;
			}
		}
		// If shared memory could not be created, then close the mutex and fail
		if (!file->SharedHandle)
		{
			if (!logged)
			{
				file->LastErrno = lastErrno;
				winLogError(RC::IOERR, file->LastErrno, "winceCreateLock3", filename);
				logged = true;
			}
			winceMutexRelease(file->Mutex);
			osCloseHandle(file->Mutex);
			file->Mutex = NULL;
			return RC::IOERR;
		}
		// Initialize the shared memory if we're supposed to
		if (init)
			memset(file->Shared, 0, sizeof(winceLock));
		winceMutexRelease(file->Mutex);
		return RC::OK;
	}

	static void winceDestroyLock(WinVFile *file)
	{
		if (file->Mutex)
		{
			// Acquire the mutex
			winceMutexAcquire(file->Mutex);
			// The following blocks should probably assert in debug mode, but they are to cleanup in case any locks remained open
			if (file->Local.Readers)
				file->Shared->Readers--;
			if (file->Local.Reserved)
				file->Shared->Reserved = FALSE;
			if (file->Local.Pending)
				file->Shared->Pending = FALSE;
			if (file->Local.Exclusive)
				file->Shared->Exclusive = FALSE;
			// De-reference and close our copy of the shared memory handle
			osUnmapViewOfFile(file->Shared);
			osCloseHandle(file->SharedHandle);
			// Done with the mutex
			winceMutexRelease(file->Mutex);    
			osCloseHandle(file->Mutex);
			file->Mutex = NULL;
		}
	}

	static BOOL winceLockFile(LPHANDLE fileHandle, DWORD fileOffsetLow, DWORD fileOffsetHigh, DWORD numberOfBytesToLockLow, DWORD numberOfBytesToLockHigh)
	{
		WinVFile *file = HANDLE_TO_WINFILE(fileHandle);
		BOOL r = FALSE;
		if (!file->Mutex) return true;
		winceMutexAcquire(file->Mutex);
		// Wanting an exclusive lock?
		if (fileOffsetLow == (DWORD)SHARED_FIRST && numberOfBytesToLockLow == (DWORD)SHARED_SIZE)
		{
			if (file->Shared->Readers == 0 && !file->Shared->Exclusive)
			{
				file->Shared->Exclusive = true;
				file->Local.Exclusive = true;
				r = TRUE;
			}
		}
		// Want a read-only lock? 
		else if (fileOffsetLow == (DWORD)SHARED_FIRST && numberOfBytesToLockLow == 1)
		{
			if (!file->Shared->Exclusive)
			{
				file->Local.Readers++;
				if (file->Local.Readers == 1)
					file->Shared->Readers++;
				r = TRUE;
			}
		}
		// Want a pending lock?
		else if (fFileOffsetLow == (DWORD)PENDING_BYTE && numberOfBytesToLockLow == 1)
		{
			// If no pending lock has been acquired, then acquire it
			if (!file->Shared->Pending) 
			{
				file->Shared->Pending = true;
				file->Local.Pending = true;
				r = TRUE;
			}
		}
		// Want a reserved lock?
		else if (fileOffsetLow == (DWORD)RESERVED_BYTE && numberOfBytesToLockLow == 1)
		{
			if (!file->Shared->Reserved)
			{
				file->Shared->Reserved = true;
				file->Local.Reserved = true;
				r = TRUE;
			}
		}
		winceMutexRelease(file->Mutex);
		return r;
	}

	static BOOL winceUnlockFile(LPHANDLE fileHandle, DWORD fileOffsetLow, DWORD fileOffsetHigh, DWORD numberOfBytesToUnlockLow, DWORD numberOfBytesToUnlockHigh)
	{
		WinVFile *file = HANDLE_TO_WINFILE(fileHandle);
		BOOL r = FALSE;
		if (!file->Mutex) return true;
		winceMutexAcquire(file->Mutex);
		// Releasing a reader lock or an exclusive lock
		if (fileOffsetLow == (DWORD)SHARED_FIRST)
		{
			// Did we have an exclusive lock?
			if (file->Local.Exclusive)
			{
				_assert(numberOfBytesToUnlockLow == (DWORD)SHARED_SIZE);
				file->Local.Exclusive = false;
				file->Shared->Exclusive = false;
				r = TRUE;
			}
			// Did we just have a reader lock?
			else if (file->Local.Readers)
			{
				_assert(numberOfBytesToUnlockLow == (DWORD)SHARED_SIZE || numberOfBytesToUnlockLow == 1);
				file->Local.Readers--;
				if (file->Local.Readers == 0)
					file->Shared->Readers--;
				r = TRUE;
			}
		}
		// Releasing a pending lock
		else if (fileOffsetLow == (DWORD)PENDING_BYTE && numberOfBytesToUnlockLow == 1)
		{
			if (file->Local.Pending)
			{
				file->Local.Pending = false;
				file->Shared->Pending = false;
				r = TRUE;
			}
		}
		// Releasing a reserved lock
		else if (fileOffsetLow == (DWORD)RESERVED_BYTE && numberOfBytesToUnlockLow == 1)
		{
			if (file->Local.Reserved)
			{
				file->Local.Reserved = false;
				file->Shared->Reserved = false;
				r = TRUE;
			}
		}
		winceMutexRelease(file->Mutex);
		return r;
	}

#endif
#pragma endregion

#pragma region Locking

	static BOOL winLockFile(LPHANDLE fileHandle, DWORD flags, DWORD offsetLow, DWORD offsetHigh, DWORD numBytesLow, DWORD numBytesHigh)
	{
#if OS_WINCE
		// NOTE: Windows CE is handled differently here due its lack of the Win32 API LockFile.
		return winceLockFile(fileHandle, offsetLow, offsetHigh, numBytesLow, numBytesHigh);
#else
		if (isNT())
		{
			OVERLAPPED ovlp;
			memset(&ovlp, 0, sizeof(OVERLAPPED));
			ovlp.Offset = offsetLow;
			ovlp.OffsetHigh = offsetHigh;
			return osLockFileEx(*fileHandle, flags, 0, numBytesLow, numBytesHigh, &ovlp);
		}
		else
			return osLockFile(*fileHandle, offsetLow, offsetHigh, numBytesLow, numBytesHigh);
#endif
	}

	static BOOL winUnlockFile(LPHANDLE fileHandle, DWORD offsetLow, DWORD offsetHigh, DWORD numBytesLow, DWORD numBytesHigh)
	{
#if OS_WINCE
		// NOTE: Windows CE is handled differently here due its lack of the Win32 API UnlockFile.
		return winceUnlockFile(fileHandle, offsetLow, offsetHigh, numBytesLow, numBytesHigh);
#else
		if (isNT())
		{
			OVERLAPPED ovlp;
			memset(&ovlp, 0, sizeof(OVERLAPPED));
			ovlp.Offset = offsetLow;
			ovlp.OffsetHigh = offsetHigh;
			return osUnlockFileEx(*fileHandle, 0, numBytesLow, numBytesHigh, &ovlp);
		}
		else
			return osUnlockFile(*fileHandle, offsetLow, offsetHigh, numBytesLow, numBytesHigh);
#endif
	}

#pragma endregion

#pragma region WinVFile

	static int seekWinFile(WinVFile *file, int64 offset)
	{
#if !OS_WINRT
		LONG upperBits = (LONG)((iOffset>>32) & 0x7fffffff); // Most sig. 32 bits of new offset
		LONG lowerBits = (LONG)(iOffset & 0xffffffff); // Least sig. 32 bits of new offset
		// API oddity: If successful, SetFilePointer() returns a dword containing the lower 32-bits of the new file-offset. Or, if it fails,
		// it returns INVALID_SET_FILE_POINTER. However according to MSDN, INVALID_SET_FILE_POINTER may also be a valid new offset. So to determine 
		// whether an error has actually occurred, it is also necessary to call GetLastError().
		DWORD dwRet = osSetFilePointer(file->H, lowerBits, &upperBits, FILE_BEGIN); // Value returned by SetFilePointer()
		DWORD lastErrno; // Value returned by GetLastError()
		if ((dwRet == INVALID_SET_FILE_POINTER && ((lastErrno = osGetLastError()) != NO_ERROR)))
		{
			file->LastErrno = lastErrno;
			winLogError(RC::IOERR_SEEK, file->LastErrno, "seekWinFile", file->Path);
			return 1;
		}
		return 0;
#else
		// Same as above, except that this implementation works for WinRT.
		LARGE_INTEGER x; // The new offset
		x.QuadPart = offset; 
		BOOL ret = osSetFilePointerEx(file->H, x, 0, FILE_BEGIN); // Value returned by SetFilePointerEx()
		if (!ret)
		{
			file->LastErrno = osGetLastError();
			winLogError(RC::IOERR_SEEK, file->LastErrno, "seekWinFile", file->Path);
			return 1;
		}
		return 0;
#endif
	}

#define MX_CLOSE_ATTEMPT 3
	RC WinVFile::Close()
	{
#ifndef OMIT_WAL
		_assert(Shm == 0);
#endif
		TRACE("CLOSE %d\n", file->H);
		_assert(H != NULL && H != INVALID_HANDLE_VALUE);
		int rc;
		int cnt = 0;
		do
		{
			rc = osCloseHandle(H);
		} while (!rc && ++cnt < MX_CLOSE_ATTEMPT && win32_Sleep(100));
#if OS_WINCE
#define WINCE_DELETION_ATTEMPTS 3
		winceDestroyLock(this);
		if (DeleteOnClose)
		{
			int cnt = 0;
			while (osDeleteFileW(DeleteOnClose) == 0 && osGetFileAttributesW(DeleteOnClose) != 0xffffffff && cnt++ < WINCE_DELETION_ATTEMPTS)
				win32_Sleep(100);  // Wait a little before trying again
			SysEx::Free(DeleteOnClose);
		}
#endif
		TRACE("CLOSE %d %s\n", H, rc ? "ok" : "failed");
		if (rc)
			H = NULL;
		OpenCounter(-1);
		return (rc ? RC::OK : winLogError(RC::IOERR_CLOSE, osGetLastError(), "winClose", Path));
	}

	RC WinVFile::Read(void *buffer, int amount, int64 offset)
	{
#if !OS_WINCE
		OVERLAPPED overlapped; // The offset for ReadFile.
#endif
		int retry = 0; // Number of retrys
		SimulateIOError(return RC::IOERR_READ);
		TRACE("READ %d lock=%d\n", H, Lock);
		DWORD read; // Number of bytes actually read from file
#if OS_WINCE
		if (seekWinFile(this, offset))
			return RC::FULL;
		while (!osReadFile(H, buffer, amount, &read, 0))
		{
#else
		memset(&overlapped, 0, sizeof(OVERLAPPED));
		overlapped.Offset = (LONG)(offset & 0xffffffff);
		overlapped.OffsetHigh = (LONG)((offset>>32) & 0x7fffffff);
		while (!osReadFile(H, buffer, amount, &read, &overlapped) && osGetLastError() != ERROR_HANDLE_EOF)
		{
#endif
			DWORD lastErrno;
			if (retryIoerr(&retry, &lastErrno)) continue;
			LastErrno = lastErrno;
			return winLogError(RC::IOERR_READ, LastErrno, "winRead", Path);
		}
		logIoerr(retry);
		if (read < (DWORD)amount)
		{
			// Unread parts of the buffer must be zero-filled
			memset(&((char *)buffer)[read], 0, amount - read);
			return RC::IOERR_SHORT_READ;
		}
		return RC::OK;
	}

	RC WinVFile::Write(const void *buffer, int amount, int64 offset)
	{
		_assert(amount > 0);
		SimulateIOError(return RC::IOERR_WRITE);
		SimulateDiskfullError(return RC::FULL);
		TRACE("WRITE %d lock=%d\n", H, Lock);
		int rc = 0; // True if error has occurred, else false
#if OS_WINCE
		rc = seekWinFile(this, offset);
		if (!rc)
		{
#else
		{
#endif
#if !OS_WINCE
			OVERLAPPED overlapped; // The offset for WriteFile.
			memset(&overlapped, 0, sizeof(OVERLAPPED));
			overlapped.Offset = (LONG)(offset & 0xffffffff);
			overlapped.OffsetHigh = (LONG)((offset>>32) & 0x7fffffff);
#endif
			uint8 *remain = (uint8 *)buffer; // Data yet to be written
			int remainLength = amount; // Number of bytes yet to be written
			DWORD write; // Bytes written by each WriteFile() call
			DWORD lastErrno = NO_ERROR; // Value returned by GetLastError()
			int retry = 0; // Number of retries
			while (remainLength > 0)
			{
#if OS_WINCE
				if (!osWriteFile(H, remain, remainLength, &write, 0)) {
#else
				if (!osWriteFile(H, remain, remainLength, &write, &overlapped)) {
#endif
					if (retryIoerr(&retry, &lastErrno)) continue;
					break;
				}
				_assert(write == 0 || write <= (DWORD)remainLength);
				if (write == 0 || write > (DWORD)remainLength)
				{
					lastErrno = osGetLastError();
					break;
				}
#if !OS_WINCE
				offset += write;
				overlapped.Offset = (LONG)(offset & 0xffffffff);
				overlapped.OffsetHigh = (LONG)((offset>>32) & 0x7fffffff);
#endif
				remain += write;
				remainLength -= write;
			}
			if (remainLength > 0)
			{
				LastErrno = lastErrno;
				rc = 1;
			}
		}
		if (rc)
		{
			if (LastErrno == OSTRACE ||  LastErrno == ERROR_DISK_FULL)
				return RC::FULL;
			return winLogError(RC::IOERR_WRITE, LastErrno, "winWrite", Path);
		}
		else
			logIoerr(retry);
		return RC::OK;
	}

	RC WinVFile::Truncate(int64 size)
	{
		RC rc = RC::OK;
		TRACE("TRUNCATE %d %lld\n", H, size);
		SimulateIOError(return RC::IOERR_TRUNCATE);
		// If the user has configured a chunk-size for this file, truncate the file so that it consists of an integer number of chunks (i.e. the
		// actual file size after the operation may be larger than the requested size).
		if (SizeChunk > 0)
			size = ((size+SizeChunk-1)/SizeChunk)*SizeChunk;
		// SetEndOfFile() returns non-zero when successful, or zero when it fails.
		if (seekWinFile(this, size))
			rc = winLogError(RC::IOERR_TRUNCATE, LastErrno, "winTruncate1", Path);
		else if (!osSetEndOfFile(H))
		{
			LastErrno = osGetLastError();
			rc = winLogError(RC::IOERR_TRUNCATE, LastErrno, "winTruncate2", Path);
		}
		TRACE("TRUNCATE %d %lld %s\n", H, size, rc ? "failed" : "ok");
		return rc;
	}

#ifdef TEST
	// Count the number of fullsyncs and normal syncs.  This is used to test that syncs and fullsyncs are occuring at the right times.
	int sync_count = 0;
	int fullsync_count = 0;
#endif

	RC WinVFile::Sync(int flags)
	{
		// Check that one of SQLITE_SYNC_NORMAL or FULL was passed
		_assert((flags&0x0F) == SYNC::NORMAL || (flags&0x0F) == SYNC::FULL);
		TRACE("SYNC %d lock=%d\n", H, Lock);
		// Unix cannot, but some systems may return SQLITE_FULL from here. This line is to test that doing so does not cause any problems.
		SimulateDiskfullError(return RC::FULL);
#ifdef TEST
		if ((flags&0x0F) == SYNC::FULL)
			fullsync_count++;
		sync_count++;
#endif
#ifdef NO_SYNC // If we compiled with the SQLITE_NO_SYNC flag, then syncing is a no-op
		return RC::OK;
#else
		BOOL rc = osFlushFileBuffers(H);
		SimulateIOError(rc = FALSE);
		if (rc)
			return RC::OK;
		LastErrno = osGetLastError();
		return winLogError(RC::IOERR_FSYNC, LastErrno, "winSync", Path);
#endif
	}

	RC WinVFile::get_FileSize(int64 *size)
	{
		RC rc = RC::OK;
		SimulateIOError(return RC::IOERR_FSTAT);
#if OS_WINRT
		{
			FILE_STANDARD_INFO info;
			if (osGetFileInformationByHandleEx(H, FileStandardInfo, &info, sizeof(info)))
				*size = info.EndOfFile.QuadPart;
			else
			{
				LastErrno = osGetLastError();
				rc = winLogError(RC::IOERR_FSTAT, LastErrno, "winFileSize", Path);
			}
		}
#else
		{
			DWORD upperBits;
			DWORD lowerBits = osGetFileSize(H, &upperBits);
			*size = (((int64)upperBits)<<32) + lowerBits;
			DWORD lastErrno;
			if (lowerBits == INVALID_FILE_SIZE && (lastErrno = osGetLastError()) != NO_ERROR)
			{
				LastErrno = lastErrno;
				rc = winLogError(RC::IOERR_FSTAT, LastErrno, "winFileSize", Path);
			}
		}
#endif
		return rc;
	}

	static int getReadLock(WinVFile *file)
	{
		int res;
		if (isNT())
		{
#if OS_WINCE
			// NOTE: Windows CE is handled differently here due its lack of the Win32 API LockFileEx.
			res = winceLockFile(&file->H, SHARED_FIRST, 0, 1, 0);
#else
			res = winLockFile(&file->H, SQLITE_LOCKFILEEX_FLAGS, SHARED_FIRST, 0, SHARED_SIZE, 0);
#endif
		}
#ifdef WIN32_HAS_ANSI
		else
		{
			int lock;
			sqlite3_randomness(sizeof(lock), &lock);
			file->SharedLockByte = (short)((lock & 0x7fffffff)%(SHARED_SIZE - 1));
			res = winLockFile(&pFile->h, LOCKFILE_FLAGS, SHARED_FIRST + file->SharedLockByte, 0, 1, 0);
		}
#endif
		if (res == 0)
			file->LastErrno = osGetLastError();
		// No need to log a failure to lock
		return res;
	}

	static int unlockReadLock(WinVFile *file)
	{
		int res;
		if (isNT())
			res = winUnlockFile(&file->H, SHARED_FIRST, 0, SHARED_SIZE, 0);
#ifdef WIN32_HAS_ANSI
		else
			res = winUnlockFile(&file->H, SHARED_FIRST + file->SharedLockByte, 0, 1, 0);
#endif
		DWORD lastErrno;
		if (res == 0 && (lastErrno = osGetLastError()) != ERROR_NOT_LOCKED)
		{
			file->LastErrno = lastErrno;
			winLogError(RC::IOERR_UNLOCK, file->LastErrno, "unlockReadLock", file->Path);
		}
		return res;
	}

	RC WinVFile::Lock(LOCK lock)
	{
		OSTRACE("LOCK %d %d was %d(%d)\n", H, lock, Lock, SharedLockByte);

		// If there is already a lock of this type or more restrictive on the OsFile, do nothing. Don't use the end_lock: exit path, as
		// sqlite3OsEnterMutex() hasn't been called yet.
		if (Lock >= lock)
			return RC::OK;

		// Make sure the locking sequence is correct
		_assert(Lock != LOCK::NO || lock == LOCK::SHARED);
		_assert(lock != LOCK::PENDING);
		_assert(lock != LOCK::RESERVED || Lock == LOCK::SHARED);

		// Lock the PENDING_LOCK byte if we need to acquire a PENDING lock or a SHARED lock.  If we are acquiring a SHARED lock, the acquisition of
		// the PENDING_LOCK byte is temporary.
		LOCK newLock = Lock; // Set pFile->locktype to this value before exiting
		int res = 1; // Result of a Windows lock call
		bool gotPendingLock = false; // True if we acquired a PENDING lock this time
		DWORD lastErrno = NO_ERROR;
		if (Lock == LOCK::NO || (lock == LOCK::EXCLUSIVE && Lock == LOCK::RESERVED))
		{
			int cnt = 3;
			while (cnt-- > 0 && (res = winLockFile(&H, LOCKFILE_FLAGS, PENDING_BYTE, 0, 1, 0)) == 0)
			{
				// Try 3 times to get the pending lock.  This is needed to work around problems caused by indexing and/or anti-virus software on Windows systems.
				// If you are using this code as a model for alternative VFSes, do not copy this retry logic.  It is a hack intended for Windows only.
				OSTRACE("could not get a PENDING lock. cnt=%d\n", cnt);
				if (cnt) win32_Sleep(1);
			}
			gotPendingLock = res;
			if (!res)
				lastErrno = osGetLastError();
		}

		// Acquire a SHARED lock
		if (lock == LOCK::SHARED && res)
		{
			_assert(Lock == LOCK::NO);
			res = getReadLock(this);
			if (res)
				newLock = LOCK::SHARED;
			else
				lastErrno = osGetLastError();
		}

		// Acquire a RESERVED lock
		if (lock == LOCK::RESERVED && res)
		{
			_assert(Lock == LOCK::SHARED);
			res = winLockFile(&H, LOCKFILE_FLAGS, RESERVED_BYTE, 0, 1, 0);
			if (res)
				newLock = LOCK::RESERVED;
			else
				lastErrno = osGetLastError();
		}

		// Acquire a PENDING lock
		if (lock == LOCK::EXCLUSIVE && res)
		{
			newLock = LOCK::PENDING;
			gotPendingLock = false;
		}

		// Acquire an EXCLUSIVE lock
		if (lock == LOCK::EXCLUSIVE && res)
		{
			_assert(Lock >= LOCK::SHARED);
			res = unlockReadLock(this);
			OSTRACE("unreadlock = %d\n", res);
			res = winLockFile(&H, LOCKFILE_FLAGS, SHARED_FIRST, 0, SHARED_SIZE, 0);
			if (res)
				newLock = LOCK::EXCLUSIVE;
			else
			{
				lastErrno = osGetLastError();
				OSTRACE("error-code = %d\n", lastErrno);
				getReadLock(this);
			}
		}

		// If we are holding a PENDING lock that ought to be released, then release it now.
		if (gotPendingLock && lock == LOCK::SHARED)
			winUnlockFile(&H, PENDING_BYTE, 0, 1, 0);

		// Update the state of the lock has held in the file descriptor then return the appropriate result code.
		RC rc;
		if (res)
			rc = RC::OK;
		else
		{
			OSTRACE("LOCK FAILED %d trying for %d but got %d\n", H, lock, newLock);
			LastErrno = lastErrno;
			rc = RC::BUSY;
		}
		Lock = newLock;
		return rc;
	}

	RC WinVFile::CheckReservedLock(int *resOut)
	{
		SimulateIOError(return RC::IOERR_CHECKRESERVEDLOCK;);
		int rc;
		if (Lock >= LOCK::RESERVED)
		{
			rc = 1;
			OSTRACE("TEST WR-LOCK %d %d (local)\n", H, rc);
		}
		else
		{
			rc = winLockFile(&H, LOCKFILEEX_FLAGS,RESERVED_BYTE, 0, 1, 0);
			if (rc)
				winUnlockFile(&H, RESERVED_BYTE, 0, 1, 0);
			rc = !rc;
			OSTRACE("TEST WR-LOCK %d %d (remote)\n", H, rc);
		}
		*resOut = rc;
		return RC::OK;
	}

	/*
	** Lower the locking level on file descriptor id to locktype.  locktype
	** must be either NO_LOCK or SHARED_LOCK.
	**
	** If the locking level of the file descriptor is already at or below
	** the requested locking level, this routine is a no-op.
	**
	** It is not possible for this routine to fail if the second argument
	** is NO_LOCK.  If the second argument is SHARED_LOCK then this routine
	** might return SQLITE_IOERR;
	*/
	static int winUnlock(sqlite3_file *id, int locktype){
		int type;
		winFile *pFile = (winFile*)id;
		int rc = SQLITE_OK;
		assert( pFile!=0 );
		assert( locktype<=SHARED_LOCK );
		OSTRACE(("UNLOCK %d to %d was %d(%d)\n", pFile->h, locktype,
			pFile->locktype, pFile->sharedLockByte));
		type = pFile->locktype;
		if( type>=EXCLUSIVE_LOCK ){
			winUnlockFile(&pFile->h, SHARED_FIRST, 0, SHARED_SIZE, 0);
			if( locktype==SHARED_LOCK && !getReadLock(pFile) ){
				/* This should never happen.  We should always be able to
				** reacquire the read lock */
				rc = winLogError(SQLITE_IOERR_UNLOCK, osGetLastError(),
					"winUnlock", pFile->zPath);
			}
		}
		if( type>=RESERVED_LOCK ){
			winUnlockFile(&pFile->h, RESERVED_BYTE, 0, 1, 0);
		}
		if( locktype==NO_LOCK && type>=SHARED_LOCK ){
			unlockReadLock(pFile);
		}
		if( type>=PENDING_LOCK ){
			winUnlockFile(&pFile->h, PENDING_BYTE, 0, 1, 0);
		}
		pFile->locktype = (u8)locktype;
		return rc;
	}

	/*
	** If *pArg is inititially negative then this is a query.  Set *pArg to
	** 1 or 0 depending on whether or not bit mask of pFile->ctrlFlags is set.
	**
	** If *pArg is 0 or 1, then clear or set the mask bit of pFile->ctrlFlags.
	*/
	static void winModeBit(winFile *pFile, unsigned char mask, int *pArg){
		if( *pArg<0 ){
			*pArg = (pFile->ctrlFlags & mask)!=0;
		}else if( (*pArg)==0 ){
			pFile->ctrlFlags &= ~mask;
		}else{
			pFile->ctrlFlags |= mask;
		}
	}

	/* Forward declaration */
	static int getTempname(int nBuf, char *zBuf);

	/*
	** Control and query of the open file handle.
	*/
	static int winFileControl(sqlite3_file *id, int op, void *pArg){
		winFile *pFile = (winFile*)id;
		switch( op ){
		case SQLITE_FCNTL_LOCKSTATE: {
			*(int*)pArg = pFile->locktype;
			return SQLITE_OK;
									 }
		case SQLITE_LAST_ERRNO: {
			*(int*)pArg = (int)pFile->lastErrno;
			return SQLITE_OK;
								}
		case SQLITE_FCNTL_CHUNK_SIZE: {
			pFile->szChunk = *(int *)pArg;
			return SQLITE_OK;
									  }
		case SQLITE_FCNTL_SIZE_HINT: {
			if( pFile->szChunk>0 ){
				sqlite3_int64 oldSz;
				int rc = winFileSize(id, &oldSz);
				if( rc==SQLITE_OK ){
					sqlite3_int64 newSz = *(sqlite3_int64*)pArg;
					if( newSz>oldSz ){
						SimulateIOErrorBenign(1);
						rc = winTruncate(id, newSz);
						SimulateIOErrorBenign(0);
					}
				}
				return rc;
			}
			return SQLITE_OK;
									 }
		case SQLITE_FCNTL_PERSIST_WAL: {
			winModeBit(pFile, WINFILE_PERSIST_WAL, (int*)pArg);
			return SQLITE_OK;
									   }
		case SQLITE_FCNTL_POWERSAFE_OVERWRITE: {
			winModeBit(pFile, WINFILE_PSOW, (int*)pArg);
			return SQLITE_OK;
											   }
		case SQLITE_FCNTL_VFSNAME: {
			*(char**)pArg = sqlite3_mprintf("win32");
			return SQLITE_OK;
								   }
		case SQLITE_FCNTL_WIN32_AV_RETRY: {
			int *a = (int*)pArg;
			if( a[0]>0 ){
				win32IoerrRetry = a[0];
			}else{
				a[0] = win32IoerrRetry;
			}
			if( a[1]>0 ){
				win32IoerrRetryDelay = a[1];
			}else{
				a[1] = win32IoerrRetryDelay;
			}
			return SQLITE_OK;
										  }
		case SQLITE_FCNTL_TEMPFILENAME: {
			char *zTFile = sqlite3MallocZero( pFile->pVfs->mxPathname );
			if( zTFile ){
				getTempname(pFile->pVfs->mxPathname, zTFile);
				*(char**)pArg = zTFile;
			}
			return SQLITE_OK;
										}
		}
		return SQLITE_NOTFOUND;
	}

	/*
	** Return the sector size in bytes of the underlying block device for
	** the specified file. This is almost always 512 bytes, but may be
	** larger for some devices.
	**
	** SQLite code assumes this function cannot fail. It also assumes that
	** if two files are created in the same file-system directory (i.e.
	** a database and its journal file) that the sector size will be the
	** same for both.
	*/
	static int winSectorSize(sqlite3_file *id){
		(void)id;
		return SQLITE_DEFAULT_SECTOR_SIZE;
	}

	/*
	** Return a vector of device characteristics.
	*/
	static int winDeviceCharacteristics(sqlite3_file *id){
		winFile *p = (winFile*)id;
		return SQLITE_IOCAP_UNDELETABLE_WHEN_OPEN |
			((p->ctrlFlags & WINFILE_PSOW)?SQLITE_IOCAP_POWERSAFE_OVERWRITE:0);
	}

#ifndef SQLITE_OMIT_WAL

	/* 
	** Windows will only let you create file view mappings
	** on allocation size granularity boundaries.
	** During sqlite3_os_init() we do a GetSystemInfo()
	** to get the granularity size.
	*/
	SYSTEM_INFO winSysInfo;

	/*
	** Helper functions to obtain and relinquish the global mutex. The
	** global mutex is used to protect the winLockInfo objects used by 
	** this file, all of which may be shared by multiple threads.
	**
	** Function winShmMutexHeld() is used to assert() that the global mutex 
	** is held when required. This function is only used as part of assert() 
	** statements. e.g.
	**
	**   winShmEnterMutex()
	**     assert( winShmMutexHeld() );
	**   winShmLeaveMutex()
	*/
	static void winShmEnterMutex(void){
		sqlite3_mutex_enter(sqlite3MutexAlloc(SQLITE_MUTEX_STATIC_MASTER));
	}
	static void winShmLeaveMutex(void){
		sqlite3_mutex_leave(sqlite3MutexAlloc(SQLITE_MUTEX_STATIC_MASTER));
	}
#ifdef SQLITE_DEBUG
	static int winShmMutexHeld(void) {
		return sqlite3_mutex_held(sqlite3MutexAlloc(SQLITE_MUTEX_STATIC_MASTER));
	}
#endif

	/*
	** Object used to represent a single file opened and mmapped to provide
	** shared memory.  When multiple threads all reference the same
	** log-summary, each thread has its own winFile object, but they all
	** point to a single instance of this object.  In other words, each
	** log-summary is opened only once per process.
	**
	** winShmMutexHeld() must be true when creating or destroying
	** this object or while reading or writing the following fields:
	**
	**      nRef
	**      pNext 
	**
	** The following fields are read-only after the object is created:
	** 
	**      fid
	**      zFilename
	**
	** Either winShmNode.mutex must be held or winShmNode.nRef==0 and
	** winShmMutexHeld() is true when reading or writing any other field
	** in this structure.
	**
	*/
	struct winShmNode {
		sqlite3_mutex *mutex;      /* Mutex to access this object */
		char *zFilename;           /* Name of the file */
		winFile hFile;             /* File handle from winOpen */

		int szRegion;              /* Size of shared-memory regions */
		int nRegion;               /* Size of array apRegion */
		struct ShmRegion {
			HANDLE hMap;             /* File handle from CreateFileMapping */
			void *pMap;
		} *aRegion;
		DWORD lastErrno;           /* The Windows errno from the last I/O error */

		int nRef;                  /* Number of winShm objects pointing to this */
		winShm *pFirst;            /* All winShm objects pointing to this */
		winShmNode *pNext;         /* Next in list of all winShmNode objects */
#ifdef SQLITE_DEBUG
		u8 nextShmId;              /* Next available winShm.id value */
#endif
	};

	/*
	** A global array of all winShmNode objects.
	**
	** The winShmMutexHeld() must be true while reading or writing this list.
	*/
	static winShmNode *winShmNodeList = 0;

	/*
	** Structure used internally by this VFS to record the state of an
	** open shared memory connection.
	**
	** The following fields are initialized when this object is created and
	** are read-only thereafter:
	**
	**    winShm.pShmNode
	**    winShm.id
	**
	** All other fields are read/write.  The winShm.pShmNode->mutex must be held
	** while accessing any read/write fields.
	*/
	struct winShm {
		winShmNode *pShmNode;      /* The underlying winShmNode object */
		winShm *pNext;             /* Next winShm with the same winShmNode */
		u8 hasMutex;               /* True if holding the winShmNode mutex */
		u16 sharedMask;            /* Mask of shared locks held */
		u16 exclMask;              /* Mask of exclusive locks held */
#ifdef SQLITE_DEBUG
		u8 id;                     /* Id of this connection with its winShmNode */
#endif
	};

	/*
	** Constants used for locking
	*/
#define WIN_SHM_BASE   ((22+SQLITE_SHM_NLOCK)*4)        /* first lock byte */
#define WIN_SHM_DMS    (WIN_SHM_BASE+SQLITE_SHM_NLOCK)  /* deadman switch */

	/*
	** Apply advisory locks for all n bytes beginning at ofst.
	*/
#define _SHM_UNLCK  1
#define _SHM_RDLCK  2
#define _SHM_WRLCK  3
	static int winShmSystemLock(
		winShmNode *pFile,    /* Apply locks to this open shared-memory segment */
		int lockType,         /* _SHM_UNLCK, _SHM_RDLCK, or _SHM_WRLCK */
		int ofst,             /* Offset to first byte to be locked/unlocked */
		int nByte             /* Number of bytes to lock or unlock */
		){
			int rc = 0;           /* Result code form Lock/UnlockFileEx() */

			/* Access to the winShmNode object is serialized by the caller */
			assert( sqlite3_mutex_held(pFile->mutex) || pFile->nRef==0 );

			/* Release/Acquire the system-level lock */
			if( lockType==_SHM_UNLCK ){
				rc = winUnlockFile(&pFile->hFile.h, ofst, 0, nByte, 0);
			}else{
				/* Initialize the locking parameters */
				DWORD dwFlags = LOCKFILE_FAIL_IMMEDIATELY;
				if( lockType == _SHM_WRLCK ) dwFlags |= LOCKFILE_EXCLUSIVE_LOCK;
				rc = winLockFile(&pFile->hFile.h, dwFlags, ofst, 0, nByte, 0);
			}

			if( rc!= 0 ){
				rc = SQLITE_OK;
			}else{
				pFile->lastErrno =  osGetLastError();
				rc = SQLITE_BUSY;
			}

			OSTRACE(("SHM-LOCK %d %s %s 0x%08lx\n", 
				pFile->hFile.h,
				rc==SQLITE_OK ? "ok" : "failed",
				lockType==_SHM_UNLCK ? "UnlockFileEx" : "LockFileEx",
				pFile->lastErrno));

			return rc;
	}

	/* Forward references to VFS methods */
	static int winOpen(sqlite3_vfs*,const char*,sqlite3_file*,int,int*);
	static int winDelete(sqlite3_vfs *,const char*,int);

	/*
	** Purge the winShmNodeList list of all entries with winShmNode.nRef==0.
	**
	** This is not a VFS shared-memory method; it is a utility function called
	** by VFS shared-memory methods.
	*/
	static void winShmPurge(sqlite3_vfs *pVfs, int deleteFlag){
		winShmNode **pp;
		winShmNode *p;
		BOOL bRc;
		assert( winShmMutexHeld() );
		pp = &winShmNodeList;
		while( (p = *pp)!=0 ){
			if( p->nRef==0 ){
				int i;
				if( p->mutex ) sqlite3_mutex_free(p->mutex);
				for(i=0; i<p->nRegion; i++){
					bRc = osUnmapViewOfFile(p->aRegion[i].pMap);
					OSTRACE(("SHM-PURGE pid-%d unmap region=%d %s\n",
						(int)osGetCurrentProcessId(), i,
						bRc ? "ok" : "failed"));
					bRc = osCloseHandle(p->aRegion[i].hMap);
					OSTRACE(("SHM-PURGE pid-%d close region=%d %s\n",
						(int)osGetCurrentProcessId(), i,
						bRc ? "ok" : "failed"));
				}
				if( p->hFile.h!=NULL && p->hFile.h!=INVALID_HANDLE_VALUE ){
					SimulateIOErrorBenign(1);
					winClose((sqlite3_file *)&p->hFile);
					SimulateIOErrorBenign(0);
				}
				if( deleteFlag ){
					SimulateIOErrorBenign(1);
					sqlite3BeginBenignMalloc();
					winDelete(pVfs, p->zFilename, 0);
					sqlite3EndBenignMalloc();
					SimulateIOErrorBenign(0);
				}
				*pp = p->pNext;
				sqlite3_free(p->aRegion);
				sqlite3_free(p);
			}else{
				pp = &p->pNext;
			}
		}
	}

	/*
	** Open the shared-memory area associated with database file pDbFd.
	**
	** When opening a new shared-memory file, if no other instances of that
	** file are currently open, in this process or in other processes, then
	** the file must be truncated to zero length or have its header cleared.
	*/
	static int winOpenSharedMemory(winFile *pDbFd){
		struct winShm *p;                  /* The connection to be opened */
		struct winShmNode *pShmNode = 0;   /* The underlying mmapped file */
		int rc;                            /* Result code */
		struct winShmNode *pNew;           /* Newly allocated winShmNode */
		int nName;                         /* Size of zName in bytes */

		assert( pDbFd->pShm==0 );    /* Not previously opened */

		/* Allocate space for the new sqlite3_shm object.  Also speculatively
		** allocate space for a new winShmNode and filename.
		*/
		p = sqlite3MallocZero( sizeof(*p) );
		if( p==0 ) return SQLITE_IOERR_NOMEM;
		nName = sqlite3Strlen30(pDbFd->zPath);
		pNew = sqlite3MallocZero( sizeof(*pShmNode) + nName + 17 );
		if( pNew==0 ){
			sqlite3_free(p);
			return SQLITE_IOERR_NOMEM;
		}
		pNew->zFilename = (char*)&pNew[1];
		sqlite3_snprintf(nName+15, pNew->zFilename, "%s-shm", pDbFd->zPath);
		sqlite3FileSuffix3(pDbFd->zPath, pNew->zFilename); 

		/* Look to see if there is an existing winShmNode that can be used.
		** If no matching winShmNode currently exists, create a new one.
		*/
		winShmEnterMutex();
		for(pShmNode = winShmNodeList; pShmNode; pShmNode=pShmNode->pNext){
			/* TBD need to come up with better match here.  Perhaps
			** use FILE_ID_BOTH_DIR_INFO Structure.
			*/
			if( sqlite3StrICmp(pShmNode->zFilename, pNew->zFilename)==0 ) break;
		}
		if( pShmNode ){
			sqlite3_free(pNew);
		}else{
			pShmNode = pNew;
			pNew = 0;
			((winFile*)(&pShmNode->hFile))->h = INVALID_HANDLE_VALUE;
			pShmNode->pNext = winShmNodeList;
			winShmNodeList = pShmNode;

			pShmNode->mutex = sqlite3_mutex_alloc(SQLITE_MUTEX_FAST);
			if( pShmNode->mutex==0 ){
				rc = SQLITE_IOERR_NOMEM;
				goto shm_open_err;
			}

			rc = winOpen(pDbFd->pVfs,
				pShmNode->zFilename,             /* Name of the file (UTF-8) */
				(sqlite3_file*)&pShmNode->hFile,  /* File handle here */
				SQLITE_OPEN_WAL | SQLITE_OPEN_READWRITE | SQLITE_OPEN_CREATE,
				0);
			if( SQLITE_OK!=rc ){
				goto shm_open_err;
			}

			/* Check to see if another process is holding the dead-man switch.
			** If not, truncate the file to zero length. 
			*/
			if( winShmSystemLock(pShmNode, _SHM_WRLCK, WIN_SHM_DMS, 1)==SQLITE_OK ){
				rc = winTruncate((sqlite3_file *)&pShmNode->hFile, 0);
				if( rc!=SQLITE_OK ){
					rc = winLogError(SQLITE_IOERR_SHMOPEN, osGetLastError(),
						"winOpenShm", pDbFd->zPath);
				}
			}
			if( rc==SQLITE_OK ){
				winShmSystemLock(pShmNode, _SHM_UNLCK, WIN_SHM_DMS, 1);
				rc = winShmSystemLock(pShmNode, _SHM_RDLCK, WIN_SHM_DMS, 1);
			}
			if( rc ) goto shm_open_err;
		}

		/* Make the new connection a child of the winShmNode */
		p->pShmNode = pShmNode;
#ifdef SQLITE_DEBUG
		p->id = pShmNode->nextShmId++;
#endif
		pShmNode->nRef++;
		pDbFd->pShm = p;
		winShmLeaveMutex();

		/* The reference count on pShmNode has already been incremented under
		** the cover of the winShmEnterMutex() mutex and the pointer from the
		** new (struct winShm) object to the pShmNode has been set. All that is
		** left to do is to link the new object into the linked list starting
		** at pShmNode->pFirst. This must be done while holding the pShmNode->mutex 
		** mutex.
		*/
		sqlite3_mutex_enter(pShmNode->mutex);
		p->pNext = pShmNode->pFirst;
		pShmNode->pFirst = p;
		sqlite3_mutex_leave(pShmNode->mutex);
		return SQLITE_OK;

		/* Jump here on any error */
shm_open_err:
		winShmSystemLock(pShmNode, _SHM_UNLCK, WIN_SHM_DMS, 1);
		winShmPurge(pDbFd->pVfs, 0);      /* This call frees pShmNode if required */
		sqlite3_free(p);
		sqlite3_free(pNew);
		winShmLeaveMutex();
		return rc;
	}

	/*
	** Close a connection to shared-memory.  Delete the underlying 
	** storage if deleteFlag is true.
	*/
	static int winShmUnmap(
		sqlite3_file *fd,          /* Database holding shared memory */
		int deleteFlag             /* Delete after closing if true */
		){
			winFile *pDbFd;       /* Database holding shared-memory */
			winShm *p;            /* The connection to be closed */
			winShmNode *pShmNode; /* The underlying shared-memory file */
			winShm **pp;          /* For looping over sibling connections */

			pDbFd = (winFile*)fd;
			p = pDbFd->pShm;
			if( p==0 ) return SQLITE_OK;
			pShmNode = p->pShmNode;

			/* Remove connection p from the set of connections associated
			** with pShmNode */
			sqlite3_mutex_enter(pShmNode->mutex);
			for(pp=&pShmNode->pFirst; (*pp)!=p; pp = &(*pp)->pNext){}
			*pp = p->pNext;

			/* Free the connection p */
			sqlite3_free(p);
			pDbFd->pShm = 0;
			sqlite3_mutex_leave(pShmNode->mutex);

			/* If pShmNode->nRef has reached 0, then close the underlying
			** shared-memory file, too */
			winShmEnterMutex();
			assert( pShmNode->nRef>0 );
			pShmNode->nRef--;
			if( pShmNode->nRef==0 ){
				winShmPurge(pDbFd->pVfs, deleteFlag);
			}
			winShmLeaveMutex();

			return SQLITE_OK;
	}

	/*
	** Change the lock state for a shared-memory segment.
	*/
	static int winShmLock(
		sqlite3_file *fd,          /* Database file holding the shared memory */
		int ofst,                  /* First lock to acquire or release */
		int n,                     /* Number of locks to acquire or release */
		int flags                  /* What to do with the lock */
		){
			winFile *pDbFd = (winFile*)fd;        /* Connection holding shared memory */
			winShm *p = pDbFd->pShm;              /* The shared memory being locked */
			winShm *pX;                           /* For looping over all siblings */
			winShmNode *pShmNode = p->pShmNode;
			int rc = SQLITE_OK;                   /* Result code */
			u16 mask;                             /* Mask of locks to take or release */

			assert( ofst>=0 && ofst+n<=SQLITE_SHM_NLOCK );
			assert( n>=1 );
			assert( flags==(SQLITE_SHM_LOCK | SQLITE_SHM_SHARED)
				|| flags==(SQLITE_SHM_LOCK | SQLITE_SHM_EXCLUSIVE)
				|| flags==(SQLITE_SHM_UNLOCK | SQLITE_SHM_SHARED)
				|| flags==(SQLITE_SHM_UNLOCK | SQLITE_SHM_EXCLUSIVE) );
			assert( n==1 || (flags & SQLITE_SHM_EXCLUSIVE)!=0 );

			mask = (u16)((1U<<(ofst+n)) - (1U<<ofst));
			assert( n>1 || mask==(1<<ofst) );
			sqlite3_mutex_enter(pShmNode->mutex);
			if( flags & SQLITE_SHM_UNLOCK ){
				u16 allMask = 0; /* Mask of locks held by siblings */

				/* See if any siblings hold this same lock */
				for(pX=pShmNode->pFirst; pX; pX=pX->pNext){
					if( pX==p ) continue;
					assert( (pX->exclMask & (p->exclMask|p->sharedMask))==0 );
					allMask |= pX->sharedMask;
				}

				/* Unlock the system-level locks */
				if( (mask & allMask)==0 ){
					rc = winShmSystemLock(pShmNode, _SHM_UNLCK, ofst+WIN_SHM_BASE, n);
				}else{
					rc = SQLITE_OK;
				}

				/* Undo the local locks */
				if( rc==SQLITE_OK ){
					p->exclMask &= ~mask;
					p->sharedMask &= ~mask;
				} 
			}else if( flags & SQLITE_SHM_SHARED ){
				u16 allShared = 0;  /* Union of locks held by connections other than "p" */

				/* Find out which shared locks are already held by sibling connections.
				** If any sibling already holds an exclusive lock, go ahead and return
				** SQLITE_BUSY.
				*/
				for(pX=pShmNode->pFirst; pX; pX=pX->pNext){
					if( (pX->exclMask & mask)!=0 ){
						rc = SQLITE_BUSY;
						break;
					}
					allShared |= pX->sharedMask;
				}

				/* Get shared locks at the system level, if necessary */
				if( rc==SQLITE_OK ){
					if( (allShared & mask)==0 ){
						rc = winShmSystemLock(pShmNode, _SHM_RDLCK, ofst+WIN_SHM_BASE, n);
					}else{
						rc = SQLITE_OK;
					}
				}

				/* Get the local shared locks */
				if( rc==SQLITE_OK ){
					p->sharedMask |= mask;
				}
			}else{
				/* Make sure no sibling connections hold locks that will block this
				** lock.  If any do, return SQLITE_BUSY right away.
				*/
				for(pX=pShmNode->pFirst; pX; pX=pX->pNext){
					if( (pX->exclMask & mask)!=0 || (pX->sharedMask & mask)!=0 ){
						rc = SQLITE_BUSY;
						break;
					}
				}

				/* Get the exclusive locks at the system level.  Then if successful
				** also mark the local connection as being locked.
				*/
				if( rc==SQLITE_OK ){
					rc = winShmSystemLock(pShmNode, _SHM_WRLCK, ofst+WIN_SHM_BASE, n);
					if( rc==SQLITE_OK ){
						assert( (p->sharedMask & mask)==0 );
						p->exclMask |= mask;
					}
				}
			}
			sqlite3_mutex_leave(pShmNode->mutex);
			OSTRACE(("SHM-LOCK shmid-%d, pid-%d got %03x,%03x %s\n",
				p->id, (int)osGetCurrentProcessId(), p->sharedMask, p->exclMask,
				rc ? "failed" : "ok"));
			return rc;
	}

	/*
	** Implement a memory barrier or memory fence on shared memory.  
	**
	** All loads and stores begun before the barrier must complete before
	** any load or store begun after the barrier.
	*/
	static void winShmBarrier(
		sqlite3_file *fd          /* Database holding the shared memory */
		){
			UNUSED_PARAMETER(fd);
			/* MemoryBarrier(); // does not work -- do not know why not */
			winShmEnterMutex();
			winShmLeaveMutex();
	}

	/*
	** This function is called to obtain a pointer to region iRegion of the 
	** shared-memory associated with the database file fd. Shared-memory regions 
	** are numbered starting from zero. Each shared-memory region is szRegion 
	** bytes in size.
	**
	** If an error occurs, an error code is returned and *pp is set to NULL.
	**
	** Otherwise, if the isWrite parameter is 0 and the requested shared-memory
	** region has not been allocated (by any client, including one running in a
	** separate process), then *pp is set to NULL and SQLITE_OK returned. If 
	** isWrite is non-zero and the requested shared-memory region has not yet 
	** been allocated, it is allocated by this function.
	**
	** If the shared-memory region has already been allocated or is allocated by
	** this call as described above, then it is mapped into this processes 
	** address space (if it is not already), *pp is set to point to the mapped 
	** memory and SQLITE_OK returned.
	*/
	static int winShmMap(
		sqlite3_file *fd,               /* Handle open on database file */
		int iRegion,                    /* Region to retrieve */
		int szRegion,                   /* Size of regions */
		int isWrite,                    /* True to extend file if necessary */
		void volatile **pp              /* OUT: Mapped memory */
		){
			winFile *pDbFd = (winFile*)fd;
			winShm *p = pDbFd->pShm;
			winShmNode *pShmNode;
			int rc = SQLITE_OK;

			if( !p ){
				rc = winOpenSharedMemory(pDbFd);
				if( rc!=SQLITE_OK ) return rc;
				p = pDbFd->pShm;
			}
			pShmNode = p->pShmNode;

			sqlite3_mutex_enter(pShmNode->mutex);
			assert( szRegion==pShmNode->szRegion || pShmNode->nRegion==0 );

			if( pShmNode->nRegion<=iRegion ){
				struct ShmRegion *apNew;           /* New aRegion[] array */
				int nByte = (iRegion+1)*szRegion;  /* Minimum required file size */
				sqlite3_int64 sz;                  /* Current size of wal-index file */

				pShmNode->szRegion = szRegion;

				/* The requested region is not mapped into this processes address space.
				** Check to see if it has been allocated (i.e. if the wal-index file is
				** large enough to contain the requested region).
				*/
				rc = winFileSize((sqlite3_file *)&pShmNode->hFile, &sz);
				if( rc!=SQLITE_OK ){
					rc = winLogError(SQLITE_IOERR_SHMSIZE, osGetLastError(),
						"winShmMap1", pDbFd->zPath);
					goto shmpage_out;
				}

				if( sz<nByte ){
					/* The requested memory region does not exist. If isWrite is set to
					** zero, exit early. *pp will be set to NULL and SQLITE_OK returned.
					**
					** Alternatively, if isWrite is non-zero, use ftruncate() to allocate
					** the requested memory region.
					*/
					if( !isWrite ) goto shmpage_out;
					rc = winTruncate((sqlite3_file *)&pShmNode->hFile, nByte);
					if( rc!=SQLITE_OK ){
						rc = winLogError(SQLITE_IOERR_SHMSIZE, osGetLastError(),
							"winShmMap2", pDbFd->zPath);
						goto shmpage_out;
					}
				}

				/* Map the requested memory region into this processes address space. */
				apNew = (struct ShmRegion *)sqlite3_realloc(
					pShmNode->aRegion, (iRegion+1)*sizeof(apNew[0])
					);
				if( !apNew ){
					rc = SQLITE_IOERR_NOMEM;
					goto shmpage_out;
				}
				pShmNode->aRegion = apNew;

				while( pShmNode->nRegion<=iRegion ){
					HANDLE hMap = NULL;         /* file-mapping handle */
					void *pMap = 0;             /* Mapped memory region */

#if SQLITE_OS_WINRT
					hMap = osCreateFileMappingFromApp(pShmNode->hFile.h,
						NULL, PAGE_READWRITE, nByte, NULL
						);
#elif defined(SQLITE_WIN32_HAS_WIDE)
					hMap = osCreateFileMappingW(pShmNode->hFile.h, 
						NULL, PAGE_READWRITE, 0, nByte, NULL
						);
#elif defined(SQLITE_WIN32_HAS_ANSI)
					hMap = osCreateFileMappingA(pShmNode->hFile.h, 
						NULL, PAGE_READWRITE, 0, nByte, NULL
						);
#endif
					OSTRACE(("SHM-MAP pid-%d create region=%d nbyte=%d %s\n",
						(int)osGetCurrentProcessId(), pShmNode->nRegion, nByte,
						hMap ? "ok" : "failed"));
					if( hMap ){
						int iOffset = pShmNode->nRegion*szRegion;
						int iOffsetShift = iOffset % winSysInfo.dwAllocationGranularity;
#if SQLITE_OS_WINRT
						pMap = osMapViewOfFileFromApp(hMap, FILE_MAP_WRITE | FILE_MAP_READ,
							iOffset - iOffsetShift, szRegion + iOffsetShift
							);
#else
						pMap = osMapViewOfFile(hMap, FILE_MAP_WRITE | FILE_MAP_READ,
							0, iOffset - iOffsetShift, szRegion + iOffsetShift
							);
#endif
						OSTRACE(("SHM-MAP pid-%d map region=%d offset=%d size=%d %s\n",
							(int)osGetCurrentProcessId(), pShmNode->nRegion, iOffset,
							szRegion, pMap ? "ok" : "failed"));
					}
					if( !pMap ){
						pShmNode->lastErrno = osGetLastError();
						rc = winLogError(SQLITE_IOERR_SHMMAP, pShmNode->lastErrno,
							"winShmMap3", pDbFd->zPath);
						if( hMap ) osCloseHandle(hMap);
						goto shmpage_out;
					}

					pShmNode->aRegion[pShmNode->nRegion].pMap = pMap;
					pShmNode->aRegion[pShmNode->nRegion].hMap = hMap;
					pShmNode->nRegion++;
				}
			}

shmpage_out:
			if( pShmNode->nRegion>iRegion ){
				int iOffset = iRegion*szRegion;
				int iOffsetShift = iOffset % winSysInfo.dwAllocationGranularity;
				char *p = (char *)pShmNode->aRegion[iRegion].pMap;
				*pp = (void *)&p[iOffsetShift];
			}else{
				*pp = 0;
			}
			sqlite3_mutex_leave(pShmNode->mutex);
			return rc;
	}

#else
# define winShmMap     0
# define winShmLock    0
# define winShmBarrier 0
# define winShmUnmap   0
#endif /* #ifndef SQLITE_OMIT_WAL */

	/*
	** This vector defines all the methods that can operate on an
	** sqlite3_file for win32.
	*/
	static const sqlite3_io_methods winIoMethod = {
		2,                              /* iVersion */
		winClose,                       /* xClose */
		winRead,                        /* xRead */
		winWrite,                       /* xWrite */
		winTruncate,                    /* xTruncate */
		winSync,                        /* xSync */
		winFileSize,                    /* xFileSize */
		winLock,                        /* xLock */
		winUnlock,                      /* xUnlock */
		winCheckReservedLock,           /* xCheckReservedLock */
		winFileControl,                 /* xFileControl */
		winSectorSize,                  /* xSectorSize */
		winDeviceCharacteristics,       /* xDeviceCharacteristics */
		winShmMap,                      /* xShmMap */
		winShmLock,                     /* xShmLock */
		winShmBarrier,                  /* xShmBarrier */
		winShmUnmap                     /* xShmUnmap */
	};

#pragma endregion

#pragma region VSystem


	/*
	** Convert a UTF-8 filename into whatever form the underlying
	** operating system wants filenames in.  Space to hold the result
	** is obtained from malloc and must be freed by the calling
	** function.
	*/
	static void *convertUtf8Filename(const char *zFilename){
		void *zConverted = 0;
		if( isNT() ){
			zConverted = utf8ToUnicode(zFilename);
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			zConverted = sqlite3_win32_utf8_to_mbcs(zFilename);
		}
#endif
		/* caller will handle out of memory */
		return zConverted;
	}

	/*
	** Create a temporary file name in zBuf.  zBuf must be big enough to
	** hold at pVfs->mxPathname characters.
	*/
	static int getTempname(int nBuf, char *zBuf){
		static char zChars[] =
			"abcdefghijklmnopqrstuvwxyz"
			"ABCDEFGHIJKLMNOPQRSTUVWXYZ"
			"0123456789";
		size_t i, j;
		int nTempPath;
		char zTempPath[MAX_PATH+2];

		/* It's odd to simulate an io-error here, but really this is just
		** using the io-error infrastructure to test that SQLite handles this
		** function failing. 
		*/
		SimulateIOError( return SQLITE_IOERR );

		memset(zTempPath, 0, MAX_PATH+2);

		if( sqlite3_temp_directory ){
			sqlite3_snprintf(MAX_PATH-30, zTempPath, "%s", sqlite3_temp_directory);
		}
#if !SQLITE_OS_WINRT
		else if( isNT() ){
			char *zMulti;
			WCHAR zWidePath[MAX_PATH];
			osGetTempPathW(MAX_PATH-30, zWidePath);
			zMulti = unicodeToUtf8(zWidePath);
			if( zMulti ){
				sqlite3_snprintf(MAX_PATH-30, zTempPath, "%s", zMulti);
				sqlite3_free(zMulti);
			}else{
				return SQLITE_IOERR_NOMEM;
			}
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			char *zUtf8;
			char zMbcsPath[MAX_PATH];
			osGetTempPathA(MAX_PATH-30, zMbcsPath);
			zUtf8 = sqlite3_win32_mbcs_to_utf8(zMbcsPath);
			if( zUtf8 ){
				sqlite3_snprintf(MAX_PATH-30, zTempPath, "%s", zUtf8);
				sqlite3_free(zUtf8);
			}else{
				return SQLITE_IOERR_NOMEM;
			}
		}
#endif
#endif

		/* Check that the output buffer is large enough for the temporary file 
		** name. If it is not, return SQLITE_ERROR.
		*/
		nTempPath = sqlite3Strlen30(zTempPath);

		if( (nTempPath + sqlite3Strlen30(SQLITE_TEMP_FILE_PREFIX) + 18) >= nBuf ){
			return SQLITE_ERROR;
		}

		for(i=nTempPath; i>0 && zTempPath[i-1]=='\\'; i--){}
		zTempPath[i] = 0;

		sqlite3_snprintf(nBuf-18, zBuf, (nTempPath > 0) ?
			"%s\\"SQLITE_TEMP_FILE_PREFIX : SQLITE_TEMP_FILE_PREFIX,
			zTempPath);
		j = sqlite3Strlen30(zBuf);
		sqlite3_randomness(15, &zBuf[j]);
		for(i=0; i<15; i++, j++){
			zBuf[j] = (char)zChars[ ((unsigned char)zBuf[j])%(sizeof(zChars)-1) ];
		}
		zBuf[j] = 0;
		zBuf[j+1] = 0;

		OSTRACE(("TEMP FILENAME: %s\n", zBuf));
		return SQLITE_OK; 
	}

	/*
	** Return TRUE if the named file is really a directory.  Return false if
	** it is something other than a directory, or if there is any kind of memory
	** allocation failure.
	*/
	static int winIsDir(const void *zConverted){
		DWORD attr;
		int rc = 0;
		DWORD lastErrno;

		if( isNT() ){
			int cnt = 0;
			WIN32_FILE_ATTRIBUTE_DATA sAttrData;
			memset(&sAttrData, 0, sizeof(sAttrData));
			while( !(rc = osGetFileAttributesExW((LPCWSTR)zConverted,
				GetFileExInfoStandard,
				&sAttrData)) && retryIoerr(&cnt, &lastErrno) ){}
			if( !rc ){
				return 0; /* Invalid name? */
			}
			attr = sAttrData.dwFileAttributes;
#if SQLITE_OS_WINCE==0
		}else{
			attr = osGetFileAttributesA((char*)zConverted);
#endif
		}
		return (attr!=INVALID_FILE_ATTRIBUTES) && (attr&FILE_ATTRIBUTE_DIRECTORY);
	}

	/*
	** Open a file.
	*/
	static int winOpen(
		sqlite3_vfs *pVfs,        /* Not used */
		const char *zName,        /* Name of the file (UTF-8) */
		sqlite3_file *id,         /* Write the SQLite file handle here */
		int flags,                /* Open mode flags */
		int *pOutFlags            /* Status return flags */
		){
			HANDLE h;
			DWORD lastErrno;
			DWORD dwDesiredAccess;
			DWORD dwShareMode;
			DWORD dwCreationDisposition;
			DWORD dwFlagsAndAttributes = 0;
#if SQLITE_OS_WINCE
			int isTemp = 0;
#endif
			winFile *pFile = (winFile*)id;
			void *zConverted;              /* Filename in OS encoding */
			const char *zUtf8Name = zName; /* Filename in UTF-8 encoding */
			int cnt = 0;

			/* If argument zPath is a NULL pointer, this function is required to open
			** a temporary file. Use this buffer to store the file name in.
			*/
			char zTmpname[MAX_PATH+2];     /* Buffer used to create temp filename */

			int rc = SQLITE_OK;            /* Function Return Code */
#if !defined(NDEBUG) || SQLITE_OS_WINCE
			int eType = flags&0xFFFFFF00;  /* Type of file to open */
#endif

			int isExclusive  = (flags & SQLITE_OPEN_EXCLUSIVE);
			int isDelete     = (flags & SQLITE_OPEN_DELETEONCLOSE);
			int isCreate     = (flags & SQLITE_OPEN_CREATE);
#ifndef NDEBUG
			int isReadonly   = (flags & SQLITE_OPEN_READONLY);
#endif
			int isReadWrite  = (flags & SQLITE_OPEN_READWRITE);

#ifndef NDEBUG
			int isOpenJournal = (isCreate && (
				eType==SQLITE_OPEN_MASTER_JOURNAL 
				|| eType==SQLITE_OPEN_MAIN_JOURNAL 
				|| eType==SQLITE_OPEN_WAL
				));
#endif

			/* Check the following statements are true: 
			**
			**   (a) Exactly one of the READWRITE and READONLY flags must be set, and 
			**   (b) if CREATE is set, then READWRITE must also be set, and
			**   (c) if EXCLUSIVE is set, then CREATE must also be set.
			**   (d) if DELETEONCLOSE is set, then CREATE must also be set.
			*/
			assert((isReadonly==0 || isReadWrite==0) && (isReadWrite || isReadonly));
			assert(isCreate==0 || isReadWrite);
			assert(isExclusive==0 || isCreate);
			assert(isDelete==0 || isCreate);

			/* The main DB, main journal, WAL file and master journal are never 
			** automatically deleted. Nor are they ever temporary files.  */
			assert( (!isDelete && zName) || eType!=SQLITE_OPEN_MAIN_DB );
			assert( (!isDelete && zName) || eType!=SQLITE_OPEN_MAIN_JOURNAL );
			assert( (!isDelete && zName) || eType!=SQLITE_OPEN_MASTER_JOURNAL );
			assert( (!isDelete && zName) || eType!=SQLITE_OPEN_WAL );

			/* Assert that the upper layer has set one of the "file-type" flags. */
			assert( eType==SQLITE_OPEN_MAIN_DB      || eType==SQLITE_OPEN_TEMP_DB 
				|| eType==SQLITE_OPEN_MAIN_JOURNAL || eType==SQLITE_OPEN_TEMP_JOURNAL 
				|| eType==SQLITE_OPEN_SUBJOURNAL   || eType==SQLITE_OPEN_MASTER_JOURNAL 
				|| eType==SQLITE_OPEN_TRANSIENT_DB || eType==SQLITE_OPEN_WAL
				);

			assert( pFile!=0 );
			memset(pFile, 0, sizeof(winFile));
			pFile->h = INVALID_HANDLE_VALUE;

#if SQLITE_OS_WINRT
			if( !sqlite3_temp_directory ){
				sqlite3_log(SQLITE_ERROR,
					"sqlite3_temp_directory variable should be set for WinRT");
			}
#endif

			/* If the second argument to this function is NULL, generate a 
			** temporary file name to use 
			*/
			if( !zUtf8Name ){
				assert(isDelete && !isOpenJournal);
				memset(zTmpname, 0, MAX_PATH+2);
				rc = getTempname(MAX_PATH+2, zTmpname);
				if( rc!=SQLITE_OK ){
					return rc;
				}
				zUtf8Name = zTmpname;
			}

			/* Database filenames are double-zero terminated if they are not
			** URIs with parameters.  Hence, they can always be passed into
			** sqlite3_uri_parameter().
			*/
			assert( (eType!=SQLITE_OPEN_MAIN_DB) || (flags & SQLITE_OPEN_URI) ||
				zUtf8Name[strlen(zUtf8Name)+1]==0 );

			/* Convert the filename to the system encoding. */
			zConverted = convertUtf8Filename(zUtf8Name);
			if( zConverted==0 ){
				return SQLITE_IOERR_NOMEM;
			}

			if( winIsDir(zConverted) ){
				sqlite3_free(zConverted);
				return SQLITE_CANTOPEN_ISDIR;
			}

			if( isReadWrite ){
				dwDesiredAccess = GENERIC_READ | GENERIC_WRITE;
			}else{
				dwDesiredAccess = GENERIC_READ;
			}

			/* SQLITE_OPEN_EXCLUSIVE is used to make sure that a new file is 
			** created. SQLite doesn't use it to indicate "exclusive access" 
			** as it is usually understood.
			*/
			if( isExclusive ){
				/* Creates a new file, only if it does not already exist. */
				/* If the file exists, it fails. */
				dwCreationDisposition = CREATE_NEW;
			}else if( isCreate ){
				/* Open existing file, or create if it doesn't exist */
				dwCreationDisposition = OPEN_ALWAYS;
			}else{
				/* Opens a file, only if it exists. */
				dwCreationDisposition = OPEN_EXISTING;
			}

			dwShareMode = FILE_SHARE_READ | FILE_SHARE_WRITE;

			if( isDelete ){
#if SQLITE_OS_WINCE
				dwFlagsAndAttributes = FILE_ATTRIBUTE_HIDDEN;
				isTemp = 1;
#else
				dwFlagsAndAttributes = FILE_ATTRIBUTE_TEMPORARY
					| FILE_ATTRIBUTE_HIDDEN
					| FILE_FLAG_DELETE_ON_CLOSE;
#endif
			}else{
				dwFlagsAndAttributes = FILE_ATTRIBUTE_NORMAL;
			}
			/* Reports from the internet are that performance is always
			** better if FILE_FLAG_RANDOM_ACCESS is used.  Ticket #2699. */
#if SQLITE_OS_WINCE
			dwFlagsAndAttributes |= FILE_FLAG_RANDOM_ACCESS;
#endif

			if( isNT() ){
#if SQLITE_OS_WINRT
				CREATEFILE2_EXTENDED_PARAMETERS extendedParameters;
				extendedParameters.dwSize = sizeof(CREATEFILE2_EXTENDED_PARAMETERS);
				extendedParameters.dwFileAttributes =
					dwFlagsAndAttributes & FILE_ATTRIBUTE_MASK;
				extendedParameters.dwFileFlags = dwFlagsAndAttributes & FILE_FLAG_MASK;
				extendedParameters.dwSecurityQosFlags = SECURITY_ANONYMOUS;
				extendedParameters.lpSecurityAttributes = NULL;
				extendedParameters.hTemplateFile = NULL;
				while( (h = osCreateFile2((LPCWSTR)zConverted,
					dwDesiredAccess,
					dwShareMode,
					dwCreationDisposition,
					&extendedParameters))==INVALID_HANDLE_VALUE &&
					retryIoerr(&cnt, &lastErrno) ){
						/* Noop */
				}
#else
				while( (h = osCreateFileW((LPCWSTR)zConverted,
					dwDesiredAccess,
					dwShareMode, NULL,
					dwCreationDisposition,
					dwFlagsAndAttributes,
					NULL))==INVALID_HANDLE_VALUE &&
					retryIoerr(&cnt, &lastErrno) ){
						/* Noop */
				}
#endif
			}
#ifdef SQLITE_WIN32_HAS_ANSI
			else{
				while( (h = osCreateFileA((LPCSTR)zConverted,
					dwDesiredAccess,
					dwShareMode, NULL,
					dwCreationDisposition,
					dwFlagsAndAttributes,
					NULL))==INVALID_HANDLE_VALUE &&
					retryIoerr(&cnt, &lastErrno) ){
						/* Noop */
				}
			}
#endif
			logIoerr(cnt);

			OSTRACE(("OPEN %d %s 0x%lx %s\n", 
				h, zName, dwDesiredAccess, 
				h==INVALID_HANDLE_VALUE ? "failed" : "ok"));

			if( h==INVALID_HANDLE_VALUE ){
				pFile->lastErrno = lastErrno;
				winLogError(SQLITE_CANTOPEN, pFile->lastErrno, "winOpen", zUtf8Name);
				sqlite3_free(zConverted);
				if( isReadWrite && !isExclusive ){
					return winOpen(pVfs, zName, id, 
						((flags|SQLITE_OPEN_READONLY) &
						~(SQLITE_OPEN_CREATE|SQLITE_OPEN_READWRITE)),
						pOutFlags);
				}else{
					return SQLITE_CANTOPEN_BKPT;
				}
			}

			if( pOutFlags ){
				if( isReadWrite ){
					*pOutFlags = SQLITE_OPEN_READWRITE;
				}else{
					*pOutFlags = SQLITE_OPEN_READONLY;
				}
			}

#if SQLITE_OS_WINCE
			if( isReadWrite && eType==SQLITE_OPEN_MAIN_DB
				&& (rc = winceCreateLock(zName, pFile))!=SQLITE_OK
				){
					osCloseHandle(h);
					sqlite3_free(zConverted);
					return rc;
			}
			if( isTemp ){
				pFile->zDeleteOnClose = zConverted;
			}else
#endif
			{
				sqlite3_free(zConverted);
			}

			pFile->pMethod = &winIoMethod;
			pFile->pVfs = pVfs;
			pFile->h = h;
			if( sqlite3_uri_boolean(zName, "psow", SQLITE_POWERSAFE_OVERWRITE) ){
				pFile->ctrlFlags |= WINFILE_PSOW;
			}
			pFile->lastErrno = NO_ERROR;
			pFile->zPath = zName;

			OpenCounter(+1);
			return rc;
	}

	/*
	** Delete the named file.
	**
	** Note that Windows does not allow a file to be deleted if some other
	** process has it open.  Sometimes a virus scanner or indexing program
	** will open a journal file shortly after it is created in order to do
	** whatever it does.  While this other process is holding the
	** file open, we will be unable to delete it.  To work around this
	** problem, we delay 100 milliseconds and try to delete again.  Up
	** to MX_DELETION_ATTEMPTs deletion attempts are run before giving
	** up and returning an error.
	*/
	static int winDelete(
		sqlite3_vfs *pVfs,          /* Not used on win32 */
		const char *zFilename,      /* Name of file to delete */
		int syncDir                 /* Not used on win32 */
		){
			int cnt = 0;
			int rc;
			DWORD attr;
			DWORD lastErrno;
			void *zConverted;
			UNUSED_PARAMETER(pVfs);
			UNUSED_PARAMETER(syncDir);

			SimulateIOError(return SQLITE_IOERR_DELETE);
			zConverted = convertUtf8Filename(zFilename);
			if( zConverted==0 ){
				return SQLITE_IOERR_NOMEM;
			}
			if( isNT() ){
				do {
#if SQLITE_OS_WINRT
					WIN32_FILE_ATTRIBUTE_DATA sAttrData;
					memset(&sAttrData, 0, sizeof(sAttrData));
					if ( osGetFileAttributesExW(zConverted, GetFileExInfoStandard,
						&sAttrData) ){
							attr = sAttrData.dwFileAttributes;
					}else{
						lastErrno = osGetLastError();
						if( lastErrno==ERROR_FILE_NOT_FOUND
							|| lastErrno==ERROR_PATH_NOT_FOUND ){
								rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
						}else{
							rc = SQLITE_ERROR;
						}
						break;
					}
#else
					attr = osGetFileAttributesW(zConverted);
#endif
					if ( attr==INVALID_FILE_ATTRIBUTES ){
						lastErrno = osGetLastError();
						if( lastErrno==ERROR_FILE_NOT_FOUND
							|| lastErrno==ERROR_PATH_NOT_FOUND ){
								rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
						}else{
							rc = SQLITE_ERROR;
						}
						break;
					}
					if ( attr&FILE_ATTRIBUTE_DIRECTORY ){
						rc = SQLITE_ERROR; /* Files only. */
						break;
					}
					if ( osDeleteFileW(zConverted) ){
						rc = SQLITE_OK; /* Deleted OK. */
						break;
					}
					if ( !retryIoerr(&cnt, &lastErrno) ){
						rc = SQLITE_ERROR; /* No more retries. */
						break;
					}
				} while(1);
			}
#ifdef SQLITE_WIN32_HAS_ANSI
			else{
				do {
					attr = osGetFileAttributesA(zConverted);
					if ( attr==INVALID_FILE_ATTRIBUTES ){
						lastErrno = osGetLastError();
						if( lastErrno==ERROR_FILE_NOT_FOUND
							|| lastErrno==ERROR_PATH_NOT_FOUND ){
								rc = SQLITE_IOERR_DELETE_NOENT; /* Already gone? */
						}else{
							rc = SQLITE_ERROR;
						}
						break;
					}
					if ( attr&FILE_ATTRIBUTE_DIRECTORY ){
						rc = SQLITE_ERROR; /* Files only. */
						break;
					}
					if ( osDeleteFileA(zConverted) ){
						rc = SQLITE_OK; /* Deleted OK. */
						break;
					}
					if ( !retryIoerr(&cnt, &lastErrno) ){
						rc = SQLITE_ERROR; /* No more retries. */
						break;
					}
				} while(1);
			}
#endif
			if( rc && rc!=SQLITE_IOERR_DELETE_NOENT ){
				rc = winLogError(SQLITE_IOERR_DELETE, lastErrno,
					"winDelete", zFilename);
			}else{
				logIoerr(cnt);
			}
			sqlite3_free(zConverted);
			OSTRACE(("DELETE \"%s\" %s\n", zFilename, (rc ? "failed" : "ok" )));
			return rc;
	}

	/*
	** Check the existence and status of a file.
	*/
	static int winAccess(
		sqlite3_vfs *pVfs,         /* Not used on win32 */
		const char *zFilename,     /* Name of file to check */
		int flags,                 /* Type of test to make on this file */
		int *pResOut               /* OUT: Result */
		){
			DWORD attr;
			int rc = 0;
			DWORD lastErrno;
			void *zConverted;
			UNUSED_PARAMETER(pVfs);

			SimulateIOError( return SQLITE_IOERR_ACCESS; );
			zConverted = convertUtf8Filename(zFilename);
			if( zConverted==0 ){
				return SQLITE_IOERR_NOMEM;
			}
			if( isNT() ){
				int cnt = 0;
				WIN32_FILE_ATTRIBUTE_DATA sAttrData;
				memset(&sAttrData, 0, sizeof(sAttrData));
				while( !(rc = osGetFileAttributesExW((LPCWSTR)zConverted,
					GetFileExInfoStandard, 
					&sAttrData)) && retryIoerr(&cnt, &lastErrno) ){}
				if( rc ){
					/* For an SQLITE_ACCESS_EXISTS query, treat a zero-length file
					** as if it does not exist.
					*/
					if(    flags==SQLITE_ACCESS_EXISTS
						&& sAttrData.nFileSizeHigh==0 
						&& sAttrData.nFileSizeLow==0 ){
							attr = INVALID_FILE_ATTRIBUTES;
					}else{
						attr = sAttrData.dwFileAttributes;
					}
				}else{
					logIoerr(cnt);
					if( lastErrno!=ERROR_FILE_NOT_FOUND && lastErrno!=ERROR_PATH_NOT_FOUND ){
						winLogError(SQLITE_IOERR_ACCESS, lastErrno, "winAccess", zFilename);
						sqlite3_free(zConverted);
						return SQLITE_IOERR_ACCESS;
					}else{
						attr = INVALID_FILE_ATTRIBUTES;
					}
				}
			}
#ifdef SQLITE_WIN32_HAS_ANSI
			else{
				attr = osGetFileAttributesA((char*)zConverted);
			}
#endif
			sqlite3_free(zConverted);
			switch( flags ){
			case SQLITE_ACCESS_READ:
			case SQLITE_ACCESS_EXISTS:
				rc = attr!=INVALID_FILE_ATTRIBUTES;
				break;
			case SQLITE_ACCESS_READWRITE:
				rc = attr!=INVALID_FILE_ATTRIBUTES &&
					(attr & FILE_ATTRIBUTE_READONLY)==0;
				break;
			default:
				assert(!"Invalid flags argument");
			}
			*pResOut = rc;
			return SQLITE_OK;
	}


	/*
	** Returns non-zero if the specified path name should be used verbatim.  If
	** non-zero is returned from this function, the calling function must simply
	** use the provided path name verbatim -OR- resolve it into a full path name
	** using the GetFullPathName Win32 API function (if available).
	*/
	static BOOL winIsVerbatimPathname(
		const char *zPathname
		){
			/*
			** If the path name starts with a forward slash or a backslash, it is either
			** a legal UNC name, a volume relative path, or an absolute path name in the
			** "Unix" format on Windows.  There is no easy way to differentiate between
			** the final two cases; therefore, we return the safer return value of TRUE
			** so that callers of this function will simply use it verbatim.
			*/
			if ( zPathname[0]=='/' || zPathname[0]=='\\' ){
				return TRUE;
			}

			/*
			** If the path name starts with a letter and a colon it is either a volume
			** relative path or an absolute path.  Callers of this function must not
			** attempt to treat it as a relative path name (i.e. they should simply use
			** it verbatim).
			*/
			if ( sqlite3Isalpha(zPathname[0]) && zPathname[1]==':' ){
				return TRUE;
			}

			/*
			** If we get to this point, the path name should almost certainly be a purely
			** relative one (i.e. not a UNC name, not absolute, and not volume relative).
			*/
			return FALSE;
	}

	/*
	** Turn a relative pathname into a full pathname.  Write the full
	** pathname into zOut[].  zOut[] will be at least pVfs->mxPathname
	** bytes in size.
	*/
	static int winFullPathname(
		sqlite3_vfs *pVfs,            /* Pointer to vfs object */
		const char *zRelative,        /* Possibly relative input path */
		int nFull,                    /* Size of output buffer in bytes */
		char *zFull                   /* Output buffer */
		){

#if defined(__CYGWIN__)
			SimulateIOError( return SQLITE_ERROR );
			UNUSED_PARAMETER(nFull);
			assert( pVfs->mxPathname>=MAX_PATH );
			assert( nFull>=pVfs->mxPathname );
			if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
				/*
				** NOTE: We are dealing with a relative path name and the data
				**       directory has been set.  Therefore, use it as the basis
				**       for converting the relative path name to an absolute
				**       one by prepending the data directory and a slash.
				*/
				char zOut[MAX_PATH+1];
				memset(zOut, 0, MAX_PATH+1);
				cygwin_conv_path(CCP_POSIX_TO_WIN_A|CCP_RELATIVE, zRelative, zOut,
					MAX_PATH+1);
				sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
					sqlite3_data_directory, zOut);
			}else{
				cygwin_conv_path(CCP_POSIX_TO_WIN_A, zRelative, zFull, nFull);
			}
			return SQLITE_OK;
#endif

#if (SQLITE_OS_WINCE || SQLITE_OS_WINRT) && !defined(__CYGWIN__)
			SimulateIOError( return SQLITE_ERROR );
			/* WinCE has no concept of a relative pathname, or so I am told. */
			/* WinRT has no way to convert a relative path to an absolute one. */
			if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
				/*
				** NOTE: We are dealing with a relative path name and the data
				**       directory has been set.  Therefore, use it as the basis
				**       for converting the relative path name to an absolute
				**       one by prepending the data directory and a backslash.
				*/
				sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
					sqlite3_data_directory, zRelative);
			}else{
				sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s", zRelative);
			}
			return SQLITE_OK;
#endif

#if !SQLITE_OS_WINCE && !SQLITE_OS_WINRT && !defined(__CYGWIN__)
			DWORD nByte;
			void *zConverted;
			char *zOut;

			/* If this path name begins with "/X:", where "X" is any alphabetic
			** character, discard the initial "/" from the pathname.
			*/
			if( zRelative[0]=='/' && sqlite3Isalpha(zRelative[1]) && zRelative[2]==':' ){
				zRelative++;
			}

			/* It's odd to simulate an io-error here, but really this is just
			** using the io-error infrastructure to test that SQLite handles this
			** function failing. This function could fail if, for example, the
			** current working directory has been unlinked.
			*/
			SimulateIOError( return SQLITE_ERROR );
			if ( sqlite3_data_directory && !winIsVerbatimPathname(zRelative) ){
				/*
				** NOTE: We are dealing with a relative path name and the data
				**       directory has been set.  Therefore, use it as the basis
				**       for converting the relative path name to an absolute
				**       one by prepending the data directory and a backslash.
				*/
				sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s\\%s",
					sqlite3_data_directory, zRelative);
				return SQLITE_OK;
			}
			zConverted = convertUtf8Filename(zRelative);
			if( zConverted==0 ){
				return SQLITE_IOERR_NOMEM;
			}
			if( isNT() ){
				LPWSTR zTemp;
				nByte = osGetFullPathNameW((LPCWSTR)zConverted, 0, 0, 0);
				if( nByte==0 ){
					winLogError(SQLITE_ERROR, osGetLastError(),
						"GetFullPathNameW1", zConverted);
					sqlite3_free(zConverted);
					return SQLITE_CANTOPEN_FULLPATH;
				}
				nByte += 3;
				zTemp = sqlite3MallocZero( nByte*sizeof(zTemp[0]) );
				if( zTemp==0 ){
					sqlite3_free(zConverted);
					return SQLITE_IOERR_NOMEM;
				}
				nByte = osGetFullPathNameW((LPCWSTR)zConverted, nByte, zTemp, 0);
				if( nByte==0 ){
					winLogError(SQLITE_ERROR, osGetLastError(),
						"GetFullPathNameW2", zConverted);
					sqlite3_free(zConverted);
					sqlite3_free(zTemp);
					return SQLITE_CANTOPEN_FULLPATH;
				}
				sqlite3_free(zConverted);
				zOut = unicodeToUtf8(zTemp);
				sqlite3_free(zTemp);
			}
#ifdef SQLITE_WIN32_HAS_ANSI
			else{
				char *zTemp;
				nByte = osGetFullPathNameA((char*)zConverted, 0, 0, 0);
				if( nByte==0 ){
					winLogError(SQLITE_ERROR, osGetLastError(),
						"GetFullPathNameA1", zConverted);
					sqlite3_free(zConverted);
					return SQLITE_CANTOPEN_FULLPATH;
				}
				nByte += 3;
				zTemp = sqlite3MallocZero( nByte*sizeof(zTemp[0]) );
				if( zTemp==0 ){
					sqlite3_free(zConverted);
					return SQLITE_IOERR_NOMEM;
				}
				nByte = osGetFullPathNameA((char*)zConverted, nByte, zTemp, 0);
				if( nByte==0 ){
					winLogError(SQLITE_ERROR, osGetLastError(),
						"GetFullPathNameA2", zConverted);
					sqlite3_free(zConverted);
					sqlite3_free(zTemp);
					return SQLITE_CANTOPEN_FULLPATH;
				}
				sqlite3_free(zConverted);
				zOut = sqlite3_win32_mbcs_to_utf8(zTemp);
				sqlite3_free(zTemp);
			}
#endif
			if( zOut ){
				sqlite3_snprintf(MIN(nFull, pVfs->mxPathname), zFull, "%s", zOut);
				sqlite3_free(zOut);
				return SQLITE_OK;
			}else{
				return SQLITE_IOERR_NOMEM;
			}
#endif
	}

#ifndef SQLITE_OMIT_LOAD_EXTENSION
	/*
	** Interfaces for opening a shared library, finding entry points
	** within the shared library, and closing the shared library.
	*/
	/*
	** Interfaces for opening a shared library, finding entry points
	** within the shared library, and closing the shared library.
	*/
	static void *winDlOpen(sqlite3_vfs *pVfs, const char *zFilename){
		HANDLE h;
		void *zConverted = convertUtf8Filename(zFilename);
		UNUSED_PARAMETER(pVfs);
		if( zConverted==0 ){
			return 0;
		}
		if( isNT() ){
#if SQLITE_OS_WINRT
			h = osLoadPackagedLibrary((LPCWSTR)zConverted, 0);
#else
			h = osLoadLibraryW((LPCWSTR)zConverted);
#endif
		}
#ifdef SQLITE_WIN32_HAS_ANSI
		else{
			h = osLoadLibraryA((char*)zConverted);
		}
#endif
		sqlite3_free(zConverted);
		return (void*)h;
	}
	static void winDlError(sqlite3_vfs *pVfs, int nBuf, char *zBufOut){
		UNUSED_PARAMETER(pVfs);
		getLastErrorMsg(osGetLastError(), nBuf, zBufOut);
	}
	static void (*winDlSym(sqlite3_vfs *pVfs,void *pH,const char *zSym))(void){
		UNUSED_PARAMETER(pVfs);
		return (void(*)(void))osGetProcAddressA((HANDLE)pH, zSym);
	}
	static void winDlClose(sqlite3_vfs *pVfs, void *pHandle){
		UNUSED_PARAMETER(pVfs);
		osFreeLibrary((HANDLE)pHandle);
	}
#else /* if SQLITE_OMIT_LOAD_EXTENSION is defined: */
#define winDlOpen  0
#define winDlError 0
#define winDlSym   0
#define winDlClose 0
#endif


	/*
	** Write up to nBuf bytes of randomness into zBuf.
	*/
	static int winRandomness(sqlite3_vfs *pVfs, int nBuf, char *zBuf){
		int n = 0;
		UNUSED_PARAMETER(pVfs);
#if defined(SQLITE_TEST)
		n = nBuf;
		memset(zBuf, 0, nBuf);
#else
		if( sizeof(SYSTEMTIME)<=nBuf-n ){
			SYSTEMTIME x;
			osGetSystemTime(&x);
			memcpy(&zBuf[n], &x, sizeof(x));
			n += sizeof(x);
		}
		if( sizeof(DWORD)<=nBuf-n ){
			DWORD pid = osGetCurrentProcessId();
			memcpy(&zBuf[n], &pid, sizeof(pid));
			n += sizeof(pid);
		}
#if SQLITE_OS_WINRT
		if( sizeof(ULONGLONG)<=nBuf-n ){
			ULONGLONG cnt = osGetTickCount64();
			memcpy(&zBuf[n], &cnt, sizeof(cnt));
			n += sizeof(cnt);
		}
#else
		if( sizeof(DWORD)<=nBuf-n ){
			DWORD cnt = osGetTickCount();
			memcpy(&zBuf[n], &cnt, sizeof(cnt));
			n += sizeof(cnt);
		}
#endif
		if( sizeof(LARGE_INTEGER)<=nBuf-n ){
			LARGE_INTEGER i;
			osQueryPerformanceCounter(&i);
			memcpy(&zBuf[n], &i, sizeof(i));
			n += sizeof(i);
		}
#endif
		return n;
	}


	/*
	** Sleep for a little while.  Return the amount of time slept.
	*/
	static int winSleep(sqlite3_vfs *pVfs, int microsec){
		sqlite3_win32_sleep((microsec+999)/1000);
		UNUSED_PARAMETER(pVfs);
		return ((microsec+999)/1000)*1000;
	}

	/*
	** The following variable, if set to a non-zero value, is interpreted as
	** the number of seconds since 1970 and is used to set the result of
	** sqlite3OsCurrentTime() during testing.
	*/
#ifdef SQLITE_TEST
	int sqlite3_current_time = 0;  /* Fake system time in seconds since 1970. */
#endif

	/*
	** Find the current time (in Universal Coordinated Time).  Write into *piNow
	** the current time and date as a Julian Day number times 86_400_000.  In
	** other words, write into *piNow the number of milliseconds since the Julian
	** epoch of noon in Greenwich on November 24, 4714 B.C according to the
	** proleptic Gregorian calendar.
	**
	** On success, return SQLITE_OK.  Return SQLITE_ERROR if the time and date 
	** cannot be found.
	*/
	static int winCurrentTimeInt64(sqlite3_vfs *pVfs, sqlite3_int64 *piNow){
		/* FILETIME structure is a 64-bit value representing the number of 
		100-nanosecond intervals since January 1, 1601 (= JD 2305813.5). 
		*/
		FILETIME ft;
		static const sqlite3_int64 winFiletimeEpoch = 23058135*(sqlite3_int64)8640000;
#ifdef SQLITE_TEST
		static const sqlite3_int64 unixEpoch = 24405875*(sqlite3_int64)8640000;
#endif
		/* 2^32 - to avoid use of LL and warnings in gcc */
		static const sqlite3_int64 max32BitValue = 
			(sqlite3_int64)2000000000 + (sqlite3_int64)2000000000 +
			(sqlite3_int64)294967296;

#if SQLITE_OS_WINCE
		SYSTEMTIME time;
		osGetSystemTime(&time);
		/* if SystemTimeToFileTime() fails, it returns zero. */
		if (!osSystemTimeToFileTime(&time,&ft)){
			return SQLITE_ERROR;
		}
#else
		osGetSystemTimeAsFileTime( &ft );
#endif

		*piNow = winFiletimeEpoch +
			((((sqlite3_int64)ft.dwHighDateTime)*max32BitValue) + 
			(sqlite3_int64)ft.dwLowDateTime)/(sqlite3_int64)10000;

#ifdef SQLITE_TEST
		if( sqlite3_current_time ){
			*piNow = 1000*(sqlite3_int64)sqlite3_current_time + unixEpoch;
		}
#endif
		UNUSED_PARAMETER(pVfs);
		return SQLITE_OK;
	}

	/*
	** Find the current time (in Universal Coordinated Time).  Write the
	** current time and date as a Julian Day number into *prNow and
	** return 0.  Return 1 if the time and date cannot be found.
	*/
	static int winCurrentTime(sqlite3_vfs *pVfs, double *prNow){
		int rc;
		sqlite3_int64 i;
		rc = winCurrentTimeInt64(pVfs, &i);
		if( !rc ){
			*prNow = i/86400000.0;
		}
		return rc;
	}

	/*
	** The idea is that this function works like a combination of
	** GetLastError() and FormatMessage() on Windows (or errno and
	** strerror_r() on Unix). After an error is returned by an OS
	** function, SQLite calls this function with zBuf pointing to
	** a buffer of nBuf bytes. The OS layer should populate the
	** buffer with a nul-terminated UTF-8 encoded error message
	** describing the last IO error to have occurred within the calling
	** thread.
	**
	** If the error message is too large for the supplied buffer,
	** it should be truncated. The return value of xGetLastError
	** is zero if the error message fits in the buffer, or non-zero
	** otherwise (if the message was truncated). If non-zero is returned,
	** then it is not necessary to include the nul-terminator character
	** in the output buffer.
	**
	** Not supplying an error message will have no adverse effect
	** on SQLite. It is fine to have an implementation that never
	** returns an error message:
	**
	**   int xGetLastError(sqlite3_vfs *pVfs, int nBuf, char *zBuf){
	**     assert(zBuf[0]=='\0');
	**     return 0;
	**   }
	**
	** However if an error message is supplied, it will be incorporated
	** by sqlite into the error message available to the user using
	** sqlite3_errmsg(), possibly making IO errors easier to debug.
	*/
	static int winGetLastError(sqlite3_vfs *pVfs, int nBuf, char *zBuf){
		UNUSED_PARAMETER(pVfs);
		return getLastErrorMsg(osGetLastError(), nBuf, zBuf);
	}

	/*
	** Initialize and deinitialize the operating system interface.
	*/
	int sqlite3_os_init(void){
		static sqlite3_vfs winVfs = {
			3,                   /* iVersion */
			sizeof(winFile),     /* szOsFile */
			MAX_PATH,            /* mxPathname */
			0,                   /* pNext */
			"win32",             /* zName */
			0,                   /* pAppData */
			winOpen,             /* xOpen */
			winDelete,           /* xDelete */
			winAccess,           /* xAccess */
			winFullPathname,     /* xFullPathname */
			winDlOpen,           /* xDlOpen */
			winDlError,          /* xDlError */
			winDlSym,            /* xDlSym */
			winDlClose,          /* xDlClose */
			winRandomness,       /* xRandomness */
			winSleep,            /* xSleep */
			winCurrentTime,      /* xCurrentTime */
			winGetLastError,     /* xGetLastError */
			winCurrentTimeInt64, /* xCurrentTimeInt64 */
			winSetSystemCall,    /* xSetSystemCall */
			winGetSystemCall,    /* xGetSystemCall */
			winNextSystemCall,   /* xNextSystemCall */
		};

		/* Double-check that the aSyscall[] array has been constructed
		** correctly.  See ticket [bb3a86e890c8e96ab] */
		assert( ArraySize(aSyscall)==74 );

#ifndef SQLITE_OMIT_WAL
		/* get memory map allocation granularity */
		memset(&winSysInfo, 0, sizeof(SYSTEM_INFO));
#if SQLITE_OS_WINRT
		osGetNativeSystemInfo(&winSysInfo);
#else
		osGetSystemInfo(&winSysInfo);
#endif
		assert(winSysInfo.dwAllocationGranularity > 0);
#endif

		sqlite3_vfs_register(&winVfs, 1);
		return SQLITE_OK; 
	}

	int sqlite3_os_end(void){ 
#if SQLITE_OS_WINRT
		if( sleepObj!=NULL ){
			osCloseHandle(sleepObj);
			sleepObj = NULL;
		}
#endif
		return SQLITE_OK;
	}


#pragma endregion

}}
#endif