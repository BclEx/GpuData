﻿// pcache1.c
#include "Core+PCache1.cu.h"
using namespace Core;

namespace Core
{
	static void BufferSetup(void *buffer, int size, int n)
	{
		if (_pcache1.IsInit)
		{
			size = SysEx_ROUNDDOWN8(size);
			_pcache1.SizeSlot = size;
			_pcache1.Slots = _pcache1.FreeSlots = n;
			_pcache1.Reserves = (n > 90 ? 10 : (n / 10 + 1));
			_pcache1.Start = buffer;
			_pcache1.Free = nullptr;
			_pcache1.UnderPressure = false;
			while (n--)
			{
				PgFreeslot *p = (PgFreeslot *)buffer;
				p->Next = _pcache1.Free;
				_pcache1.Free = p;
				buffer = (void *)&((char *)buffer)[size];
			}
			_pcache1.End = buffer;
		}
	}

	static void *Alloc(int bytes)
	{
		_assert(MutexEx::NotHeld(_pcache1.Group.Mutex));
		StatusEx::StatusSet(StatusEx::STATUS_PAGECACHE_SIZE, bytes);
		PgHdr *p = nullptr;
		if (bytes <= _pcache1.SizeSlot)
		{
			MutexEx::Enter(_pcache1.Mutex);
			p = (PgHdr1 *)_pcache1.Free;
			if (p)
			{
				_pcache1.Free = _pcache1.Free->Next;
				_pcache1.FreeSlots--;
				_pcache1.UnderPressure = (_pcache1.FreeSlots < _pcache1.Reserves);
				_assert(_pcache1.FreeSlots >= 0);
				StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_USED, 1);
			}
			MutexEx::Leave(_pcache1.Mutex);
		}
		if (!p)
		{
			// Memory is not available in the SQLITE_CONFIG_PAGECACHE pool.  Get it from sqlite3Malloc instead.
			p = SysEx::Alloc(bytes);
#ifndef DISABLE_PAGECACHE_OVERFLOW_STATS
			if (p)
			{
				int size = SysEx::AllocSize(p);
				MutexEx::Enter(_pcache1.Mutex);
				StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_OVERFLOW, size);
				MutexEx::Leave(_pcache1.Mutex);
			}
#endif
			SysEx::MemdebugSetType(p, SysEx::MEMTYPE_PCACHE);
		}
		return p;
	}

	static int Free(void *p)
	{
		int freed = 0;
		if (p == nullptr)
			return 0;
		if (p >= _pcache1.Start && p < _pcache1.End)
		{
			MutexEx::Enter(_pcache1.Mutex);
			StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_USED, -1);
			PgFreeslot *slot = (PgFreeslot *)p;
			slot->Next = pcache1.Free;
			_pcache1.Free = slot;
			_pcache1.FreeSlots++;
			_pcache1.UnderPressure = (_pcache1.FreeSlots < _pcache1.Reserves);
			_assert(pcache1.FreeSlots <= _pcache1.Slots);
			MutexEx::Leave(_pcache1.Mutex);
		}
		else
		{
			_assert(SysEx::MemdebugHasType(p, SysEx::MEMTYPE_PCACHE));
			SysEx::MemdebugSetType(p, SysEx::MEMTYPE_HEAP);
			freed = SysEx::MallocSize(p);
#ifndef DISABLE_PAGECACHE_OVERFLOW_STATS
			MutexEx::Enter(_pcache1.Mutex);
			StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_OVERFLOW, -freed);
			Mutex::Leave(_pcache1.Mutex);
#endif
			SysEx::Free(p);
		}
		return freed;
	}

#ifdef ENABLE_MEMORY_MANAGEMENT
	static int MemSize(void *p)
	{
		if (p >= pcache1.Start && p < pcache1.End)
			return pcache1.SizeSlot;
		assert(SysEx::MemdebugHasType(p, SysEx::MEMTYPE_PCACHE) );
		SysEx::MemdebugSetType(p, SysEx::MEMTYPE_HEAP);
		int size = SysEx::MallocSize(p);
		SysEx::MemdebugSetType(p, SysEx::MEMTYPE_PCACHE);
		return size;
	}
#endif

	static PgHdr1 *AllocPage(PCache1 *t)
	{
		// The group mutex must be released before pcache1Alloc() is called. This is because it may call sqlite3_release_memory(), which assumes that this mutex is not held.
		assert(MutexEx::Held(t->Group->Mutex));
		PCache1::LeaveMutex(t->Group);
		PgHdr1 *p = nullptr;
		void *pPg;
#ifdef PCACHE_SEPARATE_HEADER
		pPg = Alloc(t->SizePage);
		p = SysEx::Malloc(sizeof(PgHdr1) + t->SizeExtra);
		if (!pPg || !p)
		{
			Free(pPg);
SysEx:Free(p);
			pPg = nullptr;
		}
#else
		pPg = Alloc(sizeof(PgHdr1) + t->SizePage + t->SizeExtra);
		p = (PgHdr1 *)&((u8 *)pPg)[t->SizePage];
#endif
		PCache1::EnterMutex(t->Group);
		if (pPg)
		{
			p->Page.Buffer = pPg;
			p->Page.Extra = &p[1];
			if (t->Purgeable)
				t->Group->CurrentPages++;
			return p;
		}
		return nullptr;
	}

	static void FreePage(PgHdr1 *p)
	{
		if (SysEx::ALWAYS(p))
		{
			PCache1 *cache = p->Cache;
			assert(MutexEx::Held(p->Cache->Group->Mutex));
			Free(p->Page.Buffer);
#ifdef PCACHE_SEPARATE_HEADER
			SysEx::Free(p);
#endif
			if (cache->Purgeable)
				cache->Group->CurrentPages--;
		}
	}

	/*
	** Malloc function used by SQLite to obtain space from the buffer configured
	** using sqlite3_config(SQLITE_CONFIG_PAGECACHE) option. If no such buffer
	** exists, this function falls back to sqlite3Malloc().
	*/
	void *sqlite3PageMalloc(int sz){
		return pcache1Alloc(sz);
	}

	/*
	** Free an allocated buffer obtained from sqlite3PageMalloc().
	*/
	void sqlite3PageFree(void *p){
		pcache1Free(p);
	}


	/*
	** Return true if it desirable to avoid allocating a new page cache
	** entry.
	**
	** If memory was allocated specifically to the page cache using
	** SQLITE_CONFIG_PAGECACHE but that memory has all been used, then
	** it is desirable to avoid allocating a new page cache entry because
	** presumably SQLITE_CONFIG_PAGECACHE was suppose to be sufficient
	** for all page cache needs and we should not need to spill the
	** allocation onto the heap.
	**
	** Or, the heap is used for all page cache memory but the heap is
	** under memory pressure, then again it is desirable to avoid
	** allocating a new page cache entry in order to avoid stressing
	** the heap even further.
	*/
	static int pcache1UnderMemoryPressure(PCache1 *pCache){
		if( pcache1.nSlot && (pCache->szPage+pCache->szExtra)<=pcache1.szSlot ){
			return pcache1.bUnderPressure;
		}else{
			return sqlite3HeapNearlyFull();
		}
	}

}
