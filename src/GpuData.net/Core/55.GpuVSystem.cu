﻿// os_win.c
#define OS_GPU 1
#if OS_GPU
#include "Core.cu.h"
#include <new.h>

namespace Core
{
#pragma region Preamble

#if defined(TEST) || defined(_DEBUG)
	bool OsTrace = true;
#define OSTRACE(X, ...) if (OsTrace) { printf(X, __VA_ARGS__); }
#else
#define OSTRACE(X, ...)
#endif

#pragma endregion

#pragma region GpuVSystem

	class GpuVSystem : public VSystem
	{
	public:
		__device__ GpuVSystem() { }
		__device__ virtual RC Open(const char *path, VFile *file, OPEN flags, OPEN *outFlags);
		__device__ virtual RC Delete(const char *path, bool syncDirectory);
		__device__ virtual RC Access(const char *path, ACCESS flags, int *outRC);
		__device__ virtual RC FullPathname(const char *path, int pathOutLength, char *pathOut);

		__device__ virtual void *DlOpen(const char *filename);
		__device__ virtual void DlError(int bufLength, char *buf);
		__device__ virtual void (*DlSym(void *handle, const char *symbol))();
		__device__ virtual void DlClose(void *handle);

		__device__ virtual int Randomness(int bufLength, char *buf);
		__device__ virtual int Sleep(int microseconds);
		__device__ virtual RC CurrentTimeInt64(int64 *now);
		__device__ virtual RC CurrentTime(double *now);
		__device__ virtual RC GetLastError(int bufLength, char *buf);

		__device__ virtual RC SetSystemCall(const char *name, syscall_ptr newFunc);
		__device__ virtual syscall_ptr GetSystemCall(const char *name);
		__device__ virtual const char *NextSystemCall(const char *name);
	};

#pragma endregion

#pragma region GpuVSystem

	__device__ RC GpuVSystem::Open(const char *name, VFile *id, OPEN flags, OPEN *outFlags)
	{
		return RC::ERROR;
	}

	__device__ RC GpuVSystem::Delete(const char *filename, bool syncDir)
	{
		return RC::ERROR;
	}

	__device__ RC GpuVSystem::Access(const char *filename, ACCESS flags, int *resOut)
	{
		return RC::ERROR;
	}

	__device__ RC GpuVSystem::FullPathname(const char *relative, int fullLength, char *full)
	{
		return RC::ERROR;
	}

#ifndef OMIT_LOAD_EXTENSION
	__device__ void *GpuVSystem::DlOpen(const char *filename)
	{
		return nullptr;
	}

	__device__ void GpuVSystem::DlError(int bufLength, char *buf)
	{
	}

	__device__ void (*GpuVSystem::DlSym(void *handle, const char *symbol))()
	{
		return nullptr;
	}

	__device__ void GpuVSystem::DlClose(void *handle)
	{
	}
#else
#define winDlOpen  0
#define winDlError 0
#define winDlSym   0
#define winDlClose 0
#endif

	__device__ int GpuVSystem::Randomness(int bufLength, char *buf)
	{
		return 0;
	}

	__device__ int GpuVSystem::Sleep(int microseconds)
	{
		return 0;
	}

	__device__ RC GpuVSystem::CurrentTimeInt64(int64 *now)
	{
		return RC::ERROR;
	}

	__device__ RC GpuVSystem::CurrentTime(double *now)
	{
		return RC::ERROR;
	}

	__device__ RC GpuVSystem::GetLastError(int bufLength, char *buf)
	{
		return RC::ERROR;
	}


	__device__ RC GpuVSystem::SetSystemCall(const char *name, syscall_ptr newFunc)
	{
		return RC::ERROR;
	}
	__device__ syscall_ptr GpuVSystem::GetSystemCall(const char *name)
	{
		return nullptr;
	}
	__device__ const char *GpuVSystem::NextSystemCall(const char *name)
	{
		return nullptr;
	}

	__device__ static char _gpuVfsBuf[sizeof(GpuVSystem)];
	__device__ static GpuVSystem *_gpuVfs;
	__device__ RC VSystem::Initialize()
	{
		_gpuVfs = new (_gpuVfsBuf) GpuVSystem();
		_gpuVfs->SizeOsFile = 0;
		_gpuVfs->MaxPathname = 260;
		_gpuVfs->Name = "gpu";
		RegisterVfs(_gpuVfs, true);
		return RC::OK; 
	}

	__device__ void VSystem::Shutdown()
	{ 
	}

#pragma endregion

}
#endif