#include "hip/hip_runtime.h"
//#include "..\GpuData\Core\Core.cu.h"
#include "..\GpuData.net\Core+Pager\Core+Pager.cu.h"
using namespace Core;
using namespace Core::IO;


__device__ static void TestVFS();
__device__ static void TestPager();

__global__ void MainTest(void *heap)
{
	SysEx::Initialize();
	//TestVFS();
	TestPager();
}

void __main(cudaRuntimeHost &r)
{	
	cudaRuntimeSetHeap(r.heap);
	MainTest<<<1, 1>>>(r.heap);
}

__device__ static void TestVFS()
{
	auto vfs = VSystem::Find("gpu");
	auto file = (VFile *)SysEx::Alloc(vfs->SizeOsFile);
	auto rc = vfs->Open("C:\\T_\\Test.db", file, (VSystem::OPEN)((int)VSystem::OPEN_CREATE | (int)VSystem::OPEN_READWRITE | (int)VSystem::OPEN_MAIN_DB), nullptr);
	_printf("%d\n", rc);
	file->Write4(0, 123145);
	file->Close();
	SysEx::Free(file);
}

__device__ static int Busyhandler(void *x) { _printf("BUSY"); return -1; }

__device__ static Pager *Open(VSystem *vfs)
{
	byte dbHeader[100]; // Database header content

	auto flags = (IPager::PAGEROPEN)0;
	auto vfsFlags = (VSystem::OPEN)((int)VSystem::OPEN_CREATE | (int)VSystem::OPEN_READWRITE | (int)VSystem::OPEN_MAIN_DB);
	//
	Pager *pager;
	auto rc = Pager::Open(vfs, &pager, "memory", 0, flags, vfsFlags, nullptr);
	if (rc == RC::OK)
		rc = pager->ReadFileheader(sizeof(dbHeader), dbHeader);
	if (rc != RC::OK)
		goto _out;
	pager->SetBusyhandler(Busyhandler, nullptr);
	auto readOnly = pager->get_Readonly();
	//
	int reserves;
	auto pageSize = (uint)((dbHeader[16] << 8) | (dbHeader[17] << 16));
	if (pageSize < 512 || pageSize > MAX_PAGE_SIZE || ((pageSize - 1) & pageSize) != 0)
	{
		pageSize = 0;
		reserves = 0;
	}
	else
		reserves = dbHeader[20];
	rc = pager->SetPageSize(&pageSize, reserves);
	if (rc) goto _out;
_out:
	if (rc != RC::OK)
	{
		if (pager)
			pager->Close();
		pager = nullptr;
	}
	else
		pager->SetCacheSize(2000);
	return pager;
}

__device__ static void TestPager()
{
	auto vfs = VSystem::Find("gpu");
	auto pager = Open(vfs);
	if (pager == nullptr)
		_throw("");
	auto rc = pager->SharedLock();
	if (rc != RC::OK)
		_throw("");
	//
	IPage *p = nullptr;
	rc = pager->Acquire(1, &p, false);
	if (rc != RC::OK)
		_throw("");
	rc = pager->Begin(0, false);
	if (rc != RC::OK)
		_throw("");
	char values[] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };
	memcpy(values, p->Data, 10);
	Pager::Write(p);
	pager->CommitPhaseOne(nullptr, false);
	pager->CommitPhaseTwo();
	//
	if (pager != nullptr)
		pager->Close();
}
