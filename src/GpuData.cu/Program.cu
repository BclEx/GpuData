#include "hip/hip_runtime.h"
//#include "..\GpuData\Core\Core.cu.h"
#include "..\GpuData.net\Core+Pager\Core+Pager.cu.h"
using namespace Core;
using namespace Core::IO;

//static void TestVFS();
//static void TestPager();

__device__ static void TestVFS()
{
	VSystem *vfs = VSystem::Find("gpu");
	VFile *file = (VFile *)SysEx::Alloc(vfs->SizeOsFile);
	RC rc = vfs->Open("C:\\T_\\Test.db", file, VSystem::OPEN_CREATE | VSystem::OPEN_READWRITE | VSystem::OPEN_MAIN_DB, nullptr);
	file->Write4(0, 123145);
	file->Close();
}

__global__ void MainTest(void *heap)
{
	_runtimeSetHeap(heap);
	_printf("HERE");
	//SysEx::Initialize();
	//TestVFS();
}

void __main(cudaRuntimeHost &r)
{	
	cudaRuntimeSetHeap(r.heap);
	MainTest<<<1, 1>>>(r.heap);
	//
	//TestVFS();
	//TestPager();
}

/*
static void TestVFS()
{
	auto vfs = VSystem::Find("win32");
	auto file = (VFile *)SysEx::Alloc(vfs->SizeOsFile);
	auto rc = vfs->Open("C:\\T_\\Test.db", file, VSystem::OPEN_CREATE | VSystem::OPEN_READWRITE | VSystem::OPEN_MAIN_DB, nullptr);
	file->Write4(0, 123145);
	file->Close();
}

static int Busyhandler(void *x) { printf("BUSY"); return -1; }

static Pager *Open(VSystem *vfs)
{
	byte dbHeader[100]; // Database header content

	IPager::PAGEROPEN flags = (IPager::PAGEROPEN)0;
	VSystem::OPEN vfsFlags = VSystem::OPEN_CREATE | VSystem::OPEN_READWRITE | VSystem::OPEN_MAIN_DB;
	//
	Pager *pager;
	auto rc = Pager::Open(vfs, &pager, "C:\\T_\\Test.db", 0, flags, vfsFlags, nullptr);
	if (rc == RC::OK)
		rc = pager->ReadFileheader(sizeof(dbHeader), dbHeader);
	if (rc != RC::OK)
		goto _out;
	pager->SetBusyhandler(Busyhandler, nullptr);
	auto readOnly = pager->get_Readonly();
	//
	int reserves;
	auto pageSize = (uint)((dbHeader[16] << 8) | (dbHeader[17] << 16));
	if (pageSize < 512 || pageSize > MAX_PAGE_SIZE || ((pageSize - 1) & pageSize) != 0)
	{
		pageSize = 0;
		reserves = 0;
	}
	else
		reserves = dbHeader[20];
	rc = pager->SetPageSize(&pageSize, reserves);
	if (rc) goto _out;
_out:
	if (rc != RC::OK)
	{
		if (pager)
			pager->Close();
		pager = nullptr;
	}
	else
		pager->SetCacheSize(2000);
	return pager;
}

static void TestPager()
{
	auto vfs = VSystem::Find("win32");
	auto pager = Open(vfs);
	if (pager == nullptr)
		throw;
	auto rc = pager->SharedLock();
	if (rc != RC::OK)
		throw;
	//
	IPage *p = nullptr;
	rc = pager->Acquire(1, &p, false);
	if (rc != RC::OK)
		throw;
	rc = pager->Begin(0, false);
	if (rc != RC::OK)
		throw;
	char values[] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };
	memcpy(values, p->Data, 10);
	Pager::Write(p);
	pager->CommitPhaseOne(nullptr, false);
	pager->CommitPhaseTwo();
	//
	if (pager != nullptr)
		pager->Close();
}

void TestBitvec()
{
	int ops[] = { 5, 1, 1, 1, 0 };
	Core::Bitvec_BuiltinTest(400, ops);
}
*/